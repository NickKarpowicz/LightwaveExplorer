#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "LightwaveExplorerCore.cuh"
#include "LightwaveExplorerUtilities.h"
#include <complex>
#include <cstdlib>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hipfft/hipfft.h>
#include <mkl.h>
#include <thread>
#include <thrust/complex.h>
#define _CRT_SECTURE_NO_WARNINGS

#define THREADS_PER_BLOCK 32
#define MIN_GRIDDIM 8
#define ANGLETOLERANCE 1e-12
#define FALSE 0
#define TRUE 1
#define MAX_LOADSTRING 1024
#define TWOPI 6.2831853071795862
#define PI 3.1415926535897931
#define DEG2RAD 1.7453292519943295e-02
#define LIGHTC 2.99792458e8
#define EPS0 8.8541878128e-12
#define SIXTH 0.1666666666666667
#define THIRD 0.3333333333333333
#define KLORENTZIAN 3183.9 //(e * e / (epsilon_o * m_e)
#ifndef max
#define max(a,b)            (((a) > (b)) ? (a) : (b))
#endif
#ifndef min
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#endif

#ifdef __HIPCC__
#define FGLOBAL __global__
#define FDEVICE __device__ __host__
#define GKERN
#define RUNTYPE 0
#else
#define FGLOBAL
#define FDEVICE
#define GKERN uint3 blockIdx, uint3 threadIdx, uint3 blockDim,
#define RUNTYPE 1
#endif

#ifdef __HIPCC__
namespace deviceFunctions {
#else
namespace ordinaryFunctions {
#endif
	FDEVICE double hipCreal(thrust::complex<double> x) {
		return x.real();
	}

	FDEVICE double hipCimag(thrust::complex<double> x) {
		return x.imag();
	}


	//Inner function for the Sellmeier equation to provide the refractive indicies
	//current equation form:
	//n^2 = a[0] //background (high freq) contribution
	//      + four resonances, purely real contribution
	//      + parametrized low-frequency correction
	//      + 2 complex-valued Lorenzian contribution
	//inputs:
	//a: 22 component array of the coefficients
	//ls: lamda^2 (microns^2)
	//omega: frequency (rad/s)
	//ii: sqrt(-1)
	//kL: 3183.9 i.e. (e * e / (epsilon_o * m_e)
	FDEVICE thrust::complex<double> sellmeierSubfunctionCuda(
		double* a, double ls, double omega) {
		double realPart = a[0]
			+ (a[1] + a[2] * ls) / (ls + a[3])
			+ (a[4] + a[5] * ls) / (ls + a[6])
			+ (a[7] + a[8] * ls) / (ls + a[9])
			+ (a[10] + a[11] * ls) / (ls + a[12])
			+ a[13] * ls
			+ a[14] * ls * ls
			+ a[15] * ls * ls * ls;

		//traditional sellmeier part is not allowed to give complex values because that almost always
		//means it's out of range and causes instability
		if (realPart < 0) realPart = 1;

		return thrust::sqrt(realPart
			+ KLORENTZIAN * a[16] / thrust::complex<double>(a[17] - omega * omega, a[18] * omega)
			+ KLORENTZIAN * a[19] / thrust::complex<double>(a[20] - omega * omega, a[21] * omega));
	}

	//Sellmeier equation for refractive indicies
	FDEVICE thrust::complex<double> sellmeierCuda(
		thrust::complex<double>* ne, thrust::complex<double>* no, double* a, double f, double theta, double phi, int type, int eqn) {
		if (f == 0) return thrust::complex<double>(1.0, 0.0); //exit immediately for f=0

		double ls = 2.99792458e14 / f; //wavelength in microns
		ls *= ls; //only wavelength^2 is ever used
		double omega = TWOPI * abs(f);

		//option 0: isotropic
		if (type == 0) {
			ne[0] = sellmeierSubfunctionCuda(a, ls, omega);
			no[0] = ne[0];
			return ne[0];
		}
		//option 1: uniaxial
		else if (type == 1) {
			thrust::complex<double> na = sellmeierSubfunctionCuda(a, ls, omega);
			thrust::complex<double> nb = sellmeierSubfunctionCuda(&a[22], ls, omega);
			no[0] = na;
			ne[0] = 1.0 / thrust::sqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
			return ne[0];
		}
		else {
			//type == 2: biaxial
			// X. Yin, S. Zhang and Z. Tian, Optics and Laser Technology 39 (2007) 510 - 513.
			// I am sorry if there is a bug and you're trying to find it, i did my best.
			thrust::complex<double> na = sellmeierSubfunctionCuda(a, ls, omega);
			thrust::complex<double> nb = sellmeierSubfunctionCuda(&a[22], ls, omega);
			thrust::complex<double> nc = sellmeierSubfunctionCuda(&a[44], ls, omega);
			double cosTheta = cos(theta);
			double cosTheta2 = cosTheta * cosTheta;
			double sinTheta = sin(theta);
			double sinTheta2 = sinTheta * sinTheta;
			double sinPhi = sin(phi);
			double sinPhi2 = sinPhi * sinPhi;
			double cosPhi = cos(phi);
			double cosPhi2 = cosPhi * cosPhi;
			double realna2 = na.real() * na.real();
			double realnb2 = nb.real() * nb.real();
			thrust::complex<double> na2 = na * na;
			thrust::complex<double> nb2 = nb * nb;
			thrust::complex<double> nc2 = nc * nc;
			double delta = 0.5 * atan(-((1. / realna2 - 1. / realnb2)
				* sin(2 * phi) * cosTheta) / ((cosPhi2 / realna2 + sinPhi2 / realnb2)
					+ ((sinPhi2 / realna2 + cosPhi2 / realnb2)
						* cosTheta2 + sinTheta2 / (nc.real() * nc.real()))));
			double cosDelta = cos(delta);
			double sinDelta = sin(delta);
			ne[0] = 1.0 / thrust::sqrt(cosDelta * cosDelta * (cosTheta2 * (cosPhi2 / na2
				+ sinPhi2 / nb2) + sinTheta2 / nc2)
				+ sinDelta * sinDelta * (sinPhi2 / na2 + cosPhi2 / nb2)
				- 0.5 * sin(2 * phi) * cosTheta * sin(2 * delta) * (1. / na2 - 1. / (nb * nb)));

			no[0] = 1.0 / thrust::sqrt(sinDelta * sinDelta * (cosTheta2 * (cosPhi2 / na2
				+ sinPhi2 / nb2) + sinTheta2 / nc2)
				+ cosDelta * cosDelta * (sinPhi2 / na2 + cosPhi2 / nb2)
				+ 0.5 * sin(2 * phi) * cosTheta * sin(2 * delta) * (1. / na2 - 1. / nb2));
			return ne[0];
		}
	}

	FDEVICE double cuCModSquared(thrust::complex<double> a) {
		return a.real() * a.real() + a.imag() * a.imag();
	}

	//provide a list of nearest-3 neighbors for taking spatial derivatives
	// exploiting the fact that the radial grid is offset by 1/4 step from 0
	// this means that midpoints are available on the other side of the origin.
	// returns rho at the given index j
	FDEVICE double resolveNeighborsInOffsetRadialSymmetry(
		long long* neighbors, long long N, int j, double dr, long long Ntime, long long h) {
		if (j < N / 2) {
			neighbors[0] = (N - j - 2) * Ntime + h;
			neighbors[1] = (j + 1) * Ntime + h;
			neighbors[2] = (N - j - 1) * Ntime + h;
			neighbors[3] = (N - j) * Ntime + h;
			neighbors[4] = (j - 1) * Ntime + h;
			neighbors[5] = (N - j + 1) * Ntime + h;
			return -(dr * (j - N / 2) + 0.25 * dr);
		}
		else {
			neighbors[0] = (N - j + 1) * Ntime + h;
			neighbors[1] = (j - 1) * Ntime + h;
			neighbors[2] = (N - j) * Ntime + h;
			neighbors[3] = (N - j - 1) * Ntime + h;
			neighbors[4] = (j + 1) * Ntime + h;
			neighbors[5] = (N - j - 2) * Ntime + h;
			return dr * (j - N / 2) + 0.25 * dr;
		}
	}


	//function to find the effective crystal direction (theta,phi) for a given coordinate in k-space
	// trivial in isotropic media (Snell's law) but in birefringent media,
	// the refracted angle depends on refractive index, but refractive
	// index depends on the refracted angle. 
	//  Using conservation of the transverse momentum, we must solve the system of equations:
	//	kx1 = kx2
	//  ky1 = ky2
	//  where 
	//  kx1 = (w/c)sin(alpha_i)
	//  kx2 = (n(theta+alpha,phi+beta)w/c)*sin(alpha)
	//  ky1 = (w/c)sin(beta_i)
	//  ky2 = (n(theta+alpha,phi+beta)w/c)*sin(beta)
	//
	// The k grid is known, meaning kx1 and ky1 are givens; alpha and beta are unknowns
	// minimize n(alpha,beta)*sin(alpha) - kx1*c/w, n(alpha,beta)*sin(beta) - ky1*c/w
	//
	// starting point n = n(alpha=0, beta=0), which is the actual solution for isotropic medium
	// If isotropic, return
	// If uniaxial, solve 1D problem with n(alpha,0)
	// If biaxial, solve 2D problem
	// Use OGM1; D. Kim, J.A. Fessler, Optimized first-order methods for smooth convex minimization, arXiv:1406.5468
	FDEVICE void findBirefringentCrystalIndex(cudaParameterSet* s, double* sellmeierCoefficients, long long i, thrust::complex<double>* n1, thrust::complex<double>* n2) {
		long long j, k, h, col;
		h = 1 + i % ((*s).Nfreq - 1);
		col = i / ((*s).Nfreq - 1);
		j = col % (*s).Nspace;
		k = col / (*s).Nspace;

		double f = (*s).fStep * h;
		double kx1 = (LIGHTC / (TWOPI * f)) * (j * (*s).dk1 - (j >= ((*s).Nspace / 2)) * ((*s).dk1 * (*s).Nspace));
		double ky1 = (LIGHTC / (TWOPI * f)) * (k * (*s).dk2 - (k >= ((*s).Nspace2 / 2)) * ((*s).dk2 * (*s).Nspace2));
		//alpha is deviation from crystal Theta (x2 polarizations)
		//beta is deviation from crystal Phi
		//
		thrust::complex<double> n[4][2];
		thrust::complex<double> nW;
		sellmeierCuda(&n[0][0], &n[0][1], sellmeierCoefficients, f, sellmeierCoefficients[66], sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
		if ((*s).axesNumber == 0) {
			*n1 = n[0][0];
			*n2 = n[0][1];
			return;
		}

		double gradient[2][2];
		double alpha[2] = { asin(kx1 / n[0][0].real()),asin(kx1 / n[0][1].real()) };
		double beta[2] = { asin(ky1 / n[0][0].real()),asin(ky1 / n[0][1].real()) };

		double gradientStep = 1.0e-7;
		double gradientFactor = 0.5 / gradientStep;
		int it;
		int maxiter = 32;
		//emperical testing: 
		// converges to double precision limit in two iterations for BBO
		// converges in 32 iterations in BiBO

		double errArray[4][2];
		if ((*s).axesNumber == 1) {
			maxiter = 4;
			sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] + gradientStep, sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&n[1][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] - gradientStep, sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
			errArray[0][0] = sin(alpha[0] + gradientStep) * n[0][0].real() - kx1;
			errArray[1][0] = sin(alpha[0] - gradientStep) * n[1][0].real() - kx1;
			gradient[0][0] = gradientFactor * (errArray[0][0] - errArray[1][0]);

			for (it = 0; it < maxiter; it++) {
				if (abs(gradient[0][0]) > 1e-13) alpha[0] -= 0.5 * (errArray[0][0] + errArray[1][0]) / gradient[0][0];

				sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] + gradientStep, sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&n[1][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] - gradientStep, sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
				errArray[0][0] = sin(alpha[0] + gradientStep) * n[0][0].real() - kx1;
				errArray[1][0] = sin(alpha[0] - gradientStep) * n[1][0].real() - kx1;
				gradient[0][0] = gradientFactor * (errArray[0][0] - errArray[1][0]);
			}
			sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[1][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67], (*s).axesNumber, (*s).sellmeierType);
			*n1 = n[0][0];
			*n2 = n[1][1];
			return;
		}

		if ((*s).axesNumber == 2) {
			sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] + gradientStep, sellmeierCoefficients[67] + beta[0], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&n[1][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] - gradientStep, sellmeierCoefficients[67] + beta[0], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&n[2][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67] + beta[0] + gradientStep, (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&n[3][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67] + beta[0] - gradientStep, (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[0][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1] + gradientStep, sellmeierCoefficients[67] + beta[1], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[1][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1] - gradientStep, sellmeierCoefficients[67] + beta[1], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[2][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67] + beta[1] + gradientStep, (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[3][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67] + beta[1] - gradientStep, (*s).axesNumber, (*s).sellmeierType);
			errArray[0][0] = sin(alpha[0] + gradientStep) * n[0][0].real() - kx1;
			errArray[1][0] = sin(alpha[0] - gradientStep) * n[1][0].real() - kx1;
			errArray[2][0] = sin(beta[0] + gradientStep) * n[2][0].real() - ky1;
			errArray[3][0] = sin(beta[0] - gradientStep) * n[3][0].real() - ky1;
			errArray[0][1] = sin(alpha[1] + gradientStep) * n[0][1].real() - kx1;
			errArray[1][1] = sin(alpha[1] - gradientStep) * n[1][1].real() - kx1;
			errArray[2][1] = sin(beta[1] + gradientStep) * n[2][1].real() - ky1;
			errArray[3][1] = sin(beta[1] - gradientStep) * n[3][1].real() - ky1;
			gradient[0][0] = gradientFactor * (errArray[0][0] - errArray[1][0]);
			gradient[1][0] = gradientFactor * (errArray[2][0] - errArray[3][0]);
			gradient[0][1] = gradientFactor * (errArray[0][1] - errArray[1][1]);
			gradient[1][1] = gradientFactor * (errArray[2][1] - errArray[3][1]);

			for (it = 0; it < maxiter; it++) {
				if (abs(gradient[0][0]) > 1e-13) alpha[0] -= 0.25 * (errArray[0][0] + errArray[1][0]) / gradient[0][0];
				if (abs(gradient[1][0]) > 1e-13) beta[0] -= 0.25 * (errArray[2][0] + errArray[3][0]) / gradient[1][0];
				if (abs(gradient[0][1]) > 1e-13) alpha[1] -= 0.25 * (errArray[0][1] + errArray[1][1]) / gradient[0][1];
				if (abs(gradient[1][1]) > 1e-13) beta[1] -= 0.25 * (errArray[2][1] + errArray[3][1]) / gradient[1][1];
				sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] + gradientStep, sellmeierCoefficients[67] + beta[0], (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&n[1][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0] - gradientStep, sellmeierCoefficients[67] + beta[0], (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&n[2][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67] + beta[0] + gradientStep, (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&n[3][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67] + beta[0] - gradientStep, (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&nW, &n[0][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1] + gradientStep, sellmeierCoefficients[67] + beta[1], (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&nW, &n[1][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1] - gradientStep, sellmeierCoefficients[67] + beta[1], (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&nW, &n[2][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67] + beta[1] + gradientStep, (*s).axesNumber, (*s).sellmeierType);
				sellmeierCuda(&nW, &n[3][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67] + beta[1] - gradientStep, (*s).axesNumber, (*s).sellmeierType);
				errArray[0][0] = sin(alpha[0] + gradientStep) * n[0][0].real() - kx1;
				errArray[1][0] = sin(alpha[0] - gradientStep) * n[1][0].real() - kx1;
				errArray[2][0] = sin(beta[0] + gradientStep) * n[2][0].real() - ky1;
				errArray[3][0] = sin(beta[0] - gradientStep) * n[3][0].real() - ky1;
				errArray[0][1] = sin(alpha[1] + gradientStep) * n[0][1].real() - kx1;
				errArray[1][1] = sin(alpha[1] - gradientStep) * n[1][1].real() - kx1;
				errArray[2][1] = sin(beta[1] + gradientStep) * n[2][1].real() - ky1;
				errArray[3][1] = sin(beta[1] - gradientStep) * n[3][1].real() - ky1;
				gradient[0][0] = gradientFactor * (errArray[0][0] - errArray[1][0]);
				gradient[1][0] = gradientFactor * (errArray[2][0] - errArray[3][0]);
				gradient[0][1] = gradientFactor * (errArray[0][1] - errArray[1][1]);
				gradient[1][1] = gradientFactor * (errArray[2][1] - errArray[3][1]);
			}
			sellmeierCuda(&n[0][0], &nW, sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[0], sellmeierCoefficients[67] + beta[0], (*s).axesNumber, (*s).sellmeierType);
			sellmeierCuda(&nW, &n[1][1], sellmeierCoefficients, f, sellmeierCoefficients[66] + alpha[1], sellmeierCoefficients[67] + beta[1], (*s).axesNumber, (*s).sellmeierType);
			*n1 = n[0][0];
			*n2 = n[1][1];
			return;
		}

	}

	FDEVICE void findBirefingentCrystalAngle(double* alphaE, double* alphaO, long long j, double f, double* sellmeierCoefficients, cudaParameterSet* s) {
		//Find walkoff angle, starting from zero
		// in the case of an extraordinary axis, the angle of propagation is related to the transverse
		// momentum in a complicated way:
		// sin(theta) * n(theta) = delta k * c/omega
		// theta depends on the refractive index, and the refractive index depends on theta
		// so we solve numerically
		double dAlpha = 0.1;
		double nePlus, neMinus;
		double err, errPlus, errMinus;
		thrust::complex<double> ne, no;


		thrust::complex<double> ii = thrust::complex<double>(0, 1);
		double crystalTheta = sellmeierCoefficients[66];
		double crystalPhi = sellmeierCoefficients[67];
		double kStep = sellmeierCoefficients[70];
		double tol = sellmeierCoefficients[72];
		double dk = j * kStep - (j >= ((*s).Nspace / 2)) * (kStep * (*s).Nspace); //frequency grid in transverse direction
		double rhs = LIGHTC * dk / (TWOPI * f);

		//if not biaxial, the o-axis can be solved analytically.
		sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
			crystalTheta, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
		*alphaO = asin(rhs / no.real());
		if ((*s).axesNumber == 2) {
			sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
				crystalTheta + *alphaO, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
			nePlus = no.real();
			err = abs(nePlus * sin(*alphaO) - rhs);

			int iters = 0;
			errPlus = 2;
			errMinus = 2;
			while (err > tol && iters < 2048) {
				iters++;

				sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
					crystalTheta + *alphaO + dAlpha, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
				nePlus = no.real();
				errPlus = abs(nePlus * sin(*alphaO + dAlpha) - rhs);

				sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
					crystalTheta + *alphaO - dAlpha, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
				neMinus = no.real();
				errMinus = abs(neMinus * sin(*alphaO - dAlpha) - rhs);

				//Basic hill climbing algorithm
				//calculate the error at theta +/- dTheta
				// if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
				// if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
				// if theta has lowest error, step size is too large, dTheta /= 2;
				if (errPlus < err && errPlus < errMinus) {
					*alphaO += dAlpha;
					err = errPlus;
				}
				else if (errMinus < err) {
					*alphaO -= dAlpha;
					err = errMinus;
				}
				else {
					dAlpha *= 0.5;
				}

			}
		}

		//find the extraordinary angle if the crystal isn't isotropic
		*alphaE = *alphaO;
		if ((*s).axesNumber > 0) {
			sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
				crystalTheta + *alphaE, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
			nePlus = ne.real();
			err = abs(nePlus * sin(*alphaE) - rhs);

			int iters = 0;
			errPlus = 2;
			errMinus = 2;
			dAlpha = 0.1;
			while (err > tol && iters < 2048) {
				iters++;

				sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
					crystalTheta + *alphaE + dAlpha, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
				nePlus = ne.real();
				errPlus = abs(nePlus * sin(*alphaE + dAlpha) - rhs);

				sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
					crystalTheta + *alphaE - dAlpha, crystalPhi, (*s).axesNumber, (*s).sellmeierType);
				neMinus = ne.real();
				errMinus = abs(neMinus * sin(*alphaE - dAlpha) - rhs);

				//Basic hill climbing algorithm
				//calculate the error at theta +/- dTheta
				// if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
				// if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
				// if theta has lowest error, step size is too large, dTheta /= 2;
				if (errPlus < err && errPlus < errMinus) {
					*alphaE += dAlpha;
					err = errPlus;
				}
				else if (errMinus < err) {
					*alphaE -= dAlpha;
					err = errMinus;
				}
				else {
					dAlpha *= 0.5;
				}
			}
		}
	}
}

#ifdef __HIPCC__
using namespace deviceFunctions;
#else
using namespace ordinaryFunctions;
#endif
FGLOBAL void millersRuleNormalizationKernel(GKERN cudaParameterSet* s, double* sellmeierCoefficients, double* referenceFrequencies) {
	if (!(*s).isUsingMillersRule) {
		return;
	}
	size_t i;
	double chi11[7];
	double chi12[7];
	thrust::complex<double> ne, no;
	for (i = 0; i < 7; i++) {
		if (referenceFrequencies[i] == 0) {
			chi11[i] = 100000.0;
			chi12[i] = 100000.0;
		}
		else {
			sellmeierCuda(&ne, &no, sellmeierCoefficients, referenceFrequencies[i], sellmeierCoefficients[66], sellmeierCoefficients[67], (int)sellmeierCoefficients[69], 0);
			chi11[i] =ne.real() *ne.real() - 1;
			chi12[i] =no.real() *no.real() - 1;
		}
	}

	//normalize chi2 tensor values
	(*s).chi2Tensor[0] /= chi11[0] * chi11[1] * chi11[2];
	(*s).chi2Tensor[1] /= chi11[0] * chi11[1] * chi12[2];
	(*s).chi2Tensor[2] /= chi11[0] * chi12[1] * chi11[2];
	(*s).chi2Tensor[3] /= chi11[0] * chi12[1] * chi12[2];
	(*s).chi2Tensor[4] /= chi12[0] * chi12[1] * chi11[2];
	(*s).chi2Tensor[5] /= chi12[0] * chi12[1] * chi12[2];

	//normalize chi3 tensor values
	for (char i = 0; i < 81; i++) {
		(*s).chi3Tensor[i] /= chi11[3] * chi11[4] * chi11[5] * chi11[6];
	}
}

FGLOBAL void totalSpectrumKernel(GKERN thrust::complex<double>* fieldGrid1, thrust::complex<double>* fieldGrid2, double gridStep, size_t Ntime, size_t Nspace, double* spectrum) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t j;
	double beamCenter1 = 0.;
	double beamCenter2 = 0.;
	double beamTotal1 = 0.;
	double beamTotal2 = 0.;
	double a, x;

	//find beam centers
	for (j = 0; j < Nspace; j++) {
		x = gridStep * j;
		a = cuCModSquared(fieldGrid1[i + j * Ntime]);
		beamTotal1 += a;
		beamCenter1 += x * a;
		a = cuCModSquared(fieldGrid2[i + j * Ntime]);
		beamTotal2 += a;
		beamCenter2 += x * a;
	}
	if (beamTotal1 > 0) {
		beamCenter1 /= beamTotal1;
	}
	if (beamTotal2 > 0) {
		beamCenter2 /= beamTotal2;
	}


	//Integrate total beam power, assuming radially-symmetric beam around
	//the center
	beamTotal1 = 0.;
	beamTotal2 = 0.;
	for (j = 0; j < Nspace; j++) {
		x = gridStep * j;
		beamTotal1 += PI * abs(x - beamCenter1) * cuCModSquared(fieldGrid1[i + j * Ntime]);
		beamTotal2 += PI * abs(x - beamCenter2) * cuCModSquared(fieldGrid2[i + j * Ntime]);
	}
	beamTotal1 *= gridStep / Ntime;
	beamTotal2 *= gridStep / Ntime;

	//put the values into the output spectrum
	spectrum[i] = beamTotal1;
	spectrum[i + Ntime] = beamTotal2;
	spectrum[i + 2 * Ntime] = beamTotal1 + beamTotal2;
}

FGLOBAL void totalSpectrum3DKernel(GKERN thrust::complex<double>* fieldGrid1, thrust::complex<double>* fieldGrid2, double gridStep, size_t Ntime, size_t Nspace, double* spectrum) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t j;

	double beamTotal1 = 0.;
	double beamTotal2 = 0.;
	//Integrate total beam power
	beamTotal1 = 0.;
	beamTotal2 = 0.;
	for (j = 0; j < Nspace; j++) {
		beamTotal1 += cuCModSquared(fieldGrid1[i + j * Ntime]);
		beamTotal2 += cuCModSquared(fieldGrid2[i + j * Ntime]);
	}
	beamTotal1 *= gridStep * gridStep / Ntime;
	beamTotal2 *= gridStep * gridStep / Ntime;

	//put the values into the output spectrum
	spectrum[i] = beamTotal1;
	spectrum[i + Ntime] = beamTotal2;
	spectrum[i + 2 * Ntime] = beamTotal1 + beamTotal2;
}

//rotate the field around the propagation axis (basis change)
FGLOBAL void rotateFieldKernel(GKERN thrust::complex<double>* Ein1, thrust::complex<double>* Ein2, thrust::complex<double>* Eout1,
	thrust::complex<double>* Eout2, double rotationAngle) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	Eout1[i] = cos(rotationAngle) * Ein1[i] - sin(rotationAngle) * Ein2[i];
	Eout2[i] = sin(rotationAngle) * Ein1[i] + cos(rotationAngle) * Ein2[i];
}



FGLOBAL void radialLaplacianKernel(GKERN cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j = i / (*s).Ntime; //spatial coordinate
	long long h = i % (*s).Ntime; //temporal coordinate
	long long neighbors[6];

	//zero at edges of grid
	if (j<3 || j>((*s).Nspace - 4)) {
		(*s).gridRadialLaplacian1[i] = 0.;
		(*s).gridRadialLaplacian2[i] = 0.;
	}
	else {
		double rho = resolveNeighborsInOffsetRadialSymmetry(neighbors, (*s).Nspace, j, (*s).dx, (*s).Ntime, h);
		rho = -1.0 / rho;
		(*s).gridRadialLaplacian1[i] = rho * ((*s).firstDerivativeOperation[0] * (*s).gridETime1[neighbors[0]]
			+ (*s).firstDerivativeOperation[1] * (*s).gridETime1[neighbors[1]]
			+ (*s).firstDerivativeOperation[2] * (*s).gridETime1[neighbors[2]]
			+ (*s).firstDerivativeOperation[3] * (*s).gridETime1[neighbors[3]]
			+ (*s).firstDerivativeOperation[4] * (*s).gridETime1[neighbors[4]]
			+ (*s).firstDerivativeOperation[5] * (*s).gridETime1[neighbors[5]]);
		(*s).gridRadialLaplacian2[i] = rho * ((*s).firstDerivativeOperation[0] * (*s).gridETime2[neighbors[0]]
			+ (*s).firstDerivativeOperation[1] * (*s).gridETime2[neighbors[1]]
			+ (*s).firstDerivativeOperation[2] * (*s).gridETime2[neighbors[2]]
			+ (*s).firstDerivativeOperation[3] * (*s).gridETime2[neighbors[3]]
			+ (*s).firstDerivativeOperation[4] * (*s).gridETime2[neighbors[4]]
			+ (*s).firstDerivativeOperation[5] * (*s).gridETime2[neighbors[5]]);
	}

}
//Expand the information contained in the radially-symmetric beam in the offset grid
// representation.
// The grid is offset from the origin; rather than ...-2 -1 0 1 2... etc, which would
// contain redundant information (the symmetry means that -1 and -1 are equivalent)
// the grid is at the points -1.75 -0.75 0.25 1.25 2.25, etc.
// the grid spacing is the same, but now the two sides of the origin contain different
// information. This has effectively doubled the resolution of the nonlinear
// polarization. 
// We make use of this by expanding into the full-resolution beam on the grid
// -2.25 -1.75 -1.25 -0.75 -0.25 0.25 0.75 1.25 1.75 2.25...
// after FFT, we can discard the high frequencies. Thus we have downsampled
// in such a way as to avoid aliasing, which inside the simulation is most
// likely the appear (and cause instability) in the nonlinear terms.
FGLOBAL void expandCylindricalBeam(GKERN cudaParameterSet* s, double* polarization1, double* polarization2) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t j = i / (*s).Ntime; //spatial coordinate
	size_t k = i % (*s).Ntime; //temporal coordinate

	//positions on the expanded grid corresponding the the current index
	size_t pos1 = 2 * ((*s).Nspace - j - 1) * (*s).Ntime + k;
	size_t pos2 = (2 * j + 1) * (*s).Ntime + k;

	//reuse memory allocated for the radial Laplacian, casting complex double
	//to a 2x larger double real grid
	double* expandedBeam1 = (double*)(*s).gridRadialLaplacian1;
	double* expandedBeam2 = expandedBeam1 + 2 * (*s).Ngrid;

	expandedBeam1[pos1] = polarization1[i];
	expandedBeam1[pos2] = polarization1[i];
	expandedBeam2[pos1] = polarization2[i];
	expandedBeam2[pos2] = polarization2[i];
}



//prepare propagation constants for the simulation, when it is taking place on a Cartesian grid
//note that the sellmeier coefficients have extra values appended to the end
//to give info about the current simulation
FGLOBAL void applyFresnelLossKernel(GKERN double* sellmeierCoefficients1, double* sellmeierCoefficients2, cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	double alpha1, alpha2, alphaO1, alphaO2;
	long long j, k;
	long long Ntime = (*s).Ntime;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> ne1, no1, ne2, no2, n0;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);
	j = i / Ntime; //spatial coordinate
	k = i % Ntime; //temporal coordinate
	thrust::complex<double> ii = thrust::complex<double>(0, 1);
	double crystalTheta = sellmeierCoefficients1[66];
	double crystalPhi = sellmeierCoefficients1[67];
	double fStep = sellmeierCoefficients1[71];

	//frequency being resolved by current thread
	double f = k * fStep;


	findBirefingentCrystalAngle(&alpha1, &alphaO1, j, f, sellmeierCoefficients1, s);
	findBirefingentCrystalAngle(&alpha2, &alphaO2, j, f, sellmeierCoefficients2, s);
	//walkoff angle has been found, generate the rest of the grids


	sellmeierCuda(&ne1, &no1, sellmeierCoefficients1, f,
		crystalTheta + 0*alpha1, crystalPhi, axesNumber, sellmeierType);
	sellmeierCuda(&n0, &no1, sellmeierCoefficients1, f,
		crystalTheta + 0*alphaO1, crystalPhi, axesNumber, sellmeierType);
	if (isnan(ne1.real()) || isnan(no1.real())) {
		ne1 = thrust::complex<double>(1, 0);
		no1 = thrust::complex<double>(1, 0);
	}


	sellmeierCuda(&ne2, &no2, sellmeierCoefficients2, f,
		crystalTheta + alpha2, crystalPhi, axesNumber, sellmeierType);
	sellmeierCuda(&n0, &no2, sellmeierCoefficients2, f,
		crystalTheta + alphaO2, crystalPhi, axesNumber, sellmeierType);
	if (isnan(ne2.real()) || isnan(no2.real())) {
		ne2 = thrust::complex<double>(1, 0);
		no2 = thrust::complex<double>(1, 0);
	}

	thrust::complex<double> ts = 2 * ne1 * cos(alpha1) / (ne1 * cos(alpha1) + ne2 * cos(alpha2));
	thrust::complex<double> tp = 2 * ne1 * cos(alpha1) / (ne2 * cos(alpha1) + ne1 * cos(alpha2));
	if (isnan(ts.real()) || isnan(ts.imag())) ts = thrust::complex<double>(0, 0);
	if (isnan(tp.real()) || isnan(tp.imag())) ts = thrust::complex<double>(0, 0);
	(*s).gridEFrequency1[i] = ts * (*s).gridEFrequency1[i];
	(*s).gridEFrequency2[i] = tp * (*s).gridEFrequency2[i];
}


FGLOBAL void apertureKernel(GKERN cudaParameterSet* s, double radius, double activationParameter) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k, col;

	col = i / (*s).Ntime;
	j = col % (*s).Nspace;
	k = col / (*s).Nspace;
	double r;
	if ((*s).is3D) {
		double x = ((*s).dx * (j - (*s).Nspace / 2.0));
		double y = ((*s).dx * (k - (*s).Nspace2 / 2.0));
		r = sqrt(x * x + y * y);
	} 
	else {
		r = abs((*s).dx * ((double)j - (*s).Nspace / 2.0) - 0.25 * (*s).dx);
	}

	double a = 1.0 - (1.0 / (1.0 + exp( - activationParameter*(r - radius)/(*s).dx)));

	//if (r>radius) a = 0;
	(*s).gridETime1[i] *= a;
	(*s).gridETime2[i] *= a;
}

FGLOBAL void parabolicMirrorKernel(GKERN cudaParameterSet* s, double focus) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k, h, col;
	h = 1 + i % ((*s).Nfreq - 1);
	col = i / ((*s).Nfreq - 1);
	i = h + col * (*s).Nfreq;
	j = col % (*s).Nspace;
	k = col / (*s).Nspace;

	double w = TWOPI * h * (*s).fStep;
	double r;
	if ((*s).is3D) {
		double x = ((*s).dx * (j - (*s).Nspace / 2.0));
		double y = ((*s).dx * (k - (*s).Nspace2 / 2.0));
		r = sqrt(x * x + y * y);
	}
	else {
		r = abs((*s).dx * ((double)j - (*s).Nspace / 2.0) - 0.25 * (*s).dx);
	}


	

	thrust::complex<double>	u = thrust::exp(thrust::complex<double>(0.0,
		w * r * r * (0.5 / focus) / LIGHTC));


	(*s).gridEFrequency1[i] = u * (*s).gridEFrequency1[i];
	(*s).gridEFrequency2[i] = u * (*s).gridEFrequency2[i];
}

FGLOBAL void sphericalMirrorKernel(GKERN cudaParameterSet* s, double ROC) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k, h, col;
	h = 1 + i % ((*s).Nfreq - 1);
	col = i / ((*s).Nfreq - 1);
	i = h + col * (*s).Nfreq;
	j = col % (*s).Nspace;
	k = col / (*s).Nspace;

	double w = TWOPI * h * (*s).fStep;
	double r;
	if ((*s).is3D) {
		double x = ((*s).dx * (j - (*s).Nspace / 2.0));
		double y = ((*s).dx * (k - (*s).Nspace2 / 2.0));
		r = sqrt(x * x + y * y);
	}
	else {
		r = abs((*s).dx * ((double)j - (*s).Nspace / 2.0) - 0.25 * (*s).dx);
	}

	bool isNegative = signbit(ROC);
	ROC = abs(ROC);
	thrust::complex<double> u = thrust::complex<double>(0.0, 0.0);
	if (r < ROC) {
		u = thrust::exp(thrust::complex<double>(0.0, 
			2.0 * pow(-1,isNegative)*w*ROC*((sqrt(1.0 - r * r / (ROC * ROC))) - 1.0)/LIGHTC));
	}

	(*s).gridEFrequency1[i] = u * (*s).gridEFrequency1[i];
	(*s).gridEFrequency2[i] = u * (*s).gridEFrequency2[i];
}

FGLOBAL void applyLinearPropagationKernel(GKERN double* sellmeierCoefficients, double thickness, cudaParameterSet *s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, h, k, col;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> ne, no, n0, n0o;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);
	h = 1 + i % ((*s).Nfreq - 1);
	col = i / ((*s).Nfreq - 1);
	i = h + col * ((*s).Nfreq);
	j = col % (*s).Nspace;
	k = col / (*s).Nspace;
	thrust::complex<double> ii = thrust::complex<double>(0, 1);
	double crystalTheta = sellmeierCoefficients[66];
	double crystalPhi = sellmeierCoefficients[67];



	//frequency being resolved by current thread
	double f = h * (*s).fStep;
	double omega = TWOPI * f;
	findBirefringentCrystalIndex(s, sellmeierCoefficients, threadIdx.x + blockIdx.x * blockDim.x, &ne, &no);
	double dk1 = j * (*s).dk1 - (j >= ((*s).Nspace / 2)) * ((*s).dk1 * (*s).Nspace);
	double dk2 = k * (*s).dk2 - (k >= ((*s).Nspace2 / 2)) * ((*s).dk2 * (*s).Nspace2);
	if (!(*s).is3D)dk2 = 0.0;
	//if ((*s).isCylindric) dk2 = dk1;
	sellmeierCuda(&n0, &n0o, sellmeierCoefficients, (*s).f0,
		crystalTheta, crystalPhi, axesNumber, sellmeierType);
	if (isnan(ne.real()) || isnan(no.real())) {
		ne = thrust::complex<double>(1, 0);
		no = thrust::complex<double>(1, 0);
	}

	thrust::complex<double> ke = ne * omega / LIGHTC;
	thrust::complex<double> ko = no * omega / LIGHTC;
	double k0 = (n0 * omega / LIGHTC).real();
	double kze = (thrust::sqrt(ke * ke - dk1 * dk1 - dk2 * dk2)).real();
	double kzo = (thrust::sqrt(ko * ko - dk1 * dk1 - dk2 * dk2)).real();

	thrust::complex<double> ts = thrust::exp(ii * (k0 - kze) * thickness);
	thrust::complex<double> tp = thrust::exp(ii * (k0 - kzo) * thickness);
	if (isnan(ts.real()) || isnan(ts.imag())) ts = thrust::complex<double>(0, 0);
	if (isnan(tp.real()) || isnan(tp.imag())) tp = thrust::complex<double>(0, 0);
	(*s).gridEFrequency1[i] = ts * (*s).gridEFrequency1[i];
	(*s).gridEFrequency2[i] = tp * (*s).gridEFrequency2[i];
}


//prepare propagation constants for the simulation, when it is taking place on a Cartesian grid
//note that the sellmeier coefficients have extra values appended to the end
//to give info about the current simulation
FGLOBAL void prepareCartesianGridsKernel(GKERN double* sellmeierCoefficients, cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> ne, no, n0;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);
	j = i / ((*s).Nfreq-1); //spatial coordinate
	k = 1 + (i % ((*s).Nfreq-1)); //temporal coordinate
	i = k + j * (*s).Nfreq;
	thrust::complex<double> ii = thrust::complex<double>(0, 1);
	double crystalTheta = sellmeierCoefficients[66];
	double crystalPhi = sellmeierCoefficients[67];
	double kStep = sellmeierCoefficients[70];
	double fStep = sellmeierCoefficients[71];

	//frequency being resolved by current thread
	double f = -k * fStep;

	//transverse wavevector being resolved
	double dk = j * kStep - (j >= ((*s).Nspace / 2)) * (kStep * (*s).Nspace); //frequency grid in transverse direction
	sellmeierCuda(&n0, &no, sellmeierCoefficients, abs((*s).f0),
		crystalTheta, crystalPhi, axesNumber, sellmeierType);
	findBirefringentCrystalIndex(s, sellmeierCoefficients, threadIdx.x + blockIdx.x * blockDim.x, &ne, &no);

	//walkoff angle has been found, generate the rest of the grids



	if (isnan(ne.real()) || isnan(no.real())) {
		ne = thrust::complex<double>(1, 0);
		no = thrust::complex<double>(1, 0);
	}

	thrust::complex<double> k0 = thrust::complex<double>(TWOPI * n0.real() * f / LIGHTC, 0);
	thrust::complex<double> ke = TWOPI * ne * f / LIGHTC;
	thrust::complex<double> ko = TWOPI * no * f / LIGHTC;


	thrust::complex<double> chi11 = thrust::complex<double>(1.0, 0);
	thrust::complex<double> chi12 = thrust::complex<double>(1.0, 0);
	if ((*s).isUsingMillersRule) {
		chi11 = (*s).chiLinear1[k];
		chi12 = (*s).chiLinear2[k];
	}
	else {
		chi11 = thrust::complex<double>(1, 0);
		chi12 = thrust::complex<double>(1, 0);
	}

	if (abs(dk) < thrust::abs(ke)) {
		(*s).gridPropagationFactor1[i] = ii * (ke - k0 - dk * dk / (2. * ke.real())) * (*s).h;
		if (isnan(((*s).gridPropagationFactor1[i]).real())) {
			(*s).gridPropagationFactor1[i] = cuZero;
		}

		(*s).gridPropagationFactor2[i] = ii * (ko - k0 - dk * dk / (2. * ko.real())) * (*s).h;
		if (isnan(((*s).gridPropagationFactor2[i]).real())) {
			(*s).gridPropagationFactor2[i] = cuZero;
		}

		(*s).gridPolarizationFactor1[i] = ii * chi11 * (TWOPI * f) / (2. *ne.real() * LIGHTC) * (*s).h;
		(*s).gridPolarizationFactor2[i] = ii * chi12 * (TWOPI * f) / (2. *no.real() * LIGHTC) * (*s).h;
	}

	else {
		(*s).gridPropagationFactor1[i] = cuZero;
		(*s).gridPropagationFactor2[i] = cuZero;
		(*s).gridPolarizationFactor1[i] = cuZero;
		(*s).gridPolarizationFactor2[i] = cuZero;
	}

}

//prepare propagation constants for the simulation, when it is taking place on a Cartesian grid
//note that the sellmeier coefficients have extra values appended to the end
//to give info about the current simulation
FGLOBAL void prepare3DGridsKernel(GKERN double* sellmeierCoefficients, cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long col,j, k, l;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> ne, no, n0;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);
	col = i / ((*s).Nfreq-1); //spatial coordinate
	j = 1+i % ((*s).Nfreq-1); // frequency coordinate
	i = j + col * (*s).Nfreq;
	k = col % (*s).Nspace;
	l = col / (*s).Nspace;

	thrust::complex<double> ii = thrust::complex<double>(0, 1);
	double crystalTheta = sellmeierCoefficients[66];
	double crystalPhi = sellmeierCoefficients[67];

	//frequency being resolved by current thread
	double f = -j * (*s).fStep;

	//transverse wavevector being resolved
	double dk1 = k * (*s).dk1 -(k >= ((*s).Nspace / 2)) * ((*s).dk1 * (*s).Nspace); //frequency grid in x direction
	double dk2 = l * (*s).dk2 - (l >= ((*s).Nspace2 / 2)) * ((*s).dk2 * (*s).Nspace2); //frequency grid in y direction
	sellmeierCuda(&n0, &no, sellmeierCoefficients, abs((*s).f0),
		crystalTheta, crystalPhi, axesNumber, sellmeierType);
	findBirefringentCrystalIndex(s, sellmeierCoefficients, threadIdx.x + blockIdx.x * blockDim.x, &ne, &no);



	if (isnan(ne.real()) || isnan(no.real())) {
		ne = thrust::complex<double>(1, 0);
		no = thrust::complex<double>(1, 0);
	}

	thrust::complex<double> k0 = thrust::complex<double>(TWOPI * n0.real() * f / LIGHTC, 0);
	thrust::complex<double> ke = TWOPI * ne * f / LIGHTC;
	thrust::complex<double> ko = TWOPI * no * f / LIGHTC;


	thrust::complex<double> chi11 = thrust::complex<double>(1.0, 0);
	thrust::complex<double> chi12 = thrust::complex<double>(1.0, 0);
	if ((*s).isUsingMillersRule) {
		chi11 = (*s).chiLinear1[j];
		chi12 = (*s).chiLinear2[j];
	}
	else {
		chi11 = thrust::complex<double>(1, 0);
		chi12 = thrust::complex<double>(1, 0);
	}

	if (max(abs(dk1),abs(dk2)) < thrust::abs(ke)) {
		(*s).gridPropagationFactor1[i] = ii * (ke - k0 - (dk1 * dk1 + dk2 * dk2) / (2. * ke.real())) * (*s).h;
		if (isnan(hipCreal((*s).gridPropagationFactor1[i]))) {
			(*s).gridPropagationFactor1[i] = cuZero;
		}

		(*s).gridPropagationFactor2[i] = ii * (ko - k0 - (dk1 * dk1 + dk2 * dk2) / (2. * ko.real())) * (*s).h;
		if (isnan(hipCreal((*s).gridPropagationFactor2[i]))) {
			(*s).gridPropagationFactor2[i] = cuZero;
		}

		(*s).gridPolarizationFactor1[i] = ii * chi11 * (TWOPI * f) / (2. *ne.real() * LIGHTC) * (*s).h;
		(*s).gridPolarizationFactor2[i] = ii * chi12 * (TWOPI * f) / (2. *no.real() * LIGHTC) * (*s).h;
	}

	else {
		(*s).gridPropagationFactor1[i] = cuZero;
		(*s).gridPropagationFactor2[i] = cuZero;
		(*s).gridPolarizationFactor1[i] = cuZero;
		(*s).gridPolarizationFactor2[i] = cuZero;
	}

}

FGLOBAL void getChiLinearKernel(GKERN cudaParameterSet* s, double* sellmeierCoefficients) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);


	double crystalTheta = sellmeierCoefficients[66];
	double crystalPhi = sellmeierCoefficients[67];
	double fStep = sellmeierCoefficients[71];

	thrust::complex<double> ne, no, n0;

	//frequency being resolved by current thread
	double f = i * fStep;
	sellmeierCuda(&n0, &no, sellmeierCoefficients, abs((*s).f0), crystalTheta, crystalPhi, axesNumber, sellmeierType);
	sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, axesNumber, sellmeierType);
	if (isnan(ne.real()) || isnan(no.real())) {
		ne = thrust::complex<double>(1, 0);
		no = thrust::complex<double>(1, 0);
	}


	(*s).chiLinear1[i] = -1. + ne * ne;
	(*s).chiLinear2[i] = -1. + no * no;
	if ((hipCreal((*s).chiLinear1[i]) == 0) || (hipCreal((*s).chiLinear2[i]) == 0) || isnan(hipCreal((*s).chiLinear1[i])) || isnan(hipCreal((*s).chiLinear2[i]))) {
		(*s).chiLinear1[i] = thrust::complex<double>(1, 0);
		(*s).chiLinear2[i] = thrust::complex<double>(1, 0);
	}

}
//prepare the propagation constants under the assumption of cylindrical symmetry of the beam
FGLOBAL void prepareCylindricGridsKernel(GKERN double* sellmeierCoefficients, cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k;
	long long Nspace = (*s).Nspace;
	int axesNumber = (*s).axesNumber;
	int sellmeierType = (*s).sellmeierType;
	thrust::complex<double> cuZero = thrust::complex<double>(0, 0);
	j = i / ((*s).Nfreq-1); //spatial coordinate
	k = 1 + i % ((*s).Nfreq-1); //temporal coordinate
	i = k + j * (*s).Nfreq;


	thrust::complex<double> ii = thrust::complex<double>(0, 1);
	double crystalTheta = sellmeierCoefficients[66];
	double crystalPhi = sellmeierCoefficients[67];
	double kStep = sellmeierCoefficients[70];
	double fStep = sellmeierCoefficients[71];

	thrust::complex<double> ne, no, n0;

	//frequency being resolved by current thread
	double f = -k * fStep;

	//transverse wavevector being resolved
	double dk = j * kStep - (j >= (Nspace / 2)) * (kStep * Nspace); //frequency grid in transverse direction
	sellmeierCuda(&n0, &no, sellmeierCoefficients, abs((*s).f0), crystalTheta, crystalPhi, axesNumber, sellmeierType);
	sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, axesNumber, sellmeierType);
	if (isnan(ne.real()) || isnan(no.real())) {
		ne = thrust::complex<double>(1, 0);
		no = thrust::complex<double>(1, 0);
	}

	thrust::complex<double> k0 = thrust::complex<double>(TWOPI * n0.real() * f / LIGHTC, 0);
	thrust::complex<double> ke = TWOPI * ne * f / LIGHTC;
	thrust::complex<double> ko = TWOPI * no * f / LIGHTC;

	thrust::complex<double> chi11 = (*s).chiLinear1[k];
	thrust::complex<double> chi12 = (*s).chiLinear2[k];
	if (!(*s).isUsingMillersRule) {
		chi11 = thrust::complex<double>(1, 0);
		chi12 = thrust::complex<double>(1, 0);
	}

	if (abs(dk) <= min(thrust::abs(ke), thrust::abs(ko))) {
		(*s).gridPropagationFactor1[i] = ii * (ke - k0 - dk * dk / (2. * ke.real())) * (*s).h;
		(*s).gridPropagationFactor1Rho1[i] = ii * (1 / (chi11 * 2. * ke.real())) * (*s).h;
		if (isnan(hipCreal((*s).gridPropagationFactor1[i]))) {
			(*s).gridPropagationFactor1[i] = cuZero;
			(*s).gridPropagationFactor1Rho1[i] = cuZero;
		}

		(*s).gridPropagationFactor2[i] = ii * (ko - k0 - dk * dk / (2. * ko.real())) * (*s).h;
		(*s).gridPropagationFactor1Rho2[i] = ii * (1 / (chi12 * 2. * ko.real())) * (*s).h;
		if (isnan(hipCreal((*s).gridPropagationFactor2[i]))) {
			(*s).gridPropagationFactor2[i] = cuZero;
			(*s).gridPropagationFactor1Rho2[i] = cuZero;
		}
		//factor of 0.5 comes from doubled grid size in cylindrical symmetry mode after expanding the beam
		(*s).gridPolarizationFactor1[i] = 0.5 * chi11 * ii * (TWOPI * f) / (2. *ne.real() * LIGHTC) * (*s).h;
		(*s).gridPolarizationFactor2[i] = 0.5 * chi12 * ii * (TWOPI * f) / (2. *no.real() * LIGHTC) * (*s).h;


	}

	else {
		(*s).gridPropagationFactor1[i] = cuZero;
		(*s).gridPropagationFactor2[i] = cuZero;
		(*s).gridPolarizationFactor1[i] = cuZero;
		(*s).gridPolarizationFactor2[i] = cuZero;
		(*s).gridPropagationFactor1[i] = cuZero;
		(*s).gridPropagationFactor1Rho2[i] = cuZero;
	}
}

//replaces E with its complex conjugate
FGLOBAL void conjugateKernel(GKERN thrust::complex<double>* E) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	E[i] = thrust::conj(E[i]);
}

FGLOBAL void realToComplexKernel(GKERN double* in, thrust::complex<double>* out) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	out[i] = thrust::complex<double>(in[i], 0.0);
}

FGLOBAL void complexToRealKernel(GKERN thrust::complex<double>* in, double* out) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	out[i] = hipCreal(in[i]);
}

FGLOBAL void materialPhaseKernel(GKERN double df, size_t Ntime, double* a, double f01, double f02, 
	double thickness1, double thickness2, double* phase1, double* phase2) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	//frequency being resolved by current thread
	double f = i * df;
	if (i >= Ntime / 2) {
		f -= df * Ntime;
	}

	//give phase shift relative to group velocity (approximated 
	// with low-order finite difference) so the pulse doesn't move
	thrust::complex<double> ne, no, no0, n0p, n0m;
	sellmeierCuda(&ne, &no, a, abs(f), 0, 0, 0, 0);
	f *= -TWOPI;
	sellmeierCuda(&ne, &no0, a, f01, 0, 0, 0, 0);
	sellmeierCuda(&ne, &n0p, a, f01 + 1e11, 0, 0, 0, 0);
	sellmeierCuda(&ne, &n0m, a, f01 - 1e11, 0, 0, 0, 0);
	no0 = no0 + f01 * (n0p - n0m) / 2e11;
	phase1[i] = thickness1 * f * hipCreal(no - no0) / LIGHTC;
	sellmeierCuda(&ne, &no0, a, f02, 0, 0, 0, 0);
	sellmeierCuda(&ne, &n0p, a, f02 + 1e11, 0, 0, 0, 0);
	sellmeierCuda(&ne, &n0m, a, f02 - 1e11, 0, 0, 0, 0);
	no0 = no0 + f02 * (n0p - n0m) / 2e11;
	phase2[i] = thickness2 * f * hipCreal(no - no0) / LIGHTC;

}
//replaces NaN values with 0
FGLOBAL void fixnanKernel(GKERN thrust::complex<double>* E) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	if (isnan(hipCreal(E[i])) || isnan(hipCimag(E[i]))) {
		E[i] = thrust::complex<double>(0., 0.);
	}
}

//calculate the nonlinear polarization, after FFT to get the field
//in the time domain
FGLOBAL void nonlinearPolarizationKernel(GKERN cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	double Ex = (*s).fftNorm * (*s).gridETime1[i];
	double Ey = (*s).fftNorm * (*s).gridETime2[i];

	double Ex2 = Ex * Ex;
	double Ey2 = Ey * Ey;
	(*s).gridPolarizationTime1[i] = 0.;
	(*s).gridPolarizationTime2[i] = 0.;

	//The d2eff tensor has the form
	// | d_xxx d_xyx d_yyx |
	// | d_xxy d_xyy d_yyy |
	if ((*s).nonlinearSwitches[0] == 1) {
		(*s).gridPolarizationTime1[i] += (*s).chi2Tensor[0] * Ex2 + (*s).chi2Tensor[2] * Ex * Ey + (*s).chi2Tensor[4] * Ey2;
		(*s).gridPolarizationTime2[i] += (*s).chi2Tensor[1] * Ex2 + (*s).chi2Tensor[3] * Ex * Ey + (*s).chi2Tensor[5] * Ey2;
	}

	//resolve the full chi3 matrix when (*s).nonlinearSwitches[1]==1
	if ((*s).nonlinearSwitches[1] == 1) {

		//rotate field into crystal frame
		double E3[3] = { (*s).rotationForward[0] * Ex + (*s).rotationForward[1] * Ey,
			(*s).rotationForward[3] * Ex + (*s).rotationForward[4] * Ey,
			(*s).rotationForward[6] * Ex + (*s).rotationForward[7] * Ey };

		//loop over tensor element X_abcd
		//i hope the compiler unrolls this, but no way am I writing that out by hand
		unsigned char a, b, c, d;
		double P3[3] = { 0 };
		for (a = 0; a < 3; a++) {
			for (b = 0; b < 3; b++) {
				for (c = 0; c < 3; c++) {
					for (d = 0; d < 3; d++) {
						P3[d] += (*s).chi3Tensor[a + 3 * b + 9 * c + 27 * d] * E3[a] * E3[b] * E3[c];
					}
				}
			}
		}

		//rotate back into simulation frame
		(*s).gridPolarizationTime1[i] += (*s).rotationBackward[0] * P3[0] + (*s).rotationBackward[1] * P3[1] + (*s).rotationBackward[2] * P3[2];
		(*s).gridPolarizationTime2[i] += (*s).rotationBackward[3] * P3[0] + (*s).rotationBackward[4] * P3[1] + (*s).rotationBackward[5] * P3[2];
	}
	//using only one value of chi3, under assumption of centrosymmetry
	if ((*s).nonlinearSwitches[1] == 2) {
		double Esquared = (*s).chi3Tensor[0] * (Ex2 + Ey2);
		(*s).gridPolarizationTime1[i] += Ex * Esquared;
		(*s).gridPolarizationTime2[i] += Ey * Esquared;
	}
}


//Plasma response with time-dependent carrier density
//This polarization needs a different factor in the nonlinear wave equation
//to account for the integration
//plasmaParameters vector:
// 0    e^2/m_eff
// 1    gamma_drude
// 2    ionization rate/E^N
// 3    absorption strength
//equation for the plasma current:
//J_drude(t) = (e/m)*exp(-gamma*t)*\int_-infty^t dt' exp(gamma*t)*N(t)*E(t)
//J_absorption(t) = beta*E^(2*Nphot-2)*E
//plasmaParameters[0] is the nonlinear absorption parameter
	//nonlinearSwitches[3] is Nphotons-2
	//plasmaParameters[2] is the 1/photon energy, translating the loss of power
	//from the field to the number of free carriers
	//extra factor of (dt^2e^2/(m*photon energy*eo) included as it is needed for the amplitude
	//of the plasma current
FGLOBAL void plasmaCurrentKernel(GKERN cudaParameterSet* s) {
	long long j = threadIdx.x + blockIdx.x * blockDim.x;
	j *= (*s).Ntime;
	double N = 0;
	double integralx = 0;
	double integraly = 0;
	double* expMinusGammaT = &(*s).expGammaT[(*s).Ntime];
	double w, Esquared, Ex, Ey, a;
	long long k;
	unsigned char p;
	unsigned char pMax = (unsigned char)(*s).nonlinearSwitches[3];
	double Jx, Jy;
	for (k = 0; k < (*s).Ntime; k++) {
		Ex = (*s).gridETime1[j] * (*s).fftNorm;
		Ey = (*s).gridETime2[j] * (*s).fftNorm;
		Esquared = Ex * Ex + Ey * Ey;
		w = (*s).plasmaParameters[0] * Esquared;
		for (p = 0; p < pMax; p++) {
			w *= Esquared;
		}

		Jx = w * Ex;
		Jy = w * Ey;

		N += (*s).plasmaParameters[2] * (Jx * Ex + Jy * Ey);
		a = N * (*s).expGammaT[k];
		integralx += a * Ex;
		integraly += a * Ey;
		(*s).gridPolarizationTime1[j] = Jx + expMinusGammaT[k] * integralx;
		(*s).gridPolarizationTime2[j] = Jy + expMinusGammaT[k] * integraly;
		j++;
	}
}

FGLOBAL void updateKwithPolarizationKernel(GKERN cudaParameterSet* sP) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long h = 1 + i % ((*sP).Nfreq - 1); //temporal coordinate
	long long j = i / ((*sP).Nfreq - 1); //spatial coordinate
	i = h + j * ((*sP).Nfreq);
	h += (j + ((*sP).isCylindric * (j > ((*sP).Nspace / 2))) * (*sP).Nspace) * (*sP).Nfreq;

	(*sP).k1[i] = (*sP).k1[i] + (*sP).gridPolarizationFactor1[i] * (*sP).workspace1[h];
	(*sP).k2[i] = (*sP).k2[i] + (*sP).gridPolarizationFactor2[i] * (*sP).workspace2P[h];
}

FGLOBAL void updateKwithPlasmaKernel(GKERN cudaParameterSet* sP) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long h = 1 + i % ((*sP).Nfreq - 1); //temporal coordinate
	long long j = i / ((*sP).Nfreq - 1); //spatial coordinate
	i = h + j * ((*sP).Nfreq);

	thrust::complex<double> jfac = thrust::complex<double>(0, -1.0 / (h * (*sP).fStep));
	h += (j + ((*sP).isCylindric * (j > ((*sP).Nspace / 2))) * (*sP).Nspace) * (*sP).Nfreq;


	if ((*sP).isUsingMillersRule) {
		(*sP).k1[i] = (*sP).k1[i] + jfac * (*sP).gridPolarizationFactor1[i] * (*sP).workspace1[h] / (*sP).chiLinear1[i % ((*sP).Nfreq)].real();
		(*sP).k2[i] = (*sP).k2[i] + jfac * (*sP).gridPolarizationFactor2[i] * (*sP).workspace2P[h] / (*sP).chiLinear2[i % ((*sP).Nfreq)].real();
	}
	else {
		(*sP).k1[i] = (*sP).k1[i] + jfac * (*sP).gridPolarizationFactor1[i] * (*sP).workspace1[h];
		(*sP).k2[i] = (*sP).k2[i] + jfac * (*sP).gridPolarizationFactor2[i] * (*sP).workspace2P[h];
	}
}

//Main kernel for RK4 propagation of the field
FGLOBAL void rkKernel(GKERN cudaParameterSet* sP, uint8_t stepNumber) {
	long long iC = threadIdx.x + blockIdx.x * blockDim.x;
	long long h = 1 + iC % ((*sP).Nfreq - 1); //frequency coordinate

	iC = h + (iC / ((*sP).Nfreq - 1)) * (*sP).Nfreq;
	if (h == 1) {
		(*sP).k1[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).k2[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).gridEFrequency1[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).gridEFrequency2[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).gridEFrequency1Next1[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).gridEFrequency1Next2[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).workspace1[iC - 1] = thrust::complex<double>(0., 0.);
		(*sP).workspace2[iC - 1] = thrust::complex<double>(0., 0.);
	}
	thrust::complex<double> estimate1, estimate2;

	if ((*sP).isCylindric) {
		(*sP).k1[iC] = (*sP).k1[iC] + (*sP).gridPropagationFactor1Rho1[iC] * (*sP).workspace1[iC];
		(*sP).k2[iC] = (*sP).k2[iC] + (*sP).gridPropagationFactor1Rho2[iC] * (*sP).workspace2[iC];
	}

	//generate the estimates and do the weighted sum to get the grid at the next step
	//with weights determined by the step number
	switch (stepNumber) {
	case 0:
		estimate1 = (*sP).gridEFrequency1[iC] + 0.5 * (*sP).k1[iC];
		estimate2 = (*sP).gridEFrequency2[iC] + 0.5 * (*sP).k2[iC];
		(*sP).gridEFrequency1Next1[iC] = SIXTH * (*sP).k1[iC] + (*sP).gridEFrequency1[iC];
		(*sP).gridEFrequency1Next2[iC] = SIXTH * (*sP).k2[iC] + (*sP).gridEFrequency2[iC];
		if ((*sP).isUsingMillersRule) {
			(*sP).workspace1[iC] = (*sP).chiLinear1[h] * estimate1;
			(*sP).workspace2[iC] = (*sP).chiLinear2[h] * estimate2;
		}
		else {
			(*sP).workspace1[iC] = estimate1;
			(*sP).workspace2[iC] = estimate2;
		}
		(*sP).k1[iC] = (*sP).gridPropagationFactor1[iC] * estimate1;
		(*sP).k2[iC] = (*sP).gridPropagationFactor2[iC] * estimate2;
		break;
	case 1:
		estimate1 = (*sP).gridEFrequency1[iC] + 0.5 * (*sP).k1[iC];
		estimate2 = (*sP).gridEFrequency2[iC] + 0.5 * (*sP).k2[iC];
		(*sP).gridEFrequency1Next1[iC] = (*sP).gridEFrequency1Next1[iC] + THIRD * (*sP).k1[iC];
		(*sP).gridEFrequency1Next2[iC] = (*sP).gridEFrequency1Next2[iC] + THIRD * (*sP).k2[iC];
		if ((*sP).isUsingMillersRule) {
			(*sP).workspace1[iC] = (*sP).chiLinear1[h] * estimate1;
			(*sP).workspace2[iC] = (*sP).chiLinear2[h] * estimate2;
		}
		else {
			(*sP).workspace1[iC] = estimate1;
			(*sP).workspace2[iC] = estimate2;
		}
		(*sP).k1[iC] = (*sP).gridPropagationFactor1[iC] * estimate1;
		(*sP).k2[iC] = (*sP).gridPropagationFactor2[iC] * estimate2;
		break;
	case 2:
		estimate1 = (*sP).gridEFrequency1[iC] + (*sP).k1[iC];
		estimate2 = (*sP).gridEFrequency2[iC] + (*sP).k2[iC];
		(*sP).gridEFrequency1Next1[iC] = (*sP).gridEFrequency1Next1[iC] + THIRD * (*sP).k1[iC];
		(*sP).gridEFrequency1Next2[iC] = (*sP).gridEFrequency1Next2[iC] + THIRD * (*sP).k2[iC];
		if ((*sP).isUsingMillersRule) {
			(*sP).workspace1[iC] = (*sP).chiLinear1[h] * estimate1;
			(*sP).workspace2[iC] = (*sP).chiLinear2[h] * estimate2;
		}
		else {
			(*sP).workspace1[iC] = estimate1;
			(*sP).workspace2[iC] = estimate2;
		}
		(*sP).k1[iC] = (*sP).gridPropagationFactor1[iC] * estimate1;
		(*sP).k2[iC] = (*sP).gridPropagationFactor2[iC] * estimate2;
		break;
	case 3:
		(*sP).gridEFrequency1[iC] = (*sP).gridEFrequency1Next1[iC] + SIXTH * (*sP).k1[iC];
		(*sP).gridEFrequency2[iC] = (*sP).gridEFrequency1Next2[iC] + SIXTH * (*sP).k2[iC];
		if ((*sP).isUsingMillersRule) {
			(*sP).workspace1[iC] = (*sP).chiLinear1[h] * (*sP).gridEFrequency1[iC];
			(*sP).workspace2[iC] = (*sP).chiLinear2[h] * (*sP).gridEFrequency2[iC];
		}
		else {
			(*sP).workspace1[iC] = (*sP).gridEFrequency1[iC];
			(*sP).workspace2[iC] = (*sP).gridEFrequency2[iC];
		}
		(*sP).k1[iC] = (*sP).gridPropagationFactor1[iC] * (*sP).gridEFrequency1[iC];
		(*sP).k2[iC] = (*sP).gridPropagationFactor2[iC] * (*sP).gridEFrequency2[iC];
		break;
	}
}

FGLOBAL void beamNormalizeKernel(GKERN cudaParameterSet* s, double* rawSum, double* pulse, double pulseEnergy) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	double normFactor = sqrt(pulseEnergy / ((*s).Ntime * (*rawSum)));
	pulse[i] *= normFactor;
}

FGLOBAL void addDoubleArraysKernel(GKERN double* A, double* B) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	A[i] += B[i];
}

FGLOBAL void beamGenerationKernel2D(GKERN thrust::complex<double>* pulse, double* pulseSum, cudaParameterSet* s, double frequency, double bandwidth,
	int sgOrder, double cep, double delay, double gdd, double tod,
	bool hasLoadedField, thrust::complex<double>* loadedField, double* materialPhase,
	double w0, double z0, double x0, double beamAngle,
	double polarizationAngle, double circularity,
	double* sellmeierCoefficients, double crystalTheta, double crystalPhi, int sellmeierType) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, h;
	h = 1 + i % ((*s).Nfreq - 1);
	j = i / ((*s).Nfreq - 1);
	i = h + j * ((*s).Nfreq);
	double f = h * (*s).fStep;
	double w = TWOPI * (f - frequency);

	//supergaussian pulse spectrum, if no input pulse specified
	thrust::complex<double> specfac = thrust::complex<double>(-pow((f - frequency) / bandwidth, sgOrder), 0);

	thrust::complex<double> specphase = thrust::complex<double>(0,
		-(cep
			+ TWOPI * f * (delay - 0.5 * (*s).dt * (*s).Ntime)
			+ 0.5 * gdd * w * w
			+ tod * w * w * w / 6.0
			+ materialPhase[h]));
	specfac = thrust::exp(specfac + specphase);

	if (hasLoadedField) {
		specfac = loadedField[h] * thrust::exp(specphase);
	}
	thrust::complex<double> ne, no;
	sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, sellmeierType, 0);


	double ko = TWOPI * no.real() * f / LIGHTC;
	double zR = PI * w0 * w0 * ne.real() * f / LIGHTC;
	if (f == 0) {
		zR = 1e3;
	}
	double rB = (x0 - (*s).dx * (j - (*s).Nspace / 2.0) - 0.25 * (*s).dx);
	double r = rB * cos(beamAngle) - z0 * sin(beamAngle);
	double z = rB * sin(beamAngle) + z0 * cos(beamAngle);

	double wz = w0 * sqrt(1 + (z * z / (zR * zR)));
	double Rz = z * (1. + (zR * zR / (z * z)));

	if (z == 0) {
		Rz = 1.0e15;
	}
	double phi = atan(z / zR);
	thrust::complex<double> Eb = (w0 / wz) * thrust::exp(thrust::complex<double>(0., 1.) * (ko * (z - z0) + ko * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
	Eb = Eb * specfac;
	if (isnan(cuCModSquared(Eb)) || f <= 0) {
		Eb = thrust::complex<double>(0., 0.);
	}

	pulse[i] = thrust::complex<double>(cos(polarizationAngle), -circularity * sin(polarizationAngle)) * Eb;
	pulse[i + (*s).NgridC] = thrust::complex<double>(sin(polarizationAngle), circularity * cos(polarizationAngle)) * Eb;
	double pointEnergy = abs(r) * (cuCModSquared(pulse[i]) + cuCModSquared(pulse[i + (*s).NgridC]));
	pointEnergy *= 2 * PI * LIGHTC * EPS0 * (*s).dx * (*s).dt;
	//two factors of two cancel here - there should be one for the missing frequency plane, but the sum is over x instead of r
	//accordingly we already multiplied by two
#ifdef __HIPCC__
	atomicAdd(pulseSum, pointEnergy);
#else
	*pulseSum += pointEnergy; //NOT THREAD SAFE, RUN CPU CODE ON SINGLE THREAD
#endif
}

FGLOBAL void beamGenerationKernel3D(GKERN thrust::complex<double>* pulse, double* pulseSum, cudaParameterSet* s, double frequency, double bandwidth,
	int sgOrder, double cep, double delay, double gdd, double tod,
	bool hasLoadedField, thrust::complex<double>* loadedField, double* materialPhase,
	double w0, double z0, double y0, double x0, double beamAngle, double beamAnglePhi,
	double polarizationAngle, double circularity,
	double* sellmeierCoefficients, double crystalTheta, double crystalPhi, int sellmeierType) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long j, k, h, col;
	h = 1 + i % ((*s).Nfreq - 1);
	col = i / ((*s).Nfreq - 1);
	i = h + col * ((*s).Nfreq);
	j = col % (*s).Nspace;
	k = col / (*s).Nspace;
	double f = h * (*s).fStep;
	double w = TWOPI * (f - frequency);

	//supergaussian pulse spectrum, if no input pulse specified
	thrust::complex<double> specfac = thrust::complex<double>(-pow((f - frequency) / bandwidth, sgOrder), 0);

	thrust::complex<double> specphase = thrust::complex<double>(0,
		-(cep
			+ TWOPI * f * (delay - 0.5 * (*s).dt * (*s).Ntime)
			+ 0.5 * gdd * w * w
			+ tod * w * w * w / 6.0
			+ materialPhase[h]));
	specfac = thrust::exp(specfac + specphase);

	if (hasLoadedField) {
		specfac = loadedField[h] * thrust::exp(specphase);
	}
	thrust::complex<double> ne, no;
	sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, sellmeierType, 0);


	double ko = TWOPI * no.real() * f / LIGHTC;
	double zR = PI * w0 * w0 * ne.real() * f / LIGHTC;
	if (f == 0) {
		zR = 1e3;
	}
	double xo = ((*s).dx * (j - (*s).Nspace / 2.0)) - x0;
	double yo = ((*s).dx * (k - (*s).Nspace2 / 2.0)) - y0;
	double zo = z0;
	double cB = cos(beamAngle);
	double cA = cos(beamAnglePhi);
	double sB = sin(beamAngle);
	double sA = sin(beamAnglePhi);
	double x = cB * xo + sA * sB * yo + sA * sB * zo;
	double y = cA * yo - sA * zo;
	double z = -sB * xo + sA * cB * yo + cA * cB * zo;
	double r = sqrt(x * x + y * y);

	double wz = w0 * sqrt(1 + (z * z / (zR * zR)));
	double Rz = 1.0e15;
	if (z != 0.0) {
		Rz = z * (1. + (zR * zR / (z * z)));
	}

	double phi = atan(z / zR);
	thrust::complex<double> Eb = (w0 / wz) * thrust::exp(thrust::complex<double>(0., 1.) * (ko * (z - z0) + ko * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
	Eb = Eb * specfac;
	if (isnan(cuCModSquared(Eb)) || f <= 0) {
		Eb = thrust::complex<double>(0., 0.);
	}

	pulse[i] = thrust::complex<double>(cos(polarizationAngle), -circularity * sin(polarizationAngle)) * Eb;
	pulse[i + (*s).NgridC] = thrust::complex<double>(sin(polarizationAngle), circularity * cos(polarizationAngle)) * Eb;
	double pointEnergy = (cuCModSquared(pulse[i]) + cuCModSquared(pulse[i + (*s).NgridC]));
	pointEnergy *= 2 * LIGHTC * EPS0 * (*s).dx * (*s).dx * (*s).dt;
	//factor 2 accounts for the missing negative frequency plane
#ifdef __HIPCC__
	atomicAdd(pulseSum, pointEnergy);
#else
	*pulseSum += pointEnergy; //NOT THREAD SAFE, RUN CPU CODE ON SINGLE THREAD
#endif
}

//Take absolute value of complex array
FGLOBAL void absKernel(GKERN double* absOut, thrust::complex<double>* complexIn) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	absOut[i] = thrust::abs(complexIn[i]);
}

//Apply fft normalization
//Take absolute value of complex array
FGLOBAL void fftNormalizeKernel(GKERN thrust::complex<double>* A, size_t fftSize) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	A[i] = A[i] / fftSize;
}


//Apply fft normalization
#ifdef __HIPCC__
__global__ void multiplyByConstantKernel(
#else
void multiplyByConstantKernel(uint3 blockIdx, uint3 threadIdx, uint3 blockDim,
#endif
	thrust::complex<double>* A, double val) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	A[i] = val * A[i];
}

FGLOBAL void multiplyByConstantKernelD(GKERN double* A, double val) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	A[i] = val * A[i];
}


//element-wise B*A = C;
FGLOBAL void multiplicationKernel(GKERN thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* C) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	C[i] = B[i] * A[i];
}

FGLOBAL void multiplicationKernelCompactVector(GKERN thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* C, cudaParameterSet* s) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	long long h = i % (*s).Nfreq; //temporal coordinate

	C[i] = A[h] * B[i];
}

FGLOBAL void multiplicationKernelCompact(GKERN thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* C) {
	long long i = threadIdx.x + blockIdx.x * blockDim.x;
	C[i] = A[i] * B[i];
}


namespace {
	simulationParameterSet* fittingSet;
	simulationParameterSet* fittingReferenceSet;
	int				runRK4Step(cudaParameterSet* sH, cudaParameterSet* sD, uint8_t stepNumber);
	int				preparePropagation2DCartesian(simulationParameterSet* s, cudaParameterSet sc);
	int				preparePropagation3DCylindric(simulationParameterSet* s, cudaParameterSet sc);
	int             preparePropagation3D(simulationParameterSet* s, cudaParameterSet sc);
	int             getTotalSpectrum(simulationParameterSet* sCPU, cudaParameterSet* sc);
	int				rotateField(simulationParameterSet* s, double rotationAngle);
	void            runFittingIteration(int* m, int* n, double* fittingValues, double* fittingFunction);
//	int             resolveSequence(int currentIndex, simulationParameterSet* s, crystalEntry* db);
	int				prepareElectricFieldArrays(simulationParameterSet* s, cudaParameterSet* sc);
	int             applyLinearPropagation(simulationParameterSet* s, int materialIndex, double thickness);
	int             fillRotationMatricies(simulationParameterSet* sCPU, cudaParameterSet* s);
	int             deallocateCudaParameterSet(cudaParameterSet* s);
	int             initializeCudaParameterSet(simulationParameterSet* sCPU, cudaParameterSet* s);
	//generate the rotation matricies for translating between the beam coordinates and
	//crystal coordinates

//My weird bilingual wrapper template that lets me either call CUDA kernels
//normally on the GPU, or process them on the CPU
//This is why kernel declarations have FGLOBAL in front of them
//instead of the usual __global__ tag
	template<typename Function, typename... Args>
	void flexLaunch(unsigned int Nblock, unsigned int Nthread, hipStream_t stream, Function func, Args... args) {
#ifdef __HIPCC__
		func <<<Nblock, Nthread, 0, stream >>> (args...);
#else
		uint3 tIdx;
		uint3 bIdx;
		uint3 bDim;
		bDim.x = Nthread;

#pragma omp parallel for private(tIdx,bIdx)
		for (int i = 0; i < (int)Nthread; i++) {
			tIdx.x = (unsigned int)i;
			for (unsigned int j = 0; j < Nblock; j++) {
				bIdx.x = j;
				func(bIdx, tIdx, bDim, args...);
			}
		}
#endif
	}

	int flexMemset(void* ptr, int value, size_t count) {
#ifdef __HIPCC__
		hipMemset(ptr, value, count);
#else
		memset(ptr, value, count);
#endif
		return 0;
	}
	int flexCalloc(void** ptr, size_t N, size_t elementSize) {
#ifdef __HIPCC__
		int err = hipMalloc(ptr, N * elementSize);
		flexMemset(*ptr, 0, N * elementSize);
		return err;
#else
		(*ptr) = calloc(N, elementSize);
		return (int)((*ptr) == NULL);
#endif
	}

	int flexFree(void* block) {
#ifdef __HIPCC__
		hipFree(block);
#else
		free(block);
#endif
		return 0;
	}

	int flexMemcpy(void* dst, void* src, size_t count, hipMemcpyKind kind) {
#ifdef __HIPCC__
		hipMemcpy(dst, src, count, kind);
#else
		memcpy(dst, src, count);
#endif
		return 0;
	}

	int combinedFFT(cudaParameterSet* s, void* input, void* output, int type) {
		type += RUNTYPE * 5;
		switch (type) {
		case 0:
			hipfftExecD2Z((*s).fftPlanD2Z, (hipfftDoubleReal*)input, (hipfftDoubleComplex*)output);
			break;
		case 1:
			hipfftExecZ2D((*s).fftPlanZ2D, (hipfftDoubleComplex*)input, (hipfftDoubleReal*)output);
			break;
		case 2:
			hipfftExecD2Z((*s).fftPlan1DD2Z, (hipfftDoubleReal*)input, (hipfftDoubleComplex*)output);
			break;
		case 3:
			hipfftExecZ2D((*s).fftPlan1DZ2D, (hipfftDoubleComplex*)input, (hipfftDoubleReal*)output);
			break;
		case 4:
			hipfftExecD2Z((*s).doublePolfftPlan, (hipfftDoubleReal*)input, (hipfftDoubleComplex*)output);
			break;
		case 5:
			DftiComputeForward((*s).mklPlanD2Z, input, output);
			break;
		case 6:
			DftiComputeBackward((*s).mklPlanZ2D, input, output);
			break;
		case 7:
			DftiComputeForward((*s).mklPlan1DD2Z, input, output);
			break;
		case 8:
			DftiComputeBackward((*s).mklPlan1DZ2D, input, output);
			break;
		case 9:
			DftiComputeForward((*s).mklPlanDoublePolfft, input, output);
			break;
		}

		return 0;
	}

	int prepareElectricFieldArrays(simulationParameterSet* s, cudaParameterSet* sc) {
		
		//run the beam generation single-threaded on CPU to avoid race condition
		unsigned int beamBlocks = (*sc).Nblock / 2;
		unsigned int beamThreads = (*sc).Nthread;
		if (RUNTYPE == 1) {
			beamBlocks = beamBlocks * beamThreads;
			beamThreads = 1;
		}
			
		cudaParameterSet* scDevice;
		flexCalloc((void**)&scDevice, 1, sizeof(cudaParameterSet));
		flexMemcpy(scDevice, sc, sizeof(cudaParameterSet), hipMemcpyHostToDevice);
		if ((*s).isFollowerInSequence && !(*s).isReinjecting) {
			flexMemcpy((*sc).gridETime1, (*s).ExtOut, 2 * (*s).Ngrid * sizeof(double), hipMemcpyHostToDevice);
			//hipfftExecD2Z((*sc).fftPlanD2Z, (*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridEFrequency1);
			combinedFFT(sc, (*sc).gridETime1, (*sc).gridEFrequency1, 0);
			//Copy the field into the temporary array
			flexMemcpy((*sc).gridEFrequency1Next1, (*sc).gridEFrequency1, 2 * (*sc).NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToDevice);

			if ((*sc).isUsingMillersRule) {
				//multiplicationKernelCompactVector << <(unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2*MIN_GRIDDIM, 0, (*sc).CUDAStream >> > ((*sc).chiLinear1, (*sc).gridEFrequency1Next1, (*sc).workspace1, scDevice);
				flexLaunch((unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2 * MIN_GRIDDIM, (*sc).CUDAStream, multiplicationKernelCompactVector, (*sc).chiLinear1, (*sc).gridEFrequency1Next1, (*sc).workspace1, scDevice);
			}
			else {
				flexMemcpy((*sc).workspace1, (*sc).gridEFrequency1Next1, 2 * sizeof(thrust::complex<double>) * (*sc).NgridC, hipMemcpyDeviceToDevice);
			}

			//multiplicationKernelCompact << <(unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2* MIN_GRIDDIM, 0, (*sc).CUDAStream >> > ((*sc).gridPropagationFactor1, (*sc).gridEFrequency1Next1, (*sc).k1);
			flexLaunch((unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2 * MIN_GRIDDIM, (*sc).CUDAStream, multiplicationKernelCompact, (*sc).gridPropagationFactor1, (*sc).gridEFrequency1Next1, (*sc).k1);
			flexMemcpy((*sc).gridEFrequency1Next1, (*sc).gridEFrequency1, 2 * (*sc).NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToDevice);
			flexFree(scDevice);
			return 0;
		}
		double* materialPhase1CUDA, * materialPhase2CUDA;
		thrust::complex<double>* loadedField1, * loadedField2;

		flexCalloc((void**)&loadedField1, (*sc).Ntime, sizeof(thrust::complex<double>));
		flexCalloc((void**)&loadedField2, (*sc).Ntime, sizeof(thrust::complex<double>));

		//get the material phase
		double* materialCoefficientsCUDA, * sellmeierPropagationMedium;
		//NOTE TO SELF: add second phase material


		if ((*s).field1IsAllocated) {
			flexMemcpy(loadedField1, (*s).loadedField1, (*s).Ntime * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
		}
		if ((*s).field2IsAllocated) {
			flexMemcpy(loadedField2, (*s).loadedField2, (*s).Ntime * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
		}
		flexCalloc((void**)&materialCoefficientsCUDA, 66, sizeof(double));
		flexCalloc((void**)&sellmeierPropagationMedium, 66, sizeof(double));
		flexCalloc((void**)&materialPhase1CUDA, (*s).Ntime, sizeof(double));
		flexCalloc((void**)&materialPhase2CUDA, (*s).Ntime, sizeof(double));
		flexMemcpy(materialCoefficientsCUDA, (*s).crystalDatabase[(*s).phaseMaterialIndex1].sellmeierCoefficients, 66 * sizeof(double), hipMemcpyHostToDevice);
		flexMemcpy(sellmeierPropagationMedium, (*s).crystalDatabase[(*s).materialIndex].sellmeierCoefficients, 66 * sizeof(double), hipMemcpyHostToDevice);
		//materialPhaseKernel<<<(unsigned int)(*s).Ntime, 1, 0, (*sc).CUDAStream>>> ((*s).fStep, (*s).Ntime, materialCoefficientsCUDA, (*s).frequency1, (*s).frequency2, (*s).phaseMaterialThickness1, (*s).phaseMaterialThickness2, materialPhase1CUDA, materialPhase2CUDA);
		flexLaunch((unsigned int)(*s).Ntime, 1, (*sc).CUDAStream, materialPhaseKernel, (*s).fStep, (*s).Ntime, materialCoefficientsCUDA, (*s).frequency1, (*s).frequency2, (*s).phaseMaterialThickness1, (*s).phaseMaterialThickness2, materialPhase1CUDA, materialPhase2CUDA);

		double* pulseSum = &materialCoefficientsCUDA[0];
		//calculate pulse 1 and store it in unused memory
		flexMemset(pulseSum, 0, sizeof(double));
		flexMemset((*sc).workspace1, 0, 2 * (*sc).NgridC * sizeof(thrust::complex<double>));
		if ((*sc).is3D) {
			//beamGenerationKernel3D << <(*sc).Nblock/2, (*sc).Nthread, 0, (*sc).CUDAStream >> > (
			//	(*sc).workspace1, pulseSum, scDevice, (*s).frequency1, (*s).bandwidth1,
			//	(*s).sgOrder1, (*s).cephase1, (*s).delay1, (*s).gdd1, (*s).tod1,
			//	(*s).field1IsAllocated, loadedField1, materialPhase1CUDA, (*s).beamwaist1,
			//	(*s).z01, (*s).y01, (*s).x01, (*s).propagationAngle1, (*s).propagationAnglePhi1, (*s).polarizationAngle1, (*s).circularity1,
			//	sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
			flexLaunch(beamBlocks, beamThreads, (*sc).CUDAStream, beamGenerationKernel3D,
				(*sc).workspace1, pulseSum, scDevice, (*s).frequency1, (*s).bandwidth1,
				(*s).sgOrder1, (*s).cephase1, (*s).delay1, (*s).gdd1, (*s).tod1,
				(*s).field1IsAllocated, loadedField1, materialPhase1CUDA, (*s).beamwaist1,
				(*s).z01, (*s).y01, (*s).x01, (*s).propagationAngle1, (*s).propagationAnglePhi1, (*s).polarizationAngle1, (*s).circularity1,
				sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
		}
		else {
			//beamGenerationKernel2D << <(*sc).Nblock/2, (*sc).Nthread, 0, (*sc).CUDAStream >> > (
			//	(*sc).workspace1, pulseSum, scDevice, (*s).frequency1, (*s).bandwidth1,
			//	(*s).sgOrder1, (*s).cephase1, (*s).delay1, (*s).gdd1, (*s).tod1,
			//	(*s).field1IsAllocated, loadedField1, materialPhase1CUDA, (*s).beamwaist1,
			//	(*s).z01, (*s).x01, (*s).propagationAngle1, (*s).polarizationAngle1, (*s).circularity1,
			//	sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
			flexLaunch(beamBlocks, beamThreads, (*sc).CUDAStream, beamGenerationKernel2D,
				(*sc).workspace1, pulseSum, scDevice, (*s).frequency1, (*s).bandwidth1,
				(*s).sgOrder1, (*s).cephase1, (*s).delay1, (*s).gdd1, (*s).tod1,
				(*s).field1IsAllocated, loadedField1, materialPhase1CUDA, (*s).beamwaist1,
				(*s).z01, (*s).x01, (*s).propagationAngle1, (*s).polarizationAngle1, (*s).circularity1,
				sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
		}
		

		//hipfftExecZ2D((*sc).fftPlan1DZ2D, (hipfftDoubleComplex*)(*sc).workspace1, (*sc).gridETime1);
		combinedFFT(sc, (*sc).workspace1, (*sc).gridETime1, 3);

		//std::complex<double>* TestWorkspace1 = (std::complex<double>*)calloc((*sc).NgridC * 2, sizeof(std::complex<double>));
		//double* TestField1 = (double*)calloc((*sc).Ngrid * 2, sizeof(double));
		//hipMemcpy(TestWorkspace1, (*sc).workspace1, (*sc).NgridC * 2 * 2 * sizeof(double),hipMemcpyDeviceToHost);
		//combinedFFT(sc, TestWorkspace1, TestField1, 3 + 5);
		//hipMemcpy((*sc).gridETime1, TestField1, 2 * (*sc).Ngrid * sizeof(double), hipMemcpyHostToDevice);
		//free(TestWorkspace1);
		//free(TestField1);


		//beamNormalizeKernel<<<2 * (*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream>>> (scDevice, pulseSum, (*sc).gridETime1, (*s).pulseEnergy1);
		flexLaunch(2 * (*sc).Nblock, (*sc).Nthread, (*sc).CUDAStream, beamNormalizeKernel, scDevice, pulseSum, (*sc).gridETime1, (*s).pulseEnergy1);
		flexMemcpy((*sc).gridEFrequency1Next1, (*sc).gridETime1, (*sc).Ngrid * 2 * sizeof(double), hipMemcpyDeviceToDevice);

		//calculate pulse 2
		flexMemset(pulseSum, 0, sizeof(double));
		flexMemset((*sc).workspace1, 0, 2 * (*sc).NgridC * sizeof(thrust::complex<double>));
		if ((*sc).is3D) {
			//beamGenerationKernel3D << <(*sc).Nblock/2, (*sc).Nthread, 0, (*sc).CUDAStream >> > (
			//	(*sc).workspace1, pulseSum, scDevice, (*s).frequency2, (*s).bandwidth2,
			//	(*s).sgOrder2, (*s).cephase2, (*s).delay2, (*s).gdd2, (*s).tod2,
			//	(*s).field2IsAllocated, loadedField2, materialPhase2CUDA, (*s).beamwaist2,
			//	(*s).z02, (*s).y02, (*s).x02, (*s).propagationAngle2, (*s).propagationAnglePhi2, (*s).polarizationAngle2, (*s).circularity2,
			//	sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
			flexLaunch(beamBlocks, beamThreads, (*sc).CUDAStream, beamGenerationKernel3D,
				(*sc).workspace1, pulseSum, scDevice, (*s).frequency2, (*s).bandwidth2,
				(*s).sgOrder2, (*s).cephase2, (*s).delay2, (*s).gdd2, (*s).tod2,
				(*s).field2IsAllocated, loadedField2, materialPhase2CUDA, (*s).beamwaist2,
				(*s).z02, (*s).y02, (*s).x02, (*s).propagationAngle2, (*s).propagationAnglePhi2, (*s).polarizationAngle2, (*s).circularity2,
				sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
		}
		else {
			/*beamGenerationKernel2D << <(*sc).Nblock/2, (*sc).Nthread, 0, (*sc).CUDAStream >> > (
				(*sc).workspace1, pulseSum, scDevice, (*s).frequency2, (*s).bandwidth2,
				(*s).sgOrder2, (*s).cephase2, (*s).delay2, (*s).gdd2, (*s).tod2,
				(*s).field2IsAllocated, loadedField2, materialPhase2CUDA, (*s).beamwaist2,
				(*s).z02, (*s).x02, (*s).propagationAngle2, (*s).polarizationAngle2, (*s).circularity2,
				sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);*/
			flexLaunch(beamBlocks, beamThreads, (*sc).CUDAStream, beamGenerationKernel2D,
				(*sc).workspace1, pulseSum, scDevice, (*s).frequency2, (*s).bandwidth2,
				(*s).sgOrder2, (*s).cephase2, (*s).delay2, (*s).gdd2, (*s).tod2,
				(*s).field2IsAllocated, loadedField2, materialPhase2CUDA, (*s).beamwaist2,
				(*s).z02, (*s).x02, (*s).propagationAngle2, (*s).polarizationAngle2, (*s).circularity2,
				sellmeierPropagationMedium, (*s).crystalTheta, (*s).crystalPhi, (*s).sellmeierType);
		}
		

		//hipfftExecZ2D(planBeamFreqToTime, (hipfftDoubleComplex*)(*sc).workspace1, (*sc).gridETime1);
		combinedFFT(sc, (*sc).workspace1, (*sc).gridETime1, 3);

		//beamNormalizeKernel<<<2 * (*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream>>> (scDevice, pulseSum, (*sc).gridETime1, (*s).pulseEnergy2);
		flexLaunch(2 * (*sc).Nblock, (*sc).Nthread, (*sc).CUDAStream, beamNormalizeKernel, scDevice, pulseSum, (*sc).gridETime1, (*s).pulseEnergy2);

		//add the pulses
		//addDoubleArraysKernel<<<2 * (*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream>>> ((*sc).gridETime1, (double*)(*sc).gridEFrequency1Next1);
		flexLaunch(2 * (*sc).Nblock, (*sc).Nthread, (*sc).CUDAStream, addDoubleArraysKernel, (*sc).gridETime1, (double*)(*sc).gridEFrequency1Next1);
		if ((*s).isReinjecting) {
			flexMemcpy((*sc).workspace1, (*s).ExtOut, 2 * (*s).Ngrid * sizeof(double), hipMemcpyHostToDevice);
			//addDoubleArraysKernel << <2 * (*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream >> > ((*sc).gridETime1, (double*)(*sc).workspace1);
			flexLaunch(2 * (*sc).Nblock, (*sc).Nthread, (*sc).CUDAStream, addDoubleArraysKernel, (*sc).gridETime1, (double*)(*sc).workspace1);
		}
		//fft onto frequency grid

		//hipfftExecD2Z((*sc).fftPlanD2Z, (*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridEFrequency1);
		combinedFFT(sc, (*sc).gridETime1, (*sc).gridEFrequency1, 0);



		//Copy the field into the temporary array
		flexMemcpy((*sc).gridEFrequency1Next1, (*sc).gridEFrequency1, 2 * (*sc).NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToDevice);

		if ((*sc).isUsingMillersRule && !(*sc).forceLinear) {
			//multiplicationKernelCompactVector<<<(unsigned int)((*sc).NgridC/ MIN_GRIDDIM), 2* MIN_GRIDDIM, 0, (*sc).CUDAStream>>> ((*sc).chiLinear1, (*sc).gridEFrequency1Next1, (*sc).workspace1, scDevice);
			flexLaunch((unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2 * MIN_GRIDDIM, (*sc).CUDAStream, multiplicationKernelCompactVector, (*sc).chiLinear1, (*sc).gridEFrequency1Next1, (*sc).workspace1, scDevice);
		}
		else {
			flexMemcpy((*sc).workspace1, (*sc).gridEFrequency1Next1, 2 * sizeof(thrust::complex<double>) * (*sc).NgridC, hipMemcpyDeviceToDevice);
		}

		//multiplicationKernelCompact<<<(unsigned int)((*sc).NgridC/ MIN_GRIDDIM), 2* MIN_GRIDDIM, 0, (*sc).CUDAStream>>> ((*sc).gridPropagationFactor1, (*sc).gridEFrequency1Next1, (*sc).k1);
		flexLaunch((unsigned int)((*sc).NgridC / MIN_GRIDDIM), 2 * MIN_GRIDDIM, (*sc).CUDAStream, multiplicationKernelCompact, (*sc).gridPropagationFactor1, (*sc).gridEFrequency1Next1, (*sc).k1);
		flexMemcpy((*sc).gridEFrequency1Next1, (*sc).gridEFrequency1, 2 * (*sc).NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToDevice);

		flexFree(materialPhase1CUDA);
		flexFree(materialPhase2CUDA);
		flexFree(materialCoefficientsCUDA);
		flexFree(sellmeierPropagationMedium);
		flexFree(loadedField1);
		flexFree(loadedField2);
		flexFree(scDevice);

		return 0;
	}
	int applyFresnelLoss(simulationParameterSet* s, int materialIndex1, int materialIndex2) {
		cudaParameterSet sc;
		initializeCudaParameterSet(s, &sc);
		double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
		memcpy(sellmeierCoefficientsAugmentedCPU, (*s).crystalDatabase[materialIndex1].sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		double* sellmeierCoefficients1;
		double* sellmeierCoefficients2;
		flexCalloc((void**)&sellmeierCoefficients1, 74, sizeof(double));
		flexCalloc((void**)&sellmeierCoefficients2, 74, sizeof(double));
		flexMemcpy(sellmeierCoefficients1, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		memcpy(sellmeierCoefficientsAugmentedCPU, (*s).crystalDatabase[materialIndex2].sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		flexMemcpy(sellmeierCoefficients2, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);

		hipDeviceSynchronize();

		flexMemcpy(sc.gridEFrequency1, (*s).EkwOut, 2 * (*s).NgridC * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

		//applyFresnelLossKernel<<<sc.Nblock, sc.Nthread, 0, sc.CUDAStream>>> (sellmeierCoefficients1, sellmeierCoefficients2, sc);

		//transform final result
		//fixnanKernel<<<(unsigned int)(2 * sc.NgridC/ MIN_GRIDDIM), 2* MIN_GRIDDIM, 0, sc.CUDAStream>>> (sc.gridEFrequency1);

		//hipfftExecZ2D(sc.fftPlanZ2D, (hipfftDoubleComplex*)sc.gridEFrequency1, sc.gridETime1);
		combinedFFT(&sc, sc.gridEFrequency1, sc.gridETime1, 1);
		//multiplyByConstantKernelD<<<2 * sc.Nblock, sc.Nthread, 0, sc.CUDAStream>>> (sc.gridETime1, 1.0 / sc.Ngrid);
		flexLaunch(2 * sc.Nblock, sc.Nthread, sc.CUDAStream, multiplyByConstantKernelD, sc.gridETime1, 1.0 / sc.Ngrid);
		//copy the field arrays from the GPU to CPU memory
		flexMemcpy((*s).ExtOut, sc.gridETime1, 2 * (*s).Ngrid * sizeof(double), hipMemcpyDeviceToHost);
		flexMemcpy((*s).EkwOut, sc.gridEFrequency1, 2 * (*s).Ngrid * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

		free(sellmeierCoefficientsAugmentedCPU);
		flexFree(sellmeierCoefficients1);
		flexFree(sellmeierCoefficients2);
		deallocateCudaParameterSet(&sc);
		return 0;
	}

	int applyAperature(simulationParameterSet* sCPU, double diameter, double activationParameter) {
		cudaParameterSet s;
		initializeCudaParameterSet(sCPU, &s);
		flexMemcpy(s.gridETime1, (*sCPU).ExtOut, 2 * s.Ngrid * sizeof(double), hipMemcpyHostToDevice);

		cudaParameterSet* sDevice;
		flexCalloc((void**)&sDevice, 1, sizeof(cudaParameterSet));
		flexMemcpy(sDevice, &s, sizeof(cudaParameterSet), hipMemcpyHostToDevice);
		//apertureKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream>>>(sDevice, 0.5 * diameter, activationParameter);
		flexLaunch(s.Nblock, s.Nthread, s.CUDAStream, apertureKernel, sDevice, 0.5 * diameter, activationParameter);
		//hipfftExecD2Z(s.fftPlanD2Z, s.gridETime1, (hipfftDoubleComplex*)s.gridEFrequency1);
		combinedFFT(&s, s.gridETime1, s.gridEFrequency1, 0);
		flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * s.Ngrid * sizeof(double), hipMemcpyDeviceToHost);
		flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * s.NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
		getTotalSpectrum(sCPU, &s);
		deallocateCudaParameterSet(&s);
		flexFree(sDevice);
		return 0;
	}

	int applySphericalMirror(simulationParameterSet* sCPU, double ROC) {
		cudaParameterSet s;
		initializeCudaParameterSet(sCPU, &s);

		cudaParameterSet* sDevice;
		flexCalloc((void**)&sDevice, 1, sizeof(cudaParameterSet));
		flexMemcpy(sDevice, &s, sizeof(cudaParameterSet), hipMemcpyHostToDevice);

		flexMemcpy(s.gridETime1, (*sCPU).ExtOut, 2 * s.Ngrid * sizeof(double), hipMemcpyHostToDevice);
		//hipfftExecD2Z(planBeamTimeToFreq, s.gridETime1, (hipfftDoubleComplex*)s.gridEFrequency1);
		combinedFFT(&s, s.gridETime1, s.gridEFrequency1, 2);
		//sphericalMirrorKernel << <s.Nblock/2, s.Nthread, 0, s.CUDAStream >> > (sDevice, ROC);
		flexLaunch(s.Nblock / 2, s.Nthread, s.CUDAStream, sphericalMirrorKernel, sDevice, ROC);
		//hipfftExecZ2D(planBeamFreqToTime, (hipfftDoubleComplex*)s.gridEFrequency1, s.gridETime1);
		combinedFFT(&s, s.gridEFrequency1, s.gridETime1, 3);
		//multiplyByConstantKernelD<<<2*s.Nblock ,s.Nthread, 0, s.CUDAStream>>>(s.gridETime1, 1.0 / s.Ntime);
		flexLaunch(2 * s.Nblock, s.Nthread, s.CUDAStream, multiplyByConstantKernelD, s.gridETime1, 1.0 / s.Ntime);
		//hipfftExecD2Z(s.fftPlanD2Z, s.gridETime1, (hipfftDoubleComplex*)s.gridEFrequency1);
		combinedFFT(&s, s.gridETime1, s.gridEFrequency1, 0);
		flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * s.Ngrid * sizeof(double), hipMemcpyDeviceToHost);
		flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * s.NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
		getTotalSpectrum(sCPU, &s);
		deallocateCudaParameterSet(&s);
		flexFree(sDevice);

		return 0;
	}

	int applyParabolicMirror(simulationParameterSet* sCPU, double focus) {
		cudaParameterSet s;
		initializeCudaParameterSet(sCPU, &s);

		cudaParameterSet* sDevice;
		flexCalloc((void**)&sDevice, 1, sizeof(cudaParameterSet));
		flexMemcpy(sDevice, &s, sizeof(cudaParameterSet), hipMemcpyHostToDevice);

		flexMemcpy(s.gridETime1, (*sCPU).ExtOut, 2 * s.Ngrid * sizeof(double), hipMemcpyHostToDevice);
		//hipfftExecD2Z(planBeamTimeToFreq, s.gridETime1, (hipfftDoubleComplex*)s.gridEFrequency1);
		combinedFFT(&s, s.gridETime1, s.gridEFrequency1, 2);
		//parabolicMirrorKernel << <s.Nblock / 2, s.Nthread, 0, s.CUDAStream >> > (sDevice, focus);
		flexLaunch(s.Nblock / 2, s.Nthread, s.CUDAStream, parabolicMirrorKernel, sDevice, focus);
		//hipfftExecZ2D(planBeamFreqToTime, (hipfftDoubleComplex*)s.gridEFrequency1, s.gridETime1);
		combinedFFT(&s, s.gridEFrequency1, s.gridETime1, 3);
		//multiplyByConstantKernelD << <2 * s.Nblock, s.Nthread, 0, s.CUDAStream >> > (s.gridETime1, 1.0 / s.Ntime);
		flexLaunch(2 * s.Nblock, s.Nthread, s.CUDAStream, multiplyByConstantKernelD, s.gridETime1, 1.0 / s.Ntime);
		//hipfftExecD2Z(s.fftPlanD2Z, s.gridETime1, (hipfftDoubleComplex*)s.gridEFrequency1);
		combinedFFT(&s, s.gridETime1, s.gridEFrequency1, 0);
		flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * s.Ngrid * sizeof(double), hipMemcpyDeviceToHost);
		flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * s.NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
		getTotalSpectrum(sCPU, &s);
		deallocateCudaParameterSet(&s);
		flexFree(sDevice);
		return 0;
	}

	int applyLinearPropagation(simulationParameterSet* sCPU, int materialIndex, double thickness) {
		cudaParameterSet s;
		initializeCudaParameterSet(sCPU, &s);


		flexMemcpy(s.gridEFrequency1, (*sCPU).EkwOut, s.NgridC * 2 * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);



		double* sellmeierCoefficients = (double*)s.gridEFrequency1Next1;
		//construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
		double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
		memcpy(sellmeierCoefficientsAugmentedCPU, (*sCPU).crystalDatabase[materialIndex].sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*sCPU).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*sCPU).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*sCPU).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*sCPU).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*sCPU).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*sCPU).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		flexMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);
		s.axesNumber = (*sCPU).crystalDatabase[materialIndex].axisType;
		s.sellmeierType = (*sCPU).crystalDatabase[materialIndex].sellmeierType;
		cudaParameterSet* sDevice;
		flexCalloc((void**)&sDevice, 1, sizeof(cudaParameterSet));
		flexMemcpy(sDevice, &s, sizeof(cudaParameterSet), hipMemcpyHostToDevice);



		//applyLinearPropagationKernel<<<s.Nblock/2, s.Nthread, 0, s.CUDAStream>>>(sellmeierCoefficients, thickness, sDevice);
		flexLaunch(s.Nblock / 2, s.Nthread, s.CUDAStream, applyLinearPropagationKernel, sellmeierCoefficients, thickness, sDevice);
		flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, s.NgridC * 2 * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
		//hipfftExecZ2D(s.fftPlanZ2D, (hipfftDoubleComplex*)s.gridEFrequency1, s.gridETime1);
		combinedFFT(&s, s.gridEFrequency1, s.gridETime1, 1);
		//multiplyByConstantKernelD<<<2*s.Nblock,s.Nthread,0,s.CUDAStream>>>(s.gridETime1, 1.0 / s.Ngrid);
		flexLaunch(2 * s.Nblock, s.Nthread, s.CUDAStream, multiplyByConstantKernelD, s.gridETime1, 1.0 / s.Ngrid);

		flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * s.Ngrid * sizeof(double), hipMemcpyDeviceToHost);

		deallocateCudaParameterSet(&s);
		flexFree(sDevice);
		return 0;
	}

	int preparePropagation2DCartesian(simulationParameterSet* s, cudaParameterSet sc) {
		//recycle allocated device memory for the grids needed
		double* sellmeierCoefficients = (double*)sc.gridEFrequency1Next1;

		double* referenceFrequencies;
		flexCalloc((void**)&referenceFrequencies, 7, sizeof(double));
		flexMemcpy(referenceFrequencies, (*s).crystalDatabase[(*s).materialIndex].nonlinearReferenceFrequencies, 7 * sizeof(double), hipMemcpyHostToDevice);

		//construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
		double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
		memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		flexMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);

		//prepare the propagation grids
		cudaParameterSet* sD;
		flexCalloc((void**)&sD, 1, sizeof(cudaParameterSet));
		flexMemcpy(sD, &sc, sizeof(cudaParameterSet), hipMemcpyHostToDevice);
		//getChiLinearKernel<<<(unsigned int)sc.Nfreq, 1, 0, sc.CUDAStream>>> (sD, sellmeierCoefficients);
		//prepareCartesianGridsKernel<<<sc.Nblock/2, sc.Nthread, 0, sc.CUDAStream>>> (sellmeierCoefficients, sD);
		//millersRuleNormalizationKernel<<<1, 1, 0, sc.CUDAStream>>> (sD, sellmeierCoefficients, referenceFrequencies);
		flexLaunch(sc.Nfreq, 1, sc.CUDAStream, getChiLinearKernel, sD, sellmeierCoefficients);
		flexLaunch(sc.Nblock / 2, sc.Nthread, sc.CUDAStream, prepareCartesianGridsKernel, sellmeierCoefficients, sD);
		flexLaunch(1, 1, sc.CUDAStream, millersRuleNormalizationKernel, sD, sellmeierCoefficients, referenceFrequencies);
		hipDeviceSynchronize();
		flexFree(sD);

		//clean up
		flexMemset(sc.gridEFrequency1Next1, 0, 2 * (*s).NgridC * sizeof(thrust::complex<double>));

		free(sellmeierCoefficientsAugmentedCPU);
		flexFree(referenceFrequencies);
		return 0;
	}



	int preparePropagation3D(simulationParameterSet* s, cudaParameterSet sc) {
		//recycle allocated device memory for the grids needed
		double* sellmeierCoefficients = (double*)sc.gridEFrequency1Next1;

		double* referenceFrequencies;
		flexCalloc((void**)&referenceFrequencies, 7, sizeof(double));
		flexMemcpy(referenceFrequencies, (*s).crystalDatabase[(*s).materialIndex].nonlinearReferenceFrequencies, 7 * sizeof(double), hipMemcpyHostToDevice);

		//construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
		double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
		memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		flexMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);

		//prepare the propagation grids
		cudaParameterSet* sD;
		flexCalloc((void**)&sD, 1, sizeof(cudaParameterSet));
		flexMemcpy(sD, &sc, sizeof(cudaParameterSet), hipMemcpyHostToDevice);

		flexLaunch(sc.Nfreq, 1, sc.CUDAStream, getChiLinearKernel, sD, sellmeierCoefficients);
		flexLaunch(sc.Nblock / 2, sc.Nthread, sc.CUDAStream, prepare3DGridsKernel, sellmeierCoefficients, sD);
		flexLaunch(1, 1, sc.CUDAStream, millersRuleNormalizationKernel, sD, sellmeierCoefficients, referenceFrequencies);
		flexFree(sD);

		//clean up
		flexMemset(sc.gridEFrequency1Next1, 0, 2 * (*s).NgridC * sizeof(thrust::complex<double>));

		free(sellmeierCoefficientsAugmentedCPU);
		flexFree(referenceFrequencies);
		return 0;
	}

	int preparePropagation3DCylindric(simulationParameterSet* s, cudaParameterSet sc) {
		//recycle allocated device memory for the grids needed
		double* sellmeierCoefficients = (double*)sc.gridEFrequency1Next1;
		double* referenceFrequencies;
		flexCalloc((void**)&referenceFrequencies, 7, sizeof(double));
		flexMemcpy(referenceFrequencies, (*s).crystalDatabase[(*s).materialIndex].nonlinearReferenceFrequencies, 7 * sizeof(double), hipMemcpyHostToDevice);

		//construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
		double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
		memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
		sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
		sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
		sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
		sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
		sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
		sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
		sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
		flexMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		//prepare the propagation grids
		cudaParameterSet* sD;
		flexCalloc((void**)&sD, 1, sizeof(cudaParameterSet));
		flexMemcpy(sD, &sc, sizeof(cudaParameterSet), hipMemcpyHostToDevice);
		//getChiLinearKernel<<< (unsigned int)sc.Nfreq, 1, 0, sc.CUDAStream>>> (sD, sellmeierCoefficients);
		//prepareCylindricGridsKernel<<<sc.Nblock/2, sc.Nthread, 0, sc.CUDAStream>>> (sellmeierCoefficients, sD);
		//millersRuleNormalizationKernel<<<1, 1, 0, sc.CUDAStream>>> (sD, sellmeierCoefficients, referenceFrequencies);
		flexLaunch(sc.Nfreq, 1, sc.CUDAStream, getChiLinearKernel, sD, sellmeierCoefficients);
		flexLaunch(sc.Nblock / 2, sc.Nthread, sc.CUDAStream, prepareCylindricGridsKernel, sellmeierCoefficients, sD);
		flexLaunch(1, 1, sc.CUDAStream, millersRuleNormalizationKernel, sD, sellmeierCoefficients, referenceFrequencies);

		//hipDeviceSynchronize();
		flexFree(sD);
		//hipDeviceSynchronize();

		//clean up
		flexMemset(sc.gridEFrequency1Next1, 0, 2 * (*s).NgridC * sizeof(thrust::complex<double>));
		flexFree(referenceFrequencies);
		free(sellmeierCoefficientsAugmentedCPU);
		return 0;
	}





	//Rotate the field on the GPU
	//Allocates memory and copies from CPU, then copies back to CPU and deallocates
	// - inefficient but the general principle is that only the CPU memory is preserved
	// after simulations finish... and this only runs at the end of the simulation
	int rotateField(simulationParameterSet* s, double rotationAngle) {
		cudaParameterSet sc;
		initializeCudaParameterSet(s, &sc);
		thrust::complex<double>* Ein1, * Eout1, * Ein2, * Eout2;
		Ein1 = sc.gridEFrequency1;
		Ein2 = sc.gridEFrequency2;
		Eout1 = sc.gridEFrequency1Next1;
		Eout2 = sc.gridEFrequency1Next2;

		//retrieve/rotate the field from the CPU memory
		flexMemcpy(Ein1, (*s).EkwOut, 2 * (*s).NgridC * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
		//rotateFieldKernel<<<(unsigned int)(sc.NgridC / MIN_GRIDDIM), MIN_GRIDDIM, 0, sc.CUDAStream>>> (Ein1, Ein2, Eout1, Eout2, rotationAngle);
		flexLaunch((unsigned int)(sc.NgridC / MIN_GRIDDIM), MIN_GRIDDIM, sc.CUDAStream, rotateFieldKernel, Ein1, Ein2, Eout1, Eout2, rotationAngle);
		flexMemcpy((*s).EkwOut, Eout1, 2 * (*s).NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

		//transform back to time
		//hipfftExecZ2D(sc.fftPlanZ2D, (hipfftDoubleComplex*)Eout1, sc.gridETime1);
		combinedFFT(&sc, Eout1, sc.gridETime1, 1);
		//multiplyByConstantKernelD<<<2 * sc.Nblock, sc.Nthread, 0, sc.CUDAStream>>> (sc.gridETime1, 1.0 / sc.Ngrid);
		flexLaunch(2 * sc.Nblock, sc.Nthread, sc.CUDAStream, multiplyByConstantKernelD, sc.gridETime1, 1.0 / sc.Ngrid);
		flexMemcpy((*s).ExtOut, sc.gridETime1, 2 * (*s).Ngrid * sizeof(double), hipMemcpyDeviceToHost);

		//update spectrum
		getTotalSpectrum(s, &sc);

		deallocateCudaParameterSet(&sc);
		return 0;
	}
	int fillRotationMatricies(simulationParameterSet* sCPU, cudaParameterSet* s) {
		double cosT = cos((*sCPU).crystalTheta);
		double sinT = sin((*sCPU).crystalTheta);
		double cosP = cos((*sCPU).crystalPhi);
		double sinP = sin((*sCPU).crystalPhi);
		double forward[9] =
		{ cosP, sinP, 0, -cosT * sinP, cosT * cosP, sinT, sinT * sinP, -sinT * cosP, cosT };

		//reverse direction (same array contents)
		sinT *= -1;
		sinP *= -1;
		double backward[9] =
		{ cosP, sinP, 0, -cosT * sinP, cosT * cosP, sinT, sinT * sinP, -sinT * cosP, cosT };

		memcpy((*s).rotationForward, forward, 9 * sizeof(double));
		memcpy((*s).rotationBackward, backward, 9 * sizeof(double));
		return 0;
	}

	int initializeCudaParameterSet(simulationParameterSet* sCPU, cudaParameterSet* s) {
		//initialize and take values from the struct handed over by the dispatcher
		hipStreamCreate(&(*s).CUDAStream);
		unsigned long long i;
		(*s).Ntime = (*sCPU).Ntime;
		(*s).Nspace = (*sCPU).Nspace;
		(*s).Nspace2 = (*sCPU).Nspace2;
		(*s).is3D = (*sCPU).is3D;
		(*s).Nfreq = ((*s).Ntime / 2 + 1);
		(*s).Ngrid = (*s).Ntime * (*s).Nspace * (*s).Nspace2;
		(*s).NgridC = (*s).Nfreq * (*s).Nspace * (*s).Nspace2; //size of the positive frequency side of the grid
		(*s).fftNorm = 1.0 / (*s).Ngrid;
		(*s).dt = (*sCPU).tStep;
		(*s).dx = (*sCPU).rStep;
		(*s).dk1 = TWOPI / ((*sCPU).Nspace * (*sCPU).rStep);
		(*s).dk2 = TWOPI / ((*sCPU).Nspace2 * (*sCPU).rStep);
		(*s).fStep = (*sCPU).fStep;
		(*s).Nsteps = (size_t)round((*sCPU).crystalThickness / (*sCPU).propagationStep);
		(*s).h = (*sCPU).crystalThickness / ((*s).Nsteps); //adjust step size so that thickness can be varied continuously by fitting
		(*s).axesNumber = (*sCPU).axesNumber;
		(*s).sellmeierType = (*sCPU).sellmeierType;
		(*s).f0 = (*sCPU).frequency1;
		(*s).Nthread = THREADS_PER_BLOCK;
		(*s).Nblock = (int)((*s).Ngrid / THREADS_PER_BLOCK);
		(*s).NblockC = (int)((*s).NgridC / THREADS_PER_BLOCK);
		(*s).isCylindric = (*sCPU).isCylindric;
		(*s).forceLinear = (*sCPU).forceLinear;
		(*s).isNonLinear = ((*sCPU).nonlinearSwitches[0] + (*sCPU).nonlinearSwitches[1]) > 0;
		(*s).isUsingMillersRule = ((*sCPU).crystalDatabase[(*sCPU).materialIndex].nonlinearReferenceFrequencies[0]) != 0;



		size_t beamExpansionFactor = 1;
		if ((*s).isCylindric) {
			beamExpansionFactor = 2;
		}
		fillRotationMatricies(sCPU, s);

		//GPU allocations
		//
		// currently 8 large grids, meaning memory use is approximately
		// 64 bytes per grid point (8 grids x 2 polarizations x 4ouble precision)
		// plus a little bit for additional constants/workspaces/etc
		int memErrors = 0;
		memErrors += flexCalloc((void**)&(*s).gridETime1, 2 * (*s).Ngrid, sizeof(double));
		memErrors += flexCalloc((void**)&(*s).gridPolarizationTime1, 2 * (*s).Ngrid, sizeof(double));
		memErrors += flexCalloc((void**)&(*s).workspace1, beamExpansionFactor * 2 * (*s).NgridC, sizeof(std::complex<double>));
		memErrors += flexCalloc((void**)&(*s).gridEFrequency1, 2 * (*s).NgridC, sizeof(std::complex<double>));
		memErrors += flexCalloc((void**)&(*s).gridPropagationFactor1, 2 * (*s).NgridC, sizeof(std::complex<double>));
		memErrors += flexCalloc((void**)&(*s).gridPolarizationFactor1, 2 * (*s).NgridC, sizeof(std::complex<double>));
		memErrors += flexCalloc((void**)&(*s).gridEFrequency1Next1, 2 * (*s).NgridC, sizeof(std::complex<double>));
		memErrors += flexCalloc((void**)&(*s).k1, 2 * (*s).NgridC, sizeof(std::complex<double>));

		//cylindric sym grids
		if ((*s).isCylindric) {
			memErrors += flexCalloc((void**)&(*s).gridPropagationFactor1Rho1, 4 * (*s).NgridC, sizeof(std::complex<double>));
			memErrors += flexCalloc((void**)&(*s).gridRadialLaplacian1, 4 * (*s).Ngrid, sizeof(std::complex<double>));
		}

		//smaller helper grids
		memErrors += flexCalloc((void**)&(*s).expGammaT, 2 * (*s).Ntime, sizeof(double));
		double* expGammaTCPU = (double*)malloc(2 * sizeof(double) * (*s).Ntime);
		memErrors += flexCalloc((void**)&(*s).chiLinear1, 2 * (*s).Nfreq, sizeof(std::complex<double>));
		for (i = 0; i < (*s).Ntime; i++) {
			expGammaTCPU[i] = exp((*s).dt * i * (*sCPU).drudeGamma);
			expGammaTCPU[i + (*s).Ntime] = exp(-(*s).dt * i * (*sCPU).drudeGamma);
		}
		flexMemcpy((*s).expGammaT, expGammaTCPU, 2 * sizeof(double) * (*s).Ntime, hipMemcpyHostToDevice);
		free(expGammaTCPU);

		memErrors += flexCalloc((void**)&(*s).chi3Tensor, 81, sizeof(double));

		(*sCPU).memoryError = memErrors;
		if (memErrors > 0) {
			return memErrors;
		}

		//second polarization grids are to pointers within the first polarization
		//to have contiguous memory
		(*s).gridETime2 = (*s).gridETime1 + (*s).Ngrid;
		(*s).workspace2 = (*s).workspace1 + (*s).NgridC;
		(*s).gridPolarizationTime2 = (*s).gridPolarizationTime1 + (*s).Ngrid;
		(*s).workspace2P = (*s).workspace1 + beamExpansionFactor * (*s).NgridC;
		(*s).k2 = (*s).k1 + (*s).NgridC;
		(*s).chiLinear2 = (*s).chiLinear1 + (*s).Nfreq;
		(*s).gridRadialLaplacian2 = (*s).gridRadialLaplacian1 + (*s).Ngrid;
		(*s).gridPropagationFactor1Rho2 = (*s).gridPropagationFactor1Rho1 + (*s).NgridC;
		(*s).gridPolarizationFactor2 = (*s).gridPolarizationFactor1 + (*s).NgridC;
		(*s).gridEFrequency1Next2 = (*s).gridEFrequency1Next1 + (*s).NgridC;
		(*s).gridPropagationFactor2 = (*s).gridPropagationFactor1 + (*s).NgridC;
		(*s).gridEFrequency2 = (*s).gridEFrequency1 + (*s).NgridC;


		//prepare effective nonlinearity tensors and put them on the GPU

		double firstDerivativeOperation[6] = { -1. / 60.,  3. / 20., -3. / 4.,  3. / 4.,  -3. / 20., 1. / 60. };
		for (i = 0; i < 6; i++) {
			firstDerivativeOperation[i] *= (-2.0 / ((*s).Ngrid * (*s).dx));
		}

		//set nonlinearSwitches[3] to the number of photons needed to overcome bandgap
		(*sCPU).nonlinearSwitches[3] = (int)ceil((*sCPU).bandGapElectronVolts * 241.79893e12 / (*sCPU).frequency1) - 2;
		double plasmaParametersCPU[6] = { 0 };

		if ((*sCPU).nonlinearAbsorptionStrength > 0.) {
			(*s).hasPlasma = TRUE;
			(*s).isNonLinear = TRUE;
		}
		else {
			(*s).hasPlasma = FALSE;
		}

		if ((*s).forceLinear) {
			(*s).hasPlasma = FALSE;
			(*s).isNonLinear = FALSE;
		}
		plasmaParametersCPU[0] = (*sCPU).nonlinearAbsorptionStrength; //nonlinear absorption strength parameter
		plasmaParametersCPU[1] = (*sCPU).drudeGamma; //gamma
		if ((*sCPU).nonlinearAbsorptionStrength > 0.) {
			plasmaParametersCPU[2] = (*sCPU).tStep * (*sCPU).tStep
				* 2.817832e-08 / (1.6022e-19 * (*sCPU).bandGapElectronVolts * (*sCPU).effectiveMass); // (dt^2)*e* e / (m * band gap));
		}
		else {
			plasmaParametersCPU[2] = 0;
		}

		calcEffectiveChi2Tensor((*sCPU).deffTensor, (*sCPU).chi2Tensor, (*sCPU).crystalTheta, (*sCPU).crystalPhi);
		memcpy((*s).chi2Tensor, (*sCPU).deffTensor, 9 * sizeof(double));
		memcpy((*s).nonlinearSwitches, (*sCPU).nonlinearSwitches, 4 * sizeof(int));

		flexMemcpy((*s).chi3Tensor, (*sCPU).chi3Tensor, 81 * sizeof(double), hipMemcpyHostToDevice);
		memcpy((*s).absorptionParameters, (*sCPU).absorptionParameters, 6 * sizeof(double));
		memcpy((*s).plasmaParameters, plasmaParametersCPU, 6 * sizeof(double));
		memcpy((*s).firstDerivativeOperation, firstDerivativeOperation, 6 * sizeof(double));


		//prepare FFT plans
		//explicitly make different plans for GPU or CPU (most other parts of the code can be universal,
		//but not this, since the libraries are different).
		if (RUNTYPE == 0) {
			size_t workSize;
			hipfftPlan1d(&(*s).fftPlan1DD2Z, (int)(*s).Ntime, HIPFFT_D2Z, 2 * (int)((*s).Nspace * (*s).Nspace2));
			hipfftPlan1d(&(*s).fftPlan1DZ2D, (int)(*s).Ntime, HIPFFT_Z2D, 2 * (int)((*s).Nspace * (*s).Nspace2));
			hipfftSetStream((*s).fftPlan1DD2Z, (*s).CUDAStream);
			hipfftSetStream((*s).fftPlan1DZ2D, (*s).CUDAStream);
			if ((*s).is3D) {
				int cufftSizes1[] = { (int)(*s).Nspace2, (int)(*s).Nspace, (int)(*s).Ntime };
				hipfftCreate(&(*s).fftPlanD2Z);
				hipfftGetSizeMany((*s).fftPlanD2Z, 3, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);
				hipfftMakePlanMany((*s).fftPlanD2Z, 3, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);

				hipfftCreate(&(*s).fftPlanZ2D);
				hipfftGetSizeMany((*s).fftPlanZ2D, 3, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_Z2D, 2, &workSize);
				hipfftMakePlanMany((*s).fftPlanZ2D, 3, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_Z2D, 2, &workSize);
			}
			else {
				int cufftSizes1[] = { (int)(*s).Nspace, (int)(*s).Ntime };

				hipfftCreate(&(*s).fftPlanD2Z);
				hipfftGetSizeMany((*s).fftPlanD2Z, 2, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);
				hipfftMakePlanMany((*s).fftPlanD2Z, 2, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);

				hipfftCreate(&(*s).fftPlanZ2D);
				hipfftGetSizeMany((*s).fftPlanZ2D, 2, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_Z2D, 2, &workSize);
				hipfftMakePlanMany((*s).fftPlanZ2D, 2, cufftSizes1, NULL, 0, 0, 0, 0, 0, HIPFFT_Z2D, 2, &workSize);

				if ((*s).isCylindric) {
					int cufftSizes2[] = { 2 * (int)(*s).Nspace, (int)(*s).Ntime };
					hipfftCreate(&(*s).doublePolfftPlan);
					hipfftGetSizeMany((*s).doublePolfftPlan, 2, cufftSizes2, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);
					hipfftMakePlanMany((*s).doublePolfftPlan, 2, cufftSizes2, NULL, 0, 0, 0, 0, 0, HIPFFT_D2Z, 2, &workSize);
					hipfftSetStream((*s).doublePolfftPlan, (*s).CUDAStream);
				}
			}
			hipfftSetStream((*s).fftPlanD2Z, (*s).CUDAStream);
			hipfftSetStream((*s).fftPlanZ2D, (*s).CUDAStream);
		}
		//else {
			DftiCreateDescriptor(&(*s).mklPlan1DD2Z, DFTI_DOUBLE, DFTI_REAL, 1, (*s).Ntime);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_NUMBER_OF_TRANSFORMS, (*s).Nspace * (*s).Nspace2);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_INPUT_DISTANCE, (*s).Ntime);
			DftiSetValue((*s).mklPlan1DD2Z, DFTI_OUTPUT_DISTANCE, (*s).Nfreq);
			DftiCommitDescriptor((*s).mklPlan1DD2Z);

			DftiCreateDescriptor(&(*s).mklPlan1DZ2D, DFTI_DOUBLE, DFTI_REAL, 1, (*s).Ntime);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_NUMBER_OF_TRANSFORMS, (*s).Nspace * (*s).Nspace2);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_INPUT_DISTANCE, (*s).Nfreq);
			DftiSetValue((*s).mklPlan1DZ2D, DFTI_OUTPUT_DISTANCE, (*s).Ntime);
			DftiCommitDescriptor((*s).mklPlan1DZ2D);

			if ((*s).is3D) {
				MKL_LONG mklSizes[] = { (MKL_LONG)(*s).Nspace, (MKL_LONG)(*s).Nspace2, (MKL_LONG)(*s).Ntime };
				MKL_LONG mklStrides[4] = { 0, (MKL_LONG)(*s).Ntime / 2 + 1, (MKL_LONG)(*s).Nspace, 1 };
				DftiCreateDescriptor(&(*s).mklPlanD2Z, DFTI_DOUBLE, DFTI_REAL, 3, mklSizes);
				DftiSetValue((*s).mklPlanD2Z, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
				DftiSetValue((*s).mklPlanD2Z, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
				DftiSetValue((*s).mklPlanD2Z, DFTI_OUTPUT_STRIDES, mklStrides);
				DftiSetValue((*s).mklPlanD2Z, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
				DftiSetValue((*s).mklPlanD2Z, DFTI_NUMBER_OF_TRANSFORMS, 2);
				DftiSetValue((*s).mklPlanD2Z, DFTI_INPUT_DISTANCE, (*s).Ngrid);
				DftiSetValue((*s).mklPlanD2Z, DFTI_OUTPUT_DISTANCE, (*s).NgridC);
				DftiCommitDescriptor((*s).mklPlanD2Z);

				DftiCreateDescriptor(&(*s).mklPlanZ2D, DFTI_DOUBLE, DFTI_REAL, 3, mklSizes);
				DftiSetValue((*s).mklPlanZ2D, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
				DftiSetValue((*s).mklPlanZ2D, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
				DftiSetValue((*s).mklPlanZ2D, DFTI_INPUT_STRIDES, mklStrides);
				DftiSetValue((*s).mklPlanZ2D, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
				DftiSetValue((*s).mklPlanZ2D, DFTI_NUMBER_OF_TRANSFORMS, 2);
				DftiSetValue((*s).mklPlanZ2D, DFTI_INPUT_DISTANCE, (*s).NgridC);
				DftiSetValue((*s).mklPlanZ2D, DFTI_OUTPUT_DISTANCE, (*s).Ngrid);
				DftiCommitDescriptor((*s).mklPlanZ2D);
			}
			else {
				MKL_LONG mklSizes[] = { (MKL_LONG)(*s).Nspace, (MKL_LONG)(*s).Ntime};
				MKL_LONG mklStrides[4] = { 0, (MKL_LONG)(*s).Ntime / 2 + 1, 1, 1 };
				
				DftiCreateDescriptor(&(*s).mklPlanD2Z, DFTI_DOUBLE, DFTI_REAL, 2, mklSizes);
				DftiSetValue((*s).mklPlanD2Z, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
				DftiSetValue((*s).mklPlanD2Z, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
				DftiSetValue((*s).mklPlanD2Z, DFTI_OUTPUT_STRIDES, mklStrides);
				DftiSetValue((*s).mklPlanD2Z, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
				DftiSetValue((*s).mklPlanD2Z, DFTI_NUMBER_OF_TRANSFORMS, 2);
				DftiSetValue((*s).mklPlanD2Z, DFTI_INPUT_DISTANCE, (*s).Ngrid);
				DftiSetValue((*s).mklPlanD2Z, DFTI_OUTPUT_DISTANCE, (*s).NgridC);
				DftiCommitDescriptor((*s).mklPlanD2Z);

				DftiCreateDescriptor(&(*s).mklPlanZ2D, DFTI_DOUBLE, DFTI_REAL, 2, mklSizes);
				DftiSetValue((*s).mklPlanZ2D, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
				DftiSetValue((*s).mklPlanZ2D, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
				DftiSetValue((*s).mklPlanZ2D, DFTI_INPUT_STRIDES, mklStrides);
				DftiSetValue((*s).mklPlanZ2D, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
				DftiSetValue((*s).mklPlanZ2D, DFTI_NUMBER_OF_TRANSFORMS, 2);
				DftiSetValue((*s).mklPlanZ2D, DFTI_INPUT_DISTANCE, (*s).NgridC);
				DftiSetValue((*s).mklPlanZ2D, DFTI_OUTPUT_DISTANCE, (*s).Ngrid);
				DftiCommitDescriptor((*s).mklPlanZ2D);

				if ((*s).isCylindric) {
					mklSizes[0] *= 2;
					DftiCreateDescriptor(&(*s).mklPlanDoublePolfft, DFTI_DOUBLE, DFTI_REAL, 2, mklSizes);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_CONJUGATE_EVEN_STORAGE, DFTI_COMPLEX_COMPLEX);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_OUTPUT_STRIDES, mklStrides);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_NUMBER_OF_USER_THREADS, THREADS_PER_BLOCK);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_NUMBER_OF_TRANSFORMS, 2);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_INPUT_DISTANCE, 2*(*s).Ngrid);
					DftiSetValue((*s).mklPlanDoublePolfft, DFTI_OUTPUT_DISTANCE, 2*(*s).NgridC);
					DftiCommitDescriptor((*s).mklPlanDoublePolfft);
				}

			}

		//}
		
		
		

		return 0;
	}

	int deallocateCudaParameterSet(cudaParameterSet* s) {
		flexFree((*s).gridETime1);
		flexFree((*s).workspace1);
		flexFree((*s).gridEFrequency1);
		flexFree((*s).gridPropagationFactor1);
		if ((*s).isCylindric) {
			flexFree((*s).gridPropagationFactor1Rho1);
			flexFree((*s).gridRadialLaplacian1);
		}
		flexFree((*s).gridPolarizationFactor1);
		flexFree((*s).gridEFrequency1Next1);
		flexFree((*s).k1);
		flexFree((*s).gridPolarizationTime1);
		flexFree((*s).chi3Tensor);
		flexFree((*s).expGammaT);
		flexFree((*s).chiLinear1);

		if (RUNTYPE == 0) {
			hipfftDestroy((*s).fftPlanD2Z);
			hipfftDestroy((*s).fftPlanZ2D);
			hipfftDestroy((*s).fftPlan1DD2Z);
			hipfftDestroy((*s).fftPlan1DZ2D);
			if ((*s).isCylindric) {
				hipfftDestroy((*s).doublePolfftPlan);
			}
			hipStreamDestroy((*s).CUDAStream);
		}
		else {
			DftiFreeDescriptor(&(*s).mklPlan1DD2Z);
			DftiFreeDescriptor(&(*s).mklPlanD2Z);
			DftiFreeDescriptor(&(*s).mklPlanZ2D);
			if((*s).isCylindric)DftiFreeDescriptor(&(*s).mklPlanDoublePolfft);
		}


		//flexFree(s);
		return 0;
	}

	//function to run a RK4 time step
//stepNumber is the sub-step index, from 0 to 3
	int runRK4Step(cudaParameterSet* sH, cudaParameterSet* sD, uint8_t stepNumber) {

		
		//operations involving FFT
		if ((*sH).isNonLinear || (*sH).isCylindric) {
			//perform inverse FFT to get time-space electric field
			//hipfftExecZ2D((*sH).fftPlanZ2D, (hipfftDoubleComplex*)(*sH).workspace1, (*sH).gridETime1);
			combinedFFT(sH, (hipfftDoubleComplex*)(*sH).workspace1, (*sH).gridETime1, 1);
			if ((*sH).isNonLinear) {
				//nonlinearPolarizationKernel << <(*sH).Nblock, (*sH).Nthread, 0, (*sH).CUDAStream >> > (sD);
				flexLaunch((*sH).Nblock, (*sH).Nthread, (*sH).CUDAStream, nonlinearPolarizationKernel, sD);
				if ((*sH).isCylindric) {
					//expandCylindricalBeam << < (*sH).Nblock, (*sH).Nthread, 0, (*sH).CUDAStream >> >
					//	(sD, (*sH).gridPolarizationTime1, (*sH).gridPolarizationTime2);
					flexLaunch((*sH).Nblock, (*sH).Nthread, (*sH).CUDAStream, expandCylindricalBeam, sD, (*sH).gridPolarizationTime1, (*sH).gridPolarizationTime2);
					//hipfftExecD2Z((*sH).doublePolfftPlan, (double*)(*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1);
					combinedFFT(sH, (*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1, 4);
				}
				else {
					//hipfftExecD2Z((*sH).fftPlanD2Z, (*sH).gridPolarizationTime1, (hipfftDoubleComplex*)(*sH).workspace1);
					combinedFFT(sH, (*sH).gridPolarizationTime1, (hipfftDoubleComplex*)(*sH).workspace1, 0);
				}
				//updateKwithPolarizationKernel << <(*sH).Nblock / 2, (*sH).Nthread, 0, (*sH).CUDAStream >> > (sD);
				flexLaunch((*sH).Nblock / 2, (*sH).Nthread, (*sH).CUDAStream, updateKwithPolarizationKernel, sD);
			}

			if ((*sH).hasPlasma) {
				//plasmaCurrentKernel << <(unsigned int)(((*sH).Nspace2 * (*sH).Nspace) / MIN_GRIDDIM), MIN_GRIDDIM, 0, (*sH).CUDAStream >> > (sD);
				flexLaunch((unsigned int)(((*sH).Nspace2 * (*sH).Nspace) / MIN_GRIDDIM), MIN_GRIDDIM, (*sH).CUDAStream, plasmaCurrentKernel, sD);
				if ((*sH).isCylindric) {
					//expandCylindricalBeam << < (*sH).Nblock, (*sH).Nthread, 0, (*sH).CUDAStream >> >
					//	(sD, (*sH).gridPolarizationTime1, (*sH).gridPolarizationTime2);
					flexLaunch((*sH).Nblock, (*sH).Nthread, (*sH).CUDAStream, expandCylindricalBeam, sD, (*sH).gridPolarizationTime1, (*sH).gridPolarizationTime2);
					//hipfftExecD2Z((*sH).doublePolfftPlan, (double*)(*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1);
					combinedFFT(sH, (*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1, 4);
				}
				else {
					//hipfftExecD2Z((*sH).fftPlanD2Z, (*sH).gridPolarizationTime1, (hipfftDoubleComplex*)(*sH).workspace1);
					combinedFFT(sH, (*sH).gridPolarizationTime1, (hipfftDoubleComplex*)(*sH).workspace1, 0);
				}
				//updateKwithPlasmaKernel << <(*sH).Nblock / 2, (*sH).Nthread, 0, (*sH).CUDAStream >> > (sD);
				flexLaunch((*sH).Nblock / 2, (*sH).Nthread, (*sH).CUDAStream, updateKwithPlasmaKernel, sD);
			}

			if ((*sH).isCylindric) {
				//radialLaplacianKernel << <(*sH).Nblock, (*sH).Nthread, 0, (*sH).CUDAStream >> > (sD);
				flexLaunch((*sH).Nblock, (*sH).Nthread, (*sH).CUDAStream, radialLaplacianKernel, sD);
				//hipfftExecD2Z((*sH).fftPlanD2Z, (*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1);
				combinedFFT(sH, (*sH).gridRadialLaplacian1, (hipfftDoubleComplex*)(*sH).workspace1, 0);
			}
		}

		//advance an RK4 step
		flexLaunch((*sH).Nblock / 2, (*sH).Nthread, (*sH).CUDAStream, rkKernel, sD, stepNumber);
		return 0;
	}

	void runFittingIteration(int* m, int* n, double* fittingValues, double* fittingFunction) {
		int i;
		int fitLocation;
		double referenceValue;
		//pointers to values that can be scanned in batch mode
		double* targets[36] = { 0,
			&(*fittingSet).pulseEnergy1, &(*fittingSet).pulseEnergy2, &(*fittingSet).frequency1, &(*fittingSet).frequency2,
			&(*fittingSet).bandwidth1, &(*fittingSet).bandwidth2, &(*fittingSet).cephase1, &(*fittingSet).cephase2,
			&(*fittingSet).delay1, &(*fittingSet).delay2, &(*fittingSet).gdd1, &(*fittingSet).gdd2,
			&(*fittingSet).tod1, &(*fittingSet).tod2, &(*fittingSet).phaseMaterialThickness1, &(*fittingSet).phaseMaterialThickness2,
			&(*fittingSet).beamwaist1, &(*fittingSet).beamwaist2,
			&(*fittingSet).x01, &(*fittingSet).x02, &(*fittingSet).z01, &(*fittingSet).z02,
			&(*fittingSet).propagationAngle1, &(*fittingSet).propagationAngle2, &(*fittingSet).polarizationAngle1, &(*fittingSet).polarizationAngle2,
			&(*fittingSet).circularity1, &(*fittingSet).circularity2, &(*fittingSet).crystalTheta, &(*fittingSet).crystalPhi,
			&(*fittingSet).nonlinearAbsorptionStrength, &(*fittingSet).drudeGamma, &(*fittingSet).effectiveMass, &(*fittingSet).crystalThickness,
			&(*fittingSet).propagationStep };

		double* references[36] = { 0,
		&(*fittingReferenceSet).pulseEnergy1, &(*fittingReferenceSet).pulseEnergy2, &(*fittingReferenceSet).frequency1, &(*fittingReferenceSet).frequency2,
		&(*fittingReferenceSet).bandwidth1, &(*fittingReferenceSet).bandwidth2, &(*fittingReferenceSet).cephase1, &(*fittingReferenceSet).cephase2,
		&(*fittingReferenceSet).delay1, &(*fittingReferenceSet).delay2, &(*fittingReferenceSet).gdd1, &(*fittingReferenceSet).gdd2,
		&(*fittingReferenceSet).tod1, &(*fittingReferenceSet).tod2, &(*fittingReferenceSet).phaseMaterialThickness1, &(*fittingReferenceSet).phaseMaterialThickness2,
		&(*fittingReferenceSet).beamwaist1, &(*fittingReferenceSet).beamwaist2,
		&(*fittingReferenceSet).x01, &(*fittingReferenceSet).x02, &(*fittingReferenceSet).z01, &(*fittingReferenceSet).z02,
		&(*fittingReferenceSet).propagationAngle1, &(*fittingReferenceSet).propagationAngle2, &(*fittingReferenceSet).polarizationAngle1, &(*fittingReferenceSet).polarizationAngle2,
		&(*fittingReferenceSet).circularity1, &(*fittingReferenceSet).circularity2, &(*fittingReferenceSet).crystalTheta, &(*fittingReferenceSet).crystalPhi,
		&(*fittingReferenceSet).nonlinearAbsorptionStrength, &(*fittingReferenceSet).drudeGamma, &(*fittingReferenceSet).effectiveMass, &(*fittingReferenceSet).crystalThickness,
		&(*fittingReferenceSet).propagationStep };


		for (i = 0; i < *n; i++) {
			fitLocation = (int)round((*fittingSet).fittingArray[3 * i]);
			referenceValue = *references[fitLocation];
			if (referenceValue == 0.0) {
				referenceValue = 1.;
			}
			*targets[fitLocation] = fittingValues[i] * referenceValue;
		}
		if ((*fittingSet).isInSequence) {
			solveNonlinearWaveEquationSequence(fittingSet);
			(*fittingSet).isFollowerInSequence = FALSE;
		}
		else {
			solveNonlinearWaveEquation(fittingSet);
		}


		//mode 0: maximize total spectrum in ROI
		if ((*fittingSet).fittingMode == 0) {
			for (i = 0; i < *m; i++) {
				fittingFunction[i] = (1.0e8 / ((*fittingSet).totalSpectrum[2 * (*fittingSet).Nfreq + (*fittingSet).fittingROIstart + i]));
			}
		}
		//mode 1: maximize s-polarized spectrum in ROI
		if ((*fittingSet).fittingMode == 1) {
			for (i = 0; i < *m; i++) {
				fittingFunction[i] = (1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).fittingROIstart + i]));
			}
		}
		//mode 2: maximize p-polarized spectrum in ROI
		if ((*fittingSet).fittingMode == 2) {
			for (i = 0; i < *m; i++) {
				fittingFunction[i] = (1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).Nfreq + (*fittingSet).fittingROIstart + i]));
			}
		}
		//mode 3: match total spectrum to reference given in ascii file
		if ((*fittingSet).fittingMode == 3) {
			double maxSim = 0;
			double maxRef = 0;
			double sumSim = 0;
			double sumRef = 0;
			double* simSpec = &(*fittingSet).totalSpectrum[2 * (*fittingSet).Nfreq + (*fittingSet).fittingROIstart];
			double* refSpec = &(*fittingSet).fittingReference[(*fittingSet).fittingROIstart];
			for (i = 0; i < *m; i++) {
				maxSim = max(maxSim, simSpec[i]);
				maxRef = max(maxRef, refSpec[i]);
				sumSim += simSpec[i];
				sumRef += refSpec[i];
			}

			if (maxSim == 0) {
				maxSim = 1;
			}
			if (maxRef == 0) {
				maxRef = 1;
			}

			double sumFF = 0;
			for (i = 0; i < *m; i++) {
				fittingFunction[i] = log10(1e5 * refSpec[i] / maxRef) - log10(1e5 * simSpec[i] / maxSim);
				sumFF += fittingFunction[i];
				//fittingFunction[i] = 1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).Ntime + (*fittingSet).fittingROIstart + i]);
			}
			sumFF /= *m;
			for (i = 0; i < *m; i++) {
				fittingFunction[i] -= sumFF;
			}
		}


		return;
	}

	int getTotalSpectrum(simulationParameterSet* sCPU, cudaParameterSet* sc) {

		flexMemset((*sc).workspace1, 0, 2 * (*sc).NgridC * sizeof(thrust::complex<double>));
		//hipfftExecD2Z(plan1, (*sc).gridETime1, (hipfftDoubleComplex*)(*sc).workspace1);
		combinedFFT(sc, (*sc).gridETime1, (*sc).workspace1, 2);
		if ((*sc).is3D) {
			//totalSpectrum3DKernel << <(unsigned int)(*sCPU).Nfreq, 1, 0, (*sc).CUDAStream >> > ((*sc).workspace1, (*sc).workspace2, (*sCPU).rStep, (*sCPU).Ntime / 2 + 1, (*sCPU).Nspace * (*sCPU).Nspace2, (*sc).gridPolarizationTime1);
			flexLaunch((unsigned int)(*sCPU).Nfreq, 1, (*sc).CUDAStream, totalSpectrum3DKernel, (*sc).workspace1, (*sc).workspace2, (*sCPU).rStep, (*sCPU).Ntime / 2 + 1, (*sCPU).Nspace * (*sCPU).Nspace2, (*sc).gridPolarizationTime1);
		}
		else {
			//totalSpectrumKernel << <(unsigned int)(*sCPU).Nfreq, 1, 0, (*sc).CUDAStream >> > ((*sc).workspace1, (*sc).workspace2, (*sCPU).rStep, (*sCPU).Ntime / 2 + 1, (*sCPU).Nspace, (*sc).gridPolarizationTime1);
			flexLaunch((*sCPU).Nfreq, 1, (*sc).CUDAStream, totalSpectrumKernel, (*sc).workspace1, (*sc).workspace2, (*sCPU).rStep, (*sCPU).Ntime / 2 + 1, (*sCPU).Nspace, (*sc).gridPolarizationTime1);
		}

		hipDeviceSynchronize();
		flexMemcpy((*sCPU).totalSpectrum, (*sc).gridPolarizationTime1, 3 * (*sCPU).Nfreq * sizeof(double), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		return 0;
	}


}
//END OF NAMESPACE

#ifdef __HIPCC__
unsigned long runFitting(simulationParameterSet* sCPU) {
#else
unsigned long runFittingCPU(simulationParameterSet * sCPU) {
#endif
	int n = (int)(*sCPU).Nfitting;
	int m = (int)(*sCPU).fittingROIsize;
	fittingReferenceSet = sCPU;
	fittingSet = (simulationParameterSet*)malloc((*sCPU).Nsims * sizeof(simulationParameterSet));
	memcpy(fittingSet, sCPU, (*sCPU).Nsims * sizeof(simulationParameterSet));

	double commonPrecision = (*sCPU).fittingPrecision;
	const double eps[6] = { commonPrecision,commonPrecision,commonPrecision,commonPrecision,commonPrecision,commonPrecision }; /* set precisions for stop-criteria */
	double jacobianPrecision = commonPrecision;
	double* x = (double*)mkl_malloc(sizeof(double) * n, 64);
	double* fittingValues = (double*)mkl_malloc(sizeof(double) * m, 64);
	double* fjac = (double*)mkl_malloc(sizeof(double) * m * n, 64);
	double* lowerBounds = (double*)mkl_malloc(sizeof(double) * n, 64);
	double* upperBounds = (double*)mkl_malloc(sizeof(double) * n, 64);
	const int maxIterations = max((*sCPU).fittingMaxIterations, 2);
	const int maxTrialIterations = max(maxIterations / 10, 2);
	/* initial step bound */
	double rs = 0.0;
	int RCI_Request;
	int successful;

	int iter;
	int stopCriterion;
	double inputResiduals = 0.0, outputResiduals = 0.0;
	_TRNSPBC_HANDLE_t handle;
	int i;
	int error = 0;

	//initial guess and bounds
	for (i = 0; i < n; i++) {
		x[i] = 1.;
		upperBounds[i] = (*fittingSet).fittingArray[3 * i + 2];
		lowerBounds[i] = (*fittingSet).fittingArray[3 * i + 1];
	}

	//initialize fitting function and jacobian
	for (i = 0; i < m; i++) {
		fittingValues[i] = 0.0;
	}
	for (i = 0; i < m * n; i++) {
		fjac[i] = 0.0;
	}

	error += dtrnlspbc_init(&handle, &n, &m, x, lowerBounds, upperBounds, eps, &maxIterations, &maxTrialIterations, &rs) != TR_SUCCESS;
	size_t currentIteration = 0;
	if (error == 0) {
		RCI_Request = 0;
		successful = 0;
		while (successful == 0 && (*sCPU).imdone[0] != 2 && currentIteration < maxIterations)
		{
			currentIteration++;
			if (dtrnlspbc_solve(&handle, fittingValues, fjac, &RCI_Request) != TR_SUCCESS)
			{
				successful = -1;
			}

			//check convergence
			if (RCI_Request > -7 && RCI_Request < -1) successful = 1;

			//recalculate
			if (RCI_Request == 1)
			{
				runFittingIteration(&m, &n, x, fittingValues);
			}

			//make jacobian
			if (RCI_Request == 2)
			{
				djacobi(runFittingIteration, &n, &m, fjac, x, &jacobianPrecision);
			}
		}
	}


	dtrnlspbc_get(&handle, &iter, &stopCriterion, &inputResiduals, &outputResiduals);
	memcpy(sCPU, fittingSet, (*fittingSet).Nsims * sizeof(simulationParameterSet));

	solveNonlinearWaveEquation(sCPU);

	//free memory
	dtrnlspbc_delete(&handle);
	mkl_free(upperBounds);
	mkl_free(lowerBounds);
	mkl_free(fjac);
	mkl_free(fittingValues);
	mkl_free(x);
	MKL_Free_Buffers();
	free(fittingSet);
	return 0;
}

#ifdef __HIPCC__
unsigned long solveNonlinearWaveEquation(void* lpParam) {
	simulationParameterSet* sCPU = (simulationParameterSet*)lpParam;
	hipSetDevice((*sCPU).assignedGPU);
#else
unsigned long solveNonlinearWaveEquationCPU(void* lpParam) {
	simulationParameterSet* sCPU = (simulationParameterSet*)lpParam;
#endif
	size_t i;
	cudaParameterSet* sDevice;
	cudaParameterSet s;
	memset(&s, 0, sizeof(cudaParameterSet));
	if(initializeCudaParameterSet(sCPU, &s)) return 1;

	//prepare the propagation arrays
	if (s.is3D) {
		preparePropagation3D(sCPU, s);
	}
	else if (s.isCylindric) {
		preparePropagation3DCylindric(sCPU, s);
	}
	else {
		preparePropagation2DCartesian(sCPU, s);
	}
	prepareElectricFieldArrays(sCPU, &s);
	double canaryPixel = 0;
	double* canaryPointer = &s.gridETime1[s.Ntime / 2 + s.Ntime * (s.Nspace / 2 + s.Nspace * (s.Nspace2 / 2))];

	flexCalloc((void**)&sDevice, 1, sizeof(cudaParameterSet));
	flexMemcpy(sDevice, &s, sizeof(cudaParameterSet), hipMemcpyHostToDevice);

	//Core propagation loop
	for (i = 0; i < s.Nsteps; i++) {

		//RK4
		runRK4Step(&s, sDevice, 0);
		runRK4Step(&s, sDevice, 1);
		runRK4Step(&s, sDevice, 2);
		runRK4Step(&s, sDevice, 3);
#ifdef __HIPCC__
		hipMemcpyAsync(&canaryPixel, canaryPointer, sizeof(double), hipMemcpyDeviceToHost);
#else
		//canaryPixel = *canaryPointer;
#endif
		if (isnan(canaryPixel)) {
			break;
		}

		if ((*sCPU).imdone[0] == 2) {
			break;
		}

		if ((*sCPU).imdone[0] == 3) {
			//copy the field arrays from the GPU to CPU memory if requested by the UI
			flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * (*sCPU).Ngrid * sizeof(double), hipMemcpyDeviceToHost);
			flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * (*sCPU).Ngrid * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

			(*sCPU).imdone[0] = 0;
		}
		(*(*sCPU).progressCounter)++;
	}

	////give the result to the CPU
	flexMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * s.NgridC * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
	

	//hipfftExecZ2D(s.fftPlanZ2D, (hipfftDoubleComplex*)s.gridEFrequency1, s.gridETime1);
	combinedFFT(&s, s.gridEFrequency1, s.gridETime1, 1);

	flexLaunch((int)(s.Ngrid / MIN_GRIDDIM), 2 * MIN_GRIDDIM, s.CUDAStream, multiplyByConstantKernelD, s.gridETime1, 1.0 / s.Ngrid);
	//multiplyByConstantKernelD<<<(int)(s.Ngrid / MIN_GRIDDIM), 2* MIN_GRIDDIM, 0, s.CUDAStream>>> (s.gridETime1, 1.0 / s.Ngrid);
	flexMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * (*sCPU).Ngrid * sizeof(double), hipMemcpyDeviceToHost);

	getTotalSpectrum(sCPU, &s);

	deallocateCudaParameterSet(&s);
	flexFree(sDevice);
	(*sCPU).imdone[0] = 1;
	return isnan(canaryPixel);
}

#ifdef __HIPCC__
int resolveSequence(int currentIndex, simulationParameterSet* s, crystalEntry* db) {
#else
int resolveSequenceCPU(int currentIndex, simulationParameterSet * s, crystalEntry * db) {
#endif

	double* offsetArray = &(*s).sequenceArray[11 * currentIndex];
	int error = 0;
	//sequence format
	//0: step type
	int stepType = (int)offsetArray[0];
	int materialIndex = 0;
	double thickness = 0;
	// 
	// if stepType == 0, normal propagation
	//1: material index
	//2: theta,
	//3: phi, 
	//4: NL absorption
	//5: Band gap
	//6: Drude relaxation
	//7: Effective mass
	//8: Crystal thickness
	//9: Propagation step size
	//10: rotation angle
	//
	// if stepType == 1, linear propagation
	// same parameters as 0, but only 1,2,3,8, and 10 matter
	//
	// if stepType == 2, fresnel loss
	// 1: incidence material index
	// 2: transmission material index
	// other parameters don't matter
	// 
	// if stepType == 3, spherical mirror
	// 1: ROC (m)
	//
	// if stepType == 4, parabolic mirror
	// 1: focus (m)
	// 
	// if stepType == 5, aperture
	// 1: diameter (m)
	// 2: activation parameter p (function is 1 - 1/(1 + exp(-p*(r-radius)))
	//
	// if stepType == 6, loop back to start (handled by solveNonlinearWaveEquationSequence())
	// 1: counter (counts down to zero)
	//
	// if stepType == 7, reinjection, same as 0, but input fields are added to current fields.

	switch (stepType) {
	case 7:
		(*s).isReinjecting = TRUE;
	case 0:
		if ((int)offsetArray[1] != -1) (*s).materialIndex = (int)offsetArray[1];
		if ((int)offsetArray[2] != -1) (*s).crystalTheta = DEG2RAD * offsetArray[2];
		if ((int)offsetArray[3] != -1) (*s).crystalPhi = DEG2RAD * offsetArray[3];
		if ((int)offsetArray[4] != -1) (*s).nonlinearAbsorptionStrength = offsetArray[4];
		if ((int)offsetArray[5] != -1) (*s).bandGapElectronVolts = offsetArray[5];
		if ((int)offsetArray[6] != -1) (*s).drudeGamma = offsetArray[6];
		if ((int)offsetArray[7] != -1) (*s).effectiveMass = offsetArray[7];
		if ((int)offsetArray[8] != -1) (*s).crystalThickness = 1e-6 * offsetArray[8];
		if ((int)offsetArray[9] != -1) (*s).propagationStep = 1e-9 * offsetArray[9];
		if ((int)offsetArray[8] != -1) (*s).Npropagation
			= (size_t)(1e-6 * offsetArray[8] / (*s).propagationStep);
		if (currentIndex > 0) {
			(*s).isFollowerInSequence = TRUE;
		}
		(*s).chi2Tensor = db[(*s).materialIndex].d;
		(*s).chi3Tensor = db[(*s).materialIndex].chi3;
		(*s).nonlinearSwitches = db[(*s).materialIndex].nonlinearSwitches;
		(*s).absorptionParameters = db[(*s).materialIndex].absorptionParameters;
		(*s).sellmeierCoefficients = db[(*s).materialIndex].sellmeierCoefficients;

		(*s).sellmeierType = db[(*s).materialIndex].sellmeierType;
		(*s).axesNumber = db[(*s).materialIndex].axisType;

#ifdef __HIPCC__
		error = solveNonlinearWaveEquation(s);
#else
		error = solveNonlinearWaveEquationCPU(s);
#endif
		if (offsetArray[10] != 0.0) {
			rotateField(s, DEG2RAD * offsetArray[10]);
		}

		if ((*s).memoryError > 0) {
			printf("Warning: device memory error (%i).\n", (*s).memoryError);
		}
		return error;

	case 1:
		if ((*s).isCylindric) {
			if ((int)offsetArray[1] != -1) (*s).materialIndex = (int)offsetArray[1];
			if ((int)offsetArray[2] != -1) (*s).crystalTheta = DEG2RAD * offsetArray[2];
			if ((int)offsetArray[3] != -1) (*s).crystalPhi = DEG2RAD * offsetArray[3];
			if ((int)offsetArray[4] != -1) (*s).nonlinearAbsorptionStrength = offsetArray[4];
			if ((int)offsetArray[5] != -1) (*s).bandGapElectronVolts = offsetArray[5];
			if ((int)offsetArray[6] != -1) (*s).drudeGamma = offsetArray[6];
			if ((int)offsetArray[7] != -1) (*s).effectiveMass = offsetArray[7];
			if ((int)offsetArray[8] != -1) (*s).crystalThickness = 1e-6 * offsetArray[8];
			if ((int)offsetArray[9] != -1) (*s).propagationStep = 1e-9 * offsetArray[9];
			if ((int)offsetArray[8] != -1 && (int)offsetArray[8] != -1) (*s).Npropagation
				= (size_t)(1e-6 * offsetArray[8] / (*s).propagationStep);
			if (currentIndex > 0) {
				(*s).isFollowerInSequence = TRUE;
			}
			(*s).chi2Tensor = db[(*s).materialIndex].d;
			(*s).chi3Tensor = db[(*s).materialIndex].chi3;
			(*s).nonlinearSwitches = db[(*s).materialIndex].nonlinearSwitches;
			(*s).absorptionParameters = db[(*s).materialIndex].absorptionParameters;
			(*s).sellmeierCoefficients = db[(*s).materialIndex].sellmeierCoefficients;
			(*s).sellmeierType = db[(*s).materialIndex].sellmeierType;
			(*s).axesNumber = db[(*s).materialIndex].axisType;
			(*s).forceLinear = TRUE;
#ifdef __HIPCC__
			error = solveNonlinearWaveEquation(s);
#else
			error = solveNonlinearWaveEquationCPU(s);
#endif
		}
		else {
			if ((int)offsetArray[1] != -1) (*s).materialIndex = (int)offsetArray[1];
			if ((int)offsetArray[2] != -1) (*s).crystalTheta = DEG2RAD * offsetArray[2];
			if ((int)offsetArray[3] != -1) (*s).crystalPhi = DEG2RAD * offsetArray[3];
			thickness = 1.0e-6 * offsetArray[8];
			if (offsetArray[8] == -1) {
				thickness = (*s).crystalThickness;
			}
			materialIndex = (int)offsetArray[1];
			if (offsetArray[1] == -1) {
				materialIndex = (*s).materialIndex;
			}
			applyLinearPropagation(s, materialIndex, thickness);
		}

		if (offsetArray[10] != 0.0) {
			rotateField(s, DEG2RAD * offsetArray[10]);
		}
		return 0;

	case 2:
		if ((int)offsetArray[1] != -1) (*s).materialIndex = (int)offsetArray[1];
		if ((int)offsetArray[2] != -1) (*s).crystalTheta = DEG2RAD * offsetArray[2];
		if ((int)offsetArray[3] != -1) (*s).crystalPhi = DEG2RAD * offsetArray[3];
		applyFresnelLoss(s, (int)offsetArray[4], (int)offsetArray[5]);
		return 0;
	case 3:
		applySphericalMirror(s, offsetArray[8]);
		if (offsetArray[10] != 0.0) {
			rotateField(s, DEG2RAD * offsetArray[10]);
		}
		return 0;
	case 4:
		applyParabolicMirror(s, offsetArray[8]);
		if (offsetArray[10] != 0.0) {
			rotateField(s, DEG2RAD * offsetArray[10]);
		}
		return 0;
	case 5:
		applyAperature(s, offsetArray[1], offsetArray[2]);
		if (offsetArray[10] != 0.0) {
			rotateField(s, DEG2RAD * offsetArray[10]);
		}
		return 0;
	}



	return 1;
	}

#ifdef __HIPCC__
unsigned long solveNonlinearWaveEquationSequence(void* lpParam) {
#else
unsigned long solveNonlinearWaveEquationSequenceCPU(void* lpParam) {
#endif
	simulationParameterSet* sCPU = (simulationParameterSet*)lpParam;
	simulationParameterSet* sCPUbackup = (simulationParameterSet*)calloc(1, sizeof(simulationParameterSet));
	memcpy(sCPUbackup, sCPU, sizeof(simulationParameterSet));
	int k;
	int error = 0;
	for (k = 0; k < (*sCPU).Nsequence; k++) {
		if ((int)round((*sCPU).sequenceArray[k * 11]) == 6
			&& ((int)round((*sCPU).sequenceArray[k * 11 + 1])) > 0) {
			(*sCPUbackup).sequenceArray[k * 11 + 1] -= 1.0;
			(*sCPUbackup).isFollowerInSequence = TRUE;
			k = 0;
		}
#ifdef __HIPCC__
		error = resolveSequence(k, sCPU, (*sCPU).crystalDatabase);
#else
		error = resolveSequenceCPU(k, sCPU, (*sCPU).crystalDatabase);
#endif
		
		if (error) break;
		memcpy(sCPU, sCPUbackup, sizeof(simulationParameterSet));
	}
	free(sCPUbackup);
	return error;
}

#ifdef __HIPCC__
int main(int argc, char* argv[]) {
#else
int mainCPU(int argc, char* argv[]) {
#endif
	int i, j;
	int CUDAdevice;
	int CUDAdeviceCount = 0;
	size_t progressCounter = 0;
	hipGetDeviceCount(&CUDAdeviceCount);
	hipError_t cuErr = hipGetDevice(&CUDAdevice);
	struct hipDeviceProp_t activeCUDADeviceProp;
	if (cuErr == hipSuccess) {
		printf("Found %i GPU(s): \n", CUDAdeviceCount);
		for (i = 0; i < CUDAdeviceCount; i++) {
			cuErr = hipGetDeviceProperties(&activeCUDADeviceProp, CUDAdevice);
			printf("%s\r\n", activeCUDADeviceProp.name);
			printf(" Memory: %lli MB; Multiprocessors: %i\n",
				activeCUDADeviceProp.totalGlobalMem / (1024 * 1024), activeCUDADeviceProp.multiProcessorCount);
		}
	}
	else {
		printf("No GPU found.\n");
		return 1;
	}

	if (argc < 2) {
		printf("no input file specified.\n");
		return 2;
	}

	// allocate databases, main structs
	simulationParameterSet* sCPU = (simulationParameterSet*)calloc(512, sizeof(simulationParameterSet));
	crystalEntry* crystalDatabasePtr = (crystalEntry*)calloc(512, sizeof(crystalEntry));
	(*sCPU).crystalDatabase = crystalDatabasePtr;
	(*sCPU).progressCounter = &progressCounter;
	// read crystal database
	if (readCrystalDatabase(crystalDatabasePtr) == -2) {
		return 11;
	}
	if ((*crystalDatabasePtr).numberOfEntries == 0) {
		printf("Could not read crystal database.\n");
		free(sCPU);
		free(crystalDatabasePtr);
		return 12;
	}
	printf("Read %i crystal database entries:\n", (*crystalDatabasePtr).numberOfEntries);
	for (j = 0; j < (*crystalDatabasePtr).numberOfEntries; j++) {
		printf("Material %i name: %ls", j, crystalDatabasePtr[j].crystalNameW);
	}

	// read from settings file
	if (readInputParametersFile(sCPU, crystalDatabasePtr, argv[1]) == 1) {
		printf("Could not read input file.\n");
		free(sCPU);
		free(crystalDatabasePtr);
		return 13;
	}

	allocateGrids(sCPU);
	if (loadPulseFiles(sCPU) == 1) {
		printf("Could not read pulse file.\n");
		free((*sCPU).imdone);
		free((*sCPU).deffTensor);
		free((*sCPU).loadedField1);
		free((*sCPU).loadedField2);
		free(sCPU);
		free(crystalDatabasePtr);
		return 14;
	}

	readSequenceString(sCPU);
	printf("Found %i steps in sequence\n", (*sCPU).Nsequence);
	readFittingString(sCPU);
	configureBatchMode(sCPU);

	auto simulationTimerBegin = std::chrono::high_resolution_clock::now();

	// run simulations
	if ((*sCPU).isInFittingMode) {
		if ((*sCPU).fittingMode == 3) {
			if (loadReferenceSpectrum((*sCPU).fittingPath, sCPU)) {
				printf("Could not load reference spectrum!\n");
				free((*sCPU).imdone);
				free((*sCPU).deffTensor);
				free((*sCPU).loadedField1);
				free((*sCPU).loadedField2);
				free((*sCPU).ExtOut);
				free((*sCPU).EkwOut);
				free((*sCPU).totalSpectrum);
				free((*sCPU).fittingReference);
				free(sCPU);
				free(crystalDatabasePtr);
				return 10;
			}
		}
		printf("Running in fitting mode -- I don't know how long this will take!\n");
		runFitting(sCPU);

		auto simulationTimerEnd = std::chrono::high_resolution_clock::now();
		printf("Finished after %8.4lf s. \n",
			1e-6 * (double)(std::chrono::duration_cast<std::chrono::microseconds>(simulationTimerEnd - simulationTimerBegin).count()));

		saveDataSet(sCPU, crystalDatabasePtr, (*sCPU).outputBasePath, FALSE);

		free((*sCPU).imdone);
		free((*sCPU).deffTensor);
		free((*sCPU).loadedField1);
		free((*sCPU).loadedField2);
		free((*sCPU).ExtOut);
		free((*sCPU).EkwOut);
		free((*sCPU).totalSpectrum);
		free((*sCPU).fittingReference);
		free(sCPU);
		free(crystalDatabasePtr);

		return 0;
	}
	std::thread* threadBlock = (std::thread*)calloc((*sCPU).Nsims * (*sCPU).Nsims2, sizeof(std::thread));
	size_t maxThreads = min(CUDAdeviceCount, (*sCPU).Nsims * (*sCPU).Nsims2);
	for (j = 0; j < (*sCPU).Nsims * (*sCPU).Nsims2; j++) {

		sCPU[j].assignedGPU = j % CUDAdeviceCount;
		if (j >= maxThreads) {
			if (threadBlock[j - maxThreads].joinable()) {
				threadBlock[j - maxThreads].join();
			}
		}

		if ((*sCPU).isInSequence) {
			threadBlock[j] = std::thread(solveNonlinearWaveEquationSequence, &sCPU[j]);
		}
		else {
			threadBlock[j] = std::thread(solveNonlinearWaveEquation, &sCPU[j]);
		}
	}

	for (i = 0; i < (*sCPU).Nsims * (*sCPU).Nsims2; i++) {
		if (sCPU[i].memoryError > 0) {
			printf("Warning: device memory error (%i).\n", sCPU[i].memoryError);
		}
		if (threadBlock[i].joinable()) {
			threadBlock[i].join();
		}
	}

	auto simulationTimerEnd = std::chrono::high_resolution_clock::now();
	printf("Finished after %8.4lf s. \n",
		1e-6 * (double)(std::chrono::duration_cast<std::chrono::microseconds>(simulationTimerEnd - simulationTimerBegin).count()));


	saveDataSet(sCPU, crystalDatabasePtr, (*sCPU).outputBasePath, FALSE);
	//free
	free(threadBlock);
	free((*sCPU).imdone);
	free((*sCPU).deffTensor);
	free((*sCPU).loadedField1);
	free((*sCPU).loadedField2);
	free((*sCPU).ExtOut);
	free((*sCPU).EkwOut);
	free((*sCPU).totalSpectrum);
	free(sCPU);
	free(crystalDatabasePtr);
	return 0;
}