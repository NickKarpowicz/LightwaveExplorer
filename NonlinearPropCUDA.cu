#include "hip/hip_runtime.h"
#include "NonlinearPropCUDA.cuh"
#include "framework.h"
#include<complex>
#include<cstdlib>
#include<math.h>
#include "MPQ_Nonlinear_Propagation.h"
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "qr_solve.hpp"
#include "MPQ_Nonlinear_Propagation.h"
//#include <complex.h>

#define THREADS_PER_BLOCK 64
#define MAX_LOADSTRING 1024
//overload the math operators for cuda complex numbers so this code fits inside the observable universe
__device__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b) { return hipCmul(a, b); }
__device__ hipDoubleComplex operator+(hipDoubleComplex a, hipDoubleComplex b) { return hipCadd(a, b); }
__device__ hipDoubleComplex operator+(double a, hipDoubleComplex b) { return hipCadd(make_hipDoubleComplex(a, 0.0), b); }
__device__ hipDoubleComplex operator+(hipDoubleComplex a, double b) { return hipCadd(a, make_hipDoubleComplex(b, 0.0)); }
__device__ hipDoubleComplex operator-(hipDoubleComplex a, hipDoubleComplex b) { return hipCsub(a, b); }
__device__ hipDoubleComplex operator-(double a, hipDoubleComplex b) { return hipCsub(make_hipDoubleComplex(a, 0.0), b); }
__device__ hipDoubleComplex operator/(hipDoubleComplex a, double b) { return hipCdiv(a, make_hipDoubleComplex(b, 0.0)); }
__device__ hipDoubleComplex operator/(double b, hipDoubleComplex a) { return hipCdiv(make_hipDoubleComplex(b, 0.0), a); }
__device__ hipDoubleComplex operator*(hipDoubleComplex a, double b) { return hipCmul(a, make_hipDoubleComplex(b, 0.0)); }
__device__ hipDoubleComplex operator*(double b, hipDoubleComplex a) { return hipCmul(a, make_hipDoubleComplex(b, 0.0)); }


//complex exponential function for CUDA
__device__ __forceinline__ hipDoubleComplex cuCexpd(hipDoubleComplex z)
{
    hipDoubleComplex res;
    double t = exp(z.x);
    res.y = sin(z.y);
    res.x = cos(z.y);
    res.x *= t;
    res.y *= t;
    return res;
}

//copy and paste from
// https://forums.developer.nvidia.com/t/additional-cucomplex-functions-cucnorm-cucsqrt-cucexp-and-some-complex-double-functions/36892 
__device__ hipDoubleComplex cuCsqrt(hipDoubleComplex x)
{
    double radius = hipCabs(x);
    double cosA = x.x / radius;
    hipDoubleComplex out;
    out.x = sqrt(radius * (cosA + 1.0) / 2.0);
    out.y = sqrt(radius * (1.0 - cosA) / 2.0);
    // signbit should be false if x.y is negative
    if (signbit(x.y))
        out.y *= -1.0;

    return out;
}


__device__ hipDoubleComplex sellmeierCuda(hipDoubleComplex* ne, hipDoubleComplex* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return make_hipDoubleComplex(1.0,0.0); //exit immediately for f=0
    
    double c = 2.99792458e8; //speed of light
    double l = 1e6 * c / f; //wavelength in microns
    double ls = l * l;
    hipDoubleComplex ii = make_hipDoubleComplex(0.0, 1.0);
    double pi = 3.14159265358979323846264338327950288;
    double omega = 2 * pi * abs(f);
    double kL = 3183.9; //(e * e / (e_o *m_e)
    hipDoubleComplex one = make_hipDoubleComplex(1.0, 0);
    hipDoubleComplex na = one;
    hipDoubleComplex nb = one;
    //option 0: isotropic
    if (type == 0) {
        ne[0] = make_hipDoubleComplex(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6])
            + (a[7] + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls, 0.0);
        if (hipCreal(ne[0]) < 1) {
            ne[0] = one;
        }
        ne[0] = ne[0] + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega);
        ne[0] = hipConj(cuCsqrt(ne[0]));
        if (isnan(hipCreal(ne[0]))) {
            ne[0] = one;
        }

        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {
        
        na = cuCsqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega));
        
        a = &a[22];
        nb = cuCsqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega));
        no[0] = na;
        ne[0] = 1.0 / cuCsqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return ne[0];
    }
    else {
        //later, implement biaxial crystals, for now just return 1;
        return one;
    }
}
__global__ void radialLaplacianKernel(struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j = i / s.Ntime; //spatial coordinate
    long long k = i - j * s.Ntime; //temporal coordinate
    double rho = s.dx * j - (s.dx / 2) * s.Nspace;

    //zero at edges of grid and at origin
    if (abs(rho<1e-15) || j<3 || j>(s.Nspace-4)) {
        s.gridRadialLaplacian1[i] = make_hipDoubleComplex(0, 0);
        s.gridRadialLaplacian2[i] = make_hipDoubleComplex(0, 0);
    }
    else {
        s.gridRadialLaplacian1[i] = (s.firstDerivativeOperation[0] * s.gridETime[i - 3 * s.Ntime]
            + s.firstDerivativeOperation[1] * s.gridETime[i - 2 * s.Ntime]
            + s.firstDerivativeOperation[2] * s.gridETime[i - s.Ntime]
            + s.firstDerivativeOperation[3] * s.gridETime[i + s.Ntime]
            + s.firstDerivativeOperation[4] * s.gridETime[i + 2 * s.Ntime]
            + s.firstDerivativeOperation[5] * s.gridETime[i + 3 * s.Ntime])/rho;
        s.gridRadialLaplacian2[i] = (s.firstDerivativeOperation[0] * s.gridETime2[i - 3 * s.Ntime]
            + s.firstDerivativeOperation[1] * s.gridETime2[i - 2 * s.Ntime]
            + s.firstDerivativeOperation[2] * s.gridETime2[i - s.Ntime]
            + s.firstDerivativeOperation[3] * s.gridETime2[i + s.Ntime]
            + s.firstDerivativeOperation[4] * s.gridETime2[i + 2 * s.Ntime]
            + s.firstDerivativeOperation[5] * s.gridETime2[i + 3 * s.Ntime]) / rho;
    }
}

__global__ void prepareCartesianGridsKernel(double* theta, double* sellmeierCoefficients, struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j, k;
    long long Ntime = s.Ntime;
    long long Nspace = s.Nspace;
    int axesNumber = s.axesNumber;
    int sellmeierType = s.sellmeierType;
    double c = 2.99792458e8; //speed of light
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    hipDoubleComplex cuZero = make_hipDoubleComplex(0, 0);
    j = i / Ntime; //spatial coordinate
    k = i - j*Ntime; //temporal coordinate
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double fStep = sellmeierCoefficients[71];
    double tol = sellmeierCoefficients[72];
    double dTheta = 0.1;
    double err, errPlus, errMinus;
    
    hipDoubleComplex ne, no, n0;
    double nePlus, neMinus;



    //frequency being resolved by current thread
    double f = k * fStep;
	if (k >= Ntime / 2) {
		f -= fStep * Ntime;
	}
	f *= -1;

    //transverse wavevector being resolved
	double dk = j * kStep - (j >= (Nspace / 2)) * (kStep * Nspace); //frequency grid in transverse direction


    //Find walkoff angle, starting from zero
    theta[i] = 0;
    double rhs = 2.99792458e8 * dk / (2 * 3.14159265358979323846264338327950288 * f);
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta + theta[i], crystalPhi, axesNumber, sellmeierType);    
    nePlus = hipCreal(ne);
    err = abs(nePlus * sin(theta[i]) - rhs);

    int iters = 0;
    errPlus = 2;
    errMinus = 2;
    while (err > tol && iters < 2048) {
        iters++;

        sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta + theta[i] + dTheta, crystalPhi, axesNumber, sellmeierType);
        nePlus = hipCreal(ne);
        errPlus = abs(nePlus * sin(theta[i] + dTheta) - rhs);

        sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta + theta[i] - dTheta, crystalPhi, axesNumber, sellmeierType);
        neMinus = hipCreal(ne);
        errMinus = abs(neMinus * sin(theta[i] - dTheta) - rhs);

        //Basic hill climbing algorithm
        //calculate the error at theta +/- dTheta
        // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
        // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
        // if theta has lowest error, step size is too large, dTheta /= 2;
        if (errPlus < err && errPlus < errMinus) {
            theta[i] += dTheta;
            err = errPlus;
        }
        else if (errMinus < err) {
            theta[i] -= dTheta;
            err = errMinus;
        }
        else {
            dTheta *= 0.5;
        }
        
    }


    //walkoff angle has been found, generate the rest of the grids
    f = k * fStep;
    if (k >= Ntime / 2) {
        f -= fStep * Ntime;
    }
    f *= -1;

    sellmeierCuda(&n0, &no, sellmeierCoefficients, abs(s.f0), crystalTheta, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta + theta[i], crystalPhi, axesNumber, sellmeierType);
    if (isnan(hipCreal(ne)) || isnan(hipCreal(no))) {
        ne = make_hipDoubleComplex(1,0);
        no = make_hipDoubleComplex(1, 0);
    }
    s.ne[i] = ne;
    s.no[i] = no;

    hipDoubleComplex k0 = make_hipDoubleComplex(2 * pi * hipCreal(n0) * f / c,0);
    hipDoubleComplex ke = 2 * pi * ne * f / c;
    hipDoubleComplex ko = 2 * pi * no * f / c;

    if (hipCreal(ke) < 0 && hipCreal(ko) < 0) {
        s.gridPropagationFactor[i] = ii * (ke - k0 + dk * dk / (2. * hipCreal(ke))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor[i]))) {
            s.gridPropagationFactor[i] = cuZero;
        }

        s.gridPropagationFactor2[i] = ii * (ko - k0 + dk * dk / (2. * hipCreal(ko))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor2[i]))) {
            s.gridPropagationFactor2[i] = cuZero;
        }

        int posf = (int)(f < -20e12);
        s.gridPolarizationFactor[i] = ii * (posf * 2 * pi * f) / (2. * hipCreal(ne) * c) * s.h;
        s.gridPolarizationFactor2[i] = ii * (posf * 2 * pi * f) / (2. * hipCreal(no) * c) * s.h;
    }
    
    else {
        s.gridPropagationFactor[i] = cuZero;
        s.gridPropagationFactor2[i] = cuZero;
        s.gridPolarizationFactor[i] = cuZero;
        s.gridPolarizationFactor2[i] = cuZero;
    }
    
    
}

__global__ void prepareCylindricGridsKernel(double* sellmeierCoefficients, struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j, k;
    long long Ntime = s.Ntime;
    long long Nspace = s.Nspace;
    int axesNumber = s.axesNumber;
    int sellmeierType = s.sellmeierType;
    double c = 2.99792458e8; //speed of light
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    hipDoubleComplex cuZero = make_hipDoubleComplex(0, 0);
    j = i / Ntime; //spatial coordinate
    k = i - j * Ntime; //temporal coordinate
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double fStep = sellmeierCoefficients[71];
    double tol = sellmeierCoefficients[72];
    double dTheta = 0.1;
    double err, errPlus, errMinus;

    hipDoubleComplex ne, no, n0;

    //frequency being resolved by current thread
    double f = k * fStep;
    if (k >= Ntime / 2) {
        f -= fStep * Ntime;
    }
    f *= -1;

    //transverse wavevector being resolved
    double dk = j * kStep - (j >= (Nspace / 2)) * (kStep * Nspace); //frequency grid in transverse direction



    sellmeierCuda(&n0, &no, sellmeierCoefficients, abs(s.f0), crystalTheta, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, axesNumber, sellmeierType);
    if (isnan(hipCreal(ne)) || isnan(hipCreal(no))) {
        ne = make_hipDoubleComplex(1, 0);
        no = make_hipDoubleComplex(1, 0);
    }
    s.ne[i] = ne;
    s.no[i] = no;

    hipDoubleComplex k0 = make_hipDoubleComplex(2 * pi * hipCreal(n0) * f / c, 0);
    hipDoubleComplex ke = 2 * pi * ne * f / c;
    hipDoubleComplex ko = 2 * pi * no * f / c;

    if (hipCreal(ke) < 0 && hipCreal(ko) < 0) {
        s.gridPropagationFactor[i] = ii * (ke - k0 + dk * dk / (2. * hipCreal(ke))) * s.h;
        s.gridPropagationFactorRho1[i] = ii * (1 / (2. * hipCreal(ke))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor[i]))) {
            s.gridPropagationFactor[i] = cuZero;
            s.gridPropagationFactorRho1[i] = cuZero;
        }

        s.gridPropagationFactor2[i] = ii * (ko - k0 + dk * dk / (2. * hipCreal(ko))) * s.h;
        s.gridPropagationFactorRho2[i] = ii * (1 / (2. * hipCreal(ko))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor2[i]))) {
            s.gridPropagationFactor2[i] = cuZero;
            s.gridPropagationFactorRho2[i] = cuZero;
        }

        int posf = (int)(f < -20e12);
        s.gridPolarizationFactor[i] = ii * (posf * 2 * pi * f) / (2. * hipCreal(ne) * c) * s.h;
        s.gridPolarizationFactor2[i] = ii * (posf * 2 * pi * f) / (2. * hipCreal(no) * c) * s.h;
    }

    else {
        s.gridPropagationFactor[i] = cuZero;
        s.gridPropagationFactor2[i] = cuZero;
        s.gridPolarizationFactor[i] = cuZero;
        s.gridPolarizationFactor2[i] = cuZero;
        s.gridPropagationFactor[i] = cuZero;
        s.gridPropagationFactorRho2[i] = cuZero;
    }


}
//replaces E with its complex conjugate
__global__ void conjugateKernel(hipDoubleComplex* E) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    E[i] = hipConj(E[i]);
}

//replaces NaN values with 0
__global__ void fixnanKernel(hipDoubleComplex* E) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (isnan(hipCreal(E[i])) || isnan(hipCimag(E[i]))) {
        E[i] = make_hipDoubleComplex(0., 0.);
    }
}
__global__ void nonlinearpolarizationKernel(struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    double Ex = hipCreal(s.gridETime[i]) / s.propagationInts[0];
    double Ey = hipCreal(s.gridETime2[i]) / s.propagationInts[0];
    s.gridPolarizationTime[i] = 0.;
    s.gridPolarizationTime2[i] = 0.;

    //The d2eff tensor has the form
    // | d_xxx d_xyx d_yyx |
    // | d_xxy d_xyy d_yyy |
    if (s.nonlinearSwitches[0] == 1) {
        s.gridPolarizationTime[i] += s.chi2Tensor[0] * Ex * Ex + s.chi2Tensor[2] * Ex * Ey + s.chi2Tensor[4] * Ey * Ey;
        s.gridPolarizationTime2[i] += s.chi2Tensor[1] * Ex * Ex + s.chi2Tensor[3] * Ex * Ey + s.chi2Tensor[5] * Ey * Ey;
    }
    
    //to be implemented: full chi3 matrix on s.nonlinearSwitches[1]==1

    //using only one value of chi3, under assumption of centrosymmetry
    if (s.nonlinearSwitches[1] == 2) {
        s.gridPolarizationTime[i] += s.chi3Tensor[0] * (Ex * Ex * Ex + Ey * Ey * Ex / 3.);
        s.gridPolarizationTime2[i] += s.chi3Tensor[0] * (Ey * Ey * Ey + Ex * Ex * Ey / 3.);
    }

    //Nonlinear absorption
    if (s.nonlinearSwitches[2] == 1) {
        
        double Exi = hipCimag(s.gridETime[i]) / s.propagationInts[0];
        double Eyi = hipCimag(s.gridETime2[i]) / s.propagationInts[0];
        double fieldAmp2 = Exi * Exi + Eyi * Eyi;
        int j;
        for (j = 0; j < s.nonlinearSwitches[3]; j++) {
            Exi *= fieldAmp2;
            Eyi *= fieldAmp2;
        }
        s.gridPolarizationTime[i] += s.absorptionParameters[1] * Exi;
        s.gridPolarizationTime2[i] += s.absorptionParameters[1] * Eyi;
    }
}

//Plasma response with time-dependent carrier density
//This polarization needs a different factor in the nonlinear wave equation
//to account for the integration
//TO BE IMPLEMENTED!
__global__ void plasmaCurrentKernel(struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    int j,k,l;
    double N = 0;
    double integralx = 0;
    double integraly = 0;
    double t, w, Esquared, Ex, Ey;
    for (j = 0; j < s.Ntime; j++) {
        Ex = hipCreal(s.gridETime[l]);
        Ey = hipCreal(s.gridETime2[l]);
        l = j + i * s.Nspace;
        t = j * s.dt;
        Esquared = Ex * Ex + Ey * Ey;
        w = Esquared;
        for (k = 0; k < s.nonlinearSwitches[3]; k++) {
            w *= Esquared;
        }
        N += w;
        integralx += exp(-s.plasmaParameters[1] * t) * N * Ex;
        integraly += exp(-s.plasmaParameters[1] * t) * N * Ey;
        s.plasmaCurrent1[l] = s.plasmaParameters[0] * exp(-s.plasmaParameters[1] * t) * integralx;
        s.plasmaCurrent2[l] = s.plasmaParameters[0] * exp(-s.plasmaParameters[1] * t) * integraly;
    }
}


//Main kernel for RK4 propagation of the field
__global__ void rkKernel(struct cudaLoop s, int stepNumber) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j = i / s.Ntime; //spatial coordinate
    long long h = i - j * s.Ntime; //temporal coordinate


    if (s.isCylindric) {
        s.gridRadialLaplacian1[i] = s.gridPropagationFactorRho1[i] * s.k1[i];
        s.gridRadialLaplacian2[i] = s.gridPropagationFactorRho2[i] * s.k2[i];
    }

    //polarization is stored in a reduced format by cuFFT because the FFT is from real to complex, meaning if the output grid
    //were to be N_time x N_space, half of the points would be redundant. The extra steps below are to determine where in the grid the 
    //current point sits. Essentially, if in the negative frequency quadrants, reverse the frequency and take complex conjugate of the 
    //value
    if (h > s.propagationInts[3]) {
        h = s.Ntime - h;
        j = s.Nspace - j;
        h += j * s.propagationInts[3];
        s.k1[i] = s.gridPropagationFactor[i] * s.gridETemp[i] +s.gridPolarizationFactor[i] * hipConj(s.gridPolarizationFrequency[i]);
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * hipConj(s.gridPolarizationFrequency2[i]);
    }
    else {
        h += j * s.propagationInts[3];
        s.k1[i] = s.gridPropagationFactor[i] * s.gridETemp[i] +s.gridPolarizationFactor[i] * s.gridPolarizationFrequency[h];
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * s.gridPolarizationFrequency2[h];
    }
    if (s.isCylindric) {
        s.k1[i] = s.k1[i] + s.gridRadialLaplacian1[i];
        s.k2[i] = s.k2[i] + s.gridRadialLaplacian2[i];
    }

    //in the first substep, first construct the next intermediate field value
    //which will be used in the next substep. 
    if (stepNumber == 0) {
        s.gridETemp[i] = s.gridEFrequency[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];
       
        s.gridEFrequencyNext1[i] = s.k1[i] / 6 + s.gridEFrequency[i];
        s.gridEFrequencyNext2[i] = s.k2[i] / 6 + s.gridEFrequency2[i];
    }

    //in the next substep, again construct the next intermediate field and add k/3 to solution
    else if (stepNumber == 1) {
        s.gridETemp[i] = s.gridEFrequency[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];

        s.gridEFrequencyNext1[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 3;
        s.gridEFrequencyNext2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 3;

    }

    //same action as previous substep, except the weight of k in the intermediate solution is 1 instead of 0.5
    else if (stepNumber == 2) {
        s.gridETemp[i] = s.gridEFrequency[i] + s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + s.k2[i];
        s.gridEFrequencyNext1[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 3;
        s.gridEFrequencyNext2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 3;
    }

    //last substep. Solution is now complete and may be copied directly into the field arrays
    else {
        s.gridEFrequency[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 6;
        s.gridEFrequency2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 6;
        s.gridETemp[i] = s.gridEFrequency[i];
        s.gridETemp2[i] = s.gridEFrequency2[i];
    }

}


//Take absolute value of complex array
__global__ void absKernel(double* absOut, hipDoubleComplex* complexIn) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    absOut[i] = hipCabs(complexIn[i]);
}

//Apply fft normalization
__global__ void fftNormalizeKernel(hipDoubleComplex* A, long long* fftSize) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = A[i] / fftSize[0];
}

DWORD WINAPI propagationLoop(LPVOID lpParam) {

    //the struct s contains most of the simulation variables and pointers
    struct cudaLoop s;
    struct propthread* sCPU = (struct propthread*)lpParam;


    //initialize and take values from the struct handed over by the dispatcher
    long long i;
    s.Ntime = (*sCPU).Ntime;
    s.Nspace = (*sCPU).Nspace;
    s.dt = (*sCPU).tStep;
    s.dx = (*sCPU).rStep;
    s.fStep = (*sCPU).fStep;
    s.h = (*sCPU).propagationStep;
    s.Nsteps = (*sCPU).Npropagation;
    s.Ngrid = s.Ntime * s.Nspace;
    s.axesNumber = (*sCPU).axesNumber;
    s.sellmeierType = (*sCPU).sellmeierType;
    s.f0 = (*sCPU).frequency1;
    s.Nthread = THREADS_PER_BLOCK;
    s.Nblock = s.Ngrid / THREADS_PER_BLOCK;
    s.isCylindric =(*sCPU).isCylindric;
    s.isNonLinear = ((*sCPU).nonlinearSwitches[0] + (*sCPU).nonlinearSwitches[1] + (*sCPU).nonlinearSwitches[2]) > 0;
    (*sCPU).nonlinearSwitches[3] = (int)ceil((*sCPU).absorptionParameters[0] * 241.79893e12 / (*sCPU).frequency1) - 1;
    //CPU allocations
    std::complex<double>* gridPropagationFactorCPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));
    std::complex<double>* gridPolarizationFactorCPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));

    //GPU allocations
    int memErrors = 0;
    memErrors += hipMalloc((void**)&s.gridETime, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridETime2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridETemp, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridETemp2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequency, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequency2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactor, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPolarizationFactor, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactor2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPolarizationFactor2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactorRho1, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactorRho2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridRadialLaplacian1, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridRadialLaplacian2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequencyNext1, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequencyNext2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.k1, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.k2, sizeof(hipDoubleComplex) * s.Ngrid);
    //the following two should have a size (s.Ntime / 2 + 1) * s.Nspace, but I get overruns during
    //the ffts if they're not larger. If I figure this out, it will save a complex grid worth of memory...
    memErrors += hipMalloc((void**)&s.gridPolarizationFrequency, sizeof(hipDoubleComplex) * s.Ngrid); 
    memErrors += hipMalloc((void**)&s.gridPolarizationFrequency2, sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPolarizationTime, sizeof(double) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPolarizationTime2, sizeof(double) * s.Ngrid);

    memErrors += hipMalloc((void**)&s.chi2Tensor, sizeof(double) * 9);
    memErrors += hipMalloc((void**)&s.firstDerivativeOperation, sizeof(double) * 6);
    memErrors += hipMalloc((void**)&s.chi3Tensor, sizeof(double) * 81);
    memErrors += hipMalloc((void**)&s.nonlinearSwitches, sizeof(int) * 4);
    memErrors += hipMalloc((void**)&s.absorptionParameters, sizeof(double) * 6);
    memErrors += hipMalloc((void**)&s.propagationInts, sizeof(long long) * 4);
    (*sCPU).memoryError = memErrors;

    //prepare effective nonlinearity tensors and put them on the GPU
    long long propagationIntsCPU[4] = { s.Ngrid, s.Ntime, s.Nspace, (s.Ntime / 2 + 1) };
    double firstDerivativeOperation[6] = { -(1/s.dx) / 60, (1 / s.dx) * 3 / 20, (1 / s.dx) * -3 / 4, (1 / s.dx) * 3 / 4, (1 / s.dx) * -3 / 20, (1 / s.dx) / 60 };
    deff((*sCPU).deffTensor, (*sCPU).chi2Tensor, (*sCPU).crystalTheta, (*sCPU).crystalPhi);
    hipMemcpy(s.chi2Tensor, (*sCPU).deffTensor, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.nonlinearSwitches, (*sCPU).nonlinearSwitches, 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(s.propagationInts, propagationIntsCPU, 4 * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(s.chi3Tensor, (*sCPU).chi3Tensor, 27 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.absorptionParameters, (*sCPU).absorptionParameters, 6 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.firstDerivativeOperation, firstDerivativeOperation, 6 * sizeof(double), hipMemcpyHostToDevice);

    //prepare FFT plans
    hipfftPlan2d(&s.fftPlan, s.Nspace, s.Ntime, HIPFFT_Z2Z);
    hipfftPlan2d(&s.polfftPlan, s.Nspace, s.Ntime, HIPFFT_D2Z);

    //prepare the propagation arrays
    if (s.isCylindric) {
        preparepropagation3Dcylindric(sCPU, s);
    }
    else {
        preparepropagation2Dcartesian(sCPU, s);
    }
    

    //generate the pulses, either through pulsegenerator() if this is the first in the series, or by copying
    //the output of the last simulation in the sequence
    if ((*sCPU).isFollowerInSequence) {
        hipMemcpy(s.gridETime, (*sCPU).ExtOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridETime2, &(*sCPU).ExtOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency, (*sCPU).EkwOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency2, &(*sCPU).EkwOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    }
    else {
        pulsegenerator(sCPU, &s);
    }
    
    //Copy the field into the temporary array
    hipMemcpy(s.gridETemp, s.gridEFrequency, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    hipMemcpy(s.gridETemp2, s.gridEFrequency2, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    //Core propagation loop
    for (i = 0; i < s.Nsteps; i++) {
        //calculate k1
        rkstep(s, 0);
        //calculate k2
        rkstep(s, 1);
        //calculate k3
        rkstep(s, 2);
        //calculate k4
        rkstep(s, 3);

        if ((*sCPU).imdone[0] == 2) {
            break;
        }
    }
    hipDeviceSynchronize();

    //transform final result
    fixnanKernel<<<s.Nblock, s.Nthread>>>(s.gridEFrequency);
    fixnanKernel << <s.Nblock, s.Nthread >> > (s.gridEFrequency2);
    hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridEFrequency, (hipfftDoubleComplex*)s.gridETime, HIPFFT_BACKWARD);
    hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridEFrequency2, (hipfftDoubleComplex*)s.gridETime2, HIPFFT_BACKWARD);
    fftNormalizeKernel<<<s.Nblock, s.Nthread >>>(s.gridETime, s.propagationInts);
    fftNormalizeKernel<<<s.Nblock, s.Nthread >>>(s.gridETime2, s.propagationInts);
    hipDeviceSynchronize();

    //copy the field arrays from the GPU to CPU memory
    hipMemcpy((*sCPU).ExtOut, s.gridETime, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*sCPU).EkwOut, s.gridEFrequency, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*sCPU).ExtOut[s.Ngrid], s.gridETime2, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*sCPU).EkwOut[s.Ngrid], s.gridEFrequency2, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    //Free GPU memory
    hipFree(s.propagationInts);
    hipFree(s.nonlinearSwitches);
    hipFree(s.absorptionParameters);
    hipFree(s.gridETime); 
    hipFree(s.gridETemp);
    hipFree(s.gridPolarizationFrequency);
    hipFree(s.gridEFrequency);
    hipFree(s.gridPropagationFactor);
    hipFree(s.gridPropagationFactorRho1);
    hipFree(s.gridPropagationFactorRho2);
    hipFree(s.gridRadialLaplacian1);
    hipFree(s.gridRadialLaplacian2);
    hipFree(s.firstDerivativeOperation);
    hipFree(s.gridPolarizationFactor);
    hipFree(s.gridEFrequencyNext1);
    hipFree(s.k1);
    hipFree(s.gridPolarizationTime);
    hipFree(s.gridETime2);
    hipFree(s.gridETemp2);
    hipFree(s.gridPolarizationFrequency2);
    hipFree(s.gridEFrequency2);
    hipFree(s.gridPropagationFactor2);
    hipFree(s.gridPolarizationFactor2);
    hipFree(s.gridEFrequencyNext2);
    hipFree(s.k2);
    hipFree(s.gridPolarizationTime2);
    hipFree(s.chi2Tensor);
    hipFree(s.chi3Tensor);
    hipfftDestroy(s.fftPlan);
    hipfftDestroy(s.polfftPlan);

    //Free CPU memory
    free(gridPropagationFactorCPU);
    free(gridPolarizationFactorCPU);
    
    return 0;
}

//function to run a RK4 time step
//stepNumber is the sub-step index, from 0 to 3
int rkstep(struct cudaLoop s, int stepNumber) {

    //operations involving FFT
    if (s.isNonLinear || s.isCylindric) {
        //perform inverse FFT to get time-space electric field
        hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridETemp, (hipfftDoubleComplex*)s.gridETime, HIPFFT_BACKWARD);
        hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridETemp2, (hipfftDoubleComplex*)s.gridETime2, HIPFFT_BACKWARD);
        
        if (s.isNonLinear) {
            nonlinearpolarizationKernel << <s.Nblock, s.Nthread >> > (s);
            hipfftExecD2Z(s.polfftPlan, s.gridPolarizationTime, (hipfftDoubleComplex*)s.gridPolarizationFrequency);
            hipfftExecD2Z(s.polfftPlan, s.gridPolarizationTime2, (hipfftDoubleComplex*)s.gridPolarizationFrequency2);
        }

        if (s.isCylindric) {
            hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridRadialLaplacian1, (hipfftDoubleComplex*)s.k1, HIPFFT_FORWARD);
            hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridRadialLaplacian2, (hipfftDoubleComplex*)s.k2, HIPFFT_FORWARD);
            radialLaplacianKernel<<<s.Nblock, s.Nthread >>>(s);
        }
    }

    //calculate k
    rkKernel<<<s.Nblock, s.Nthread >>>(s, stepNumber);
    
    return 0;
}

int pulsegenerator(struct propthread* s, struct cudaLoop *sc) {
    long long i,j;
    double rB, zB, r, z; //r and z in the Beam and lab coordinates, respectively.
    double w0, wz, zR, Rz, phi; //Gaussian beam parameters
    double theta = 0; //rotation angle of the current beam
    double pulseSum = 0;
    std::complex<double> ne, no, n0; //active refractive index;
    double f, w; //active frequency;
    double pulseEnergySum;
    std::complex<double> ko, k0, specfac, specphase;
    double c = 2.99792458e8; //speed of light
    double eps0 = 8.8541878128e-12; //vacuum permittivity
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    std::complex<double> *pulse1, *pulse2, *pulse1f, *pulse2f;
    hipfftHandle plan1;
    hipfftHandle plan2;
    pulse1 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse1f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    std::complex<double> Eb;
    std::complex<double> ii(0, 1);



    std::complex<double> polFactor1, polFactor2; //complex phase/amplitude factors for the polarization components
    sellmeier(&n0, &no, (*s).sellmeierCoefficients, (*s).frequency1, (*s).crystalTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
    (*s).neref = real(n0);
    (*s).noref = imag(n0);


    //define pulse 1 in mixed space
    // Gaussian beam in x
    // Spectrum in frequency domain (supergaussian with phase terms)
    polFactor1 = cos((*s).polarizationAngle1) - ii * (*s).circularity1 * sin((*s).polarizationAngle1);
    polFactor2 = sin((*s).polarizationAngle1) + ii * (*s).circularity1 * cos((*s).polarizationAngle1);
    theta = (*s).propagationAngle1;
    zB = (*s).z01;
    w0 = (*s).beamwaist1;

    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency1);
        
        //supergaussian pulse spectrum, if no input pulse specified
        specfac = (f - (*s).frequency1)/(*s).bandwidth1;
        for (j = 0; j < (*s).sgOrder1; j++) {
            specfac *= specfac;
        }
        specphase = ii * ((*s).cephase1 + w * (*s).delay1 - (*s).gdd1 * w * w - (*s).tod1 * w * w * w);
        specfac = exp(-specfac - specphase);

        if ((*s).field1IsAllocated) {
            specfac = (*s).loadedField1[i] * exp(-specphase);
        }

        ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
        no = (*s).refractiveIndex2[i + (*s).Ntime * j];
        ko = 2 * pi * no * f / c;
        k0 = 2 * pi * real(n0) * f / c;
        zR = pi * w0 * w0 * real(ne) * f / c;
        if (f == 0) {
            zR = 1e3;
        }

        for (j = 0; j < (*s).Nspace; j++) {
            rB = (*s).x01 + (*s).rStep * j - (*s).Nspace* (*s).rStep / 2.;
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);
            
            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));
            
            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            //z = 0;
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z-zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cmodulussquared(Eb)) || f<=0) {
                Eb = 0;
            }
            
            pulse1[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse1[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r)*(real(ne)*cmodulussquared(pulse1[i + (*s).Ntime * j]) + real(no)*cmodulussquared(pulse1[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }
    
    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime, pulse1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse1[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftPlan1d(&plan1, (*sc).Ntime, HIPFFT_Z2Z, (*sc).Nspace);
    hipfftPlan2d(&plan2, (*sc).Nspace, (*sc).Ntime, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridETemp, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency, (hipfftDoubleComplex*)(*sc).gridETime, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel<<<(*sc).Nblock, (*sc).Nthread >>>((*sc).gridETime);
    conjugateKernel<<<(*sc).Nblock, (*sc).Nthread >>>((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse1, (*sc).gridETime, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse1f, (*sc).gridEFrequency, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= 59.958 * pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy1/pulseSum)/(*s).Ngrid;
    
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse1[i] = pulse1[i] * pulseEnergySum;
        pulse1f[i] = pulse1f[i] * pulseEnergySum;
    }
    hipDeviceSynchronize();

    //do same for pulse 2 here
    pulseSum = 0;
    polFactor1 = cos((*s).polarizationAngle2) - ii * (*s).circularity2 * sin((*s).polarizationAngle2);
    polFactor2 = sin((*s).polarizationAngle2) + ii * (*s).circularity2 * cos((*s).polarizationAngle2);
    theta = (*s).propagationAngle2;
    zB = (*s).z02;
    w0 = (*s).beamwaist2;

    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency2);

        //supergaussian pulse spectrum, if no input pulse specified
        specfac = (f - (*s).frequency2) / (*s).bandwidth2;
        for (j = 0; j < (*s).sgOrder1; j++) {
            specfac *= specfac;
        }
        specphase = ii * ((*s).cephase2 + w * (*s).delay2 - (*s).gdd2 * w * w - (*s).tod2 * w * w * w);
        specfac = exp(-specfac - specphase);

        if ((*s).field2IsAllocated) {
            specfac = (*s).loadedField2[i] * exp(-specphase);
        }


        ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
        no = (*s).refractiveIndex2[i + (*s).Ntime * j];
        ko = 2 * pi * no * f / c;
        k0 = 2 * pi * real(n0) * f / c;
        zR = pi * w0 * w0 * real(ne) * f / c;
        if (f == 0) {
            zR = 1e3;
        }

        for (j = 0; j < (*s).Nspace; j++) {

            rB = (*s).x01 + (*s).rStep * j - (*s).Nspace * (*s).rStep / 2.;
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);

            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));

            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z - zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cmodulussquared(Eb)) || f <= 0) {
                Eb = 0;
            }

            pulse2[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse2[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r) * (real(ne) * cmodulussquared(pulse2[i + (*s).Ntime * j]) + real(no) * cmodulussquared(pulse2[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }

    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime, pulse2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse2[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftPlan1d(&plan1, (*sc).Ntime, HIPFFT_Z2Z, (*sc).Nspace);
    hipfftPlan2d(&plan2, (*sc).Nspace, (*sc).Ntime, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridETemp, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency, (hipfftDoubleComplex*)(*sc).gridETime, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel << <(*sc).Nblock, (*sc).Nthread >> > ((*sc).gridETime);
    conjugateKernel << <(*sc).Nblock, (*sc).Nthread >> > ((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse2, (*sc).gridETime, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse2f, (*sc).gridEFrequency, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= 59.958 * pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy2 / pulseSum) / (*s).Ngrid;

    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse2[i] = pulse2[i] * pulseEnergySum;
        pulse2f[i] = pulse2f[i] * pulseEnergySum;
    }
    hipDeviceSynchronize();

    //make the combined fields
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        (*s).Ext[i] = pulse1[i] + pulse2[i];
        (*s).Ekw[i] = pulse1f[i] + pulse2f[i];
    }
    //Copy the grids back to the GPU
    hipMemcpy((*sc).gridETime, (*s).Ext, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &(*s).Ext[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency, (*s).Ekw, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency2, &(*s).Ekw[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftDestroy(plan1);
    hipfftDestroy(plan2);


    return 0;
}

int preparepropagation2Dcartesian(struct propthread* s, struct cudaLoop sc) {
    //recycle allocated device memory for the grids needed
    double* alphaGPU = (double*)sc.gridEFrequencyNext1;
    double* sellmeierCoefficients = (double*)sc.k1;
    sc.ne = sc.gridEFrequencyNext2;
    sc.no = sc.k2;

    //construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep; 
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66+8) * sizeof(double), hipMemcpyHostToDevice);

    //prepare the propagation grids
    prepareCartesianGridsKernel <<<sc.Nblock, sc.Nthread >>> (alphaGPU, sellmeierCoefficients, sc);
    hipDeviceSynchronize();

    //copy the retrieved refractive indicies to the cpu
    hipMemcpy((*s).refractiveIndex1, sc.ne, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*s).refractiveIndex2, sc.no, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    
    //clean up
    hipMemset(sc.gridEFrequency, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.gridEFrequency2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    free(sellmeierCoefficientsAugmentedCPU);
    return 0;
}

int preparepropagation3Dcylindric(struct propthread* s, struct cudaLoop sc) {
    //recycle allocated device memory for the grids needed
    double* alphaGPU = (double*)sc.gridEFrequencyNext1;
    double* sellmeierCoefficients = (double*)sc.k1;
    sc.ne = sc.gridEFrequencyNext2;
    sc.no = sc.k2;

    //construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);

    //prepare the propagation grids
    prepareCylindricGridsKernel << <sc.Nblock, sc.Nthread >> > (sellmeierCoefficients, sc);
    hipDeviceSynchronize();

    //copy the retrieved refractive indicies to the cpu
    hipMemcpy((*s).refractiveIndex1, sc.ne, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*s).refractiveIndex2, sc.no, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);


    //clean up
    hipMemset(sc.gridEFrequency, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.gridEFrequency2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    free(sellmeierCoefficientsAugmentedCPU);
    return 0;
}

double thetasearch(struct propthread* s, double dk, double f, double tol) {
    double theta=0;
    double dTheta = 0.1;
    double err, errPlus, errMinus;
    double rhs = 2.99792458e8 * dk / (2 * 3.14159265358979323846264338327950288 * f);
    std::complex<double> ne, no;
    double nePlus, neMinus;
    f = abs(f);
    sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
    nePlus = real(ne);
    err = abs(nePlus * sin(theta) - rhs);
    int iters = 0;
    while (err > tol && iters<65536) {
        iters++;

        sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta + dTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
        nePlus = real(ne);
        errPlus = abs(nePlus * sin(theta+dTheta) - rhs);

        sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta - dTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
        neMinus = real(ne);
        errMinus = abs(neMinus * sin(theta-dTheta) - rhs);

        //Basic hill climbing algorithm
        //calculate the error at theta +/- dTheta
        // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
        // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
        // if theta has lowest error, step size is too large, dTheta /= 2;
        if (errPlus < err && errPlus < errMinus) {
            theta += dTheta;
            err = errPlus;
        }
        else if (errMinus < err) {
            theta -= dTheta;
            err = errMinus;
        }
        else {
            dTheta *= 0.5;
        }
    }
    return theta;
}
//rearrange a matrix from row major order to column major (not used, maybe broken)
int swaprc(double* M, int dim1, int dim2) {
    double* Ms = (double*)malloc(dim1 * dim2 * sizeof(double));
    int i, j;
    for (i = 0; i < dim1; i++) {
        for (j = 0; j < dim2; j++) {
            Ms[i + j * dim1] = M[j + i * dim2];
        }
    }
    free(Ms);
    return 0;
}

int deff(double* defftensor, double* dtensor, double theta, double phi) {
    double delta = 0.; //this angle is used for biaxial crystals, but I'm ignorning it for the moment
    int i, j, k;
    //Rotation matrix between the angles of the electric field and the crystal axes
    double R[] = { cos(theta) * cos(phi) * cos(delta) - sin(phi) * sin(delta), cos(theta) * sin(phi) * cos(delta) + cos(phi) * sin(delta),
        -sin(theta) * cos(delta), -cos(theta) * cos(phi) * sin(delta) - sin(phi) * cos(delta),
        -cos(theta) * sin(phi) * sin(delta) + cos(phi) * cos(delta), sin(theta) * sin(delta) };

    //Matrix to translate the mixed field matrix in the reduced notation into the crystalline frame
    double Ore[] = { R[0] * R[0], R[1] * R[1], R[2] * R[2], 2 * R[1] * R[2], 2 * R[0] * R[2], 2 * R[0] * R[1],
        2 * R[0] * R[3], 2 * R[1] * R[4], 2 * R[2] * R[5], 2 * (R[4] * R[2] + R[1] * R[5]), 2 * (R[3] * R[2] + R[0] * R[5]), 2 * (R[3] * R[1] + R[0] * R[4]),
        R[3] * R[3], R[4] * R[4], R[5] * R[5], 2 * R[4] * R[5], 2 * R[3] * R[5], 2 * R[3] * R[4]
};

    //The deff tensor is given by the equation R deff = d Ore, solve for deff, find d Ore first
    double dOre[9] = { 0 };
    for (i = 0; i < 3; i++) {
        for (j = 0; j < 3; j++) {
            for (k = 0; k < 6; k++) {
                dOre[i + 3 * j] += dtensor[i + 3 * k] * Ore[k + 6 * j];
            }
        }
    }
    //Least squares solution to get the deff tensor
    qr_solve_mat(3, 2, 3, R, dOre, defftensor);

    //correct cross-terms
    for (i = 2; i < 4; i++) {
        defftensor[i] *= 0.5;
    }

    for (i = 0; i < 6; i++) {
        defftensor[i] *= 2e-12; //change from pm/V to m/V and multiply by 2 for chi(2) instead of d
    }
    return 0;
}
//c implementation of fftshift, working on complex double precision
//A is the input array, B is the output
//dim1: column length
//dim2: row length
int fftshiftZ(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j-div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//same as fftshiftZ, but flips the output array columns
int fftshiftZflip(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1-i-1) + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i + div1 + dim1 * (j - div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//sellmeier equation
//outputs are pointers ne and no
//a is a 16-value array containing the coefficients
//f is frequency (Hz)
//theta is the crystal angle
//phi is the other crystal angle (currently unused because biaxials haven't been implemented)
//type is the kind of crystal (0: isotropic, 1: uniaxial, 2:biaxial) 
//eqn will switch to a different equation, in the future, currently not implemented
//current equation form:
//n^2 = a[0] //background (high freq) contribution
//      + (a[1] + a[2] * lambda^2) / (lambda^2 + a[3]) + (a[4] + a[5] * lambda^2)/ (lambda^2 + a[6]) //two resonances, purely real contribution
//      + (a[7] + a[8] * lambda^2) / (lambda^2 + a[9]) + (a[10] + a[11] * lambda^2) / (lambda^2 + a[12]) //two more resonances
//      + a[13] * lambda^2 + a[14] * lambda^4 + a[15] * lambda^6 //parametrized low-frequency correction
//      + 4*pi*e^2*a[16]/(a[17] - omega^2 + i * a[18] * omega) // complex-valued Lorenzian contribution (a[17] to zero for Drude)
//      + 4*pi*e^2*a[19]/(a[20] - omega^2 + i * a[21] * omega) // complex-valued Lorenzian contribution (a[21] to zero for Drude)
std::complex<double> sellmeier(std::complex<double>* ne, std::complex<double>* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return 1; //exit immediately for f=0

    double c = 2.99792458e8; //speed of light
    double l = 1e6*c / f; //wavelength in microns
    double ls = l * l;
    std::complex<double> ii(0, 1);
    double pi = 3.14159265358979323846264338327950288;
    double omega = 2*pi*abs(f);
    double kL = 3183.9; //(e * e / (e_o *m_e)
    //option 0: isotropic
    if (type == 0) {
        ne[0] = a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6])
            + (a[7] + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls;
        if (real(ne[0]) < 1) {
            ne[0] = 1.;
        }
        ne[0] += kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega);
        ne[0] = conj(sqrt(ne[0]));
        if (isnan(real(ne[0]))) {
            ne[0] = 1;
        }
        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {
        std::complex<double> na = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega)));
        a = &a[22];
        std::complex<double> nb = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega)));
        if (isnan(real(na)) || isnan(real(nb))) {
            no[0] = 1;
            ne[0] = 1;
            return 1;
        }
        no[0] = na;
        ne[0] = 1.0 / sqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return na;
    }
    else {
        //later, implement biaxial crystals, for now just return 1;
        return 1;
    }
}

int loadfrogspeck(char* frogFilePath, std::complex<double>* Egrid, long long Ntime, double fStep, double gateLevel, int fieldIndex) {
    FILE* fp;
    int maxFileSize = 16384;
    double wavelength, R, phi, complexX, complexY, f, f0, f1, fmax;
    int i, k0, k1;
    double c = 1e9*2.99792458e8; //for conversion of wavelength in nm to frequency
    double df = 0;
    double fmin = 0;
    int currentRow = 0;
    std::complex<double>* E = (std::complex<double>*)calloc(maxFileSize, sizeof(std::complex<double>));

    //read the data
    fp = fopen(frogFilePath, "r");
    while (fscanf(fp, "%lf %lf %lf %lf %lf", &wavelength, &R, &phi, &complexX, &complexY) == 5 && currentRow < maxFileSize) {
        //get the complex field from the data
        E[currentRow].real(complexX);
        E[currentRow].imag(complexY);

        //keep track of the frequency step of the grid (running sum, divide by number of rows at end to get average)
        if (currentRow > 0) df += c / wavelength - fmax;

        //keep track of the highest frequency in the data
        fmax = c / wavelength;
        
        //store the lowest frequency in the data
        if (currentRow == 0) fmin = fmax;
        
        currentRow++;
    }
    fclose(fp);

    //return an error if nothing was loaded
    if (currentRow == 0) {
        free(E);
        return -1;
    }

    df /= currentRow; //average frequency step

    //interpolate the FROG data onto the simulation grid
    
    //fill the simulation grid based on the data
    for (i = 0; i < Ntime; i++) {

        //frequency grid used in the simulation
        f = i * fStep;
        if (i >= Ntime / 2) {
            f -= fStep * Ntime;
        }
        f *= -1;

        k0 = (int)floor((f - fmin) / df);
        k1 = (int)ceil((f - fmin) / df);
        if (k0 < 0 || k1 >= currentRow) {
            Egrid[i] = 0; //field is zero outside of data range
        }
        else {
            f0 = fmin + k0 * df;
            f1 = fmin + k1 * df;
            Egrid[i] = (E[k0] * (f1 - f) + E[k1] * (f - f0)) / df; //linear interpolation
            Egrid[i] *= (abs(Egrid[i]) > gateLevel);
        }
    }

    free(E);
    return currentRow;
}

