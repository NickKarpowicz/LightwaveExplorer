#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "NonlinearPropCUDA.cuh"
#include <complex>
#include <cstdlib>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <mkl.h>
#include <thread>

//fitting parameter set as global variable
simulationParameterSet* fittingSet;
simulationParameterSet* fittingReferenceSet;

#define THREADS_PER_BLOCK 32
#define FALSE 0
#define TRUE 1
#define MAX_LOADSTRING 1024

#ifndef max
#define max(a,b)            (((a) > (b)) ? (a) : (b))
#endif

#ifndef min
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#endif

//overload the math operators for cuda complex numbers so this code fits inside the observable universe
__device__ __forceinline__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b) { return hipCmul(a, b); }
__device__ __forceinline__ hipDoubleComplex operator*(hipDoubleComplex a, double b) { return make_hipDoubleComplex(a.x * b, a.y * b); }
__device__ __forceinline__ hipDoubleComplex operator*(double b, hipDoubleComplex a) { return make_hipDoubleComplex(a.x * b, a.y * b); }
__device__ __forceinline__ hipDoubleComplex operator+(hipDoubleComplex a, hipDoubleComplex b) { return hipCadd(a, b); }
__device__ __forceinline__ hipDoubleComplex operator+(double a, hipDoubleComplex b) { return make_hipDoubleComplex(b.x + a, b.y); }
__device__ __forceinline__ hipDoubleComplex operator+(hipDoubleComplex a, double b) { return make_hipDoubleComplex(a.x + b, a.y); }
__device__ __forceinline__ hipDoubleComplex operator-(hipDoubleComplex a, hipDoubleComplex b) { return hipCsub(a, b); }
__device__ __forceinline__ hipDoubleComplex operator-(double a, hipDoubleComplex b) { return make_hipDoubleComplex(a-b.x, -b.y); }
__device__ __forceinline__ hipDoubleComplex operator-(hipDoubleComplex a, double b) { return make_hipDoubleComplex(a.x-b,a.y); }
__device__ __forceinline__ hipDoubleComplex operator/(hipDoubleComplex b, hipDoubleComplex a) { return hipCdiv(b, a); }
__device__ __forceinline__ hipDoubleComplex operator/(hipDoubleComplex a, double b) { return make_hipDoubleComplex(a.x / b, a.y / b); }
__device__  hipDoubleComplex operator/(double b, hipDoubleComplex a) {
    double divbByDenominator = b/(a.x * a.x + a.y * a.y);
    return make_hipDoubleComplex(a.x * divbByDenominator, -a.y * divbByDenominator);
}

//complex exponential function for CUDA
__device__ hipDoubleComplex cuCexpd(hipDoubleComplex z){
    double r = exp(z.x);
    return make_hipDoubleComplex(r*cos(z.y),r*sin(z.y));
}

//sqrt for complex doubles on CUDA, copy and paste from
// https://forums.developer.nvidia.com/t/additional-cucomplex-functions-cucnorm-cucsqrt-cucexp-and-some-complex-double-functions/36892 
__device__ hipDoubleComplex cuCsqrt(hipDoubleComplex x)
{
    double radius = hipCabs(x);
    double cosA = x.x / radius;
    hipDoubleComplex out;
    out.x = sqrt(radius * (cosA + 1.0) / 2.0);
    out.y = sqrt(radius * (1.0 - cosA) / 2.0);
    // signbit should be false if x.y is negative
    if (signbit(x.y))
        out.y *= -1.0;

    return out;
}

//Inner function for the Sellmeier equation to provide the refractive indicies
//current equation form:
//n^2 = a[0] //background (high freq) contribution
//      + four resonances, purely real contribution
//      + parametrized low-frequency correction
//      + 2 complex-valued Lorenzian contribution
//inputs:
//a: 22 component array of the coefficients
//ls: lamda^2 (microns^2)
//omega: frequency (rad/s)
//ii: sqrt(-1)
//kL: 3183.9 i.e. (e * e / (epsilon_o * m_e)
__device__ hipDoubleComplex sellmeierSubfunctionCuda(
    double* a, double ls, double omega, hipDoubleComplex ii, double kL) {
    double realPart = a[0]
        + (a[1] + a[2] * ls) / (ls + a[3])
        + (a[4] + a[5] * ls) / (ls + a[6])
        + (a[7] + a[8] * ls) / (ls + a[9])
        + (a[10] + a[11] * ls) / (ls + a[12])
        + a[13] * ls
        + a[14] * ls * ls
        + a[15] * ls * ls * ls;

    //traditional sellmeier part is not allowed to give complex values because that almost always
    //means it's out of range and causes instability
    if (realPart < 0) realPart = 1;

    return cuCsqrt(realPart
        + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
        + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega));
}

//Sellmeier equation for refractive indicies
__device__ hipDoubleComplex sellmeierCuda(
    hipDoubleComplex* ne, hipDoubleComplex* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return make_hipDoubleComplex(1.0, 0.0); //exit immediately for f=0

    double ls = 2.99792458e14 / f; //wavelength in microns
    ls *= ls; //only wavelength^2 is ever used
    hipDoubleComplex ii = make_hipDoubleComplex(0.0, 1.0);
    double omega = 6.28318530718 * abs(f);
    double kL = 3183.9; //(e * e / (epsilon_o * m_e)


    //option 0: isotropic
    if (type == 0) {
        ne[0] = sellmeierSubfunctionCuda(a, ls, omega, ii, kL);
        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {

        hipDoubleComplex na = sellmeierSubfunctionCuda(a, ls, omega, ii, kL);
        hipDoubleComplex nb = sellmeierSubfunctionCuda(&a[22], ls, omega, ii, kL);
        no[0] = na;
        ne[0] = 1.0 / cuCsqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return ne[0];
    }
    else {
        //type == 2: biaxial
        // X. Yin, S. Zhang and Z. Tian, Optics and Laser Technology 39 (2007) 510 - 513.
        // I am sorry if there is a bug and you're trying to find it, i did my best.
        hipDoubleComplex na = sellmeierSubfunctionCuda(a, ls, omega, ii, kL);
        hipDoubleComplex nb = sellmeierSubfunctionCuda(&a[22], ls, omega, ii, kL);
        hipDoubleComplex nc = sellmeierSubfunctionCuda(&a[44], ls, omega, ii, kL);
        double cosTheta = cos(theta);
        double cosTheta2 = cosTheta * cosTheta;
        double sinTheta = sin(theta);
        double sinTheta2 = sinTheta * sinTheta;
        double sinPhi = sin(phi);
        double sinPhi2 = sinPhi * sinPhi;
        double cosPhi = cos(phi);
        double cosPhi2 = cosPhi * cosPhi;
        double realna2 = hipCreal(na) * hipCreal(na);
        double realnb2 = hipCreal(nb) * hipCreal(nb);

        double delta = 0.5 * atan(-((1. / realna2 - 1. / realnb2)
            * sin(2 * phi) * cosTheta) / ((cosPhi2 / realna2 + sinPhi2 / realnb2)
                + ((sinPhi2 / realna2 + cosPhi2 / realnb2)
                    * cosTheta2 + sinTheta2 / (hipCreal(nc) * hipCreal(nc)))));

        ne[0] = 1.0 / cuCsqrt(cos(delta) * cos(delta) * (cosTheta2 * (cosPhi2 / (na * na)
            + sinPhi2 / (nb * nb)) + sinTheta2 / (nc * nc))
            + sin(delta) * sin(delta) * (sinPhi2 / (na * na) + cosPhi2 / (nb * nb))
            - 0.5 * sin(2 * phi) * cosTheta * sin(2 * delta) * (1. / (na * na) - 1. / (nb * nb)));

        no[0] = 1.0 / cuCsqrt(sin(delta) * sin(delta) * (cosTheta2 * (cosPhi2 / (na * na)
            + sinPhi2 / (nb * nb)) + sinTheta2 / (nc * nc))
            + cos(delta) * cos(delta) * (sinPhi2 / (na * na) + cosPhi2 / (nb * nb))
            + 0.5 * sin(2 * phi) * cosTheta * sin(2 * delta) * (1. / (na * na) - 1. / (nb * nb)));
        return ne[0];
    }
}
__global__ void millersRuleNormalizationKernel(cudaParameterSet s, double* sellmeierCoefficients, double* referenceFrequencies) {
    if (!s.isUsingMillersRule) {
        return;
    }
    size_t i;
    double chi11[7];
    double chi12[7];
    hipDoubleComplex ne, no;
    for (i = 0; i < 7; i++) {
        if (referenceFrequencies[i] == 0) {
            chi11[i] = 100000.0;
            chi12[i] = 100000.0;
        }
        else {
            sellmeierCuda(&ne, &no, sellmeierCoefficients, referenceFrequencies[i], sellmeierCoefficients[66], sellmeierCoefficients[67], (int)sellmeierCoefficients[69], 0);
            chi11[i] = hipCreal(ne) * hipCreal(ne) - 1;
            chi12[i] = hipCreal(no) * hipCreal(no) - 1;
        }

    }

    //normalize chi2 tensor values
    s.chi2Tensor[0] /= chi11[0] * chi11[1] * chi11[2];
    s.chi2Tensor[1] /= chi11[0] * chi11[1] * chi12[2];
    s.chi2Tensor[2] /= chi11[0] * chi12[1] * chi11[2];
    s.chi2Tensor[3] /= chi11[0] * chi12[1] * chi12[2];
    s.chi2Tensor[4] /= chi12[0] * chi12[1] * chi11[2];
    s.chi2Tensor[5] /= chi12[0] * chi12[1] * chi12[2];

    //normalize chi3 tensor values
    // note that currently full chi3 isn't implemented so
    // this only applies to the first element, chi3_1111 under
    // the assumption of centrosymmetry
    s.chi3Tensor[0] /= chi11[3] * chi11[4] * chi11[5] * chi11[6];

}

__device__ __forceinline__ double cuCModSquared(hipDoubleComplex a) {
    return a.x * a.x + a.y * a.y;
}
__global__ void totalSpectrumKernel(hipDoubleComplex* fieldGrid1, hipDoubleComplex* fieldGrid2, double gridStep, size_t Ntime, size_t Nspace, double* spectrum) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j;
    double beamCenter1 = 0.;
    double beamCenter2 = 0.;
    double beamTotal1 = 0.;
    double beamTotal2 = 0.;
    double a, x;

    //find beam centers
    for (j = 0; j < Nspace; j++) {
        x = gridStep * j;
        a = cuCModSquared(fieldGrid1[i + j * Ntime]);
        beamTotal1 += a;
        beamCenter1 += x * a;
        a = cuCModSquared(fieldGrid2[i + j * Ntime]);
        beamTotal2 += a;
        beamCenter2 += x * a;
    }
    if (beamTotal1 > 0) {
        beamCenter1 /= beamTotal1;
    }
    if (beamTotal2 > 0) {
        beamCenter2 /= beamTotal2;
    }
    

    //Integrate total beam power, assuming radially-symmetric beam around
    //the center
    beamTotal1 = 0.;
    beamTotal2 = 0.;
    for (j = 0; j < Nspace; j++) {
        x = gridStep * j;
        beamTotal1 += abs(x - beamCenter1) * cuCModSquared(fieldGrid1[i + j * Ntime]);
        beamTotal2 += abs(x - beamCenter2) * cuCModSquared(fieldGrid2[i + j * Ntime]);
    }
    beamTotal1 *= gridStep/Ntime;
    beamTotal2 *= gridStep/Ntime;

    //put the values into the output spectrum
    spectrum[i] = beamTotal1;
    spectrum[i + Ntime] = beamTotal2;
    spectrum[i + 2 * Ntime] = beamTotal1 + beamTotal2;
}

//rotate the field around the propagation axis (basis change)
__global__ void rotateFieldKernel(
    hipDoubleComplex* Ein1, hipDoubleComplex* Ein2, hipDoubleComplex* Eout1, 
    hipDoubleComplex* Eout2, double rotationAngle) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    Eout1[i] = cos(rotationAngle) * Ein1[i] - sin(rotationAngle) * Ein2[i];
    Eout2[i] = sin(rotationAngle) * Ein1[i] + cos(rotationAngle) * Ein2[i];
}

//provide a list of nearest-3 neighbors for taking spatial derivatives
// exploiting the fact that the radial grid is offset by 1/4 step from 0
// this means that midpoints are available on the other side of the origin.
// returns rho at the given index j
__device__ __forceinline__ double resolveNeighborsInOffsetRadialSymmetry(
    long long* neighbors, long long N, int j, double dr, long long Ntime, long long h) {
	if (j < N / 2) {
		neighbors[0] = (N - j - 2) * Ntime + h;
		neighbors[1] = (j + 1) * Ntime + h;
		neighbors[2] = (N - j - 1) * Ntime + h;
		neighbors[3] = (N - j) * Ntime + h;
		neighbors[4] = (j - 1) * Ntime + h;
		neighbors[5] = (N - j + 1) * Ntime + h;
		return -(dr * (j - N / 2) + 0.25 * dr);
	}
	else {
		neighbors[0] = (N - j + 1) * Ntime + h;
		neighbors[1] = (j - 1) * Ntime + h;
		neighbors[2] = (N - j) * Ntime + h;
		neighbors[3] = (N - j - 1) * Ntime + h;
		neighbors[4] = (j + 1) * Ntime + h;
		neighbors[5] = (N - j - 2) * Ntime + h;
		return dr * (j - N / 2) + 0.25 * dr;
	}
}

__global__ void radialLaplacianKernel(cudaParameterSet s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j = i / s.Ntime; //spatial coordinate
    long long h = i % s.Ntime; //temporal coordinate
    long long neighbors[6];
    
    //zero at edges of grid
    if (j<3 || j>(s.Nspace - 4)) {
        s.gridRadialLaplacian1[i] = make_hipDoubleComplex(0, 0);
        s.gridRadialLaplacian2[i] = make_hipDoubleComplex(0, 0);
    }
    else {
        double rho = resolveNeighborsInOffsetRadialSymmetry(neighbors, s.Nspace, j, s.dx, s.Ntime, h);
        rho = -1.0 / rho;
        s.gridRadialLaplacian1[i] = rho * (s.firstDerivativeOperation[0] * s.gridETime1[neighbors[0]]
            + s.firstDerivativeOperation[1] * s.gridETime1[neighbors[1]]
            + s.firstDerivativeOperation[2] * s.gridETime1[neighbors[2]]
            + s.firstDerivativeOperation[3] * s.gridETime1[neighbors[3]]
            + s.firstDerivativeOperation[4] * s.gridETime1[neighbors[4]]
            + s.firstDerivativeOperation[5] * s.gridETime1[neighbors[5]]);
        s.gridRadialLaplacian2[i] = rho * (s.firstDerivativeOperation[0] * s.gridETime2[neighbors[0]]
            + s.firstDerivativeOperation[1] * s.gridETime2[neighbors[1]]
            + s.firstDerivativeOperation[2] * s.gridETime2[neighbors[2]]
            + s.firstDerivativeOperation[3] * s.gridETime2[neighbors[3]]
            + s.firstDerivativeOperation[4] * s.gridETime2[neighbors[4]]
            + s.firstDerivativeOperation[5] * s.gridETime2[neighbors[5]]);
    }

}
//Expand the information contained in the radially-symmetric beam in the offset grid
// representation.
// The grid is offset from the origin; rather than ...-2 -1 0 1 2... etc, which would
// contain redundant information (the symmetry means that -1 and -1 are equivalent)
// the grid is at the points -1.75 -0.75 0.25 1.25 2.25, etc.
// the grid spacing is the same, but now the two sides of the origin contain different
// information. This has effectively doubled the resolution of the nonlinear
// polarization. 
// We make use of this by expanding into the full-resolution beam on the grid
// -2.25 -1.75 -1.25 -0.75 -0.25 0.25 0.75 1.25 1.75 2.25...
// after FFT, we can discard the high frequencies. Thus we have downsampled
// in such a way as to avoid aliasing, which inside the simulation is most
// likely the appear (and cause instability) in the nonlinear terms.
__global__ void expandCylindricalBeam(cudaParameterSet s, double* polarization1, double* polarization2) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j = i / s.Ntime; //spatial coordinate
    size_t k = i % s.Ntime; //temporal coordinate

    //positions on the expanded grid corresponding the the current index
    size_t pos1 = 2 * (s.Nspace - j - 1) * s.Ntime + k;
    size_t pos2 = (2 * j + 1) * s.Ntime + k;

    //reuse memory allocated for the radial Laplacian, casting complex double
    //to a 2x larger double real grid
    double* expandedBeam1 = (double*)s.gridRadialLaplacian1;
    double* expandedBeam2 = expandedBeam1 + 2 * s.Ngrid;

    expandedBeam1[pos1] = polarization1[i];
    expandedBeam1[pos2] = polarization1[i];
    expandedBeam2[pos1] = polarization2[i];
    expandedBeam2[pos2] = polarization2[i];
}
__device__ void findBirefingentCrystalAngle(double* alphaE, double* alphaO, long long j, double f, double* sellmeierCoefficients, cudaParameterSet s) {
    //Find walkoff angle, starting from zero
    // in the case of an extraordinary axis, the angle of propagation is related to the transverse
    // momentum in a complicated way:
    // sin(theta) * n(theta) = delta k * c/omega
    // theta depends on the refractive index, and the refractive index depends on theta
    // so we solve numerically
    double dAlpha = 0.1;
    double nePlus, neMinus;
    double err, errPlus, errMinus;
    hipDoubleComplex ne, no;
    double twoPi = 6.283185307179586;
    
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double tol = sellmeierCoefficients[72];
    double dk = j * kStep - (j >= (s.Nspace / 2)) * (kStep * s.Nspace); //frequency grid in transverse direction
    double rhs = 2.99792458e8 * dk / (twoPi * f);

    //if not biaxial, the o-axis can be solved analytically.
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
        crystalTheta, crystalPhi, s.axesNumber, s.sellmeierType);
    *alphaO = asin(rhs / hipCreal(no));
    if(s.axesNumber == 2){
        sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
            crystalTheta + *alphaO, crystalPhi, s.axesNumber, s.sellmeierType);
        nePlus = hipCreal(no);
        err = abs(nePlus * sin(*alphaO) - rhs);

        int iters = 0;
        errPlus = 2;
        errMinus = 2;
        while (err > tol && iters < 2048) {
            iters++;

            sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
                crystalTheta + *alphaO + dAlpha, crystalPhi, s.axesNumber, s.sellmeierType);
            nePlus = hipCreal(no);
            errPlus = abs(nePlus * sin(*alphaO + dAlpha) - rhs);

            sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
                crystalTheta + *alphaO - dAlpha, crystalPhi, s.axesNumber, s.sellmeierType);
            neMinus = hipCreal(no);
            errMinus = abs(neMinus * sin(*alphaO - dAlpha) - rhs);

            //Basic hill climbing algorithm
            //calculate the error at theta +/- dTheta
            // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
            // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
            // if theta has lowest error, step size is too large, dTheta /= 2;
            if (errPlus < err && errPlus < errMinus) {
                *alphaO += dAlpha;
                err = errPlus;
            }
            else if (errMinus < err) {
                *alphaO -= dAlpha;
                err = errMinus;
            }
            else {
                dAlpha *= 0.5;
            }

        }
    }

    //find the extraordinary angle if the crystal isn't isotropic
    *alphaE = *alphaO;
    if (s.axesNumber > 0) {
        sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
            crystalTheta + *alphaE, crystalPhi, s.axesNumber, s.sellmeierType);
        nePlus = hipCreal(ne);
        err = abs(nePlus * sin(*alphaE) - rhs);

        int iters = 0;
        errPlus = 2;
        errMinus = 2;
        dAlpha = 0.1;
        while (err > tol && iters < 2048) {
            iters++;

            sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
                crystalTheta + *alphaE + dAlpha, crystalPhi, s.axesNumber, s.sellmeierType);
            nePlus = hipCreal(ne);
            errPlus = abs(nePlus * sin(*alphaE + dAlpha) - rhs);

            sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f),
                crystalTheta + *alphaE - dAlpha, crystalPhi, s.axesNumber, s.sellmeierType);
            neMinus = hipCreal(ne);
            errMinus = abs(neMinus * sin(*alphaE - dAlpha) - rhs);

            //Basic hill climbing algorithm
            //calculate the error at theta +/- dTheta
            // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
            // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
            // if theta has lowest error, step size is too large, dTheta /= 2;
            if (errPlus < err && errPlus < errMinus) {
                *alphaE += dAlpha;
                err = errPlus;
            }
            else if (errMinus < err) {
                *alphaE -= dAlpha;
                err = errMinus;
            }
            else {
                dAlpha *= 0.5;
            }

        }
    }
    
    
}


//prepare propagation constants for the simulation, when it is taking place on a Cartesian grid
//note that the sellmeier coefficients have extra values appended to the end
//to give info about the current simulation
__global__ void applyFresnelLossKernel(double* sellmeierCoefficients1, double* sellmeierCoefficients2, cudaParameterSet s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    double alpha1, alpha2, alphaO1, alphaO2;
    long long j, k;
    long long Ntime = s.Ntime;
    int axesNumber = s.axesNumber;
    int sellmeierType = s.sellmeierType;
    hipDoubleComplex ne1, no1, ne2, no2, n0;
    hipDoubleComplex cuZero = make_hipDoubleComplex(0, 0);
    j = i / Ntime; //spatial coordinate
    k = i % Ntime; //temporal coordinate
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients1[66];
    double crystalPhi = sellmeierCoefficients1[67];
    double fStep = sellmeierCoefficients1[71];

    //frequency being resolved by current thread
    double f = k * fStep;
    if (k >= Ntime / 2) {
        f -= fStep * Ntime;
    }
    f *= -1;

    findBirefingentCrystalAngle(&alpha1, &alphaO1, j, f, sellmeierCoefficients1, s);
    findBirefingentCrystalAngle(&alpha2, &alphaO2, j, f, sellmeierCoefficients2, s);
    //walkoff angle has been found, generate the rest of the grids


    sellmeierCuda(&ne1, &no1, sellmeierCoefficients1, abs(f),
        crystalTheta + alpha1, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&n0, &no1, sellmeierCoefficients1, abs(f),
        crystalTheta + alphaO1, crystalPhi, axesNumber, sellmeierType);
    if (isnan(hipCreal(ne1)) || isnan(hipCreal(no1))) {
        ne1 = make_hipDoubleComplex(1, 0);
        no1 = make_hipDoubleComplex(1, 0);
    }


    sellmeierCuda(&ne2, &no2, sellmeierCoefficients2, abs(f),
        crystalTheta + alpha2, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&n0, &no2, sellmeierCoefficients2, abs(f),
        crystalTheta + alphaO2, crystalPhi, axesNumber, sellmeierType);
    if (isnan(hipCreal(ne2)) || isnan(hipCreal(no2))) {
        ne2 = make_hipDoubleComplex(1, 0);
        no2 = make_hipDoubleComplex(1, 0);
    }

    hipDoubleComplex ts = 2 * ne1 * cos(alpha1) / (ne1 * cos(alpha1) + ne2 * cos(alpha2));
    hipDoubleComplex tp = 2 * ne1 * cos(alpha1) / (ne2 * cos(alpha1) + ne1 * cos(alpha2));
    if (isnan(ts.x) || isnan(ts.y)) ts = make_hipDoubleComplex(0, 0);
    if (isnan(tp.x) || isnan(tp.y)) ts = make_hipDoubleComplex(0, 0);
    s.gridEFrequency1[i] = ts * s.gridEFrequency1[i];
    s.gridEFrequency2[i] = tp * s.gridEFrequency2[i];
}

//prepare propagation constants for the simulation, when it is taking place on a Cartesian grid
//note that the sellmeier coefficients have extra values appended to the end
//to give info about the current simulation
__global__ void prepareCartesianGridsKernel(double* sellmeierCoefficients, cudaParameterSet s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    double alpha, alphaO;
    long long j, k;
    long long Ntime = s.Ntime;
    double twoPi = 6.283185307179586;
    int axesNumber = s.axesNumber;
    int sellmeierType = s.sellmeierType;
    double c = 2.99792458e8; //speed of light
    hipDoubleComplex ne, no, n0;
    hipDoubleComplex cuZero = make_hipDoubleComplex(0, 0);
    j = i / Ntime; //spatial coordinate
    k = i % Ntime; //temporal coordinate
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double fStep = sellmeierCoefficients[71];

    //frequency being resolved by current thread
    double f = k * fStep;
    if (k >= Ntime / 2) {
        f -= fStep * Ntime;
    }
    f *= -1;

    //transverse wavevector being resolved
    double dk = j * kStep - (j >= (s.Nspace / 2)) * (kStep * s.Nspace); //frequency grid in transverse direction
    
    findBirefingentCrystalAngle(&alpha, &alphaO, j, f, sellmeierCoefficients, s);

    //walkoff angle has been found, generate the rest of the grids

    sellmeierCuda(&n0, &no, sellmeierCoefficients, abs(s.f0), 
        crystalTheta, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), 
        crystalTheta + alpha, crystalPhi, axesNumber, sellmeierType);
    if (s.axesNumber == 2) {
        hipDoubleComplex nDummy;
        sellmeierCuda(&nDummy, &no, sellmeierCoefficients, abs(f),
            crystalTheta + alphaO, crystalPhi, axesNumber, sellmeierType);
    }
    if (isnan(hipCreal(ne)) || isnan(hipCreal(no))) {
        ne = make_hipDoubleComplex(1, 0);
        no = make_hipDoubleComplex(1, 0);
    }
    s.ne[i] = ne;
    s.no[i] = no;

    hipDoubleComplex k0 = make_hipDoubleComplex(twoPi * hipCreal(n0) * f / c, 0);
    hipDoubleComplex ke = twoPi * ne * f / c;
    hipDoubleComplex ko = twoPi * no * f / c;

    if (s.isUsingMillersRule) {
        s.chiLinear1[i] = -1. + ne * ne;
        s.chiLinear2[i] = -1. + no * no;
        if ((hipCreal(s.chiLinear1[i]) == 0) || (hipCreal(s.chiLinear2[i]) == 0) || isnan(hipCreal(s.chiLinear1[i])) || isnan(hipCreal(s.chiLinear2[i]))) {
            s.chiLinear1[i] = make_hipDoubleComplex(1, 0);
            s.chiLinear2[i] = make_hipDoubleComplex(1, 0);
        }
    }
    else {
        s.chiLinear1[i] = make_hipDoubleComplex(1, 0);
        s.chiLinear2[i] = make_hipDoubleComplex(1, 0);
    }

    if (hipCreal(ke) < 0 && hipCreal(ko) < 0) {
        s.gridPropagationFactor1[i] = ii * (ke - k0 - dk * dk / (2. * hipCreal(ke))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor1[i]))) {
            s.gridPropagationFactor1[i] = cuZero;
        }

        s.gridPropagationFactor2[i] = ii * (ko - k0 - dk * dk / (2. * hipCreal(ko))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor2[i]))) {
            s.gridPropagationFactor2[i] = cuZero;
        }

        s.gridPolarizationFactor1[i] = ii * s.chiLinear1[i] * (twoPi * f) / (2. * hipCreal(ne) * c) * s.h;
        s.gridPolarizationFactor2[i] = ii * s.chiLinear2[i] * (twoPi * f) / (2. * hipCreal(no) * c) * s.h;
    }

    else {
        s.gridPropagationFactor1[i] = cuZero;
        s.gridPropagationFactor2[i] = cuZero;
        s.gridPolarizationFactor1[i] = cuZero;
        s.gridPolarizationFactor2[i] = cuZero;
    }
}
    
//prepare the propagation constants under the assumption of cylindrical symmetry of the beam
__global__ void prepareCylindricGridsKernel(double* sellmeierCoefficients, cudaParameterSet s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j, k;
    long long Ntime = s.Ntime;
    long long Nspace = s.Nspace;
    int axesNumber = s.axesNumber;
    int sellmeierType = s.sellmeierType;
    double c = 2.99792458e8; //speed of light
    double twoPi = 2 * 3.14159265358979323846264338327950288;
    hipDoubleComplex cuZero = make_hipDoubleComplex(0, 0);
    j = i / Ntime; //spatial coordinate
    k = i % Ntime; //temporal coordinate
    hipDoubleComplex ii = make_hipDoubleComplex(0, 1);
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double fStep = sellmeierCoefficients[71];

    hipDoubleComplex ne, no, n0;

    //frequency being resolved by current thread
    double f = k * fStep;
    if (k >= Ntime / 2) {
        f -= fStep * Ntime;
    }
    f *= -1;

    //transverse wavevector being resolved
    double dk = j * kStep - (j >= (Nspace / 2)) * (kStep * Nspace); //frequency grid in transverse direction
    sellmeierCuda(&n0, &no, sellmeierCoefficients, abs(s.f0), crystalTheta, crystalPhi, axesNumber, sellmeierType);
    sellmeierCuda(&ne, &no, sellmeierCoefficients, abs(f), crystalTheta, crystalPhi, axesNumber, sellmeierType);
    if (isnan(hipCreal(ne)) || isnan(hipCreal(no))) {
        ne = make_hipDoubleComplex(1, 0);
        no = make_hipDoubleComplex(1, 0);
    }
    s.ne[i] = ne;
    s.no[i] = no;

    hipDoubleComplex k0 = make_hipDoubleComplex(twoPi * hipCreal(n0) * f / c, 0);
    hipDoubleComplex ke = twoPi * ne * f / c;
    hipDoubleComplex ko = twoPi * no * f / c;

    if (s.isUsingMillersRule) {
        s.chiLinear1[i] = -1. + ne * ne;
        s.chiLinear2[i] = -1. + no * no;
        if ((hipCreal(s.chiLinear1[i]) == 0) || (hipCreal(s.chiLinear2[i]) == 0) || isnan(hipCreal(s.chiLinear1[i])) || isnan(hipCreal(s.chiLinear2[i]))) {
            s.chiLinear1[i] = make_hipDoubleComplex(1, 0);
            s.chiLinear2[i] = make_hipDoubleComplex(1, 0);
        }
    }
    else {
        s.chiLinear1[i] = make_hipDoubleComplex(1, 0);
        s.chiLinear2[i] = make_hipDoubleComplex(1, 0);
    }

    if (hipCreal(ke) < 0 && hipCreal(ko) < 0 && abs(dk) < hipCabs(ke)) {
        s.gridPropagationFactor1[i] = ii * (ke - k0 - dk * dk / (2. * hipCreal(ke))) * s.h;
        s.gridPropagationFactor1Rho1[i] = ii * (1 / (s.chiLinear1[i] *2. * hipCreal(ke))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor1[i]))) {
            s.gridPropagationFactor1[i] = cuZero;
            s.gridPropagationFactor1Rho1[i] = cuZero;
        }

        s.gridPropagationFactor2[i] = ii * (ko - k0 - dk * dk / (2. * hipCreal(ko))) * s.h;
        s.gridPropagationFactor1Rho2[i] = ii * (1 / (s.chiLinear2[i] * 2. * hipCreal(ko))) * s.h;
        if (isnan(hipCreal(s.gridPropagationFactor2[i]))) {
            s.gridPropagationFactor2[i] = cuZero;
            s.gridPropagationFactor1Rho2[i] = cuZero;
        }
        //factor of 0.5 comes from doubled grid size in cylindrical symmetry mode after expanding the beam
        s.gridPolarizationFactor1[i] = 0.5 * s.chiLinear1[i] * ii * (twoPi * f) / (2. * hipCreal(ne) * c) * s.h;
        s.gridPolarizationFactor2[i] = 0.5 * s.chiLinear2[i] * ii * (twoPi * f) / (2. * hipCreal(no) * c) * s.h;


    }

    else {
        s.gridPropagationFactor1[i] = cuZero;
        s.gridPropagationFactor2[i] = cuZero;
        s.gridPolarizationFactor1[i] = cuZero;
        s.gridPolarizationFactor2[i] = cuZero;
        s.gridPropagationFactor1[i] = cuZero;
        s.gridPropagationFactor1Rho2[i] = cuZero;
    }


}

//replaces E with its complex conjugate
__global__ void conjugateKernel(hipDoubleComplex* E) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    E[i] = hipConj(E[i]);
}

__global__ void materialPhaseKernel(double df, size_t Ntime, double* a, double f01, double f02, double thickness1,  double thickness2, double* phase1, double* phase2) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    //frequency being resolved by current thread
    double f = i * df;
    if (i >= Ntime / 2) {
        f -= df * Ntime;
    }
    
    //give phase shift relative to group velocity (approximated 
    // with low-order finite difference) so the pulse doesn't move
    hipDoubleComplex ne, no, no0, n0p, n0m;
    sellmeierCuda(&ne, &no, a, abs(f), 0, 0, 0, 0);
    f *= -6.28318530718;
    sellmeierCuda(&ne, &no0, a, f01, 0, 0, 0, 0);
    sellmeierCuda(&ne, &n0p, a, f01+1e11, 0, 0, 0, 0);
    sellmeierCuda(&ne, &n0m, a, f01-1e11, 0, 0, 0, 0);
    no0 = no0 + f01 * (n0p - n0m) / 2e11;
    phase1[i] = thickness1 * f * hipCreal(no - no0) / 2.99792458e8;
    sellmeierCuda(&ne, &no0, a, f02, 0, 0, 0, 0);
    sellmeierCuda(&ne, &n0p, a, f02 + 1e11, 0, 0, 0, 0);
    sellmeierCuda(&ne, &n0m, a, f02 - 1e11, 0, 0, 0, 0);
    no0 = no0 + f02 * (n0p - n0m) / 2e11;
    phase2[i] = thickness2 * f * hipCreal(no - no0) / 2.99792458e8;
    
}
//replaces NaN values with 0
__global__ void fixnanKernel(hipDoubleComplex* E) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (isnan(hipCreal(E[i])) || isnan(hipCimag(E[i]))) {
        E[i] = make_hipDoubleComplex(0., 0.);
    }
}

//calculate the nonlinear polarization, after FFT to get the field
//in the time domain
__global__ void nonlinearPolarizationKernel(cudaParameterSet s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    double Ex = 2 * hipCreal(s.gridETime1[i]) / s.propagationInts[0];
    double Ey = 2 * hipCreal(s.gridETime2[i]) / s.propagationInts[0];

    double Ex2 = Ex * Ex;
    double Ey2 = Ey * Ey;
    s.gridPolarizationTime1[i] = 0.;
    s.gridPolarizationTime2[i] = 0.;

    //The d2eff tensor has the form
    // | d_xxx d_xyx d_yyx |
    // | d_xxy d_xyy d_yyy |
    if (s.nonlinearSwitches[0] == 1) {
        s.gridPolarizationTime1[i] += s.chi2Tensor[0] * Ex2 + s.chi2Tensor[2] * Ex * Ey + s.chi2Tensor[4] * Ey2;
        s.gridPolarizationTime2[i] += s.chi2Tensor[1] * Ex2 + s.chi2Tensor[3] * Ex * Ey + s.chi2Tensor[5] * Ey2;
    }
    
    //to be implemented: full chi3 matrix on s.nonlinearSwitches[1]==1

    //using only one value of chi3, under assumption of centrosymmetry
    if (s.nonlinearSwitches[1] == 2) {
        s.gridPolarizationTime1[i] += s.chi3Tensor[0] * (Ex2 * Ex + (1. / 3.) * Ey2 * Ex);
        s.gridPolarizationTime2[i] += s.chi3Tensor[0] * (Ey2 * Ey + (1. / 3.) * Ex2 * Ey);
    }
}


//Plasma response with time-dependent carrier density
//This polarization needs a different factor in the nonlinear wave equation
//to account for the integration
//plasmaParameters vector:
// 0    e^2/m_eff
// 1    gamma_drude
// 2    ionization rate/E^N
// 3    absorption strength
//equation for the plasma current:
//J_drude(t) = (e/m)*exp(-gamma*t)*\int_-infty^t dt' exp(gamma*t)*N(t)*E(t)
//J_absorption(t) = beta*E^(2*Nphot-2)*E
__global__ void plasmaCurrentKernelPrep(cudaParameterSet s, double* workN, double* workEx) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;

    int k;
    double* workEy = &workEx[s.Ngrid];
    double w, Esquared, Ex, Ey;
    Ex = hipCreal(s.gridETime1[i]) / s.propagationInts[0];
    Ey = hipCreal(s.gridETime2[i]) / s.propagationInts[0];
    Esquared = Ex * Ex + Ey * Ey;
    //plasmaParameters[0] is the nonlinear absorption parameter
    w = s.plasmaParameters[0] * Esquared;
    //nonlinearSwitches[3] is Nphotons-2
    for (k = 0; k < s.nonlinearSwitches[3]; k++) {
        w *= Esquared;
    }
    //absorption currents
    s.gridPlasmaCurrent1[i] = w * Ex;
    s.gridPlasmaCurrent2[i] = w * Ey;

    //plasmaParameters[2] is the 1/photon energy, translating the loss of power
    //from the field to the number of free carriers
    //extra factor of (dt^2e^2/(m*photon energy*eo) included as it is needed for the amplitude
    //of the plasma current
    workN[i] = s.plasmaParameters[2] * (s.gridPlasmaCurrent1[i] * Ex + s.gridPlasmaCurrent2[i] * Ey);
    workEx[i] = Ex;
    workEy[i] = Ey;

}
__global__ void plasmaCurrentKernel2(cudaParameterSet s, double* workN, double* workEx) {
    long long j = threadIdx.x + blockIdx.x * blockDim.x;
    double N = 0;
    double integralx = 0;
    double integraly = 0;
    double* workEy = &workEx[s.Ngrid];
    double* expMinusGammaT = &s.expGammaT[s.Ntime];

    long long k, l;
    j *= s.Ntime;
    for (k = 0; k < s.Ntime; k++) {

        l = j + k;
        N += workN[l];

        integralx += s.expGammaT[k] * N * workEx[l];
        integraly += s.expGammaT[k] * N * workEy[l];


        s.gridPlasmaCurrent1[l] += expMinusGammaT[k] * integralx;
        s.gridPlasmaCurrent2[l] += expMinusGammaT[k] * integraly;
    }
}


//Main kernel for RK4 propagation of the field
__global__ void rkKernel(cudaParameterSet s, int stepNumber) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j = i / s.Ntime; //spatial coordinate
    long long h = i % s.Ntime; //temporal coordinate
    hipDoubleComplex plasmaJ1 = make_hipDoubleComplex(0, 0);
    hipDoubleComplex plasmaJ2 = make_hipDoubleComplex(0, 0);

    //note that the FFT of the radial laplacian is stored in k1 and k2
    //so that memory shouldn't be used for anything else
    if (s.isCylindric) {
        s.gridRadialLaplacian1[i] = s.gridPropagationFactor1Rho1[i] * s.k1[i];
        s.gridRadialLaplacian2[i] = s.gridPropagationFactor1Rho2[i] * s.k2[i];

        //correct for double grid size after expandCylindricalBeam()
        if (j > s.Nspace / 2) {
            j += s.Nspace;
        }
    }

    if (s.hasPlasma) {
        double f = h * s.fStep;
        long long hp = h;
        if (h >= s.Ntime / 2) {
            f -= s.fStep * s.Ntime;
        }
        f *= -6.28318530718;
        hipDoubleComplex jfac = make_hipDoubleComplex(0, 1.0 / f);
        if (h > s.propagationInts[3]) {
            hp = s.Ntime - hp;
            j = s.Nspace - j;
            if (s.isCylindric) j += s.Nspace;
            hp += j * s.propagationInts[3];

            if (f != 0) {
                plasmaJ1 = jfac * s.gridPolarizationFactor1[i] * hipConj(s.gridPlasmaCurrentFrequency1[hp]);
                plasmaJ2 = jfac * s.gridPolarizationFactor2[i] * hipConj(s.gridPlasmaCurrentFrequency2[hp]);
            }
        }
        else {
            hp += j * s.propagationInts[3];
            if (f != 0) {
                plasmaJ1 = jfac * s.gridPolarizationFactor1[i] * s.gridPlasmaCurrentFrequency1[hp];
                plasmaJ2 = jfac * s.gridPolarizationFactor2[i] * s.gridPlasmaCurrentFrequency2[hp];
            }
        }
        //correct for the presence of millers rule dispersion in gridPolarizationFactor
        plasmaJ1 = plasmaJ1 / s.chiLinear1[i];
        plasmaJ2 = plasmaJ2 / s.chiLinear2[i];

    }


    //polarization is stored in a reduced format by cuFFT because the FFT is from real to complex, meaning if the output grid
    //were to be N_time x N_space, half of the points would be redundant. The extra steps below are to determine where in the grid the 
    //current point sits. Essentially, if in the negative frequency quadrants, reverse the frequency and take complex conjugate of the 
    //value
    if (h > s.propagationInts[3]) {
        h = s.Ntime - h;
        j = s.Nspace - j;
        if (s.isCylindric) j += s.Nspace;
        h += j * s.propagationInts[3];

        s.k1[i] = s.gridPropagationFactor1[i] * s.gridETemp1[i] +s.gridPolarizationFactor1[i] * hipConj(s.gridPolarizationFrequency1[h]);
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * hipConj(s.gridPolarizationFrequency2[h]);
    }
    else {
        h += j * s.propagationInts[3];

        s.k1[i] = s.gridPropagationFactor1[i] * s.gridETemp1[i] +s.gridPolarizationFactor1[i] * s.gridPolarizationFrequency1[h];
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * s.gridPolarizationFrequency2[h];
    }
    if (s.isCylindric) {
        s.k1[i] = s.k1[i] + s.gridRadialLaplacian1[i];
        s.k2[i] = s.k2[i] + s.gridRadialLaplacian2[i];
    }

    if (s.hasPlasma) {
        s.k1[i] = s.k1[i] + plasmaJ1;
        s.k2[i] = s.k2[i] + plasmaJ2;
    }

    //in the first substep, first construct the next intermediate field value
    //which will be used in the next substep. 
    if (stepNumber == 0) {
        s.gridETemp1[i] = s.gridEFrequency1[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];
       
        s.gridEFrequency1Next1[i] = s.k1[i] / 6 + s.gridEFrequency1[i];
        s.gridEFrequency1Next2[i] = s.k2[i] / 6 + s.gridEFrequency2[i];
    }

    //in the next substep, again construct the next intermediate field and add k/3 to solution
    else if (stepNumber == 1) {
        s.gridETemp1[i] = s.gridEFrequency1[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];

        s.gridEFrequency1Next1[i] = s.gridEFrequency1Next1[i] + s.k1[i] / 3;
        s.gridEFrequency1Next2[i] = s.gridEFrequency1Next2[i] + s.k2[i] / 3;

    }

    //same action as previous substep, except the weight of k in the intermediate solution is 1 instead of 0.5
    else if (stepNumber == 2) {
        s.gridETemp1[i] = s.gridEFrequency1[i] + s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + s.k2[i];
        s.gridEFrequency1Next1[i] = s.gridEFrequency1Next1[i] + s.k1[i] / 3;
        s.gridEFrequency1Next2[i] = s.gridEFrequency1Next2[i] + s.k2[i] / 3;
    }

    //last substep. Solution is now complete and may be copied directly into the field arrays
    else {
        s.gridEFrequency1[i] = s.gridEFrequency1Next1[i] + s.k1[i] / 6;
        s.gridEFrequency2[i] = s.gridEFrequency1Next2[i] + s.k2[i] / 6;
        s.gridETemp1[i] = s.gridEFrequency1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i];
    }

}

//Take absolute value of complex array
__global__ void absKernel(double* absOut, hipDoubleComplex* complexIn) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    absOut[i] = hipCabs(complexIn[i]);
}

//Apply fft normalization
__global__ void fftNormalizeKernel(hipDoubleComplex* A, long long* fftSize) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = A[i] / fftSize[0];
}

//element-wise B*A = C;
__global__ void multiplicationKernel(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    C[i] = B[i] * A[i];
}

//main function for running on CLI
int main(int argc, char *argv[]) {
    int i, j;
    int CUDAdevice;
    int CUDAdeviceCount = 0;
    hipGetDeviceCount(&CUDAdeviceCount);
    hipError_t cuErr = hipGetDevice(&CUDAdevice);
    struct hipDeviceProp_t activeCUDADeviceProp;
    if (cuErr == hipSuccess) {
        printf("Found %i GPU(s): \n", CUDAdeviceCount);
        for (i = 0; i < CUDAdeviceCount; i++) {
            cuErr = hipGetDeviceProperties(&activeCUDADeviceProp, CUDAdevice);
            printf("%s\r\n", activeCUDADeviceProp.name);
            printf(" Memory: %lli MB; Multiprocessors: %i\n", 
                activeCUDADeviceProp.totalGlobalMem / (1024 * 1024), activeCUDADeviceProp.multiProcessorCount);
        }
    }
    else {
        printf("No GPU found.\n");
        return 1;
    }
    
    if (argc < 2) {
        printf("no input file specified.\n");
        return 2;
    }

    // allocate databases, main structs
    simulationParameterSet* sCPU = (simulationParameterSet*)calloc(512, sizeof(simulationParameterSet));
    crystalEntry* crystalDatabasePtr = (crystalEntry*)calloc(512, sizeof(crystalEntry));
    (*sCPU).crystalDatabase = crystalDatabasePtr;

    // read crystal database
    if (readCrystalDatabase(crystalDatabasePtr) == -2) {
        return 11;
    }
    if ((*crystalDatabasePtr).numberOfEntries == 0) {
        printf("Could not read crystal database.\n");
        free(sCPU);
        free(crystalDatabasePtr);
        return 12;
    }
    printf("Read %i crystal database entries:\n", (*crystalDatabasePtr).numberOfEntries);
    for (j = 0; j < (*crystalDatabasePtr).numberOfEntries; j++) {
        printf("Material %i name: %ls", j, crystalDatabasePtr[j].crystalNameW);
    }
  
    // read from settings file
    if (readInputParametersFile(sCPU, crystalDatabasePtr, argv[1]) == 1) {
        printf("Could not read input file.\n");
        free(sCPU);
        free(crystalDatabasePtr);
        return 13;
    }

    allocateGrids(sCPU);
    if (loadPulseFiles(sCPU) == 1) {
        printf("Could not read pulse file.\n");
        free((*sCPU).refractiveIndex1);
        free((*sCPU).refractiveIndex2);
        free((*sCPU).imdone);
        free((*sCPU).deffTensor);
        free((*sCPU).loadedField1);
        free((*sCPU).loadedField2);
        free(sCPU);
        free(crystalDatabasePtr);
        return 14;
    }

    readSequenceString(sCPU);
    printf("Found %i steps in sequence\n", (*sCPU).Nsequence);
    readFittingString(sCPU);
    configureBatchMode(sCPU);

    auto simulationTimerBegin = std::chrono::high_resolution_clock::now();

    // run simulations
    if ((*sCPU).isInFittingMode) {
        if ((*sCPU).fittingMode == 3) {
            if (loadReferenceSpectrum((*sCPU).fittingPath, sCPU)) {
                printf("Could not load reference spectrum!\n");
                free((*sCPU).refractiveIndex1);
                free((*sCPU).refractiveIndex2);
                free((*sCPU).imdone);
                free((*sCPU).deffTensor);
                free((*sCPU).loadedField1);
                free((*sCPU).loadedField2);
                free((*sCPU).Ext);
                free((*sCPU).Ekw);
                free((*sCPU).ExtOut);
                free((*sCPU).EkwOut);
                free((*sCPU).totalSpectrum);
                free(sCPU);
                free(crystalDatabasePtr);
                return 10;
            }
        }
        printf("Running in fitting mode -- I don't know how long this will take!\n");
        runFitting(sCPU);
        
        auto simulationTimerEnd = std::chrono::high_resolution_clock::now();
        printf("Finished after %8.4lf s. \n",
            1e-6 * (double)(std::chrono::duration_cast<std::chrono::microseconds>(simulationTimerEnd - simulationTimerBegin).count()));
        
        saveDataSet(sCPU, crystalDatabasePtr, (*sCPU).outputBasePath, FALSE);
        //free
        free((*sCPU).refractiveIndex1);
        free((*sCPU).refractiveIndex2);
        free((*sCPU).imdone);
        free((*sCPU).deffTensor);
        free((*sCPU).loadedField1);
        free((*sCPU).loadedField2);
        free((*sCPU).Ext);
        free((*sCPU).Ekw);
        free((*sCPU).ExtOut);
        free((*sCPU).EkwOut);
        free((*sCPU).totalSpectrum);
        free(sCPU);
        free(crystalDatabasePtr);

        return 0;
    }
    std::thread *threadBlock = (std::thread*)calloc((*sCPU).Nsims, sizeof(std::thread));
    size_t maxThreads = min(CUDAdeviceCount, (*sCPU).Nsims);
    for (j = 0; j < (*sCPU).Nsims; j++) {

        sCPU[j].assignedGPU = j % CUDAdeviceCount;
        if (j >= maxThreads) {
            if (threadBlock[j - maxThreads].joinable()) {
                threadBlock[j - maxThreads].join();
            }
        }

        if ((*sCPU).isInSequence) {
            threadBlock[j] = std::thread(solveNonlinearWaveEquationSequence, &sCPU[j]);
        }
        else {
            threadBlock[j] = std::thread(solveNonlinearWaveEquation, &sCPU[j]);
        }
    }
    
	for (i = 0; i < (*sCPU).Nsims; i++) {
        if (sCPU[i].memoryError > 0) {
            printf("Warning: device memory error (%i).\n", sCPU[i].memoryError);
        }
		if (threadBlock[i].joinable()) {
			threadBlock[i].join();
		}
	}
    
    auto simulationTimerEnd = std::chrono::high_resolution_clock::now();
    printf("Finished after %8.4lf s. \n", 
        1e-6 * (double)(std::chrono::duration_cast<std::chrono::microseconds>(simulationTimerEnd - simulationTimerBegin).count()));


    saveDataSet(sCPU, crystalDatabasePtr, (*sCPU).outputBasePath, FALSE);
    //free
    free(threadBlock);
    free((*sCPU).refractiveIndex1);
    free((*sCPU).refractiveIndex2);
    free((*sCPU).imdone);
    free((*sCPU).deffTensor);
    free((*sCPU).loadedField1);
    free((*sCPU).loadedField2);
    free((*sCPU).Ext);
    free((*sCPU).Ekw);
    free((*sCPU).ExtOut);
    free((*sCPU).EkwOut);
    free((*sCPU).totalSpectrum);
    free(sCPU);
    free(crystalDatabasePtr);
    return 0;
}

unsigned long solveNonlinearWaveEquationSequence(void* lpParam) {
    simulationParameterSet* sCPU = (simulationParameterSet*)lpParam;
    simulationParameterSet* sCPUbackup = (simulationParameterSet*)calloc(1, sizeof(simulationParameterSet));
    memcpy(sCPUbackup, sCPU, sizeof(simulationParameterSet));
    int k;
    for (k = 0; k < (*sCPU).Nsequence; k++) {
        resolveSequence(k, sCPU, (*sCPU).crystalDatabase);
        memcpy(sCPU, sCPUbackup, sizeof(simulationParameterSet));
    }
    free(sCPUbackup);
    return 0;
}
//main thread of the nonlinear wave equation implemented on CUDA
unsigned long solveNonlinearWaveEquation(void* lpParam) {

    //the struct s contains most of the simulation variables and pointers
    cudaParameterSet s;
    simulationParameterSet* sCPU = (simulationParameterSet*)lpParam;
    hipSetDevice((*sCPU).assignedGPU);
    hipStreamCreate(&s.CUDAStream);

    //initialize and take values from the struct handed over by the dispatcher
    unsigned long long i;
    s.Ntime = (*sCPU).Ntime;
    s.Nspace = (*sCPU).Nspace;
    s.dt = (*sCPU).tStep;
    s.dx = (*sCPU).rStep;
    s.fStep = (*sCPU).fStep;
    s.Nsteps = (size_t)round((*sCPU).crystalThickness / (*sCPU).propagationStep);
    s.h = (*sCPU).crystalThickness / (s.Nsteps); //adjust step size so that thickness can be varied continuously by fitting
    s.Ngrid = s.Ntime * s.Nspace;
    s.axesNumber = (*sCPU).axesNumber;
    s.sellmeierType = (*sCPU).sellmeierType;
    s.f0 = (*sCPU).frequency1;
    s.Nthread = THREADS_PER_BLOCK;
    s.Nblock = (int)(s.Ngrid / THREADS_PER_BLOCK);
    s.isCylindric =(*sCPU).isCylindric;
    s.isNonLinear = ((*sCPU).nonlinearSwitches[0] + (*sCPU).nonlinearSwitches[1]) > 0;
    s.isUsingMillersRule = ((*sCPU).crystalDatabase[(*sCPU).materialIndex].nonlinearReferenceFrequencies[0]) != 0;

    size_t beamExpansionFactor = 1;
    if (s.isCylindric) {
        beamExpansionFactor++;
    }
    //CPU allocations
    std::complex<double>* gridPropagationFactor1CPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));
    std::complex<double>* gridPolarizationFactor1CPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));
    
    //GPU allocations
    //I shouldn't need all these memsets but, they make me feel better
    int memErrors = 0;
    memErrors += hipMalloc((void**)&s.gridETime1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridETime1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridETemp1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridETemp1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequency1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridEFrequency1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactor1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridPropagationFactor1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPolarizationFactor1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridPolarizationFactor1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPropagationFactor1Rho1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridPropagationFactor1Rho1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridRadialLaplacian1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridRadialLaplacian1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridEFrequency1Next1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.gridEFrequency1Next1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.chiLinear1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.chiLinear1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.k1, 2 * sizeof(hipDoubleComplex) * s.Ngrid);
    hipMemset(s.k1, 0, 2 * sizeof(hipDoubleComplex) * s.Ngrid);

    memErrors += hipMalloc((void**)&s.gridPolarizationFrequency1, sizeof(hipDoubleComplex) * 2 * (beamExpansionFactor * s.Nspace * (s.Ntime / 2 + 1)));
    hipMemset(s.gridPolarizationFrequency1, 0, sizeof(hipDoubleComplex) * 2 * (beamExpansionFactor * s.Nspace * (s.Ntime / 2 + 1)));
    memErrors += hipMalloc((void**)&s.gridPlasmaCurrentFrequency1, sizeof(hipDoubleComplex) * 2 * (2 * s.Nspace * (s.Ntime / 2 + 1)));
    hipMemset(s.gridPlasmaCurrentFrequency1, 0, sizeof(hipDoubleComplex) * 2 * (2 * s.Nspace * (s.Ntime / 2 + 1)));
    memErrors += hipMalloc((void**)&s.gridPolarizationTime1, 2 * sizeof(double) * s.Ngrid);
    hipMemset(s.gridPolarizationTime1, 0, 2 * sizeof(double) * s.Ngrid);
    memErrors += hipMalloc((void**)&s.gridPlasmaCurrent1, 2 * sizeof(double) * s.Ngrid);
    hipMemset(s.gridPlasmaCurrent1, 0, 2 * sizeof(double) * s.Ngrid);
    

    memErrors += hipMalloc((void**)&s.expGammaT, 2 * sizeof(double) * s.Ntime);
    double* expGammaTCPU = (double*)malloc(2 * sizeof(double) * s.Ntime);
    for (i = 0; i < s.Ntime; i++) {
        expGammaTCPU[i] = exp(s.dt * i * (*sCPU).drudeGamma);
        expGammaTCPU[i + s.Ntime] = exp(-s.dt * i * (*sCPU).drudeGamma);
    }
    hipMemcpy(s.expGammaT, expGammaTCPU, 2 * sizeof(double) * s.Ntime, hipMemcpyHostToDevice);
    free(expGammaTCPU);

    memErrors += hipMalloc((void**)&s.chi2Tensor, sizeof(double) * 9);
    memErrors += hipMalloc((void**)&s.firstDerivativeOperation, sizeof(double) * 6);
    memErrors += hipMalloc((void**)&s.chi3Tensor, sizeof(double) * 81);
    memErrors += hipMalloc((void**)&s.nonlinearSwitches, sizeof(int) * 4);
    memErrors += hipMalloc((void**)&s.absorptionParameters, sizeof(double) * 6);
    memErrors += hipMalloc((void**)&s.plasmaParameters, sizeof(double) * 6);
    memErrors += hipMalloc((void**)&s.propagationInts, sizeof(long long) * 4);
    (*sCPU).memoryError = memErrors;
    if (memErrors > 0) {
        return memErrors;
    }

    //second polarization grids are to pointers within the first polarization
    //to have contiguous memory
    s.gridETime2 = s.gridETime1 + s.Ngrid;
    s.gridPlasmaCurrent2 = s.gridPlasmaCurrent1 + s.Ngrid;
    s.gridPolarizationTime2 = s.gridPolarizationTime1 + s.Ngrid;
    s.gridPlasmaCurrentFrequency2 = s.gridPlasmaCurrentFrequency1 + (beamExpansionFactor * s.Nspace * (s.Ntime / 2 + 1));
    s.gridPolarizationFrequency2 = s.gridPolarizationFrequency1 + (beamExpansionFactor * s.Nspace * (s.Ntime/2+1));
    s.k2 = s.k1 + s.Ngrid;
    s.chiLinear2 = s.chiLinear1 + s.Ngrid;
    s.gridRadialLaplacian2 = s.gridRadialLaplacian1 + s.Ngrid;
    s.gridPropagationFactor1Rho2 = s.gridPropagationFactor1Rho1 + s.Ngrid;
    s.gridPolarizationFactor2 = s.gridPolarizationFactor1 + s.Ngrid;
    s.gridEFrequency1Next2 = s.gridEFrequency1Next1 + s.Ngrid;
    s.gridPropagationFactor2 = s.gridPropagationFactor1 + s.Ngrid;
    s.gridEFrequency2 = s.gridEFrequency1 + s.Ngrid;
    s.gridETemp2 = s.gridETemp1 + s.Ngrid;

    //prepare effective nonlinearity tensors and put them on the GPU
    size_t propagationIntsCPU[4] = { s.Ngrid, s.Ntime, s.Nspace, (s.Ntime / 2 + 1) };
    double firstDerivativeOperation[6] = { -1. / 60.,  3. / 20., -3. / 4.,  3. / 4.,  -3. / 20., 1. / 60. };
    for (i = 0; i < 6; i++) {
        firstDerivativeOperation[i] *= (-2.0/(s.Ngrid * s.dx));
    }

    //set nonlinearSwitches[3] to the number of photons needed to overcome bandgap
    (*sCPU).nonlinearSwitches[3] = (int)ceil((*sCPU).bandGapElectronVolts * 241.79893e12 / (*sCPU).frequency1) - 2;

    double plasmaParametersCPU[6] = { 0 };
    
    if ((*sCPU).nonlinearAbsorptionStrength > 0.) {
        s.hasPlasma = TRUE;
        s.isNonLinear = TRUE;
    }
    else {
        s.hasPlasma = FALSE;
    }
    
    plasmaParametersCPU[0] = (*sCPU).nonlinearAbsorptionStrength; //nonlinear absorption strength parameter
    plasmaParametersCPU[1] = (*sCPU).drudeGamma; //gamma
    if ((*sCPU).nonlinearAbsorptionStrength > 0.) {
        //plasmaParametersCPU[2] = (1. / 8.8541878128e-12) * (*sCPU).tStep * (*sCPU).tStep
            //* 2.817832e-08 / (1.6022e-19 * (*sCPU).bandGapElectronVolts * (*sCPU).effectiveMass); // (dt^2)*e* e / (m * band gap));
        plasmaParametersCPU[2] = (*sCPU).tStep * (*sCPU).tStep
            * 2.817832e-08 / (1.6022e-19 * (*sCPU).bandGapElectronVolts * (*sCPU).effectiveMass); // (dt^2)*e* e / (m * band gap));
    }
    else {
        plasmaParametersCPU[2] = 0;
    }
    

    calcEffectiveChi2Tensor((*sCPU).deffTensor, (*sCPU).chi2Tensor, (*sCPU).crystalTheta, (*sCPU).crystalPhi);
    hipMemcpy(s.chi2Tensor, (*sCPU).deffTensor, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.nonlinearSwitches, (*sCPU).nonlinearSwitches, 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(s.propagationInts, propagationIntsCPU, 4 * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(s.chi3Tensor, (*sCPU).chi3Tensor, 27 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.absorptionParameters, (*sCPU).absorptionParameters, 6 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.plasmaParameters, plasmaParametersCPU, 6 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.firstDerivativeOperation, firstDerivativeOperation, 6 * sizeof(double), hipMemcpyHostToDevice);

    //prepare FFT plans
    size_t workSize;
    int cufftSizes1[] = { (int)s.Nspace, (int)s.Ntime };
    hipfftCreate(&s.fftPlan);
    hipfftGetSizeMany(s.fftPlan, 2, cufftSizes1, NULL, NULL, NULL, NULL, NULL, NULL, HIPFFT_Z2Z, 2, &workSize);
    hipfftMakePlanMany(s.fftPlan, 2, cufftSizes1, NULL, NULL, NULL, NULL, NULL, NULL, HIPFFT_Z2Z, 2, &workSize);

    //the inputs are written out explicitly here even though 6x NULL would also do the same thing
    //this is in the hope that someday I figure out a way to do a truncated transform that doesn't
    //calculate the >f_nyquist/2 elements that I ignore
    int cufftSizes2[]{ 2*(int)s.Nspace, (int)s.Ntime };
    int istride = 1;
    int ostride = 1;
    int idist = (int)(2 * s.Ngrid);
    int odist = (int)(2 * s.Nspace * (s.Ntime / 2 + 1));
    int inembed[] = { (int)(2*s.Nspace), (int)s.Ntime };
    int onembed[] = { (int)(2*s.Nspace), (int)s.Ntime / 2 + 1 };

    hipfftCreate(&s.doublePolfftPlan);
    hipfftGetSizeMany(s.doublePolfftPlan, 2, cufftSizes2, inembed, istride, idist, onembed, ostride, odist, HIPFFT_D2Z, 2, &workSize);
    hipfftMakePlanMany(s.doublePolfftPlan, 2, cufftSizes2, inembed, istride, idist, onembed, ostride, odist, HIPFFT_D2Z, 2, &workSize);

    hipfftCreate(&s.polfftPlan);
    hipfftGetSizeMany(s.polfftPlan, 2, cufftSizes1, NULL, NULL, NULL, NULL, NULL, NULL, HIPFFT_D2Z, 2, &workSize);
    hipfftMakePlanMany(s.polfftPlan, 2, cufftSizes1, NULL, NULL, NULL, NULL, NULL, NULL, HIPFFT_D2Z, 2, &workSize);

    hipfftSetStream(s.fftPlan, s.CUDAStream);
    hipfftSetStream(s.polfftPlan, s.CUDAStream);
    hipfftSetStream(s.doublePolfftPlan, s.CUDAStream);

    //prepare the propagation arrays
    if (s.isCylindric) {
        preparePropagation3DCylindric(sCPU, s);
    }
    else {
        preparePropagation2DCartesian(sCPU, s);
    }

    //generate the pulses, either through prepareElectricFieldArrays() if this is the first in the series, or by copying
    //the output of the last simulation in the sequence
    if ((*sCPU).isFollowerInSequence) {
        hipMemcpy(s.gridETime1, (*sCPU).ExtOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridETime2, &(*sCPU).ExtOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency1, (*sCPU).EkwOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency2, &(*sCPU).EkwOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    }
    else {
        prepareElectricFieldArrays(sCPU, &s);
    }
    
    //Copy the field into the temporary array
    hipMemcpy(s.gridETemp1, s.gridEFrequency1, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    hipMemcpy(s.gridETemp2, s.gridEFrequency2, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    //Core propagation loop
    for (i = 0; i < s.Nsteps; i++) {
        
        //RK4
        runRK4Step(s, 0);
        runRK4Step(s, 1);
        runRK4Step(s, 2);
        runRK4Step(s, 3);

        if ((*sCPU).imdone[0] == 2) {
            break;
        }

        if ((*sCPU).imdone[0] == 3) {
            //copy the field arrays from the GPU to CPU memory
            hipMemcpy((*sCPU).ExtOut, s.gridETime1, 2 * (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2 * (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

            (*sCPU).imdone[0] = 0;
        }
    }

    
    //transform final result
    fixnanKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream>>>(s.gridEFrequency1);
    fixnanKernel << <s.Nblock, s.Nthread, 0, s.CUDAStream >>> (s.gridEFrequency2);
    hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridEFrequency1, (hipfftDoubleComplex*)s.gridETime1, HIPFFT_BACKWARD);
    fftNormalizeKernel << <s.Nblock, s.Nthread, 0, s.CUDAStream>>> (s.gridETime1, s.propagationInts);
    fftNormalizeKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream >>> (s.gridETime2, s.propagationInts);


    //copy the field arrays from the GPU to CPU memory
    hipMemcpy((*sCPU).ExtOut, s.gridETime1, 2*(*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*sCPU).EkwOut, s.gridEFrequency1, 2*(*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    getTotalSpectrum(sCPU, &s);

    //Free GPU memory
    hipFree(s.propagationInts);
    hipFree(s.nonlinearSwitches);
    hipFree(s.absorptionParameters);
    hipFree(s.gridETime1); 
    hipFree(s.gridETemp1);
    hipFree(s.gridPolarizationFrequency1);
    hipFree(s.gridEFrequency1);
    hipFree(s.gridPropagationFactor1);
    hipFree(s.gridPropagationFactor1Rho1);
    hipFree(s.gridRadialLaplacian1);
    hipFree(s.firstDerivativeOperation);
    hipFree(s.gridPolarizationFactor1);
    hipFree(s.gridEFrequency1Next1);
    hipFree(s.k1);
    hipFree(s.gridPolarizationTime1);
    hipFree(s.chi2Tensor);
    hipFree(s.chi3Tensor);
    hipFree(s.expGammaT);
    hipFree(s.chiLinear1);
    hipfftDestroy(s.fftPlan);
    hipfftDestroy(s.polfftPlan);
    hipfftDestroy(s.doublePolfftPlan);
    hipFree(s.plasmaParameters);
    hipFree(s.gridPlasmaCurrent1);
    hipFree(s.gridPlasmaCurrentFrequency1);
    
    hipStreamDestroy(s.CUDAStream);


    //Free CPU memory
    free(gridPropagationFactor1CPU);
    free(gridPolarizationFactor1CPU);
    (*sCPU).imdone[0] = 1;
    return 0;
}

//function to run a RK4 time step
//stepNumber is the sub-step index, from 0 to 3
int runRK4Step(cudaParameterSet s, int stepNumber) {

    //operations involving FFT
    if (s.isNonLinear || s.isCylindric) {
        //multiply by linear chi so that we have polarization (for applying Miller's rule)
        multiplicationKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream>>>(s.chiLinear1, s.gridETemp1, s.gridETime1);
        multiplicationKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream>>>(s.chiLinear2, s.gridETemp2, s.gridETime2);
        
        //perform inverse FFT to get time-space electric field
        hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridETime1, (hipfftDoubleComplex*)s.gridETime1, HIPFFT_BACKWARD);
        if (s.isNonLinear) {
            nonlinearPolarizationKernel<<<s.Nblock, s.Nthread, 0, s.CUDAStream>>>(s);

            if (s.isCylindric) {
                expandCylindricalBeam <<< s.Nblock, s.Nthread, 0, s.CUDAStream >>> 
                    (s, s.gridPolarizationTime1, s.gridPolarizationTime2);
                hipfftExecD2Z(s.doublePolfftPlan, (double*)s.gridRadialLaplacian1, (hipfftDoubleComplex*)s.gridPolarizationFrequency1);
            }
            else {
                hipfftExecD2Z(s.polfftPlan, s.gridPolarizationTime1, (hipfftDoubleComplex*)s.gridPolarizationFrequency1);
            }
        }
        if (s.hasPlasma) {
            plasmaCurrentKernelPrep <<<s.Nblock, s.Nthread, 0, s.CUDAStream >>> 
                (s, (double*)s.gridPlasmaCurrentFrequency1, (double*)s.gridPlasmaCurrentFrequency2);
            plasmaCurrentKernel2 <<<(unsigned int)s.Nspace, 1, 0, s.CUDAStream >>> 
                (s, (double*)s.gridPlasmaCurrentFrequency1, (double*)s.gridPlasmaCurrentFrequency2);
            
            if (s.isCylindric) {
                expandCylindricalBeam <<< s.Nblock, s.Nthread, 0, s.CUDAStream >>> 
                    (s, s.gridPlasmaCurrent1, s.gridPlasmaCurrent2);
                hipfftExecD2Z(s.doublePolfftPlan, (double*)s.gridRadialLaplacian1, (hipfftDoubleComplex*)s.gridPlasmaCurrentFrequency1);
            }
            else {
                hipfftExecD2Z(s.polfftPlan, s.gridPlasmaCurrent1, (hipfftDoubleComplex*)s.gridPlasmaCurrentFrequency1);
            }

        }
        
        if (s.isCylindric) {
            radialLaplacianKernel <<<s.Nblock, s.Nthread, 0, s.CUDAStream >>> (s);
            hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridRadialLaplacian1, (hipfftDoubleComplex*)s.k1, HIPFFT_FORWARD);
        }
    }

    //calculate k
    rkKernel <<<s.Nblock, s.Nthread, 0, s.CUDAStream >>> (s, stepNumber);
    return 0;
}

int prepareElectricFieldArrays(simulationParameterSet* s, cudaParameterSet *sc) {
    size_t i,j;
    double rB, zB, r, z; //r and z in the Beam and lab coordinates, respectively.
    double w0, wz, zR, Rz, phi; //Gaussian beam parameters
    double theta = 0; //rotation angle of the current beam
    double pulseSum = 0;
    std::complex<double> ne, no; //active refractive index;
    double f, w; //active frequency;
    double pulseEnergySum;
    std::complex<double> ko, specfac, specphase;
    double c = 2.99792458e8; //speed of light
    double eps0 = 8.8541878128e-12; //vacuum permittivity
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    std::complex<double> *pulse1, *pulse2, *pulse1f, *pulse2f;
    hipfftHandle plan1;
    hipfftHandle plan2;
    pulse1 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse1f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    std::complex<double> Eb;
    std::complex<double> ii(0, 1);
    std::complex<double> polFactor1, polFactor2; //complex phase/amplitude factors for the polarization components


    //define pulse 1 in mixed space
    // Gaussian beam in x
    // Spectrum in frequency domain (supergaussian with phase terms)
    polFactor1 = cos((*s).polarizationAngle1) - ii * (*s).circularity1 * sin((*s).polarizationAngle1);
    polFactor2 = sin((*s).polarizationAngle1) + ii * (*s).circularity1 * cos((*s).polarizationAngle1);
    theta = (*s).propagationAngle1;
    zB = (*s).z01;
    w0 = (*s).beamwaist1;


    //get the material phase
    double* materialCoefficientsCUDA;
    double* materialPhase1CUDA;
    double* materialPhase2CUDA;
    hipMalloc((void**)&materialCoefficientsCUDA, 66 * sizeof(double));
    hipMalloc((void**)&materialPhase1CUDA, (*s).Ntime * sizeof(double));
    hipMalloc((void**)&materialPhase2CUDA, (*s).Ntime * sizeof(double));
    hipMemcpy(materialCoefficientsCUDA, (*s).crystalDatabase[(*s).phaseMaterialIndex].sellmeierCoefficients, 66 * sizeof(double), hipMemcpyHostToDevice);
    materialPhaseKernel<<<(unsigned int)(*s).Ntime, 1, 0, (*sc).CUDAStream >> >((*s).fStep, (*s).Ntime, materialCoefficientsCUDA, (*s).frequency1, (*s).frequency2, (*s).phaseMaterialThickness1, (*s).phaseMaterialThickness2, materialPhase1CUDA, materialPhase2CUDA);
    double* materialPhase1 = (double*)calloc((*s).Ntime, sizeof(double));
    double* materialPhase2 = (double*)calloc((*s).Ntime, sizeof(double));
    hipMemcpy(materialPhase1, materialPhase1CUDA, (*s).Ntime * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(materialPhase2, materialPhase2CUDA, (*s).Ntime * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(materialPhase2CUDA);
    hipFree(materialPhase1CUDA);
    hipFree(materialCoefficientsCUDA);



    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency1);
        
        //supergaussian pulse spectrum, if no input pulse specified
        specfac = pow((f - (*s).frequency1)/(*s).bandwidth1,(*s).sgOrder1);

        specphase = ii * ((*s).cephase1 + 2*pi*f * ((*s).delay1 - 0.5*(*s).tStep*(*s).Ntime) + 0.5 * (*s).gdd1 * w * w + (*s).tod1 * w * w * w/6.0 + materialPhase1[i]);
        specfac = exp(-specfac + specphase);

        if ((*s).field1IsAllocated) {
            specfac = (*s).loadedField1[i] * exp(-specphase);
        }



        for (j = 0; j < (*s).Nspace; j++) {
            ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
            no = (*s).refractiveIndex2[i + (*s).Ntime * j];
            ko = 2 * pi * no * f / c;
            zR = pi * w0 * w0 * real(ne) * f / c;
            if (f == 0) {
                zR = 1e3;
            }
            rB = ((*s).x01 + (*s).rStep * (j - (*s).Nspace / 2.0) - 0.25*(*s).rStep);
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);
            
            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));
            
            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z-zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cModulusSquared(Eb)) || f<=0) {
                Eb = 0;
            }
            
            pulse1[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse1[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r)*(real(ne)*cModulusSquared(pulse1[i + (*s).Ntime * j]) 
                + real(no)*cModulusSquared(pulse1[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }
    
    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime1, pulse1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse1[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftPlan1d(&plan1, (int)(*sc).Ntime, HIPFFT_Z2Z, (int)(*sc).Nspace);
    hipfftSetStream(plan1, (*sc).CUDAStream);
    hipfftPlan2d(&plan2, (int)(*sc).Nspace, (int)(*sc).Ntime, HIPFFT_Z2Z);
    hipfftSetStream(plan2, (*sc).CUDAStream);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridETemp1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp1, (hipfftDoubleComplex*)(*sc).gridEFrequency1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency1, (hipfftDoubleComplex*)(*sc).gridETime1, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel<<<(*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream >>>((*sc).gridETime1);
    conjugateKernel<<<(*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream >>>((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridEFrequency1, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse1, (*sc).gridETime1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse1f, (*sc).gridEFrequency1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy1/pulseSum)/(*s).Ngrid;
    (*s).pulse1measEnergy = pulseEnergySum;
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse1[i] = pulse1[i] * pulseEnergySum;
        pulse1f[i] = pulse1f[i] * pulseEnergySum;
    }


    //do same for pulse 2 here
    pulseSum = 0;
    polFactor1 = cos((*s).polarizationAngle2) - ii * (*s).circularity2 * sin((*s).polarizationAngle2);
    polFactor2 = sin((*s).polarizationAngle2) + ii * (*s).circularity2 * cos((*s).polarizationAngle2);
    theta = (*s).propagationAngle2;
    zB = (*s).z02;
    w0 = (*s).beamwaist2;

    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency2);

        //supergaussian pulse spectrum, if no input pulse specified
        specfac = pow((f - (*s).frequency2) / (*s).bandwidth2,(*s).sgOrder2);

        specphase = ii * ((*s).cephase2 + 2*pi*f * ((*s).delay2 - 0.5*(*s).tStep*(*s).Ntime) + 0.5*(*s).gdd2 * w * w + (*s).tod2 * w * w * w/6.0 + materialPhase2[i]);
        specfac = exp(-specfac + specphase);

        if ((*s).field2IsAllocated) {
            specfac = (*s).loadedField2[i] * exp(-specphase);
        }




        for (j = 0; j < (*s).Nspace; j++) {
            ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
            no = (*s).refractiveIndex2[i + (*s).Ntime * j];
            ko = 2 * pi * no * f / c;
            zR = pi * w0 * w0 * real(ne) * f / c;
            if (f == 0) {
                zR = 1e3;
            }
            rB = ((*s).x01 + (*s).rStep * (j - (*s).Nspace / 2.0) - 0.25 * (*s).rStep);
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);

            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));

            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z - zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cModulusSquared(Eb)) || f <= 0) {
                Eb = 0;
            }

            pulse2[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse2[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r) * (real(ne) * cModulusSquared(pulse2[i + (*s).Ntime * j]) 
                + real(no) * cModulusSquared(pulse2[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }

    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime1, pulse2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse2[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridETemp1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp1, (hipfftDoubleComplex*)(*sc).gridEFrequency1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency1, (hipfftDoubleComplex*)(*sc).gridETime1, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel << <(*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream >> > ((*sc).gridETime1);
    conjugateKernel << <(*sc).Nblock, (*sc).Nthread, 0, (*sc).CUDAStream >> > ((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridEFrequency1, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse2, (*sc).gridETime1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse2f, (*sc).gridEFrequency1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy2 / pulseSum) / (*s).Ngrid;

    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse2[i] = pulse2[i] * pulseEnergySum;
        pulse2f[i] = pulse2f[i] * pulseEnergySum;
    }
    hipDeviceSynchronize();

    //make the combined fields
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        (*s).Ext[i] = pulse1[i] + pulse2[i];
        (*s).Ekw[i] = pulse1f[i] + pulse2f[i];
    }
    //Copy the grids back to the GPU
    hipMemcpy((*sc).gridETime1, (*s).Ext, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &(*s).Ext[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency1, (*s).Ekw, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency2, &(*s).Ekw[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftDestroy(plan1);
    hipfftDestroy(plan2);

    free(pulse1);
    free(pulse2);
    free(pulse1f);
    free(pulse2f);
    free(materialPhase1);
    free(materialPhase2);
    return 0;
}
int applyFresnelLoss(simulationParameterSet* s, int materialIndex1, int materialIndex2) {
    cudaParameterSet sc;
    sc.Ntime = (*s).Ntime;
    sc.Nspace = (*s).Nspace;
    sc.dt = (*s).tStep;
    sc.dx = (*s).rStep;
    sc.fStep = (*s).fStep;
    sc.h = (*s).propagationStep;
    sc.Nsteps = (*s).Npropagation;
    sc.Ngrid = (*s).Ntime * (*s).Nspace;
    sc.axesNumber = (*s).axesNumber;
    sc.sellmeierType = (*s).sellmeierType;
    sc.f0 = (*s).frequency1;
    sc.Nthread = THREADS_PER_BLOCK;
    sc.Nblock = (int)((*s).Ngrid / THREADS_PER_BLOCK);
    sc.isCylindric = (*s).isCylindric;
    sc.isNonLinear = ((*s).nonlinearSwitches[0] + (*s).nonlinearSwitches[1]) > 0;
    hipStreamCreate(&sc.CUDAStream);
    hipfftPlan2d(&sc.fftPlan, (int)sc.Nspace, (int)sc.Ntime, HIPFFT_Z2Z);
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).crystalDatabase[materialIndex1].sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    double* sellmeierCoefficients1;
    double* sellmeierCoefficients2;
    hipMalloc(&sellmeierCoefficients1, 74 * sizeof(double));
    hipMalloc(&sellmeierCoefficients2, 74 * sizeof(double));
    hipMemcpy(sellmeierCoefficients1, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).crystalDatabase[materialIndex2].sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients2, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);

    size_t propagationIntsCPU[4] = { sc.Ngrid, sc.Ntime, sc.Nspace, (sc.Ntime / 2 + 1) };
    hipMalloc((void**)&sc.propagationInts, sizeof(size_t) * 4);
    hipMemcpy(sc.propagationInts, propagationIntsCPU, 4 * sizeof(size_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipMalloc(&sc.gridEFrequency1, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc(&sc.gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc(&sc.gridETime1, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc(&sc.gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex));

    hipMemcpy(sc.gridEFrequency1, (*s).EkwOut, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(sc.gridEFrequency2, &(*s).EkwOut[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    applyFresnelLossKernel<<<sc.Nblock, sc.Nthread, 0, sc.CUDAStream>>>(sellmeierCoefficients1, sellmeierCoefficients2, sc);

    //transform final result
    fixnanKernel << <sc.Nblock, sc.Nthread, 0, sc.CUDAStream >> > (sc.gridEFrequency1);
    fixnanKernel << <sc.Nblock, sc.Nthread, 0, sc.CUDAStream >> > (sc.gridEFrequency2);
    hipfftExecZ2Z(sc.fftPlan, (hipfftDoubleComplex*)sc.gridEFrequency1, (hipfftDoubleComplex*)sc.gridETime1, HIPFFT_BACKWARD);
    hipfftExecZ2Z(sc.fftPlan, (hipfftDoubleComplex*)sc.gridEFrequency2, (hipfftDoubleComplex*)sc.gridETime2, HIPFFT_BACKWARD);
    fftNormalizeKernel << <sc.Nblock, sc.Nthread, 0, sc.CUDAStream >> > (sc.gridETime1, sc.propagationInts);
    fftNormalizeKernel << <sc.Nblock, sc.Nthread, 0, sc.CUDAStream >> > (sc.gridETime2, sc.propagationInts);

    //copy the field arrays from the GPU to CPU memory
    hipMemcpy((*s).ExtOut, sc.gridETime1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*s).EkwOut, sc.gridEFrequency1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*s).ExtOut[sc.Ngrid], sc.gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*s).EkwOut[sc.Ngrid], sc.gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);


    free(sellmeierCoefficientsAugmentedCPU);
    
    hipfftDestroy(sc.fftPlan);
    hipFree(sc.gridEFrequency1);
    hipFree(sc.gridEFrequency2);
    hipFree(sc.gridETime1);
    hipFree(sc.gridETime2);
    hipFree(sellmeierCoefficients1);
    hipFree(sellmeierCoefficients2);
    hipStreamDestroy(sc.CUDAStream);
    return 0;
}
int preparePropagation2DCartesian(simulationParameterSet* s, cudaParameterSet sc) {
    //recycle allocated device memory for the grids needed
    double* sellmeierCoefficients = (double*)sc.k1;
    sc.ne = sc.gridEFrequency1Next2;
    sc.no = sc.k2;

    double* referenceFrequencies;
    hipMalloc(&referenceFrequencies, 7 * sizeof(double));
    hipMemcpy(referenceFrequencies, (*s).crystalDatabase[(*s).materialIndex].nonlinearReferenceFrequencies, 7 * sizeof(double), hipMemcpyHostToDevice);

    //construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep; 
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66+8) * sizeof(double), hipMemcpyHostToDevice);

    //prepare the propagation grids
    prepareCartesianGridsKernel <<<sc.Nblock, sc.Nthread, 0, sc.CUDAStream >>> (sellmeierCoefficients, sc);
    millersRuleNormalizationKernel << <1, 1, 0, sc.CUDAStream >> > (sc, sellmeierCoefficients, referenceFrequencies);
    hipDeviceSynchronize();

    //copy the retrieved refractive indicies to the cpu
    hipMemcpy((*s).refractiveIndex1, sc.ne, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*s).refractiveIndex2, sc.no, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    
    //clean up
    hipMemset(sc.gridEFrequency1Next1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.gridEFrequency1Next2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    free(sellmeierCoefficientsAugmentedCPU);
    hipFree(referenceFrequencies);
    return 0;
}

int preparePropagation3DCylindric(simulationParameterSet* s, cudaParameterSet sc) {
    //recycle allocated device memory for the grids needed
    double* sellmeierCoefficients = (double*)sc.k1;
    sc.ne = sc.gridEFrequency1Next2;
    sc.no = sc.k2;
    double* referenceFrequencies;
    hipMalloc(&referenceFrequencies, 7*sizeof(double));
    hipMemcpy(referenceFrequencies, (*s).crystalDatabase[(*s).materialIndex].nonlinearReferenceFrequencies, 7 * sizeof(double), hipMemcpyHostToDevice);

    //construct augmented sellmeier coefficients used in the kernel to find the walkoff angles
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep;
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66 + 8) * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    //prepare the propagation grids
    
    prepareCylindricGridsKernel << <sc.Nblock, sc.Nthread, 0, sc.CUDAStream >> > (sellmeierCoefficients, sc);
    millersRuleNormalizationKernel<<<1, 1, 0, sc.CUDAStream>>>(sc, sellmeierCoefficients, referenceFrequencies);
    hipDeviceSynchronize();

    //copy the retrieved refractive indicies to the cpu
    hipMemcpy((*s).refractiveIndex1, sc.ne, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*s).refractiveIndex2, sc.no, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //clean up
    hipMemset(sc.gridEFrequency1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.gridEFrequency2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k1, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMemset(sc.k2, 0, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipFree(referenceFrequencies);
    free(sellmeierCoefficientsAugmentedCPU);
    return 0;
}

int calcEffectiveChi2Tensor(double* defftensor, double* dtensor, double theta, double phi) {
    double delta = 0.; //this angle is used for biaxial crystals, but I'm ignorning it for the moment
    int i, j, k;
    //Rotation matrix between the angles of the electric field and the crystal axes
    double R[] = { cos(theta) * cos(phi) * cos(delta) - sin(phi) * sin(delta), cos(theta) * sin(phi) * cos(delta) + cos(phi) * sin(delta),
        -sin(theta) * cos(delta), -cos(theta) * cos(phi) * sin(delta) - sin(phi) * cos(delta),
        -cos(theta) * sin(phi) * sin(delta) + cos(phi) * cos(delta), sin(theta) * sin(delta) };

    //Matrix to translate the mixed field matrix in the reduced notation into the crystalline frame
    double Ore[] = { R[0] * R[0], R[1] * R[1], R[2] * R[2], 2 * R[1] * R[2], 2 * R[0] * R[2], 2 * R[0] * R[1],
        2 * R[0] * R[3], 2 * R[1] * R[4], 2 * R[2] * R[5], 2 * (R[4] * R[2] + R[1] * R[5]), 2 * (R[3] * R[2] + R[0] * R[5]), 2 * (R[3] * R[1] + R[0] * R[4]),
        R[3] * R[3], R[4] * R[4], R[5] * R[5], 2 * R[4] * R[5], 2 * R[3] * R[5], 2 * R[3] * R[4]
    };



    //The deff tensor is given by the equation R deff = d Ore, solve for deff, find d Ore first
    double dOre[9] = { 0 };
    for (i = 0; i < 3; i++) {
        for (j = 0; j < 3; j++) {
            for (k = 0; k < 6; k++) {
                dOre[i + 3 * j] += dtensor[i + 3 * k] * Ore[k + 6 * j];
            }
        }
    }


    //Least squares solution to get the deff tensor
    double* work = (double*)malloc(128 * sizeof(double));
    int dgelsInfo;
    int dgelsParams[6] = { 3,2,3,3,3,64 };
    dgels("N", &dgelsParams[0], &dgelsParams[1], &dgelsParams[2], R, &dgelsParams[3], dOre, &dgelsParams[4], work, &dgelsParams[5], &dgelsInfo);
    defftensor[0] = dOre[0];
    defftensor[1] = dOre[1];
    defftensor[2] = dOre[3];
    defftensor[3] = dOre[4];
    defftensor[4] = dOre[6];
    defftensor[5] = dOre[7];
    free(work);

    //correct cross-terms
    for (i = 2; i < 4; i++) {
        defftensor[i] *= 0.5;
    }

    for (i = 0; i < 6; i++) {
        defftensor[i] *= 2e-12; //change from pm/V to m/V and multiply by 2 for chi(2) instead of d
    }
    return dgelsInfo;
}

//c implementation of fftshift, working on complex double precision
//A is the input array, B is the output
//dim1: column length
//dim2: row length
int fftshiftZ(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j-div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//same as fftshiftZ, but flips the output array columns
int fftshiftAndFilp(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1-i-1) + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i + div1 + dim1 * (j - div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//sellmeier equation
//outputs are pointers ne and no
//a is a 16-value array containing the coefficients
//f is frequency (Hz)
//theta is the crystal angle
//phi is the other crystal angle (currently unused because biaxials haven't been implemented)
//type is the kind of crystal (0: isotropic, 1: uniaxial, 2:biaxial) 
//eqn will switch to a different equation, in the future, currently not implemented
//REPLACED BY CUDA VERSION, DELETE LATER
std::complex<double> sellmeier(std::complex<double>* ne, std::complex<double>* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return 1; //exit immediately for f=0

    double c = 2.99792458e8; //speed of light
    double l = 1e6*c / f; //wavelength in microns
    double ls = l * l;
    std::complex<double> ii(0, 1);
    double pi = 3.14159265358979323846264338327950288;
    double omega = 2*pi*abs(f);
    double kL = 3183.9; //(e * e / (e_o *m_e)
    //option 0: isotropic
    if (type == 0) {
        ne[0] = a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6])
            + (a[7] + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls;
        if (real(ne[0]) < 1) {
            ne[0] = 1.;
        }
        ne[0] += kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega);
        ne[0] = conj(sqrt(ne[0]));
        if (isnan(real(ne[0]))) {
            ne[0] = 1;
        }
        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {
        std::complex<double> na = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega)));
        a = &a[22];
        std::complex<double> nb = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega)));
        if (isnan(real(na)) || isnan(real(nb))) {
            no[0] = 1;
            ne[0] = 1;
            return 1;
        }
        no[0] = na;
        ne[0] = 1.0 / sqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return na;
    }
    else {
        //later, implement biaxial crystals, for now just return 1;
        return 1;
    }
}
int loadReferenceSpectrum(char* spectrumPath, simulationParameterSet* sCPU) {
    FILE* fp = fopen(spectrumPath, "r");
    if (fp == NULL) {
        printf("Could not read reference file\r\n");
        return 1;
    }
    size_t maxFileSize = 16384;
    size_t currentRow = 0;
    double c = 1e9 * 2.99792458e8;
    double* loadedWavelengths = (double*)calloc(8192, sizeof(double));
    double* loadedFrequencies = (double*)calloc(8192, sizeof(double));
    double* loadedIntensities = (double*)calloc(8192, sizeof(double));
    double maxWavelength = 0;
    double minWavelength = 0;
    if (fp == NULL) {
        free(loadedWavelengths);
        free(loadedIntensities);
        free(loadedFrequencies);
        return -1;
    }

    while (fscanf(fp, "%lf %lf", &loadedWavelengths[currentRow], &loadedIntensities[currentRow]) == 2 && currentRow < maxFileSize) {
        if (currentRow == 0) {
            maxWavelength = loadedWavelengths[currentRow];
            minWavelength = loadedWavelengths[currentRow];
        }
        else {
            maxWavelength = max(maxWavelength, loadedWavelengths[currentRow]);
            minWavelength = min(minWavelength, loadedWavelengths[currentRow]);
        }
        //rescale to frequency spacing
        loadedIntensities[currentRow] *= loadedWavelengths[currentRow] * loadedWavelengths[currentRow];
        loadedFrequencies[currentRow] = c / loadedWavelengths[currentRow];
        currentRow++;
    }
    size_t sizeData = currentRow-1;
    size_t i, j;

    double maxFrequency = c / minWavelength;
    double minFrequency = c / maxWavelength;
    double currentFrequency = 0;
    double df;
    
    for(i = 1; i < (*sCPU).Ntime; i++) {
        currentFrequency = i * (*sCPU).fStep;
        if ((currentFrequency > minFrequency) && (currentFrequency < maxFrequency)) {
            //find the first frequency greater than the current value
            j = sizeData-1;
            while((loadedFrequencies[j] <= currentFrequency) && (j > 2)) {
                j--;
            }
            df = loadedFrequencies[j] - loadedFrequencies[j - 1];
            (*sCPU).fittingArray[i] = 
                (loadedIntensities[j-1] * (loadedFrequencies[j] - currentFrequency) 
                    + loadedIntensities[j] * (currentFrequency - loadedFrequencies[j-1])) / df; //linear interpolation
        }
    }
    
    fclose(fp);
    free(loadedWavelengths);
    free(loadedIntensities);
    free(loadedFrequencies);
    return 0;
}
int loadFrogSpeck(char* frogFilePath, std::complex<double>* Egrid, long long Ntime, double fStep, double gateLevel, int fieldIndex) {
    FILE* fp;
    int maxFileSize = 16384;
    double wavelength, R, phi, complexX, complexY, f, f0, f1, fmax;
    int i, k0, k1;
    double c = 1e9*2.99792458e8; //for conversion of wavelength in nm to frequency
    double df = 0;
    double fmin = 0;
    int currentRow = 0;
    std::complex<double>* E = (std::complex<double>*)calloc(maxFileSize, sizeof(std::complex<double>));

    //read the data
    fp = fopen(frogFilePath, "r");
    if (fp == NULL) {
        free(E);
        return -1;
    }
    while (fscanf(fp, "%lf %lf %lf %lf %lf", &wavelength, &R, &phi, &complexX, &complexY) == 5 && currentRow < maxFileSize) {
        //get the complex field from the data
        E[currentRow].real(complexX);
        E[currentRow].imag(complexY);

        //keep track of the frequency step of the grid (running sum, divide by number of rows at end to get average)
        if (currentRow > 0) df += c / wavelength - fmax;

        //keep track of the highest frequency in the data
        fmax = c / wavelength;
        
        //store the lowest frequency in the data
        if (currentRow == 0) fmin = fmax;
        
        currentRow++;
    }
    fclose(fp);

    //return an error if nothing was loaded
    if (currentRow == 0) {
        free(E);
        return -1;
    }

    df /= currentRow; //average frequency step

    //interpolate the FROG data onto the simulation grid
    
    //fill the simulation grid based on the data
    for (i = 0; i < Ntime; i++) {

        //frequency grid used in the simulation
        f = i * fStep;
        if (i >= Ntime / 2) {
            f -= fStep * (Ntime-1);
        }
        f *= -1;

        k0 = (int)floor((f - fmin) / df);
        k1 = (int)ceil((f - fmin) / df);
        if (k0 < 0 || k1 >= currentRow) {
            Egrid[i] = 0; //field is zero outside of data range
        }
        else {
            f0 = fmin + k0 * df;
            f1 = fmin + k1 * df;
            Egrid[i] = (E[k0] * (f1 - f) + E[k1] * (f - f0)) / df; //linear interpolation
            Egrid[i] *= (abs(Egrid[i]) > gateLevel);
        }
    }

    free(E);
    return currentRow;
}


//Rotate the field on the GPU
//Allocates memory and copies from CPU, then copies back to CPU and deallocates
// - inefficient but the general principle is that only the CPU memory is preserved
// after simulations finish... and this only runs at the end of the simulation
int rotateField(simulationParameterSet *s, double rotationAngle) {
    hipDoubleComplex* Ein1, * Eout1, * Ein2, * Eout2;
    hipMalloc((void**)&Ein1,  (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc((void**)&Ein2, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc((void**)&Eout1, (*s).Ngrid * sizeof(hipDoubleComplex));
    hipMalloc((void**)&Eout2, (*s).Ngrid * sizeof(hipDoubleComplex));
    unsigned int Nthread = THREADS_PER_BLOCK;
    unsigned int Nblock = (unsigned int)((*s).Ngrid / THREADS_PER_BLOCK);

    hipMemcpy(Ein1, (*s).EkwOut, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(Ein2, &(*s).EkwOut[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    rotateFieldKernel<<<Nblock, Nthread>>>(Ein1, Ein2, Eout1, Eout2, rotationAngle);

    hipMemcpy((*s).EkwOut, Eout1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*s).EkwOut[(*s).Ngrid], Eout2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipFree(Ein1);
    hipFree(Ein2);
    hipFree(Eout1);
    hipFree(Eout2);
    return 0;
}

//calculates the squard modulus of a complex number, under the assumption that the
//machine's complex number format is interleaved doubles.
//c forced to run in c++ for nostalgia reasons
double cModulusSquared(std::complex<double>complexNumber) {
    double* xy = (double*)&complexNumber;
    return xy[0] * xy[0] + xy[1] * xy[1];
}

int allocateGrids(simulationParameterSet* sCPU) {
    (*sCPU).loadedField1 = (std::complex<double>*)calloc((*sCPU).Ntime, sizeof(std::complex<double>));
    (*sCPU).loadedField2 = (std::complex<double>*)calloc((*sCPU).Ntime, sizeof(std::complex<double>));

    (*sCPU).Ext = (std::complex<double>*)calloc((*sCPU).Ngrid * 2 * (*sCPU).Nsims, sizeof(std::complex<double>));
    (*sCPU).Ekw = (std::complex<double>*)calloc((*sCPU).Ngrid * 2 * (*sCPU).Nsims, sizeof(std::complex<double>));

    (*sCPU).ExtOut = (std::complex<double>*)calloc((*sCPU).Ngrid * 2 * (*sCPU).Nsims, sizeof(std::complex<double>));
    (*sCPU).EkwOut = (std::complex<double>*)calloc((*sCPU).Ngrid * 2 * (*sCPU).Nsims, sizeof(std::complex<double>));

    (*sCPU).refractiveIndex1 = (std::complex<double>*)calloc((*sCPU).Ngrid * (*sCPU).Nsims, sizeof(std::complex<double>));
    (*sCPU).refractiveIndex2 = (std::complex<double>*)calloc((*sCPU).Ngrid * (*sCPU).Nsims, sizeof(std::complex<double>));
    (*sCPU).deffTensor = (double*)calloc(9 * (*sCPU).Nsims, sizeof(double));
    (*sCPU).totalSpectrum = (double*)calloc((*sCPU).Nsims * (*sCPU).Ntime * 3, sizeof(double));
    (*sCPU).imdone = (int*)calloc((*sCPU).Nsims, sizeof(int));
    return 0;
}


int readCrystalDatabase(crystalEntry* db) {
    int i = 0;
    double* fd;
    FILE* fp;
    fp = fopen("CrystalDatabase.txt", "r");
    if (fp == NULL) {
        return -2;
    }

    //read the entries line
    int readErrors = 0;

    while (readErrors == 0 && !feof(fp) && i < MAX_LOADSTRING) {
        readErrors += 0 != fwscanf(fp, L"Name:\n");
        fgetws(db[i].crystalNameW, 256, fp);
        readErrors += 1 != fwscanf(fp, L"Type:\n%d\n", &db[i].axisType);
        readErrors += 1 != fwscanf(fp, L"Sellmeier equation:\n%d\n", &db[i].sellmeierType);
        fd = &db[i].sellmeierCoefficients[0];
        readErrors += 22 != fwscanf(fp, L"1st axis coefficients:\n%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8], &fd[9], &fd[10], &fd[11], &fd[12], &fd[13], &fd[14], &fd[15], &fd[16], &fd[17], &fd[18], &fd[19], &fd[20], &fd[21]);
        fd = &db[i].sellmeierCoefficients[22];
        readErrors += 22 != fwscanf(fp, L"2nd axis coefficients:\n%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8], &fd[9], &fd[10], &fd[11], &fd[12], &fd[13], &fd[14], &fd[15], &fd[16], &fd[17], &fd[18], &fd[19], &fd[20], &fd[21]);
        fd = &db[i].sellmeierCoefficients[44];
        readErrors += 22 != fwscanf(fp, L"3rd axis coefficients:\n%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8], &fd[9], &fd[10], &fd[11], &fd[12], &fd[13], &fd[14], &fd[15], &fd[16], &fd[17], &fd[18], &fd[19], &fd[20], &fd[21]);
        readErrors += 0 != fwscanf(fp, L"Sellmeier reference:\n");
        fgetws(db[i].sellmeierReference, 512, fp);
        readErrors += 1 != fwscanf(fp, L"chi2 type:\n%d\n", &db[i].nonlinearSwitches[0]);
        fd = &db[i].d[0];
        readErrors += 18 != fwscanf(fp, L"d:\n%lf %lf %lf %lf %lf %lf\n%lf %lf %lf %lf %lf %lf\n%lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[3], &fd[6], &fd[9], &fd[12], &fd[15], 
            &fd[1], &fd[4], &fd[7], &fd[10], &fd[13], &fd[16], 
            &fd[2], &fd[5], &fd[8], &fd[11], &fd[14], &fd[17]);
        readErrors += 0 != fwscanf(fp, L"d reference:\n");
        fgetws(db[i].dReference, 512, fp);
        readErrors += 1 != fwscanf(fp, L"chi3 type:\n%d\n", &db[i].nonlinearSwitches[1]);
        fd = &db[i].chi3[0];
        readErrors += 9 != fwscanf(fp, L"chi3:\n%lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8]);
        fd = &db[i].chi3[9];
        readErrors += 9 != fwscanf(fp, L"%lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8]);
        fd = &db[i].chi3[18];
        readErrors += 9 != fwscanf(fp, L"%lf %lf %lf %lf %lf %lf %lf %lf %lf\n", 
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6], &fd[7], &fd[8]);
        readErrors += 0 != fwscanf(fp, L"chi3 reference:\n");
        fgetws(db[i].chi3Reference, 512, fp);
        readErrors += 0 != fwscanf(fp, L"Spectral file:\n");
        fgetws(db[i].spectralFile, 512, fp);
        fd = db[i].nonlinearReferenceFrequencies;
        readErrors += 0 != fwscanf(fp, L"Nonlinear reference frequencies:\n");
        readErrors += 7 != fwscanf(fp, L"%lf %lf %lf %lf %lf %lf %lf\n",
            &fd[0], &fd[1], &fd[2], &fd[3], &fd[4], &fd[5], &fd[6]);
        readErrors += 0 != fwscanf(fp, L"~~~crystal end~~~\n");
        if (readErrors == 0) i++;
    }
    db[0].numberOfEntries = i;
    fclose(fp);

    return i;
}

int readSequenceString(simulationParameterSet* sCPU) {
    //read the sequence string (if there is one), convert it into an array if it exists
    char sequenceString[MAX_LOADSTRING];
    strcpy(sequenceString, (*sCPU).sequenceString);
    char* tokToken = strtok(sequenceString, ";");
    int sequenceCount = sscanf(sequenceString, "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf", 
        &(*sCPU).sequenceArray[0], &(*sCPU).sequenceArray[1], &(*sCPU).sequenceArray[2], 
        &(*sCPU).sequenceArray[3], &(*sCPU).sequenceArray[4], &(*sCPU).sequenceArray[5],
        &(*sCPU).sequenceArray[6], &(*sCPU).sequenceArray[7], &(*sCPU).sequenceArray[8],
        &(*sCPU).sequenceArray[9], &(*sCPU).sequenceArray[10]);

    tokToken = strtok(NULL, ";");
    int lastread = sequenceCount;
    while (tokToken != NULL && lastread == 11) {
        lastread = sscanf(tokToken, "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf", 
            &(*sCPU).sequenceArray[sequenceCount], &(*sCPU).sequenceArray[sequenceCount + 1], 
            &(*sCPU).sequenceArray[sequenceCount + 2], &(*sCPU).sequenceArray[sequenceCount + 3], 
            &(*sCPU).sequenceArray[sequenceCount + 4], &(*sCPU).sequenceArray[sequenceCount + 5],
            &(*sCPU).sequenceArray[sequenceCount + 6], &(*sCPU).sequenceArray[sequenceCount + 7],
            &(*sCPU).sequenceArray[sequenceCount + 8], &(*sCPU).sequenceArray[sequenceCount + 9],
            &(*sCPU).sequenceArray[sequenceCount + 10]);
        if (lastread > 0) {
            sequenceCount += lastread;
        }
        tokToken = strtok(NULL, ";");
    }
    (*sCPU).Nsequence = sequenceCount / 11;
    (*sCPU).isInSequence = ((*sCPU).Nsequence > 0);

    if (!(*sCPU).isInSequence) {
        char nopeString[] = "None.";
        strcpy((*sCPU).sequenceString, nopeString);
    }
    return 0;
}

int readFittingString(simulationParameterSet* sCPU) {
    //read the fitting string (if there is one), convert it into an array if it exists
    char fittingString[MAX_LOADSTRING];
    double ROIbegin;
    double ROIend;
    strcpy(fittingString, (*sCPU).fittingString);
    char* tokToken = strtok(fittingString, ";");
    bool paramsRead = (4 == sscanf(fittingString, "%lf %lf %lf %d",
        &ROIbegin, &ROIend, &(*sCPU).fittingPrecision, &(*sCPU).fittingMaxIterations));
    (*sCPU).fittingROIstart = (size_t)(ROIbegin / (*sCPU).fStep);
    (*sCPU).fittingROIstop = (size_t)min(ROIend / (*sCPU).fStep, (*sCPU).Ntime/2);
    (*sCPU).fittingROIsize = min(max(1, (*sCPU).fittingROIstop - (*sCPU).fittingROIstart), (*sCPU).Ntime/2);
    int fittingCount = 0;
    tokToken = strtok(NULL, ";");
    int lastread = 3;
    while (tokToken != NULL && lastread == 3) {
        lastread = sscanf(tokToken, "%lf %lf %lf",
            &(*sCPU).fittingArray[fittingCount], &(*sCPU).fittingArray[fittingCount + 1],
            &(*sCPU).fittingArray[fittingCount + 2]);
        if (lastread > 0) {
            fittingCount += lastread;
        }
        tokToken = strtok(NULL, ";");
    }
    (*sCPU).Nfitting = fittingCount / 3;
    (*sCPU).isInFittingMode = (((*sCPU).Nfitting) > 0 && paramsRead);

    if (!(*sCPU).isInFittingMode) {
        char nopeString[] = "None.";
        strcpy((*sCPU).fittingString, nopeString);
    }
    return 0;
}

int configureBatchMode(simulationParameterSet* sCPU) {
    int j;
    const double pi = 3.1415926535897932384626433832795;
    if ((*sCPU).batchIndex == 0 || (*sCPU).Nsims == 1) {
        return 0;
    }

    //pointers to values that can be scanned in batch mode
    double* targets[36] = { 0,
        &(*sCPU).pulseEnergy1, &(*sCPU).pulseEnergy2, &(*sCPU).frequency1, &(*sCPU).frequency2,
        &(*sCPU).bandwidth1, &(*sCPU).bandwidth2, &(*sCPU).cephase1, &(*sCPU).cephase2,
        &(*sCPU).delay1, &(*sCPU).delay2, &(*sCPU).gdd1, &(*sCPU).gdd2,
        &(*sCPU).tod1, &(*sCPU).tod2, &(*sCPU).phaseMaterialThickness1, &(*sCPU).phaseMaterialThickness2,
        &(*sCPU).beamwaist1, &(*sCPU).beamwaist2,
        &(*sCPU).x01, &(*sCPU).x02, &(*sCPU).z01, &(*sCPU).z02,
        &(*sCPU).propagationAngle1, &(*sCPU).propagationAngle2, &(*sCPU).polarizationAngle1, &(*sCPU).polarizationAngle2,
        &(*sCPU).circularity1, &(*sCPU).circularity2, &(*sCPU).crystalTheta, &(*sCPU).crystalPhi,
        &(*sCPU).nonlinearAbsorptionStrength, &(*sCPU).drudeGamma, &(*sCPU).effectiveMass, &(*sCPU).crystalThickness,
        &(*sCPU).propagationStep };
    
    //multipliers to the Batch end value from the interface
    // (e.g. frequency in THz requires 1e12 multiplier)
    double multipliers[36] = { 0,
        1, 1, 1e12, 1e12, 
        1e12, 1e12, pi, pi, 
        1e-15, 1e-15, 1e-30, 1e-30, 
        1e-45, 1e-45, 1e-6, 1e-6,
        1e-6, 1e-6, 
        1e-6, 1e-6, 1e-6, 1e-6, 
        (pi / 180), (pi / 180), (pi / 180), (pi / 180), 
        1, 1, (pi / 180), (pi / 180), 
        1, 1e12, 1, 1e-6, 
        1e-9 };

    //Configure the struct array if in a batch
    for (j = 0; j < (*sCPU).Nsims; j++) {
        if (j > 0) {
            memcpy(&sCPU[j], sCPU, sizeof(simulationParameterSet));
        }
        
        if ((*sCPU).deffTensor != NULL) {
            sCPU[j].deffTensor = &(*sCPU).deffTensor[9 * j];;
        }

        sCPU[j].Ext = &(*sCPU).Ext[j * (*sCPU).Ngrid * 2];
        sCPU[j].Ekw = &(*sCPU).Ekw[j * (*sCPU).Ngrid * 2];
        sCPU[j].ExtOut = &(*sCPU).ExtOut[j * (*sCPU).Ngrid * 2];
        sCPU[j].EkwOut = &(*sCPU).EkwOut[j * (*sCPU).Ngrid * 2];
        sCPU[j].totalSpectrum = &(*sCPU).totalSpectrum[j * (*sCPU).Ntime * 3];

        sCPU[j].isFollowerInSequence = FALSE;
        
        // To add new modes, append values to the two arrays above, and to the combobox in the UI.
        // Cast the pointer to the original value to a pointer to a struct, 
        // increment, recast to a pointer to double and resolve then add j times the scan step size.
        *((double*)((simulationParameterSet*)targets[(*sCPU).batchIndex] + j)) += 
            j * (multipliers[(*sCPU).batchIndex] * (*sCPU).batchDestination - *targets[(*sCPU).batchIndex]) 
            / ((*sCPU).Nsims - 1);
    }
    return 0;
}
int readInputParametersFile(simulationParameterSet* sCPU, crystalEntry* crystalDatabasePtr, char* filePath) {
    FILE* textfile;
    double pi = 3.1415926535897932384626433832795;
    textfile = fopen(filePath, "r");
    if (textfile == NULL) {
        return 1;
    }
    //read parameters using fscanf:
    //recipe for programming: copy/paste the block of fprintf statements in the saveDataSet() function,
    //then find/replace:
    // fprintf->fscanf
    // (*CPU). -> &(*CPU).
    // %e -> %lf
    // &(*sCPU).sequenceString -> (*sCPU).sequenceString
    // &(*sCPU).outputBasePath -> (*sCPU).outputBasePath
    int readValueCount = 0;
    readValueCount += fscanf(textfile, "Pulse energy 1 (J): %lf\nPulse energy 2(J): %lf\nFrequency 1 (Hz): %lf\nFrequency 2 (Hz): %lf\nBandwidth 1 (Hz): %lf\nBandwidth 2 (Hz): %lf\n",
        &(*sCPU).pulseEnergy1, &(*sCPU).pulseEnergy2, &(*sCPU).frequency1, &(*sCPU).frequency2, &(*sCPU).bandwidth1, &(*sCPU).bandwidth2);
    readValueCount += fscanf(textfile, "SG order: %i\nCEP 1 (rad): %lf\nCEP 2 (rad): %lf\nDelay 1 (s): %lf\nDelay 2 (s): %lf\nGDD 1 (s^-2): %lf\nGDD 2 (s^-2): %lf\nTOD 1 (s^-3): %lf\nTOD 2(s^-3): %lf\n",
        &(*sCPU).sgOrder1, &(*sCPU).cephase1, &(*sCPU).cephase2, &(*sCPU).delay1, &(*sCPU).delay2, &(*sCPU).gdd1, &(*sCPU).gdd2, &(*sCPU).tod1, &(*sCPU).tod2);
    readValueCount += fscanf(textfile, "Phase material index: %d\nPhase material thickness 1 (mcr.): %lf\nPhase material thickness 2 (mcr.): %lf\n",
        &(*sCPU).phaseMaterialIndex, &(*sCPU).phaseMaterialThickness1, &(*sCPU).phaseMaterialThickness2);
    readValueCount += fscanf(textfile, "Beamwaist 1 (m): %lf\nBeamwaist 2 (m): %lf\nx offset 1 (m): %lf\nx offset 2 (m): %lf\nz offset 1 (m): %lf\nz offset 2 (m): %lf\nNC angle 1 (rad): %lf\nNC angle 2 (rad): %lf\n",
        &(*sCPU).beamwaist1, &(*sCPU).beamwaist2, &(*sCPU).x01, &(*sCPU).x02, &(*sCPU).z01, &(*sCPU).z02, &(*sCPU).propagationAngle1, &(*sCPU).propagationAngle2);
    readValueCount += fscanf(textfile, "Polarization 1 (rad): %lf\nPolarization 2 (rad): %lf\nCircularity 1: %lf\nCircularity 2: %lf\n",
        &(*sCPU).polarizationAngle1, &(*sCPU).polarizationAngle2, &(*sCPU).circularity1, &(*sCPU).circularity2);
    readValueCount+= fscanf(textfile, "Material index: %i\n", &(*sCPU).materialIndex);
    readValueCount += fscanf(textfile, "Crystal theta (rad): %lf\nCrystal phi (rad): %lf\nGrid width (m): %lf\ndx (m): %lf\nTime span (s): %lf\ndt (s): %lf\nThickness (m): %lf\ndz (m): %lf\n",
        &(*sCPU).crystalTheta, &(*sCPU).crystalPhi, &(*sCPU).spatialWidth, &(*sCPU).rStep, &(*sCPU).timeSpan, &(*sCPU).tStep, &(*sCPU).crystalThickness, &(*sCPU).propagationStep);
    readValueCount += fscanf(textfile, "Nonlinear absorption parameter: %lf\nBand gap (eV): %lf\nEffective mass (relative): %lf\nDrude gamma (Hz): %lf\n",
        &(*sCPU).nonlinearAbsorptionStrength, &(*sCPU).bandGapElectronVolts, &(*sCPU).effectiveMass, &(*sCPU).drudeGamma);
    readValueCount += fscanf(textfile, "Propagation mode: %i\n", &(*sCPU).symmetryType);
    readValueCount += fscanf(textfile, "Batch mode: %i\nBatch destination: %lf\nBatch steps: %lli\n",
        &(*sCPU).batchIndex, &(*sCPU).batchDestination, &(*sCPU).Nsims);
    readValueCount += fscanf(textfile, "Sequence: ");
    fgets((*sCPU).sequenceString, MAX_LOADSTRING, textfile);
    readValueCount += fscanf(textfile, "Fitting: ");
    fgets((*sCPU).fittingString, MAX_LOADSTRING, textfile);
    readValueCount += fscanf(textfile, "Fitting mode : %i\n", &(*sCPU).fittingMode);
    //readValueCount += fscanf(textfile, "Output base path: %s\n", (*sCPU).outputBasePath);
    readValueCount += fscanf(textfile, "Output base path: ");
    fgets((*sCPU).outputBasePath, MAX_LOADSTRING, textfile);
    readValueCount += fscanf(textfile, "Field 1 from file type: %i\nField 2 from file type: %i\n",
        &(*sCPU).pulse1FileType, &(*sCPU).pulse2FileType);
    //readValueCount += fscanf(textfile, "Field 1 file path: %s\n", (*sCPU).field1FilePath);
    readValueCount += fscanf(textfile, "Field 1 file path: ");
    fgets((*sCPU).field1FilePath, MAX_LOADSTRING, textfile);
    //readValueCount += fscanf(textfile, "Field 2 file path: %s\n", (*sCPU).field2FilePath);
    readValueCount += fscanf(textfile, "Field 2 file path: ");
    fgets((*sCPU).field2FilePath, MAX_LOADSTRING, textfile);

    //readValueCount += fscanf(textfile, "Fitting reference file path: %s\n", (*sCPU).fittingPath);
    readValueCount += fscanf(textfile, "Fitting reference file path: ");
    fgets((*sCPU).fittingPath, MAX_LOADSTRING, textfile);
    
    removeBreakChars((*sCPU).field1FilePath, MAX_LOADSTRING);
    removeBreakChars((*sCPU).fittingPath, MAX_LOADSTRING);
    removeBreakChars((*sCPU).field2FilePath, MAX_LOADSTRING);
    removeBreakChars((*sCPU).fittingString, MAX_LOADSTRING);
    removeBreakChars((*sCPU).sequenceString, MAX_LOADSTRING);

    //derived parameters and cleanup:
    (*sCPU).sellmeierType = 0;
    (*sCPU).axesNumber = 0;
    (*sCPU).sgOrder2 = (*sCPU).sgOrder1;
    (*sCPU).Ntime = (size_t)round((*sCPU).timeSpan / (*sCPU).tStep);
    (*sCPU).Nspace = (size_t)round((*sCPU).spatialWidth / (*sCPU).rStep);
    (*sCPU).Ngrid = (*sCPU).Ntime * (*sCPU).Nspace;
    (*sCPU).kStep = 2 * pi / ((*sCPU).Nspace * (*sCPU).rStep);
    (*sCPU).fStep = 1.0 / ((*sCPU).Ntime * (*sCPU).tStep);
    (*sCPU).Npropagation = (size_t)round((*sCPU).crystalThickness / (*sCPU).propagationStep);

    (*sCPU).isCylindric = (*sCPU).symmetryType == 1;
    if ((*sCPU).isCylindric) {
        (*sCPU).x01 = 0;
        (*sCPU).x02 = 0;
        (*sCPU).propagationAngle1 = 0;
        (*sCPU).propagationAngle2 = 0;
    }

    if ((*sCPU).batchIndex == 0 || (*sCPU).batchIndex == 4 || (*sCPU).Nsims < 1) {
        (*sCPU).Nsims = 1;
    }

    (*sCPU).field1IsAllocated = FALSE;
    (*sCPU).field2IsAllocated = FALSE;

    //crystal from database (database must be loaded!)
    (*sCPU).chi2Tensor = crystalDatabasePtr[(*sCPU).materialIndex].d;
    (*sCPU).chi3Tensor = crystalDatabasePtr[(*sCPU).materialIndex].chi3;
    (*sCPU).nonlinearSwitches = crystalDatabasePtr[(*sCPU).materialIndex].nonlinearSwitches;
    (*sCPU).absorptionParameters = crystalDatabasePtr[(*sCPU).materialIndex].absorptionParameters;
    (*sCPU).sellmeierCoefficients = crystalDatabasePtr[(*sCPU).materialIndex].sellmeierCoefficients;
    (*sCPU).sellmeierType = crystalDatabasePtr[(*sCPU).materialIndex].sellmeierType;
    (*sCPU).axesNumber = crystalDatabasePtr[(*sCPU).materialIndex].axisType;


    fclose(textfile);
    return readValueCount;
}

//print a linefeed without a carriage return so that linux systems don't complain
//about impure scripts from DOS machines
//fopen() should be called with "wb"
void unixNewLine(FILE* iostream) {
    char LF = '\x0A';
    fwrite(&LF, sizeof(char), 1, iostream);
}


int saveSlurmScript(simulationParameterSet* sCPU, int gpuType, int gpuCount) {
    FILE* textfile;
    char* stringConversionBuffer = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    wchar_t* wideStringConversionBuffer = (wchar_t*)calloc(MAX_LOADSTRING, sizeof(char));
    char* outputpath = (char*)calloc(MAX_LOADSTRING, sizeof(char));

    char* fileName = (*sCPU).outputBasePath;
    while (strchr(fileName, '\\') != NULL) {
        fileName = strchr(fileName, '\\');
        fileName++;
    }
    strcpy(outputpath, (*sCPU).outputBasePath);
    strcat(outputpath, ".slurmScript");
    textfile = fopen(outputpath, "wb");
    fprintf(textfile, "#!/bin/bash -l"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH -o ./tjob.out.%%j"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH -e ./tjob.err.%%j"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH -D ./"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH -J lightwave");  unixNewLine(textfile);
    fprintf(textfile, "#SBATCH --constraint=\"gpu\""); unixNewLine(textfile);
    if (gpuType == 0) {
        fprintf(textfile, "#SBATCH --gres=gpu:rtx5000:%i", min(gpuCount,2)); unixNewLine(textfile);
    }
    if (gpuType == 1) {
        fprintf(textfile, "#SBATCH --gres=gpu:v100:%i", min(gpuCount, 2)); unixNewLine(textfile);
    }
    if (gpuType == 2) {
        fprintf(textfile, "#SBATCH --gres=gpu:a100:%i", min(gpuCount, 4)); unixNewLine(textfile);
        fprintf(textfile, "#SBATCH --cpus-per-task=%i", 2*min(gpuCount, 4)); unixNewLine(textfile);
    }
    fprintf(textfile, "#SBATCH --mem=%lliM",1024+(18 * sizeof(double) * (*sCPU).Ngrid * max(1,(*sCPU).Nsims))/1048576);
    unixNewLine(textfile);
    fprintf(textfile, "#SBATCH --nodes=1"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH --ntasks-per-node=1"); unixNewLine(textfile);
    fprintf(textfile, "#SBATCH --time=24:00:00"); unixNewLine(textfile);
    fprintf(textfile, "module purge"); unixNewLine(textfile);
    fprintf(textfile, "module load cuda/11.2"); unixNewLine(textfile);
    fprintf(textfile, "module load mkl/2022.0"); unixNewLine(textfile);
    fprintf(textfile, "module load gcc/9"); unixNewLine(textfile);
    fprintf(textfile, "export LD_LIBRARY_PATH=$MKL_HOME/lib/intel64:$LD_LIBRARY_PATH"); unixNewLine(textfile);
    if (gpuType == 0 || gpuType == 1) {
        fprintf(textfile, "srun ./nnp %s.input > %s.out", fileName, fileName); unixNewLine(textfile);
    }
    if (gpuType == 2) {
        fprintf(textfile, "export OMP_NUM_THREADS=${SLURM_CPUS_PER_TASK}"); unixNewLine(textfile);
        fprintf(textfile, "srun ./nnp %s.input > %s.out", fileName, fileName); unixNewLine(textfile);
    }
    fclose(textfile);
    free(outputpath);
    free(wideStringConversionBuffer);
    free(stringConversionBuffer);
    return 0;
}

int saveSettingsFile(simulationParameterSet* sCPU, crystalEntry* crystalDatabasePtr) {
    int j, k;
    FILE* textfile;
    char* stringConversionBuffer = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    wchar_t* wideStringConversionBuffer = (wchar_t*)calloc(MAX_LOADSTRING, sizeof(char));
    char* outputpath = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    strcpy(outputpath, (*sCPU).outputBasePath);
    if ((*sCPU).runType > 0) {
        strcat(outputpath, ".input");
    }
    else {
        strcat(outputpath, ".txt");
    }
    
    textfile = fopen(outputpath, "w");
    fwprintf(textfile, L"Pulse energy 1 (J): %14.14e\nPulse energy 2(J): %14.14e\nFrequency 1 (Hz): %14.14e\n", 
        (*sCPU).pulseEnergy1, (*sCPU).pulseEnergy2, (*sCPU).frequency1);
    fwprintf(textfile, L"Frequency 2 (Hz): %14.14e\nBandwidth 1 (Hz): %14.14e\nBandwidth 2 (Hz): %14.14e\n", 
        (*sCPU).frequency2, (*sCPU).bandwidth1, (*sCPU).bandwidth2);
    fwprintf(textfile, L"SG order: %i\nCEP 1 (rad): %14.14e\nCEP 2 (rad): %14.14e\nDelay 1 (s): %14.14e\nDelay 2 (s): %14.14e\nGDD 1 (s^-2): %14.14e\nGDD 2 (s^-2): %14.14e\nTOD 1 (s^-3): %14.14e\nTOD 2(s^-3): %14.14e\n", 
        (*sCPU).sgOrder1, (*sCPU).cephase1, (*sCPU).cephase2, (*sCPU).delay1, (*sCPU).delay2, (*sCPU).gdd1, (*sCPU).gdd2, (*sCPU).tod1, (*sCPU).tod2);
    fwprintf(textfile, L"Phase material index: %i\nPhase material thickness 1 (mcr.): %14.14e\nPhase material thickness 2 (mcr.): %14.14e\n", 
        (*sCPU).phaseMaterialIndex, (*sCPU).phaseMaterialThickness1, (*sCPU).phaseMaterialThickness2);
    fwprintf(textfile, L"Beamwaist 1 (m): %14.14e\nBeamwaist 2 (m): %14.14e\nx offset 1 (m): %14.14e\nx offset 2 (m): %14.14e\nz offset 1 (m): %14.14e\nz offset 2 (m): %14.14e\nNC angle 1 (rad): %14.14e\nNC angle 2 (rad): %14.14e\n", 
        (*sCPU).beamwaist1, (*sCPU).beamwaist2, (*sCPU).x01, (*sCPU).x02, (*sCPU).z01, (*sCPU).z02, (*sCPU).propagationAngle1, (*sCPU).propagationAngle2);
    fwprintf(textfile, L"Polarization 1 (rad): %14.14e\nPolarization 2 (rad): %14.14e\nCircularity 1: %14.14e\nCircularity 2: %14.14e\n", 
        (*sCPU).polarizationAngle1, (*sCPU).polarizationAngle2, (*sCPU).circularity1, (*sCPU).circularity2);
    fwprintf(textfile, L"Material index: %i\n", 
        (*sCPU).materialIndex);
    fwprintf(textfile, L"Crystal theta (rad): %14.14e\nCrystal phi (rad): %14.14e\nGrid width (m): %14.14e\ndx (m): %14.14e\nTime span (s): %14.14e\ndt (s): %14.14e\nThickness (m): %14.14e\ndz (m): %14.14e\n", 
        (*sCPU).crystalTheta, (*sCPU).crystalPhi, (*sCPU).spatialWidth, (*sCPU).rStep, (*sCPU).timeSpan, (*sCPU).tStep, (*sCPU).crystalThickness, (*sCPU).propagationStep);
    fwprintf(textfile, L"Nonlinear absorption parameter: %14.14e\nBand gap (eV): %14.14e\nEffective mass (relative): %14.14e\nDrude gamma (Hz): %14.14e\n", 
        (*sCPU).nonlinearAbsorptionStrength, (*sCPU).bandGapElectronVolts, (*sCPU).effectiveMass, (*sCPU).drudeGamma);
    fwprintf(textfile, L"Propagation mode: %i\n", 
        (*sCPU).symmetryType);
    fwprintf(textfile, L"Batch mode: %i\nBatch destination: %14.14e\nBatch steps: %lli\n", 
        (*sCPU).batchIndex, (*sCPU).batchDestination, (*sCPU).Nsims);
    mbstowcs(wideStringConversionBuffer, (*sCPU).sequenceString, MAX_LOADSTRING);
    fwprintf(textfile, L"Sequence: %ls\n", wideStringConversionBuffer);
    mbstowcs(wideStringConversionBuffer, (*sCPU).fittingString, MAX_LOADSTRING);
    fwprintf(textfile, L"Fitting: %ls\n", wideStringConversionBuffer);
    fwprintf(textfile, L"Fitting mode: %i\n", (*sCPU).fittingMode);

    if ((*sCPU).runType > 0) {
        char* fileName = (*sCPU).outputBasePath;
        while (strchr(fileName, '\\') != NULL) {
            fileName = strchr(fileName, '\\');
            fileName++;
        }
        mbstowcs(wideStringConversionBuffer, fileName, strlen(fileName));
        wideStringConversionBuffer[strlen(fileName)] = L'\0';
        fwprintf(textfile, L"Output base path: %ls\n", wideStringConversionBuffer);
    }
    else {
        mbstowcs(wideStringConversionBuffer, (*sCPU).outputBasePath, MAX_LOADSTRING);
        fwprintf(textfile, L"Output base path: %ls\n", wideStringConversionBuffer);
    }

    fwprintf(textfile, L"Field 1 from file type: %i\nField 2 from file type: %i\n", (*sCPU).pulse1FileType, (*sCPU).pulse2FileType);
    mbstowcs(wideStringConversionBuffer, (*sCPU).field1FilePath, MAX_LOADSTRING);
    fwprintf(textfile, L"Field 1 file path: %ls\n", wideStringConversionBuffer);
    mbstowcs(wideStringConversionBuffer, (*sCPU).field2FilePath, MAX_LOADSTRING);
    fwprintf(textfile, L"Field 2 file path: %ls\n", wideStringConversionBuffer);
    mbstowcs(wideStringConversionBuffer, (*sCPU).fittingPath, MAX_LOADSTRING);
    fwprintf(textfile, L"Fitting reference file path: %ls\n", wideStringConversionBuffer);

    fwprintf(textfile, L"Material name: %ls\nSellmeier reference: %ls\nChi2 reference: %ls\nChi3 reference: %ls\n", crystalDatabasePtr[(*sCPU).materialIndex].crystalNameW, crystalDatabasePtr[(*sCPU).materialIndex].sellmeierReference, crystalDatabasePtr[(*sCPU).materialIndex].dReference, crystalDatabasePtr[(*sCPU).materialIndex].chi3Reference);
    fwprintf(textfile, L"Sellmeier coefficients: \n");
    for (j = 0; j < 3; j++) {
        for (k = 0; k < 22; k++) {
            fwprintf(textfile, L"%14.14e ", crystalDatabasePtr[(*sCPU).materialIndex].sellmeierCoefficients[j * 22 + k]);
        }
        fwprintf(textfile, L"\n");
    }
    fwprintf(textfile, L"Code version: 0.15 May 6, 2022\n");

    fclose(textfile);
    free(outputpath);
    free(wideStringConversionBuffer);
    free(stringConversionBuffer);
    return 0;
}

int removeBreakChars(char* cString, size_t N) {
    size_t i = 0;
    size_t r = 0;
    while (i < N) {
        if (cString[i] == '\r' || cString[i] == '\n') {
            memmove(&cString[i], &cString[i + 1], N - i - r - 1);
            cString[N - r - 1] = 0;
            r++;
        }
        else {
            i++;
        }
    }
    return 0;
}

int saveDataSet(simulationParameterSet* sCPU, crystalEntry* crystalDatabasePtr, char* outputbase, bool saveInputs) {
    int j;

    saveSettingsFile(sCPU, crystalDatabasePtr);

    //Save the results as double instead of complex
    double* saveEout = (double*)calloc((*sCPU).Ngrid * 2 * (*sCPU).Nsims, sizeof(double));
    for (j = 0; j < ((*sCPU).Ngrid * (*sCPU).Nsims * 2); j++) {
        saveEout[j] = real((*sCPU).ExtOut[j]);
    }

    char* stringConversionBuffer = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    wchar_t* wideStringConversionBuffer = (wchar_t*)calloc(MAX_LOADSTRING, sizeof(char));
    char* outputpath = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    char* outputbaseVar = strrchr(outputbase, '\\');
    if (!outputbaseVar) {
        outputbaseVar = outputbase;
    }
    else {
        outputbaseVar++;
    }
    double* matlabpadding = (double*)calloc(1024, sizeof(double));

    
    
    
    //write fields as binary
    for (j = 0; j < ((*sCPU).Ngrid * (*sCPU).Nsims * 2); j++) {
        saveEout[j] = real((*sCPU).ExtOut[j]);
    }
    FILE* ExtOutFile;
    size_t writeSize = 2 * ((*sCPU).Ngrid * (*sCPU).Nsims);
    strcpy(outputpath, outputbase);
    strcat(outputpath, "_Ext.dat");
    ExtOutFile = fopen(outputpath, "wb");
    fwrite(saveEout, sizeof(double), writeSize, ExtOutFile);
    fwrite(matlabpadding, sizeof(double), 1024, ExtOutFile);
    fclose(ExtOutFile);
    if (saveInputs) {
        for (j = 0; j < ((*sCPU).Ngrid * (*sCPU).Nsims * 2); j++) {
            saveEout[j] = real((*sCPU).Ext[j]);
        }
        FILE* ExtInFile;
        strcpy(outputpath, outputbase);
        strcat(outputpath, "_ExtIn.dat");
        ExtInFile = fopen(outputpath, "wb");
        fwrite(saveEout, sizeof(double), writeSize, ExtInFile);
        fwrite(matlabpadding, sizeof(double), 1024, ExtInFile);
        fclose(ExtInFile);
    }

    //Save the spectrum
    FILE* totalSpectrumFile;
    strcpy(outputpath, outputbase);
    strcat(outputpath, "_spectrum.dat");
    totalSpectrumFile = fopen(outputpath, "wb");
    fwrite((*sCPU).totalSpectrum, sizeof(double), 3 * (*sCPU).Ntime * (*sCPU).Nsims, totalSpectrumFile);
    fwrite(matlabpadding, sizeof(double), 1024, totalSpectrumFile);
    fclose(totalSpectrumFile);

    FILE* matlabfile;
    strcpy(outputpath, outputbase);
    strcat(outputpath, ".m");
    matlabfile = fopen(outputpath, "w");
    
    if (saveInputs) {
        fprintf(matlabfile, "fid = fopen('%s_ExtIn.dat','rb'); \n", outputbaseVar);
        fprintf(matlabfile, "%s_ExtIn = fread(fid, %lli, 'double'); \n", outputbaseVar, 2 * (*sCPU).Ngrid * (*sCPU).Nsims);
        fprintf(matlabfile, "%s_ExtIn = reshape(%s_ExtIn,[%lli %lli %lli]); \n", outputbaseVar, outputbaseVar, (*sCPU).Ntime, (*sCPU).Nspace, 2 * (*sCPU).Nsims);
        fprintf(matlabfile, "fclose(fid); \n");
    }
    
    fprintf(matlabfile, "fid = fopen('%s_Ext.dat','rb'); \n", outputbaseVar);
    fprintf(matlabfile, "%s_Ext = fread(fid, %lli, 'double'); \n", outputbaseVar, 2 * (*sCPU).Ngrid * (*sCPU).Nsims);
    fprintf(matlabfile, "%s_Ext = reshape(%s_Ext,[%lli %lli %lli]); \n", outputbaseVar, outputbaseVar, (*sCPU).Ntime, (*sCPU).Nspace, 2 * (*sCPU).Nsims);
    fprintf(matlabfile, "fclose(fid); \n");
    fprintf(matlabfile, "fid = fopen('%s_spectrum.dat','rb'); \n", outputbaseVar);
    fprintf(matlabfile, "%s_spectrum = fread(fid, %lli, 'double'); \n", outputbaseVar, 3 * (*sCPU).Ntime * (*sCPU).Nsims);
    fprintf(matlabfile, "%s_spectrum = reshape(%s_spectrum,[%lli %i %zi]); \n", outputbaseVar, outputbaseVar, (*sCPU).Ntime, 3, (*sCPU).Nsims);
    fprintf(matlabfile, "fclose(fid); \n");
    fprintf(matlabfile, "dt = %e;\ndz = %e;\ndx = %e;\ndf = %e;\n", (*sCPU).tStep, (*sCPU).propagationStep, (*sCPU).rStep, (*sCPU).fStep);
    fclose(matlabfile);
    
    //write a python script for loading the output fields in a proper shape
    char scriptfilename[MAX_LOADSTRING];
    strcpy(scriptfilename, outputbase);
    strcat(scriptfilename, ".py");
    FILE* scriptfile;
    scriptfile = fopen(scriptfilename, "w");
    fprintf(scriptfile, "#!/usr/bin/python\nimport numpy as np\n");
    fprintf(scriptfile, "dt = %e\ndz = %e\ndx = %e\ndf = %e\n", (*sCPU).tStep, (*sCPU).propagationStep, (*sCPU).rStep, (*sCPU).fStep);
    if (saveInputs) {
        fprintf(scriptfile, "%s_ExtIn = np.reshape(np.fromfile(\"", outputbaseVar);
        fprintf(scriptfile, "%s_ExtIn.dat", outputbaseVar);
        fprintf(scriptfile, "\",dtype=np.double)[0:%lli],(%lli,%lli,%lli),order='F')\n", 2 * (*sCPU).Ngrid * (*sCPU).Nsims, (*sCPU).Ntime, (*sCPU).Nspace, 2 * (*sCPU).Nsims);
    }
    fprintf(scriptfile, "%s_Ext = np.reshape(np.fromfile(\"", outputbaseVar);
    fprintf(scriptfile, "%s_Ext.dat", outputbaseVar);
    fprintf(scriptfile, "\",dtype=np.double)[0:%lli],(%lli,%lli,%lli),order='F')\n", 2 * (*sCPU).Ngrid * (*sCPU).Nsims, (*sCPU).Ntime, (*sCPU).Nspace, 2 * (*sCPU).Nsims);
    fprintf(scriptfile, "%s_spectrum = np.reshape(np.fromfile(\"", outputbaseVar);
    fprintf(scriptfile, "%s_spectrum.dat", outputbaseVar);
    fprintf(scriptfile, "\",dtype=np.double)[0:%lli],(%lli,%i,%zi),order='F')\n", 3 * (*sCPU).Ntime * (*sCPU).Nsims, (*sCPU).Ntime, 3, (*sCPU).Nsims);
    fclose(scriptfile);
    
    free(saveEout);
    free(matlabpadding);
    free(stringConversionBuffer);
    free(wideStringConversionBuffer);
    return 0;
}

int resolveSequence(int currentIndex, simulationParameterSet* s, crystalEntry* db) {
    double pi = 3.1415926535897932384626433832795;
    double* offsetArray = &(*s).sequenceArray[11 * currentIndex];
    
    //sequence format
    //0: step type
    int stepType = (int)offsetArray[0];
    // 
    // if stepType == 0, normal propagation
    //1: material index
    //2: theta,
    //3: phi, 
    //4: NL absorption
    //5: Band gap
    //6: Drude relaxation
    //7: Effective mass
    //8: Crystal thickness
    //9: Propagation step size
    //10: rotation angle
    //
    // if stepType == 1, linear propagation
    // same parameters as 0, but only 1,2,3,8, and 10 matter
    //
    // if stepType == 2, fresnel loss
    // 1: incidence material index
    // 2: transmission material index
    // other parameters don't matter
    // 
    // if stepType == 3, spherical mirror
    // 1: ROC (m)

    
    switch (stepType) {
    case 0:
        if ((int)offsetArray[1] != -1) (*s).materialIndex = (int)offsetArray[1];
        if ((int)offsetArray[2] != -1) (*s).crystalTheta = (pi / 180) * offsetArray[2];
        if ((int)offsetArray[3] != -1) (*s).crystalPhi = (pi / 180) * offsetArray[3];
        if ((int)offsetArray[4] != -1) (*s).nonlinearAbsorptionStrength = offsetArray[4];
        if ((int)offsetArray[5] != -1) (*s).bandGapElectronVolts = offsetArray[5];
        if ((int)offsetArray[6] != -1) (*s).drudeGamma = offsetArray[6];
        if ((int)offsetArray[7] != -1) (*s).effectiveMass = offsetArray[7];
        if ((int)offsetArray[8] != -1) (*s).crystalThickness = 1e-6 * offsetArray[8];
        if ((int)offsetArray[9] != -1) (*s).propagationStep = 1e-9 * offsetArray[9];
        if ((int)offsetArray[8] != -1 && (int)offsetArray[8] != -1) (*s).Npropagation 
            = (size_t)(1e-6 * offsetArray[8] / (*s).propagationStep);
        if (currentIndex > 0) {
            (*s).isFollowerInSequence = TRUE;
        }
        (*s).chi2Tensor = db[(*s).materialIndex].d;
        (*s).chi3Tensor = db[(*s).materialIndex].chi3;
        (*s).nonlinearSwitches = db[(*s).materialIndex].nonlinearSwitches;
        (*s).absorptionParameters = db[(*s).materialIndex].absorptionParameters;
        (*s).sellmeierCoefficients = db[(*s).materialIndex].sellmeierCoefficients;

        (*s).sellmeierType = db[(*s).materialIndex].sellmeierType;
        (*s).axesNumber = db[(*s).materialIndex].axisType;


        solveNonlinearWaveEquation(s);
        if (offsetArray[10] != 0.0) {
            rotateField(s, (pi / 180) * offsetArray[10]);
        }

        if ((*s).memoryError > 0) {
            printf("Warning: device memory error (%i).\n", (*s).memoryError);
        }
        return 0;
    case 1:
        //not implemented yet!
        return 1;
    case 2:
        applyFresnelLoss(s, (int)offsetArray[1], (int)offsetArray[2]);
        return 0;
    }
    


    return 1;
}

int loadPulseFiles(simulationParameterSet* sCPU) {

    //pulse type specifies if something has to be loaded to describe the pulses, or if they should be
    //synthesized later. 1: FROG .speck format; 2: EOS (not implemented yet)
    int frogLines = 0;
    int errCount = 0;
    if ((*sCPU).pulse1FileType == 1) {
        frogLines = loadFrogSpeck((*sCPU).field1FilePath, (*sCPU).loadedField1, (*sCPU).Ntime, (*sCPU).fStep, 0.0, 1);
        if (frogLines > 1) {
            (*sCPU).field1IsAllocated = TRUE;
        }
        else {
            (*sCPU).field1IsAllocated = FALSE;
            errCount++;
        }
    }

    if ((*sCPU).pulse2FileType == 1) {
        frogLines = loadFrogSpeck((*sCPU).field2FilePath, (*sCPU).loadedField2, (*sCPU).Ntime, (*sCPU).fStep, 0.0, 1);
        if (frogLines > 1) {
            (*sCPU).field2IsAllocated = TRUE;
        }
        else {
            (*sCPU).field2IsAllocated = FALSE;
            errCount++;
        }
    }
    return errCount;
}

int loadSavedFields(simulationParameterSet* sCPU, char* outputBase, bool GPUisPresent) {
    char* outputpath = (char*)calloc(MAX_LOADSTRING, sizeof(char));
    size_t writeSize = 2 * ((*sCPU).Ngrid * (*sCPU).Nsims);
    double* loadE = (double*)malloc(writeSize * sizeof(double));
    size_t j;

    //read fields as binary
    FILE* ExtOutFile;
    
    strcpy(outputpath, outputBase);
    strcat(outputpath, "_Ext.dat");
    ExtOutFile = fopen(outputpath, "rb");
    if (ExtOutFile == NULL) {
        return 1;
    }
    fread(loadE, sizeof(double), writeSize, ExtOutFile);
    fclose(ExtOutFile);
    for (j = 0; j < writeSize; j++) {
        (*sCPU).ExtOut[j] = loadE[j];
    }
    free(loadE);

    FILE* spectrumFile;
    strcpy(outputpath, outputBase);
    strcat(outputpath, "_spectrum.dat");
    spectrumFile = fopen(outputpath, "rb");
    fread((*sCPU).totalSpectrum, sizeof(double), (*sCPU).Nsims * 3 * (*sCPU).Ntime, spectrumFile);
    fclose(spectrumFile);
    free(outputpath);
    if (GPUisPresent) {
        hipfftHandle fftPlan;
        hipfftPlan2d(&fftPlan, (int)(*sCPU).Nspace, (int)(*sCPU).Ntime, HIPFFT_Z2Z);

        hipDoubleComplex* fieldGridkw;
        hipDoubleComplex* fieldGridxt;
        hipMalloc((void**)&fieldGridkw, sizeof(hipDoubleComplex) * (*sCPU).Ngrid);
        hipMalloc((void**)&fieldGridxt, sizeof(hipDoubleComplex) * (*sCPU).Ngrid);

        for (j = 0; j < 2 * (*sCPU).Nsims; j++) {
            hipMemcpy(fieldGridxt, &(*sCPU).ExtOut[j * (*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
            hipfftExecZ2Z(fftPlan, fieldGridxt, fieldGridkw, HIPFFT_FORWARD);
            hipMemcpy(&(*sCPU).EkwOut[j * (*sCPU).Ngrid], fieldGridkw, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        }
        hipFree(fieldGridkw);
        hipFree(fieldGridxt);
        hipfftDestroy(fftPlan);
    }
    else {
        DFTI_DESCRIPTOR_HANDLE dftiHandle = NULL;
        MKL_LONG fftDimensions[2] = { (long)(*sCPU).Nspace , (long)(*sCPU).Ntime };
        MKL_LONG mklError = 0;
        mklError = DftiCreateDescriptor(&dftiHandle, DFTI_DOUBLE, DFTI_COMPLEX, 2, fftDimensions);
        DftiSetValue(dftiHandle, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
        if (mklError != DFTI_NO_ERROR) return 1;
        mklError = DftiCommitDescriptor(dftiHandle);
        if (mklError != DFTI_NO_ERROR) return 2;
        for (j = 0; j < (2 * (*sCPU).Nsims); j++) {
            mklError = DftiComputeForward(dftiHandle, &(*sCPU).ExtOut[j*(*sCPU).Ngrid], &(*sCPU).EkwOut[j*(*sCPU).Ngrid]);
            if (mklError != DFTI_NO_ERROR) return 3;
        }

        DftiFreeDescriptor(&dftiHandle);
    }


    return 0;
}


int getTotalSpectrum(simulationParameterSet* sCPU, cudaParameterSet* sc) {
    hipfftHandle plan1;
    hipfftPlan1d(&plan1, (int)(*sc).Ntime, HIPFFT_Z2Z, (int)(*sc).Nspace);
    hipfftSetStream(plan1, (*sc).CUDAStream);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETime1, (hipfftDoubleComplex*)(*sc).gridETemp1, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);

    totalSpectrumKernel<<<(unsigned int)(*sc).Ntime, 1, 0, (*sc).CUDAStream>>>((*sc).gridETemp1, (*sc).gridETemp2, (*sCPU).rStep, (*sc).Ntime, (*sc).Nspace, (*sc).gridPolarizationTime1);
    hipDeviceSynchronize();
    hipMemcpy((*sCPU).totalSpectrum, (*sc).gridPolarizationTime1, 3 * (*sc).Ntime * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipfftDestroy(plan1);
    return 0;
}

unsigned long runFitting(simulationParameterSet* sCPU) {
    int n = (int)(*sCPU).Nfitting;
    int m = (int)(*sCPU).fittingROIsize;
    fittingReferenceSet = sCPU;
    fittingSet = (simulationParameterSet*)malloc((*sCPU).Nsims * sizeof(simulationParameterSet));
    memcpy(fittingSet, sCPU, (*sCPU).Nsims * sizeof(simulationParameterSet));
    
    double commonPrecision = (*sCPU).fittingPrecision;
    const double eps[6] = { commonPrecision,commonPrecision,commonPrecision,commonPrecision,commonPrecision,commonPrecision }; /* set precisions for stop-criteria */
    double jacobianPrecision = commonPrecision;
    double* x = (double*)mkl_malloc(sizeof(double) * n, 64);
    double* fittingValues = (double*)mkl_malloc(sizeof(double) * m, 64);
    double* fjac = (double*)mkl_malloc(sizeof(double) * m * n, 64);
    double* lowerBounds = (double*)mkl_malloc(sizeof(double) * n, 64);
    double* upperBounds = (double*)mkl_malloc(sizeof(double) * n, 64);
    const int maxIterations = max((*sCPU).fittingMaxIterations,2);
    const int maxTrialIterations = max(maxIterations/10, 2);
    /* initial step bound */
    double rs = 0.0;
    int RCI_Request;
    int successful;

    int iter;
    int stopCriterion;
    double inputResiduals = 0.0, outputResiduals = 0.0;
    _TRNSPBC_HANDLE_t handle;
    int i;
    int error = 0;

    //initial guess and bounds
    for (i = 0; i < n; i++) {
        x[i] = 1.;
        upperBounds[i] = (*fittingSet).fittingArray[3 * i + 2];
        lowerBounds[i] = (*fittingSet).fittingArray[3 * i + 1];
    }

    //initialize fitting function and jacobian
    for (i = 0; i < m; i++) {
        fittingValues[i] = 0.0;
    }
    for (i = 0; i < m * n; i++) {
        fjac[i] = 0.0;
    }

    error += dtrnlspbc_init(&handle, &n, &m, x, lowerBounds, upperBounds, eps, &maxIterations, &maxTrialIterations, &rs) != TR_SUCCESS;
    size_t currentIteration = 0;
    if (error == 0) {
        RCI_Request = 0;
        successful = 0;
        while (successful == 0 && (*sCPU).imdone[0] != 2 && currentIteration < maxIterations)
        {
            currentIteration++;
            if (dtrnlspbc_solve(&handle, fittingValues, fjac, &RCI_Request) != TR_SUCCESS)
            {
                successful = -1;
            }

            //check convergence
            if (RCI_Request > -7 && RCI_Request < -1) successful = 1;

            //recalculate
            if (RCI_Request == 1)
            {
                runFittingIteration(&m, &n, x, fittingValues);
            }

            //make jacobian
            if (RCI_Request == 2)
            {
                djacobi(runFittingIteration, &n, &m, fjac, x, &jacobianPrecision);
            }
        }
    }

    /* get solution statuses
       handle            in:        TR solver handle
       iter              out:       number of iterations
       stopCriterion             out:       number of stop criterion
       inputResiduals                out:       initial residuals
       outputResiduals                out:       final residuals */
    dtrnlspbc_get(&handle, &iter, &stopCriterion, &inputResiduals, &outputResiduals);
    memcpy(sCPU, fittingSet, (*fittingSet).Nsims * sizeof(simulationParameterSet));
    //free memory
    dtrnlspbc_delete(&handle);
    mkl_free(upperBounds);
    mkl_free(lowerBounds);
    mkl_free(fjac);
    mkl_free(fittingValues);
    mkl_free(x);
    MKL_Free_Buffers();
    free(fittingSet);
    return 0;
}


void runFittingIteration(int* m, int* n, double* fittingValues, double* fittingFunction) {
    int i;
    int fitLocation;
    double referenceValue;
    //pointers to values that can be scanned in batch mode
    double* targets[36] = { 0,
        &(*fittingSet).pulseEnergy1, &(*fittingSet).pulseEnergy2, &(*fittingSet).frequency1, &(*fittingSet).frequency2,
        &(*fittingSet).bandwidth1, &(*fittingSet).bandwidth2, &(*fittingSet).cephase1, &(*fittingSet).cephase2,
        &(*fittingSet).delay1, &(*fittingSet).delay2, &(*fittingSet).gdd1, &(*fittingSet).gdd2,
        &(*fittingSet).tod1, &(*fittingSet).tod2, &(*fittingSet).phaseMaterialThickness1, &(*fittingSet).phaseMaterialThickness2,
        &(*fittingSet).beamwaist1, &(*fittingSet).beamwaist2,
        &(*fittingSet).x01, &(*fittingSet).x02, &(*fittingSet).z01, &(*fittingSet).z02,
        &(*fittingSet).propagationAngle1, &(*fittingSet).propagationAngle2, &(*fittingSet).polarizationAngle1, &(*fittingSet).polarizationAngle2,
        &(*fittingSet).circularity1, &(*fittingSet).circularity2, &(*fittingSet).crystalTheta, &(*fittingSet).crystalPhi,
        &(*fittingSet).nonlinearAbsorptionStrength, &(*fittingSet).drudeGamma, &(*fittingSet).effectiveMass, &(*fittingSet).crystalThickness,
        &(*fittingSet).propagationStep };

    double* references[36] = { 0,
    &(*fittingReferenceSet).pulseEnergy1, &(*fittingReferenceSet).pulseEnergy2, &(*fittingReferenceSet).frequency1, &(*fittingReferenceSet).frequency2,
    &(*fittingReferenceSet).bandwidth1, &(*fittingReferenceSet).bandwidth2, &(*fittingReferenceSet).cephase1, &(*fittingReferenceSet).cephase2,
    &(*fittingReferenceSet).delay1, &(*fittingReferenceSet).delay2, &(*fittingReferenceSet).gdd1, &(*fittingReferenceSet).gdd2,
    &(*fittingReferenceSet).tod1, &(*fittingReferenceSet).tod2, &(*fittingReferenceSet).phaseMaterialThickness1, &(*fittingReferenceSet).phaseMaterialThickness2,
    &(*fittingReferenceSet).beamwaist1, &(*fittingReferenceSet).beamwaist2,
    &(*fittingReferenceSet).x01, &(*fittingReferenceSet).x02, &(*fittingReferenceSet).z01, &(*fittingReferenceSet).z02,
    &(*fittingReferenceSet).propagationAngle1, &(*fittingReferenceSet).propagationAngle2, &(*fittingReferenceSet).polarizationAngle1, &(*fittingReferenceSet).polarizationAngle2,
    &(*fittingReferenceSet).circularity1, &(*fittingReferenceSet).circularity2, &(*fittingReferenceSet).crystalTheta, &(*fittingReferenceSet).crystalPhi,
    &(*fittingReferenceSet).nonlinearAbsorptionStrength, &(*fittingReferenceSet).drudeGamma, &(*fittingReferenceSet).effectiveMass, &(*fittingReferenceSet).crystalThickness,
    &(*fittingReferenceSet).propagationStep };

    
    for (i = 0; i < *n; i++) {
        fitLocation = (int)round((*fittingSet).fittingArray[3 * i]);
        referenceValue = *references[fitLocation];
        if (referenceValue == 0.0) {
            referenceValue = 1.;
        }
        *targets[fitLocation] = fittingValues[i] * referenceValue;
    }
    if ((*fittingSet).isInSequence) {
        solveNonlinearWaveEquationSequence(fittingSet);
        (*fittingSet).isFollowerInSequence = FALSE;
    }
    else {
        solveNonlinearWaveEquation(fittingSet);
    }
    

    //mode 0: maximize total spectrum in ROI
    if ((*fittingSet).fittingMode == 0) {
        for (i = 0; i < *m; i++) {
            fittingFunction[i] = log10(1.0e8 / ((*fittingSet).totalSpectrum[2 * (*fittingSet).Ntime + (*fittingSet).fittingROIstart + i]));
        }
    }
    //mode 1: maximize s-polarized spectrum in ROI
    if ((*fittingSet).fittingMode == 1) {
        for (i = 0; i < *m; i++) {
            fittingFunction[i] = log10(1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).fittingROIstart + i]));
        }
    }
    //mode 2: maximize p-polarized spectrum in ROI
    if ((*fittingSet).fittingMode == 2) {
        for (i = 0; i < *m; i++) {
            fittingFunction[i] = log10(1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).Ntime + (*fittingSet).fittingROIstart + i]));
        }
    }
    //mode 3: match total spectrum to reference given in ascii file
    if ((*fittingSet).fittingMode == 3) {
        double maxSim = 0;
        double maxRef = 0;
        double* simSpec = &(*fittingSet).totalSpectrum[2 * (*fittingSet).Ntime + (*fittingSet).fittingROIstart];
        double* refSpec = &(*fittingSet).fittingArray[(*fittingSet).fittingROIstart];
        for (i = 0; i < *m; i++) {
            maxSim = max(maxSim, simSpec[i]);
            maxRef = max(maxRef, refSpec[i]);
        }
        if (maxSim == 0) {
            maxSim = 1;
        }
        if (maxRef == 0) {
            maxRef = 1;
        }
        for (i = 0; i < *m; i++) {
            fittingFunction[i] = log10(1e5*refSpec[i] / maxRef) - log10(1e5*simSpec[i] / maxSim);
            //fittingFunction[i] = 1.0e8 / ((*fittingSet).totalSpectrum[(*fittingSet).Ntime + (*fittingSet).fittingROIstart + i]);
        }
    }


    return;
}