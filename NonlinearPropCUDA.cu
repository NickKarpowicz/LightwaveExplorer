#include "hip/hip_runtime.h"
#include "NonlinearPropCUDA.cuh"
#include "framework.h"
#include<complex>
#include<cstdlib>
#include<math.h>
#include "MPQ_Nonlinear_Propagation.h"
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "qr_solve.hpp"
#include "MPQ_Nonlinear_Propagation.h"
//#include <complex.h>

#define THREADS_PER_BLOCK 64
#define MAX_LOADSTRING 1024
//overload the math operators for cuda complex numbers so this code fits inside the observable universe
__device__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b) { return hipCmul(a, b); }
__device__ hipDoubleComplex operator+(hipDoubleComplex a, hipDoubleComplex b) { return hipCadd(a, b); }
__device__ hipDoubleComplex operator+(double a, hipDoubleComplex b) { return hipCadd(make_hipDoubleComplex(a, 0.0), b); }
__device__ hipDoubleComplex operator-(hipDoubleComplex a, hipDoubleComplex b) { return hipCsub(a, b); }
__device__ hipDoubleComplex operator-(double a, hipDoubleComplex b) { return hipCsub(make_hipDoubleComplex(a, 0.0), b); }
__device__ hipDoubleComplex operator/(hipDoubleComplex a, double b) { return hipCdiv(a, make_hipDoubleComplex(b, 0.0)); }
__device__ hipDoubleComplex operator/(double b, hipDoubleComplex a) { return hipCdiv(make_hipDoubleComplex(b, 0.0), a); }
__device__ hipDoubleComplex operator*(hipDoubleComplex a, double b) { return hipCmul(a, make_hipDoubleComplex(b, 0.0)); }
__device__ hipDoubleComplex operator*(double b, hipDoubleComplex a) { return hipCmul(a, make_hipDoubleComplex(b, 0.0)); }


//complex exponential function for CUDA
__device__ __forceinline__ hipDoubleComplex cuCexpd(hipDoubleComplex z)
{
    hipDoubleComplex res;
    double t = exp(z.x);
    res.y = sin(z.y);
    res.x = cos(z.y);
    res.x *= t;
    res.y *= t;
    return res;
}

//copy and paste from
// https://forums.developer.nvidia.com/t/additional-cucomplex-functions-cucnorm-cucsqrt-cucexp-and-some-complex-double-functions/36892 
__device__ hipDoubleComplex cuCsqrt(hipDoubleComplex x)
{
    double radius = hipCabs(x);
    double cosA = x.x / radius;
    hipDoubleComplex out;
    out.x = sqrt(radius * (cosA + 1.0) / 2.0);
    out.y = sqrt(radius * (1.0 - cosA) / 2.0);
    // signbit should be false if x.y is negative
    if (signbit(x.y))
        out.y *= -1.0;

    return out;
}


__device__ hipDoubleComplex sellmeierCuda(hipDoubleComplex* ne, hipDoubleComplex* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return make_hipDoubleComplex(1.0,0.0); //exit immediately for f=0
    
    double c = 2.99792458e8; //speed of light
    double l = 1e6 * c / f; //wavelength in microns
    double ls = l * l;
    hipDoubleComplex ii = make_hipDoubleComplex(0.0, 1.0);
    double pi = 3.14159265358979323846264338327950288;
    double omega = 2 * pi * abs(f);
    double kL = 3183.9; //(e * e / (e_o *m_e)
    hipDoubleComplex one = make_hipDoubleComplex(1.0, 0);
    hipDoubleComplex na = one;
    hipDoubleComplex nb = one;
    //option 0: isotropic
    if (type == 0) {
        ne[0] = make_hipDoubleComplex(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6])
            + (a[7] + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls, 0.0);
        if (hipCreal(ne[0]) < 1) {
            ne[0] = one;
        }
        ne[0] = ne[0] + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega);
        ne[0] = hipConj(cuCsqrt(ne[0]));
        if (isnan(hipCreal(ne[0]))) {
            ne[0] = one;
        }

        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {
        
        na = cuCsqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega));
        
        a = &a[22];
        nb = cuCsqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega));
        no[0] = na;
        ne[0] = 1.0 / cuCsqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return ne[0];
    }
    else {
        //later, implement biaxial crystals, for now just return 1;
        return one;
    }
}

__global__ void thetasearchKernel(long long Ntime, long long Nspace, double* theta, double* sellmeierCoefficients, int axesNumber, int sellmeierType) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j, k;

    j = i / Ntime; //spatial coordinate
    k = i - j*Ntime; //temporal coordinate
    
    double crystalTheta = sellmeierCoefficients[66];
    double crystalPhi = sellmeierCoefficients[67];
    double kStep = sellmeierCoefficients[70];
    double fStep = sellmeierCoefficients[71];
    double tol = sellmeierCoefficients[72];

    double f = k * fStep;
	if (k >= Ntime / 2) {
		f -= fStep * Ntime;
	}
	f *= -1;
	double dk = j * kStep - (j >= (Nspace / 2)) * (kStep * Nspace); //frequency grid in transverse direction

    theta[i] = 0;
    double dTheta = 0.1;
    double err, errPlus, errMinus;
    double rhs = 2.99792458e8 * dk / (2 * 3.14159265358979323846264338327950288 * f);
    hipDoubleComplex ne, no;
    double nePlus, neMinus;
    f = abs(f);
    
    sellmeierCuda(&ne, &no, sellmeierCoefficients, f, crystalTheta + theta[i], crystalPhi, axesNumber, sellmeierType);    
    nePlus = hipCreal(ne);
    err = abs(nePlus * sin(theta[i]) - rhs);

    int iters = 0;
    errPlus = 2;
    errMinus = 2;
    while (err > tol && iters < 64) {
        iters++;

        sellmeierCuda(&ne, &no, sellmeierCoefficients, f, crystalTheta + theta[i] + dTheta, crystalPhi, axesNumber, sellmeierType);
        nePlus = hipCreal(ne);
        errPlus = abs(nePlus * sin(theta[i] + dTheta) - rhs);

        sellmeierCuda(&ne, &no, sellmeierCoefficients, f, crystalTheta + theta[i] - dTheta, crystalPhi, axesNumber, sellmeierType);
        neMinus = hipCreal(ne);
        errMinus = abs(neMinus * sin(theta[i] - dTheta) - rhs);

        //Basic hill climbing algorithm
        //calculate the error at theta +/- dTheta
        // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
        // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
        // if theta has lowest error, step size is too large, dTheta /= 2;
        if (errPlus < err && errPlus < errMinus) {
            theta[i] += dTheta;
            err = errPlus;
        }
        else if (errMinus < err) {
            theta[i] -= dTheta;
            err = errMinus;
        }
        else {
            dTheta *= 0.5;
        }
        
    }
    
    
}

//replaces E with its complex conjugate
__global__ void conjugateKernel(hipDoubleComplex* E) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    E[i] = hipConj(E[i]);
}
__global__ void nonlinearpolarizationKernel(struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    double Ex = hipCreal(s.gridETime[i]) / s.propagationInts[0];
    double Ey = hipCreal(s.gridETime2[i]) / s.propagationInts[0];
    s.gridPolarizationTime[i] = 0.;
    s.gridPolarizationTime2[i] = 0.;

    //The d2eff tensor has the form
    // | d_xxx d_xyx d_yyx |
    // | d_xxy d_xyy d_yyy |
    if (s.nonlinearSwitches[0] == 1) {
        s.gridPolarizationTime[i] += s.chi2Tensor[0] * Ex * Ex + s.chi2Tensor[2] * Ex * Ey + s.chi2Tensor[4] * Ey * Ey;
        s.gridPolarizationTime2[i] += s.chi2Tensor[1] * Ex * Ex + s.chi2Tensor[3] * Ex * Ey + s.chi2Tensor[5] * Ey * Ey;
    }
    
    //to be implemented: full chi3 matrix on s.nonlinearSwitches[1]==1

    //using only one value of chi3, under assumption of centrosymmetry
    if (s.nonlinearSwitches[1] == 2) {
        s.gridPolarizationTime[i] += s.chi3Tensor[0] * (Ex * Ex * Ex + Ey * Ey * Ex / 3.);
        s.gridPolarizationTime2[i] += s.chi3Tensor[0] * (Ey * Ey * Ey + Ex * Ex * Ey / 3.);
    }

    //Nonlinear absorption
    if (s.nonlinearSwitches[2] == 1) {
        
        double Exi = hipCimag(s.gridETime[i]) / s.propagationInts[0];
        double Eyi = hipCimag(s.gridETime2[i]) / s.propagationInts[0];
        double fieldAmp2 = Exi * Exi + Eyi * Eyi;
        int j;
        for (j = 0; j < s.nonlinearSwitches[3]; j++) {
            Exi *= fieldAmp2;
            Eyi *= fieldAmp2;
        }
        s.gridPolarizationTime[i] += s.absorptionParameters[1] * Exi;
        s.gridPolarizationTime2[i] += s.absorptionParameters[1] * Eyi;
    }
}

//Plasma response with time-dependent carrier density
//This polarization needs a different factor in the nonlinear wave equation
//to account for the integration
//TO BE IMPLEMENTED!
__global__ void plasmaCurrentKernel(struct cudaLoop s) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    int j,k,l;
    double N = 0;
    double integralx = 0;
    double integraly = 0;
    double t, w, Esquared, Ex, Ey;
    for (j = 0; j < s.Ntime; j++) {
        Ex = hipCreal(s.gridETime[l]);
        Ey = hipCreal(s.gridETime2[l]);
        l = j + i * s.Nspace;
        t = j * s.dt;
        Esquared = Ex * Ex + Ey * Ey;
        w = Esquared;
        for (k = 0; k < s.nonlinearSwitches[3]; k++) {
            w *= Esquared;
        }
        N += w;
        integralx += exp(-s.plasmaParameters[1] * t) * N * Ex;
        integraly += exp(-s.plasmaParameters[1] * t) * N * Ey;
        s.plasmaCurrent1[l] = s.plasmaParameters[0] * exp(-s.plasmaParameters[1] * t) * integralx;
        s.plasmaCurrent2[l] = s.plasmaParameters[0] * exp(-s.plasmaParameters[1] * t) * integraly;
    }
}


//Main kernel for RK4 propagation of the field
__global__ void rkKernel(struct cudaLoop s, int stepNumber) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    long long j = i / s.Ntime; //spatial coordinate
    long long h = i - j * s.Ntime; //temporal coordinate

    //polarization is stored in a reduced format by cuFFT because the FFT is from real to complex, meaning if the output grid
    //were to be N_time x N_space, half of the points would be redundant. The extra steps below are to determine where in the grid the 
    //current point sits. Essentially, if in the negative frequency quadrants, reverse the frequency and take complex conjugate of the 
    //value
    if (h > s.propagationInts[3]) {
        h = s.Ntime - h;
        j = s.Nspace - j;
        h += j * s.propagationInts[3];
        s.k1[i] = s.gridPropagationFactor[i] * s.gridETemp[i] +s.gridPolarizationFactor[i] * hipConj(s.gridPolarizationFrequency[i]);
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * hipConj(s.gridPolarizationFrequency2[i]);
    }
    else {
        h += j * s.propagationInts[3];
        s.k1[i] = s.gridPropagationFactor[i] * s.gridETemp[i] +s.gridPolarizationFactor[i] * s.gridPolarizationFrequency[h];
        s.k2[i] = s.gridPropagationFactor2[i] * s.gridETemp2[i] +s.gridPolarizationFactor2[i] * s.gridPolarizationFrequency2[h];
    }

    //in the first substep, first construct the next intermediate field value
    //which will be used in the next substep. 
    if (stepNumber == 0) {
        s.gridETemp[i] = s.gridEFrequency[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];
       
        s.gridEFrequencyNext1[i] = s.k1[i] / 6 + s.gridEFrequency[i];
        s.gridEFrequencyNext2[i] = s.k2[i] / 6 + s.gridEFrequency2[i];
    }

    //in the next substep, again construct the next intermediate field and add k/3 to solution
    else if (stepNumber == 1) {
        s.gridETemp[i] = s.gridEFrequency[i] + 0.5 * s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + 0.5 * s.k2[i];

        s.gridEFrequencyNext1[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 3;
        s.gridEFrequencyNext2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 3;

    }

    //same action as previous substep, except the weight of k in the intermediate solution is 1 instead of 0.5
    else if (stepNumber == 2) {
        s.gridETemp[i] = s.gridEFrequency[i] + s.k1[i];
        s.gridETemp2[i] = s.gridEFrequency2[i] + s.k2[i];
s.gridEFrequencyNext1[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 3;
s.gridEFrequencyNext2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 3;
    }

    //last substep. Solution is now complete and may be copied directly into the field arrays
    else {
    s.gridEFrequency[i] = s.gridEFrequencyNext1[i] + s.k1[i] / 6;
    s.gridEFrequency2[i] = s.gridEFrequencyNext2[i] + s.k2[i] / 6;
    s.gridETemp[i] = s.gridEFrequency[i];
    s.gridETemp2[i] = s.gridEFrequency2[i];
    }

}


//Take absolute value of complex array
__global__ void absKernel(double* absOut, hipDoubleComplex* complexIn) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    absOut[i] = hipCabs(complexIn[i]);
}

//Apply fft normalization
__global__ void fftNormalizeKernel(hipDoubleComplex* A, long long* fftSize) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = A[i] / fftSize[0];
}

DWORD WINAPI propagationLoop(LPVOID lpParam) {

    //the struct s contains most of the simulation variables and pointers
    struct cudaLoop s;
    struct propthread* sCPU = (struct propthread*)lpParam;


    //initialize and take values from the struct handed over by the dispatcher
    long long i;
    s.Ntime = (*sCPU).Ntime;
    s.Nspace = (*sCPU).Nspace;
    s.dt = (*sCPU).tStep;
    s.dx = (*sCPU).rStep;
    s.h = (*sCPU).propagationStep;
    s.Nsteps = (*sCPU).Npropagation;
    s.Ngrid = s.Ntime * s.Nspace;
    s.isNonLinear = ((*sCPU).nonlinearSwitches[0] + (*sCPU).nonlinearSwitches[1] + (*sCPU).nonlinearSwitches[2]) > 0;
    (*sCPU).nonlinearSwitches[3] = (int)ceil((*sCPU).absorptionParameters[0] * 241.79893e12 / (*sCPU).frequency1) - 1;
    //CPU allocations
    std::complex<double>* gridPropagationFactorCPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));
    std::complex<double>* gridPolarizationFactorCPU = (std::complex<double>*)malloc(2 * s.Ngrid * sizeof(std::complex<double>));

    //GPU allocations
    hipMalloc((void**)&s.gridETime, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridETime2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridETemp, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridETemp2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridEFrequency, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridEFrequency2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridPropagationFactor, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridPolarizationFactor, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridPropagationFactor2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridPolarizationFactor2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridEFrequencyNext1, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridEFrequencyNext2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.k1, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.k2, sizeof(hipDoubleComplex) * s.Ngrid);
    hipMalloc((void**)&s.gridPolarizationFrequency, sizeof(hipDoubleComplex) * (s.Ntime / 2 + 1) * s.Nspace);
    hipMalloc((void**)&s.gridPolarizationFrequency2, sizeof(hipDoubleComplex) * (s.Ntime / 2 + 1) * s.Nspace);
    hipMalloc((void**)&s.gridPolarizationTime, sizeof(double) * s.Ngrid);
    hipMalloc((void**)&s.gridPolarizationTime2, sizeof(double) * s.Ngrid);

    hipMalloc((void**)&s.chi2Tensor, sizeof(double) * 9);
    hipMalloc((void**)&s.chi3Tensor, sizeof(double) * 81);
    hipMalloc((void**)&s.nonlinearSwitches, sizeof(int) * 4);
    hipMalloc((void**)&s.absorptionParameters, sizeof(double) * 6);
    hipMalloc((void**)&s.propagationInts, sizeof(long long) * 4);

    hipMemset(s.gridPolarizationFrequency, 0, (s.Ntime / 2 + 1) * s.Nspace * sizeof(hipDoubleComplex));
    hipMemset(s.gridPolarizationFrequency2, 0, (s.Ntime / 2 + 1) * s.Nspace * sizeof(hipDoubleComplex));
    hipMemset(s.gridPropagationFactor, 0, s.Ngrid * sizeof(hipDoubleComplex));
    hipMemset(s.gridPropagationFactor2, 0, s.Ngrid * sizeof(hipDoubleComplex));
    hipMemset(s.gridPolarizationFactor, 0, s.Ngrid * sizeof(hipDoubleComplex));

    long long propagationIntsCPU[4] = { s.Ngrid, s.Ntime, s.Nspace, (s.Ntime / 2 + 1) };


    //prepare effective nonlinearity tensors and put them on the GPU
    deff((*sCPU).deffTensor, (*sCPU).chi2Tensor, (*sCPU).crystalTheta, (*sCPU).crystalPhi);
    hipMemcpy(s.chi2Tensor, (*sCPU).deffTensor, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.nonlinearSwitches, (*sCPU).nonlinearSwitches, 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(s.propagationInts, propagationIntsCPU, 4 * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(s.chi3Tensor, (*sCPU).chi3Tensor, 27 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(s.absorptionParameters, (*sCPU).absorptionParameters, 6 * sizeof(double), hipMemcpyHostToDevice);

    //prepare FFT plans
    hipfftPlan2d(&s.fftPlan, s.Nspace, s.Ntime, HIPFFT_Z2Z);
    hipfftPlan2d(&s.polfftPlan, s.Nspace, s.Ntime, HIPFFT_D2Z);

    //prepare the propagation arrays
    preparepropagation2Dcartesian(sCPU, &s);

    //generate the pulses - later on, add a switch to make this optional in case a field has been loaded
    if ((*sCPU).isFollowerInSequence) {
        hipMemcpy(s.gridETime, (*sCPU).ExtOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridETime2, &(*sCPU).ExtOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency, (*sCPU).EkwOut, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(s.gridEFrequency2, &(*sCPU).EkwOut[(*sCPU).Ngrid], (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    }
    else {
        pulsegenerator(sCPU, &s);
    }
    

    //Copy the field into the temporary array
    hipMemcpy(s.gridETemp, s.gridEFrequency, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
    hipMemcpy(s.gridETemp2, s.gridEFrequency2, s.Nspace * s.Ntime * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

    //Core propagation loop
    
    for (i = 0; i < s.Nsteps; i++) {
        //calculate k1
        rkstep(s, 0);
        //calculate k2
        rkstep(s, 1);
        //calculate k3
        rkstep(s, 2);
        //calculate k4
        rkstep(s, 3);

        if ((*sCPU).imdone[0] == 2) {
            break;
        }
    }
    
    //transform final result
    hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridEFrequency, (hipfftDoubleComplex*)s.gridETime, HIPFFT_BACKWARD);
    hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridEFrequency2, (hipfftDoubleComplex*)s.gridETime2, HIPFFT_BACKWARD);
    fftNormalizeKernel<<<s.Nspace, s.Ntime>>>(s.gridETime, s.propagationInts);
    fftNormalizeKernel<<<s.Nspace, s.Ntime>>>(s.gridETime2, s.propagationInts);

    //copy the field arrays from the GPU to CPU memory
    hipMemcpy((*sCPU).ExtOut, s.gridETime, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy((*sCPU).EkwOut, s.gridEFrequency, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*sCPU).ExtOut[s.Ngrid], s.gridETime2, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&(*sCPU).EkwOut[s.Ngrid], s.gridEFrequency2, (*sCPU).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(s.propagationInts);
    hipFree(s.nonlinearSwitches);
    hipFree(s.absorptionParameters);
    hipFree(s.gridETime); 
    hipFree(s.gridETemp);
    hipFree(s.gridPolarizationFrequency);
    hipFree(s.gridEFrequency);
    hipFree(s.gridPropagationFactor);
    hipFree(s.gridPolarizationFactor);
    hipFree(s.gridEFrequencyNext1);
    hipFree(s.k1);
    hipFree(s.gridPolarizationTime);
    hipFree(s.gridETime2);
    hipFree(s.gridETemp2);
    hipFree(s.gridPolarizationFrequency2);
    hipFree(s.gridEFrequency2);
    hipFree(s.gridPropagationFactor2);
    hipFree(s.gridPolarizationFactor2);
    hipFree(s.gridEFrequencyNext2);
    hipFree(s.k2);
    hipFree(s.gridPolarizationTime2);
    hipFree(s.chi2Tensor);
    hipFree(s.chi3Tensor);
    hipfftDestroy(s.fftPlan);
    hipfftDestroy(s.polfftPlan);

    //Free CPU memory
    free(gridPropagationFactorCPU);
    free(gridPolarizationFactorCPU);
    
    return 0;
}

//function to run a RK4 time step
//stepNumber is the sub-step index, from 0 to 3
int rkstep(struct cudaLoop s, int stepNumber) {

    //calculate nonlinear polarization
    if (s.isNonLinear) {
        //perform inverse FFT to get time-space electric field
        hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridETemp, (hipfftDoubleComplex*)s.gridETime, HIPFFT_BACKWARD);
        hipfftExecZ2Z(s.fftPlan, (hipfftDoubleComplex*)s.gridETemp2, (hipfftDoubleComplex*)s.gridETime2, HIPFFT_BACKWARD);
        
        //calculate nonlinear polarization
        nonlinearpolarizationKernel<<<s.Nspace, s.Ntime>>>(s);

        //FFT nonlinear polarization
        hipfftExecD2Z(s.polfftPlan, s.gridPolarizationTime, (hipfftDoubleComplex*)s.gridPolarizationFrequency);
        hipfftExecD2Z(s.polfftPlan, s.gridPolarizationTime2, (hipfftDoubleComplex*)s.gridPolarizationFrequency2);
    }

    //calculate k
    rkKernel<<<s.Nspace, s.Ntime>>>(s, stepNumber);
    
    return 0;
}

int pulsegenerator(struct propthread* s, struct cudaLoop *sc) {
    long long i,j;
    double rB, zB, r, z; //r and z in the Beam and lab coordinates, respectively.
    double w0, wz, zR, Rz, phi; //Gaussian beam parameters
    double theta = 0; //rotation angle of the current beam
    double pulseSum = 0;
    std::complex<double> ne, no, n0; //active refractive index;
    double f, w; //active frequency;
    double pulseEnergySum;
    std::complex<double> ko, k0, specfac, specphase;
    double c = 2.99792458e8; //speed of light
    double eps0 = 8.8541878128e-12; //vacuum permittivity
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    std::complex<double> *pulse1, *pulse2, *pulse1f, *pulse2f;
    hipfftHandle plan1;
    hipfftHandle plan2;
    pulse1 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse1f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    pulse2f = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    std::complex<double> Eb;
    std::complex<double> ii(0, 1);



    std::complex<double> polFactor1, polFactor2; //complex phase/amplitude factors for the polarization components
    sellmeier(&n0, &no, (*s).sellmeierCoefficients, (*s).frequency1, (*s).crystalTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
    (*s).neref = real(n0);
    (*s).noref = imag(n0);


    //define pulse 1 in mixed space
    // Gaussian beam in x
    // Spectrum in frequency domain (supergaussian with phase terms)
    polFactor1 = cos((*s).polarizationAngle1) - ii * (*s).circularity1 * sin((*s).polarizationAngle1);
    polFactor2 = sin((*s).polarizationAngle1) + ii * (*s).circularity1 * cos((*s).polarizationAngle1);
    theta = (*s).propagationAngle1;
    zB = (*s).z01;
    w0 = (*s).beamwaist1;

    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency1);
        
        //supergaussian pulse spectrum, if no input pulse specified
        specfac = (f - (*s).frequency1)/(*s).bandwidth1;
        for (j = 0; j < (*s).sgOrder1; j++) {
            specfac *= specfac;
        }
        specphase = ii * ((*s).cephase1 + w * (*s).delay1 - (*s).gdd1 * w * w - (*s).tod1 * w * w * w);
        specfac = exp(-specfac - specphase);

        if ((*s).field1IsAllocated) {
            specfac = (*s).loadedField1[i] * exp(-specphase);
        }

        ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
        no = (*s).refractiveIndex2[i + (*s).Ntime * j];
        ko = 2 * pi * no * f / c;
        k0 = 2 * pi * real(n0) * f / c;
        zR = pi * w0 * w0 * real(ne) * f / c;
        if (f == 0) {
            zR = 1e3;
        }

        for (j = 0; j < (*s).Nspace; j++) {
            rB = (*s).x01 + (*s).rStep * j - (*s).Nspace* (*s).rStep / 2.;
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);
            
            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));
            
            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            //z = 0;
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z-zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cmodulussquared(Eb)) || f<=0) {
                Eb = 0;
            }
            
            pulse1[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse1[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r)*(real(ne)*cmodulussquared(pulse1[i + (*s).Ntime * j]) + real(no)*cmodulussquared(pulse1[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }
    
    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime, pulse1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse1[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftPlan1d(&plan1, (*sc).Ntime, HIPFFT_Z2Z, (*sc).Nspace);
    hipfftPlan2d(&plan2, (*sc).Nspace, (*sc).Ntime, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridETemp, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency, (hipfftDoubleComplex*)(*sc).gridETime, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel<<<(*sc).Nspace,(*sc).Ntime>>>((*sc).gridETime);
    conjugateKernel<<<(*sc).Nspace, (*sc).Ntime>>>((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse1, (*sc).gridETime, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse1f, (*sc).gridEFrequency, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse1f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= 59.958 * pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy1/pulseSum)/(*s).Ngrid;
    
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse1[i] = pulse1[i] * pulseEnergySum;
        pulse1f[i] = pulse1f[i] * pulseEnergySum;
    }
    hipDeviceSynchronize();

    //do same for pulse 2 here
    pulseSum = 0;
    polFactor1 = cos((*s).polarizationAngle2) - ii * (*s).circularity2 * sin((*s).polarizationAngle2);
    polFactor2 = sin((*s).polarizationAngle2) + ii * (*s).circularity2 * cos((*s).polarizationAngle2);
    theta = (*s).propagationAngle2;
    zB = (*s).z02;
    w0 = (*s).beamwaist2;

    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        w = 2 * pi * (f - (*s).frequency1);

        //supergaussian pulse spectrum, if no input pulse specified
        specfac = (f - (*s).frequency2) / (*s).bandwidth2;
        for (j = 0; j < (*s).sgOrder1; j++) {
            specfac *= specfac;
        }
        specphase = ii * ((*s).cephase2 + w * (*s).delay2 - (*s).gdd2 * w * w - (*s).tod2 * w * w * w);
        specfac = exp(-specfac - specphase);

        if ((*s).field2IsAllocated) {
            specfac = (*s).loadedField2[i] * exp(-specphase);
        }


        ne = (*s).refractiveIndex1[i + (*s).Ntime * j];
        no = (*s).refractiveIndex2[i + (*s).Ntime * j];
        ko = 2 * pi * no * f / c;
        k0 = 2 * pi * real(n0) * f / c;
        zR = pi * w0 * w0 * real(ne) * f / c;
        if (f == 0) {
            zR = 1e3;
        }

        for (j = 0; j < (*s).Nspace; j++) {

            rB = (*s).x01 + (*s).rStep * j - (*s).Nspace * (*s).rStep / 2.;
            r = rB * cos(theta) - zB * sin(theta);
            z = rB * sin(theta) + zB * cos(theta);

            wz = w0 * sqrt(1 + (z * z / (zR * zR)));
            Rz = z * (1. + (zR * zR / (z * z)));

            if (z == 0) {
                Rz = 1.0e15;
            }
            phi = atan(z / zR);
            Eb = (w0 / wz) * exp(-ii * (real(ko) * (z - zB) + real(ko) * r * r / (2 * Rz) - phi) - r * r / (wz * wz));
            Eb *= specfac;
            if (isnan(cmodulussquared(Eb)) || f <= 0) {
                Eb = 0;
            }

            pulse2[i + (*s).Ntime * j] = polFactor1 * Eb;
            pulse2[i + (*s).Ntime * j + (*s).Ngrid] = polFactor2 * Eb;
            pulseSum += abs(r) * (real(ne) * cmodulussquared(pulse2[i + (*s).Ntime * j]) + real(no) * cmodulussquared(pulse2[i + (*s).Ntime * j + (*s).Ngrid]));
        }
    }

    // copy the field and propagation grids to the GPU
    hipMemcpy((*sc).gridETime, pulse2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &pulse2[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // fft along spatial dimention to get Fourier space beam
    // will take place in three steps:
    // 2D fft (x,f)->(k,t), temporary intermediate state (could be optimized out later)
    // 1D fft (k,t)->(k,f), copied to Fourier space beam
    // 2D fft (k,f)->(x,t), copied to real space beam

    hipfftPlan1d(&plan1, (*sc).Ntime, HIPFFT_Z2Z, (*sc).Nspace);
    hipfftPlan2d(&plan2, (*sc).Nspace, (*sc).Ntime, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridETemp, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency, (hipfftDoubleComplex*)(*sc).gridETime, HIPFFT_BACKWARD);

    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridETemp2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan1, (hipfftDoubleComplex*)(*sc).gridETemp2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_FORWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, (hipfftDoubleComplex*)(*sc).gridETime2, HIPFFT_BACKWARD);

    //Take the conjugate of the field because me and cufft have different ideas of time
    conjugateKernel << <(*sc).Nspace, (*sc).Ntime >> > ((*sc).gridETime);
    conjugateKernel << <(*sc).Nspace, (*sc).Ntime >> > ((*sc).gridETime2);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime, (hipfftDoubleComplex*)(*sc).gridEFrequency, HIPFFT_BACKWARD);
    hipfftExecZ2Z(plan2, (hipfftDoubleComplex*)(*sc).gridETime2, (hipfftDoubleComplex*)(*sc).gridEFrequency2, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    //Copy the GPU grids to the CPU memory
    hipMemcpy(pulse2, (*sc).gridETime, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2[(*s).Ngrid], (*sc).gridETime2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(pulse2f, (*sc).gridEFrequency, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipMemcpy(&pulse2f[(*s).Ngrid], (*sc).gridEFrequency2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    //normalize the pulse energy and set it to the input value
    pulseSum *= c * eps0;
    pulseSum *= 59.958 * pi; //59.958 is emperical factor
    pulseSum *= (*s).rStep / (*s).fStep;
    pulseEnergySum = sqrt((*s).pulseEnergy2 / pulseSum) / (*s).Ngrid;

    for (i = 0; i < (*s).Ngrid * 2; i++) {
        pulse2[i] = pulse2[i] * pulseEnergySum;
        pulse2f[i] = pulse2f[i] * pulseEnergySum;
    }
    hipDeviceSynchronize();

    //make the combined fields
    for (i = 0; i < (*s).Ngrid * 2; i++) {
        (*s).Ext[i] = pulse1[i] + pulse2[i];
        (*s).Ekw[i] = pulse1f[i] + pulse2f[i];
    }
    //Copy the grids back to the GPU
    hipMemcpy((*sc).gridETime, (*s).Ext, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridETime2, &(*s).Ext[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency, (*s).Ekw, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridEFrequency2, &(*s).Ekw[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftDestroy(plan1);
    hipfftDestroy(plan2);


    return 0;
}

int preparepropagation2Dcartesian(struct propthread* s, struct cudaLoop* sc) {
    long long i, j;
    int posf;

    //double alpha = 0; //angle wrt propagation direction of k vector
    double* alpha = (double*)calloc((*s).Ngrid, sizeof(double));
    std::complex<double> ne, no, n0; //active refractive index;
    ne = 0;
    no = 0;
    n0 = 0;
    double f, kr; //active frequency;
    std::complex<double> ke, ko, k0;
    double c = 2.99792458e8; //speed of light
    double pi = 3.14159265358979323846264338327950288; // pi to unneccessary precision
    std::complex<double>* propFactor1, * propFactor2;

    propFactor1 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));
    propFactor2 = (std::complex<double>*)calloc((*s).Ngrid * 2, sizeof(std::complex<double>));

    std::complex<double> ii(0, 1);


    sellmeier(&n0, &no, (*s).sellmeierCoefficients, (*s).frequency1, (*s).crystalTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
    (*s).neref = real(n0);
    (*s).noref = imag(n0);

    //Run the math to find the in-crystal propagation angles (needed for the treatment of walkoff)
    //on the GPU: in the future, this whole calculation should be done on GPU!
    double* alphaGPU = (double*)(*sc).gridEFrequencyNext1;
    double* sellmeierCoefficients = (double*)(*sc).k1;
    double* sellmeierCoefficientsAugmentedCPU = (double*)calloc(66 + 8, sizeof(double));
    memcpy(sellmeierCoefficientsAugmentedCPU, (*s).sellmeierCoefficients, 66 * (sizeof(double)));
    sellmeierCoefficientsAugmentedCPU[66] = (*s).crystalTheta;
    sellmeierCoefficientsAugmentedCPU[67] = (*s).crystalPhi;
    sellmeierCoefficientsAugmentedCPU[68] = (*s).axesNumber;
    sellmeierCoefficientsAugmentedCPU[69] = (*s).sellmeierType;
    sellmeierCoefficientsAugmentedCPU[70] = (*s).kStep;
    sellmeierCoefficientsAugmentedCPU[71] = (*s).fStep; 
    sellmeierCoefficientsAugmentedCPU[72] = 1.0e-12;
    hipMemcpy(sellmeierCoefficients, sellmeierCoefficientsAugmentedCPU, (66+8) * sizeof(double), hipMemcpyHostToDevice);
    thetasearchKernel <<<(*s).Nspace, (*s).Ntime >>> ((*s).Ntime, (*s).Nspace, alphaGPU, sellmeierCoefficients, (*s).axesNumber, (*s).sellmeierType);
    hipDeviceSynchronize();
    hipMemcpy(alpha, alphaGPU, (*s).Ngrid * sizeof(double), hipMemcpyDeviceToHost);
    free(sellmeierCoefficientsAugmentedCPU);
    
    ne = n0;
    for (i = 1; i < (*s).Ntime; i++) {
        f = i * (*s).fStep;
        if (i >= (*s).Ntime / 2) {
            f -= (*s).fStep * (*s).Ntime;
        }
        f *= -1;
        k0 = 2 * pi * real(n0) * f / c;

        for (j = 0; j < (*s).Nspace; j++) {
            kr = j * (*s).kStep - (j >= ((*s).Nspace / 2)) * ((*s).kStep * (*s).Nspace); //frequency grid in transverse direction
            //alpha[i + j * (*s).Ntime] = thetasearch(s, kr, f, 1e-6);
            sellmeier(&ne, &no, (*s).sellmeierCoefficients, abs(f), (*s).crystalTheta+ alpha[i + j * (*s).Ntime], (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);

            if (isnan(real(ne)) || isnan(real(no))) {
                ne = 1;
                no = 1;
            }
            (*s).refractiveIndex1[i + (*s).Ntime * j] = ne;
            (*s).refractiveIndex2[i + (*s).Ntime * j] = no;
            ke = 2 * pi * ne * f / c;
            ko =  2 * pi * no * f / c;

            if (real(ke) < 0 && real(ko) < 0) {
                propFactor1[i + (*s).Ntime * j] = ii * (ke - k0 + kr * kr / (2. * real(ke))) * (*s).propagationStep;
                if (isnan(real(propFactor1[i + (*s).Ntime * j]))) {
                    propFactor1[i + (*s).Ntime * j] = 0.0;
                }

                propFactor1[i + (*s).Ntime * j + (*s).Ngrid] = ii * (ko - k0 + kr * kr / (2. * real(ko))) * (*s).propagationStep;
                if (isnan(real(propFactor1[i + (*s).Ntime * j + (*s).Ngrid]))) {
                    propFactor1[i + (*s).Ntime * j] = 0.0;
                }

                posf = (int)(f < -20e12);
                propFactor2[i + (*s).Ntime * j] = -ii * (posf * 2 * pi * f) / (2. * real(ne) * c) * (*s).propagationStep;
                propFactor2[i + (*s).Ntime * j + (*s).Ngrid] = -ii * (posf * 2 * pi * f) / (2. * real(no) * c) * (*s).propagationStep;
            }

        }
    }

    // copy the propagation grids to the GPU
    hipMemcpy((*sc).gridPropagationFactor, propFactor1, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridPropagationFactor2, &propFactor1[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridPolarizationFactor, propFactor2, (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy((*sc).gridPolarizationFactor2, &propFactor2[(*s).Ngrid], (*s).Ngrid * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    free(propFactor1);
    free(propFactor2);
    free(alpha);
    return 0;
}

double thetasearch(struct propthread* s, double dk, double f, double tol) {
    double theta=0;
    double dTheta = 0.1;
    double err, errPlus, errMinus;
    double rhs = 2.99792458e8 * dk / (2 * 3.14159265358979323846264338327950288 * f);
    std::complex<double> ne, no;
    double nePlus, neMinus;
    f = abs(f);
    sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
    nePlus = real(ne);
    err = abs(nePlus * sin(theta) - rhs);
    int iters = 0;
    while (err > tol && iters<65536) {
        iters++;

        sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta + dTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
        nePlus = real(ne);
        errPlus = abs(nePlus * sin(theta+dTheta) - rhs);

        sellmeier(&ne, &no, (*s).sellmeierCoefficients, f, (*s).crystalTheta + theta - dTheta, (*s).crystalPhi, (*s).axesNumber, (*s).sellmeierType);
        neMinus = real(ne);
        errMinus = abs(neMinus * sin(theta-dTheta) - rhs);

        //Basic hill climbing algorithm
        //calculate the error at theta +/- dTheta
        // if theta + dTheta has lowest error, theta = theta+dTheta, err = errPlus
        // if theta - dTheta has lowest error, theta = theta-dTheta, err = errMinus
        // if theta has lowest error, step size is too large, dTheta /= 2;
        if (errPlus < err && errPlus < errMinus) {
            theta += dTheta;
            err = errPlus;
        }
        else if (errMinus < err) {
            theta -= dTheta;
            err = errMinus;
        }
        else {
            dTheta *= 0.5;
        }
    }
    return theta;
}
//rearrange a matrix from row major order to column major (not used, maybe broken)
int swaprc(double* M, int dim1, int dim2) {
    double* Ms = (double*)malloc(dim1 * dim2 * sizeof(double));
    int i, j;
    for (i = 0; i < dim1; i++) {
        for (j = 0; j < dim2; j++) {
            Ms[i + j * dim1] = M[j + i * dim2];
        }
    }
    free(Ms);
    return 0;
}

int deff(double* defftensor, double* dtensor, double theta, double phi) {
    double delta = 0.; //this angle is used for biaxial crystals, but I'm ignorning it for the moment
    int i, j, k;
    //Rotation matrix between the angles of the electric field and the crystal axes
    double R[] = { cos(theta) * cos(phi) * cos(delta) - sin(phi) * sin(delta), cos(theta) * sin(phi) * cos(delta) + cos(phi) * sin(delta),
        -sin(theta) * cos(delta), -cos(theta) * cos(phi) * sin(delta) - sin(phi) * cos(delta),
        -cos(theta) * sin(phi) * sin(delta) + cos(phi) * cos(delta), sin(theta) * sin(delta) };

    //Matrix to translate the mixed field matrix in the reduced notation into the crystalline frame
    double Ore[] = { R[0] * R[0], R[1] * R[1], R[2] * R[2], 2 * R[1] * R[2], 2 * R[0] * R[2], 2 * R[0] * R[1],
        2 * R[0] * R[3], 2 * R[1] * R[4], 2 * R[2] * R[5], 2 * (R[4] * R[2] + R[1] * R[5]), 2 * (R[3] * R[2] + R[0] * R[5]), 2 * (R[3] * R[1] + R[0] * R[4]),
        R[3] * R[3], R[4] * R[4], R[5] * R[5], 2 * R[4] * R[5], 2 * R[3] * R[5], 2 * R[3] * R[4]
};

    //The deff tensor is given by the equation R deff = d Ore, solve for deff, find d Ore first
    double dOre[9] = { 0 };
    for (i = 0; i < 3; i++) {
        for (j = 0; j < 3; j++) {
            for (k = 0; k < 6; k++) {
                dOre[i + 3 * j] += dtensor[i + 3 * k] * Ore[k + 6 * j];
            }
        }
    }
    //Least squares solution to get the deff tensor
    qr_solve_mat(3, 2, 3, R, dOre, defftensor);

    //correct cross-terms
    for (i = 2; i < 4; i++) {
        defftensor[i] *= 0.5;
    }

    for (i = 0; i < 6; i++) {
        defftensor[i] *= 2e-12; //change from pm/V to m/V and multiply by 2 for chi(2) instead of d
    }
    return 0;
}
//c implementation of fftshift, working on complex double precision
//A is the input array, B is the output
//dim1: column length
//dim2: row length
int fftshiftZ(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i + div1 + dim1 * (j-div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[i + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//same as fftshiftZ, but flips the output array columns
int fftshiftZflip(std::complex<double>* A, std::complex<double>* B, long long dim1, long long dim2) {
    long long i, j;
    long long div1 = dim1 / 2;
    long long div2 = dim2 / 2;
    //Quadrant 1
    for (i = 0; i < div1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1-i-1) + dim1 * j] = A[i + div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 2
    for (i = 0; i < div1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i + div1 + dim1 * (j - div2)];
        }
    }
    //Quadrant 3
    for (i = div1; i < dim1; i++) {
        for (j = 0; j < div2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j + div2)];
        }
    }
    //Quadrant 4
    for (i = div1; i < dim1; i++) {
        for (j = div2; j < dim2; j++) {
            B[(dim1 - i-1) + dim1 * j] = A[i - div1 + dim1 * (j - div2)];
        }
    }
    return 0;
}

//sellmeier equation
//outputs are pointers ne and no
//a is a 16-value array containing the coefficients
//f is frequency (Hz)
//theta is the crystal angle
//phi is the other crystal angle (currently unused because biaxials haven't been implemented)
//type is the kind of crystal (0: isotropic, 1: uniaxial, 2:biaxial) 
//eqn will switch to a different equation, in the future, currently not implemented
//current equation form:
//n^2 = a[0] //background (high freq) contribution
//      + (a[1] + a[2] * lambda^2) / (lambda^2 + a[3]) + (a[4] + a[5] * lambda^2)/ (lambda^2 + a[6]) //two resonances, purely real contribution
//      + (a[7] + a[8] * lambda^2) / (lambda^2 + a[9]) + (a[10] + a[11] * lambda^2) / (lambda^2 + a[12]) //two more resonances
//      + a[13] * lambda^2 + a[14] * lambda^4 + a[15] * lambda^6 //parametrized low-frequency correction
//      + 4*pi*e^2*a[16]/(a[17] - omega^2 + i * a[18] * omega) // complex-valued Lorenzian contribution (a[17] to zero for Drude)
//      + 4*pi*e^2*a[19]/(a[20] - omega^2 + i * a[21] * omega) // complex-valued Lorenzian contribution (a[21] to zero for Drude)
std::complex<double> sellmeier(std::complex<double>* ne, std::complex<double>* no, double* a, double f, double theta, double phi, int type, int eqn) {
    if (f == 0) return 1; //exit immediately for f=0

    double c = 2.99792458e8; //speed of light
    double l = 1e6*c / f; //wavelength in microns
    double ls = l * l;
    std::complex<double> ii(0, 1);
    double pi = 3.14159265358979323846264338327950288;
    double omega = 2*pi*abs(f);
    double kL = 3183.9; //(e * e / (e_o *m_e)
    //option 0: isotropic
    if (type == 0) {
        ne[0] = a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6])
            + (a[7] + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls;
        if (real(ne[0]) < 1) {
            ne[0] = 1.;
        }
        ne[0] += kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega);
        ne[0] = conj(sqrt(ne[0]));
        if (isnan(real(ne[0]))) {
            ne[0] = 1;
        }
        no[0] = ne[0];
        return ne[0];
    }
    //option 1: uniaxial
    else if (type == 1) {
        std::complex<double> na = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega + ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega + ii * a[21] * omega)));
        a = &a[22];
        std::complex<double> nb = (sqrt(a[0]
            + (a[1] + a[2] * ls) / (ls + a[3]) + (a[4] + a[5] * ls) / (ls + a[6]) + (a[7]
                + a[8] * ls) / (ls + a[9]) + (a[10] + a[11] * ls) / (ls + a[12])
            + a[13] * ls + a[14] * ls * ls + a[15] * ls * ls * ls
            + kL * a[16] / (a[17] - omega * omega - ii * a[18] * omega)
            + kL * a[19] / (a[20] - omega * omega - ii * a[21] * omega)));
        if (isnan(real(na)) || isnan(real(nb))) {
            no[0] = 1;
            ne[0] = 1;
            return 1;
        }
        no[0] = na;
        ne[0] = 1.0 / sqrt(cos(theta) * cos(theta) / (na * na) + sin(theta) * sin(theta) / (nb * nb));
        return na;
    }
    else {
        //later, implement biaxial crystals, for now just return 1;
        return 1;
    }
}

int loadfrogspeck(char* frogFilePath, std::complex<double>* Egrid, long long Ntime, double fStep, double gateLevel, int fieldIndex) {
    FILE* fp;
    int maxFileSize = 16384;
    double wavelength, R, phi, complexX, complexY, f, f0, f1, fmax;
    int i, k0, k1;
    double c = 1e9*2.99792458e8; //for conversion of wavelength in nm to frequency
    double df = 0;
    double fmin = 0;
    int currentRow = 0;
    std::complex<double>* E = (std::complex<double>*)calloc(maxFileSize, sizeof(std::complex<double>));

    //read the data
    fp = fopen(frogFilePath, "r");
    while (fscanf(fp, "%lf %lf %lf %lf %lf", &wavelength, &R, &phi, &complexX, &complexY) == 5 && currentRow < maxFileSize) {
        //get the complex field from the data
        E[currentRow].real(complexX);
        E[currentRow].imag(complexY);

        //keep track of the frequency step of the grid (running sum, divide by number of rows at end to get average)
        if (currentRow > 0) df += c / wavelength - fmax;

        //keep track of the highest frequency in the data
        fmax = c / wavelength;
        
        //store the lowest frequency in the data
        if (currentRow == 0) fmin = fmax;
        
        currentRow++;
    }
    fclose(fp);

    //return an error if nothing was loaded
    if (currentRow == 0) {
        free(E);
        return -1;
    }

    df /= currentRow; //average frequency step

    //interpolate the FROG data onto the simulation grid
    
    //fill the simulation grid based on the data
    for (i = 0; i < Ntime; i++) {

        //frequency grid used in the simulation
        f = i * fStep;
        if (i >= Ntime / 2) {
            f -= fStep * Ntime;
        }
        f *= -1;

        k0 = (int)floor((f - fmin) / df);
        k1 = (int)ceil((f - fmin) / df);
        if (k0 < 0 || k1 >= currentRow) {
            Egrid[i] = 0; //field is zero outside of data range
        }
        else {
            f0 = fmin + k0 * df;
            f1 = fmin + k1 * df;
            Egrid[i] = (E[k0] * (f1 - f) + E[k1] * (f - f0)) / df; //linear interpolation
            Egrid[i] *= (abs(Egrid[i]) > gateLevel);
        }
    }

    free(E);
    return currentRow;
}